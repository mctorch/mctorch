#include "hip/hip_runtime.h"
#include <THC/THCTensorMath.h>
#include <THC/THCGeneral.h>
#include <THC/THCAtomics.cuh>
#include <THC/THCApply.cuh>

// Compute the offsets into the given tensors for a linear index. For the 't2'
// tensor, dimension 'dim' is skipped. The tensors are assumed to have the same
// size (with the exception of 't2' in dimension 'dim').
// This version uses a static number of dimensions.
template <typename IndexType, typename Real, int Dims>
struct IndexToScatterGatherOffsets {
  static __device__ void compute(
      IndexType linearId, const int dim,
      const TensorInfo<int64_t, IndexType>& index, IndexType* indexOffset,
      const TensorInfo<Real, IndexType>& t1, IndexType* t1Offset,
      const TensorInfo<Real, IndexType>& t2, IndexType* t2Offset) {
    for (int d = Dims - 1; d >= 0; d--) {
      IndexType curDimIndex = linearId % index.sizes[d];
      *indexOffset += curDimIndex * index.strides[d];
      *t1Offset += curDimIndex * t1.strides[d];
      if (d != dim) {
        *t2Offset += curDimIndex * t2.strides[d];
      }
      linearId /= index.sizes[d];
    }
  }

  static __device__ void compute(
      IndexType linearId, const int dim,
      const TensorInfo<int64_t, IndexType>& index, IndexType* indexOffset,
      const TensorInfo<Real, IndexType>& t2, IndexType* t2Offset) {
    for (int d = Dims - 1; d >= 0; d--) {
      IndexType curDimIndex = linearId % index.sizes[d];
      *indexOffset += curDimIndex * index.strides[d];
      if (d != dim) {
        *t2Offset += curDimIndex * t2.strides[d];
      }
      linearId /= index.sizes[d];
    }
  }
};

// Same as above but using a dynamic number of dimensions.
template <typename IndexType, typename Real>
struct IndexToScatterGatherOffsets<IndexType, Real, -1> {
  static __device__ void compute(
      IndexType linearId, const int dim,
      const TensorInfo<int64_t, IndexType>& index, IndexType* indexOffset,
      const TensorInfo<Real, IndexType>& t1, IndexType* t1Offset,
      const TensorInfo<Real, IndexType>& t2, IndexType* t2Offset) {
    for (int d = index.dims - 1; d >= 0; d--) {
      IndexType curDimIndex = linearId % index.sizes[d];
      *indexOffset += curDimIndex * index.strides[d];
      *t1Offset += curDimIndex * t1.strides[d];
      if (d != dim) {
        *t2Offset += curDimIndex * t2.strides[d];
      }
      linearId /= index.sizes[d];
    }
  }

  static __device__ void compute(
      IndexType linearId, const int dim,
      const TensorInfo<int64_t, IndexType>& index, IndexType* indexOffset,
      const TensorInfo<Real, IndexType>& t2, IndexType* t2Offset) {
    for (int d = index.dims - 1; d >= 0; d--) {
      IndexType curDimIndex = linearId % index.sizes[d];
      *indexOffset += curDimIndex * index.strides[d];
      if (d != dim) {
        *t2Offset += curDimIndex * t2.strides[d];
      }
      linearId /= index.sizes[d];
    }
  }
};

template <typename IndexType, typename Real, int Dims>
#ifdef __HIP_PLATFORM_HCC__
C10_LAUNCH_BOUNDS_1(512)
#endif
__global__ void THCudaTensor_gatherKernel(
    TensorInfo<Real, IndexType> tensor,
    TensorInfo<Real, IndexType> src,
    TensorInfo<int64_t, IndexType> index,
    const int dim,
    const IndexType totalElements) {
  for (IndexType linearId = blockIdx.x * blockDim.x + threadIdx.x;
       linearId < totalElements;
       linearId += gridDim.x * blockDim.x) {
    IndexType tensorOffset = 0;
    IndexType srcOffset = 0;
    IndexType indexOffset = 0;

    IndexToScatterGatherOffsets<IndexType, Real, Dims>::compute(linearId, dim,
                                                          index, &indexOffset,
                                                          tensor, &tensorOffset,
                                                          src, &srcOffset);

    int64_t indexValue = index.data[indexOffset];
    assert(indexValue >= 0 && indexValue < src.sizes[dim]);
    srcOffset += indexValue * src.strides[dim];

    tensor.data[tensorOffset] = src.data[srcOffset];
  }
}

template <typename IndexType, typename Real, int Dims>
#ifdef __HIP_PLATFORM_HCC__
C10_LAUNCH_BOUNDS_1(512)
#endif
__global__ void THCudaTensor_scatterKernel(
    TensorInfo<Real, IndexType> tensor,
    TensorInfo<Real, IndexType> src,
    TensorInfo<int64_t, IndexType> index,
    const int dim,
    const IndexType totalElements) {
  for (IndexType linearId = blockIdx.x * blockDim.x + threadIdx.x;
       linearId < totalElements;
       linearId += gridDim.x * blockDim.x) {
    IndexType tensorOffset = 0;
    IndexType srcOffset = 0;
    IndexType indexOffset = 0;

    IndexToScatterGatherOffsets<IndexType, Real, Dims>::compute(linearId, dim,
                                                          index, &indexOffset,
                                                          src, &srcOffset,
                                                          tensor, &tensorOffset);

    int64_t indexValue = index.data[indexOffset];
    assert(indexValue >= 0 && indexValue < tensor.sizes[dim]);
    tensorOffset += indexValue * tensor.strides[dim];

    tensor.data[tensorOffset] = src.data[srcOffset];
  }
}

template <typename IndexType, typename Real, int Dims>
#ifdef __HIP_PLATFORM_HCC__
C10_LAUNCH_BOUNDS_1(512)
#endif
__global__ void THCudaTensor_scatterAddKernel(
    TensorInfo<Real, IndexType> tensor,
    TensorInfo<Real, IndexType> src,
    TensorInfo<int64_t, IndexType> index,
    const int dim,
    const IndexType totalElements) {
  for (IndexType linearId = blockIdx.x * blockDim.x + threadIdx.x;
       linearId < totalElements;
       linearId += gridDim.x * blockDim.x) {
    IndexType tensorOffset = 0;
    IndexType srcOffset = 0;
    IndexType indexOffset = 0;

    IndexToScatterGatherOffsets<IndexType, Real, Dims>::compute(linearId, dim,
                                                          index, &indexOffset,
                                                          src, &srcOffset,
                                                          tensor, &tensorOffset);

    int64_t indexValue = index.data[indexOffset];
    assert(indexValue >= 0 && indexValue < tensor.sizes[dim]);
    tensorOffset += indexValue * tensor.strides[dim];

    atomicAdd(&tensor.data[tensorOffset], src.data[srcOffset]);
  }
}

template <typename IndexType, typename Real, int Dims>
#ifdef __HIP_PLATFORM_HCC__
C10_LAUNCH_BOUNDS_1(512)
#endif
__global__ void THCudaTensor_scatterFillKernel(
    TensorInfo<Real, IndexType> tensor,
    TensorInfo<int64_t, IndexType> index,
    Real value,
    const int dim,
    const IndexType totalElements) {
  for (IndexType linearId = blockIdx.x * blockDim.x + threadIdx.x;
       linearId < totalElements;
       linearId += gridDim.x * blockDim.x) {
    IndexType tensorOffset = 0;
    IndexType indexOffset = 0;

    IndexToScatterGatherOffsets<IndexType, Real, Dims>::compute(linearId, dim,
                                                          index, &indexOffset,
                                                          tensor, &tensorOffset);

    int64_t indexValue = index.data[indexOffset];
    assert(indexValue >= 0 && indexValue < tensor.sizes[dim]);
    tensorOffset += indexValue * tensor.strides[dim];

    tensor.data[tensorOffset] = value;
  }
}

#include <THC/generic/THCTensorScatterGather.cu>
#include <THC/THCGenerateAllTypes.h>

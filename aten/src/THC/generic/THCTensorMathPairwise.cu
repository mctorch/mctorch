
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THC/generic/THCTensorMathPairwise.cu"
#else

void THCTensor_(add)(THCState *state, THCTensor *self_, THCTensor *src_, scalar_t value)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, src_));
  if (self_ == src_) {
    if (!THC_pointwiseApply1<scalar_t>(state, self_, TensorAddConstantOp<scalar_t>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src_);

    if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self_, src_, TensorAddConstantOp<scalar_t>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

void THCTensor_(sub)(THCState *state, THCTensor *self_, THCTensor *src_, scalar_t value)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, src_));
  if (self_ == src_) {
    if (!THC_pointwiseApply1<scalar_t>(state, self_, TensorSubConstantOp<scalar_t>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src_);

    if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self_, src_, TensorSubConstantOp<scalar_t>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

void THCTensor_(add_scaled)(THCState *state, THCTensor *self_, THCTensor *src_, scalar_t value, scalar_t alpha)
{
  THCTensor_(add)(state, self_, src_, value * alpha);
}

void THCTensor_(sub_scaled)(THCState *state, THCTensor *self_, THCTensor *src_, scalar_t value, scalar_t alpha)
{
  THCTensor_(sub)(state, self_, src_, value * alpha);
}

void THCTensor_(mul)(THCState *state, THCTensor *self_, THCTensor *src_, scalar_t value)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, src_));
  if (self_ == src_) {
    if (!THC_pointwiseApply1<scalar_t>(state, self_, TensorMulConstantOp<scalar_t>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src_);

    if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self_, src_, TensorMulConstantOp<scalar_t>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

void THCTensor_(div)(THCState* state, THCTensor *self_, THCTensor *src_, scalar_t value)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, src_));
  THArgCheck(value != ScalarConvert<int, scalar_t>::to(0), 3, "divide by zero");

  if (self_ == src_) {
    if (!THC_pointwiseApply1<scalar_t>(state, self_, TensorDivConstantOp<scalar_t>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src_);

    if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self_, src_, TensorDivConstantOp<scalar_t>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

void THCTensor_(lshift)(THCState* state, THCTensor *self_, THCTensor *src_, scalar_t value)
{
#if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE)
  THCTensor_(mul)(state, self_, src_, pow(2, value));
#elif defined(THC_REAL_IS_HALF)
  return THError("lshift not supported for torch.CudaHalfTensor");
#else
  if (self_ == src_) {
    if (!THC_pointwiseApply1<scalar_t>(state, self_, TensorLShiftConstantOp<scalar_t>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src_);

    if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self_, src_, TensorLShiftConstantOp<scalar_t>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
#endif
}

void THCTensor_(rshift)(THCState* state, THCTensor *self_, THCTensor *src_, scalar_t value)
{
#if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE)
  THCTensor_(mul)(state, self_, src_, pow(2, -value));
#elif defined(THC_REAL_IS_HALF)
  return THError("rshift not supported for torch.CudaHalfTensor");
#else
  if (self_ == src_) {
    if (!THC_pointwiseApply1<scalar_t>(state, self_, TensorRShiftConstantOp<scalar_t>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src_);

    if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self_, src_, TensorRShiftConstantOp<scalar_t>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
#endif
}

void THCTensor_(fmod)(THCState *state, THCTensor *self_, THCTensor *src_, scalar_t value)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, src_));
  if (self_ == src_) {
    if (!THC_pointwiseApply1<scalar_t>(state, self_, TensorFmodOp<scalar_t>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src_);

    if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self_, src_, TensorFmodOp<scalar_t>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

void THCTensor_(remainder)(THCState *state, THCTensor *self_, THCTensor *src_, scalar_t value)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, src_));
  if (self_ == src_) {
    if (!THC_pointwiseApply1<scalar_t>(state, self_, TensorRemainderOp<scalar_t>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src_);

    if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self_, src_, TensorRemainderOp<scalar_t>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

void THCTensor_(triu)(THCState *state, THCTensor *self_, THCTensor *src_, int64_t k)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, src_));
  THArgCheck(!src_->is_empty() && src_->dim() == 2, 1, "expected a matrix");

  if (self_ != src_)
    THCTensor_(resizeAs)(state, self_, src_);

  int64_t stride0 = self_->stride(0);
  int64_t stride1 = self_->stride(1);
  scalar_t *start = THCTensor_(data)(state, self_);

  TensorTriOp<scalar_t, 1> op(start, stride0, stride1, k);

  if (self_ == src_) {
    if (!THC_pointwiseApply1<scalar_t>(state, src_, op)) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {

    if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self_, src_, op)) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

int THCTensor_(equal)(THCState *state, THCTensor *self_, THCTensor *src_)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, src_));
  if (!THCTensor_(isSameSizeAs(state, self_, src_))) {
    return 0;
  }

  // This is not as efficient as TH, but the basic idea: create a buffer that stores
  // 1 if the two tensors are equal at a position, otherwise 0. If the minimum value
  // in this buffer is 1, the two tensors are equal, otherwise they are not

  THCudaByteTensor *buf = THCudaByteTensor_newWithSize(state, self_->sizes(), {});

  if (!THC_pointwiseApply3<uint8_t, scalar_t, scalar_t>(state, buf, self_, src_, TensorEQOp<scalar_t, unsigned char>())) {
    THArgCheck(false, 2, CUTORCH_DIM_WARNING);
  }

  unsigned char min = THCudaByteTensor_minall(state, buf);

  THCudaByteTensor_free(state, buf);

  return min != 0;
}

void THCTensor_(bitand)(THCState* state, THCTensor *self_, THCTensor *src_, scalar_t value)
{
#if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE) || defined(THC_REAL_IS_HALF)
  return THError("bitand only supported for integer type tensors");
#else
  if (self_ == src_) {
    if (!THC_pointwiseApply1<scalar_t>(state, self_, TensorBitAndConstantOp<scalar_t>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src_);

    if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self_, src_, TensorBitAndConstantOp<scalar_t>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
#endif
}

void THCTensor_(bitor)(THCState* state, THCTensor *self_, THCTensor *src_, scalar_t value)
{
#if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE) || defined(THC_REAL_IS_HALF)
  return THError("bitor only supported for integer type tensors");
#else
  if (self_ == src_) {
    if (!THC_pointwiseApply1<scalar_t>(state, self_, TensorBitOrConstantOp<scalar_t>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src_);

    if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self_, src_, TensorBitOrConstantOp<scalar_t>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
#endif
}

void THCTensor_(bitxor)(THCState* state, THCTensor *self_, THCTensor *src_, scalar_t value)
{
#if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE) || defined(THC_REAL_IS_HALF)
  return THError("bitxor only supported for integer type tensors");
#else
  if (self_ == src_) {
    if (!THC_pointwiseApply1<scalar_t>(state, self_, TensorBitXorConstantOp<scalar_t>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src_);

    if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self_, src_, TensorBitXorConstantOp<scalar_t>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
#endif
}

#endif

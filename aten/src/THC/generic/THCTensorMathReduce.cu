
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THC/generic/THCTensorMathReduce.cu"
#else

void THCTensor_(prod)(THCState* state, THCTensor *self, THCTensor *src, int dimension, int keepdim) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self, src));
  if (!THC_reduceDim<scalar_t>(state, self, src,
                           thrust::identity<accreal>{},
                           ReduceMultiply<accreal>{},
                           thrust::identity<accreal>{},
                           scalar_cast<accreal>(1),
                           dimension,
                           keepdim)) {
    THArgCheck(false, 2, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(cudaGetLastError());
}

#if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE) || defined(THC_REAL_IS_HALF)

void THCTensor_(renorm)(THCState *state, THCTensor* self, THCTensor* src, scalar_t value, int dimension, scalar_t maxnorm)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self, src));
  THCTensor *self_;
  THCTensor *src_ = THCTensor_(newTranspose)(state, src, dimension, 0);
  THCTensor *data = THCTensor_(newClone)(state, src_);
  int64_t numel = THCTensor_(nElement)(state, data);

  THArgCheck(dimension >= 0 && dimension < THCTensor_(nDimensionLegacyNoScalars)(state, src), 3, "invalid dimension");
  THArgCheck(THCNumerics<scalar_t>::gt(value, scalar_cast<scalar_t>(0)), 2, "non-positive-norm not supported");
  THArgCheck(THCTensor_(nDimensionLegacyNoScalars)(state, src) > 1, 1, "need at least 2 dimensions");

  if (numel > 0) {
    ptrdiff_t size = numel / THTensor_sizeLegacyNoScalars(data, 0);
    dim3 grid( THTensor_sizeLegacyNoScalars(data, 0));
    // NOTE: only with this specific number of threads can this work on GPUs with a warp size != 32 (such as AMD). Do not alter w/o changing buffer size in kernel.
    dim3 threads(32);

    THCTensor_kernel_renorm<scalar_t, accreal>
      <<<grid, threads, 0, THCState_getCurrentStream(state)>>>
      (THCTensor_(data)(state, data), scalar_cast<accreal>(value), size, scalar_cast<accreal>(maxnorm));

    cudaError_t errcode = cudaGetLastError();
    if(errcode != cudaSuccess)
      THError(cudaGetErrorString(errcode));
  }

  THCTensor_(free)(state, src_);
  self_ = THCTensor_(newTranspose)(state, data, dimension, 0);
  THCTensor_(resizeAs)(state, self, self_);
  THCTensor_(freeCopyTo)(state, self_, self);
  THCTensor_(free)(state, data);
}

void THCTensor_(std)(THCState *state, THCTensor *self_, THCTensor *src, int dimension, int biased, int keepdim)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, src));

  WelfordData<accreal, scalar_t> init;
  init.reset();
  if (!THC_reduceDim<scalar_t>(state, self_, src,
                           ModifyWelford<WelfordData<accreal, scalar_t>>{},
                           ReduceWelford<accreal, scalar_t>{},
                           VarianceWelford<accreal, scalar_t>{biased, true},
                           init,
                           dimension,
                           keepdim)) {
    THArgCheck(false, 2, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(cudaGetLastError());
}

void THCTensor_(var)(THCState *state, THCTensor *self_, THCTensor *src, int dimension, int biased, int keepdim)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, src));

  WelfordData<accreal, scalar_t> init;
  init.reset();
  if (!THC_reduceDim<scalar_t>(state, self_, src,
                           ModifyWelford<WelfordData<accreal, scalar_t>>{},
                           ReduceWelford<accreal, scalar_t>{},
                           VarianceWelford<accreal, scalar_t>{biased, false},
                           init,
                           dimension,
                           keepdim)) {
    THArgCheck(false, 2, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(cudaGetLastError());
}

accreal THCTensor_(stdall)(THCState *state, THCTensor *self, int biased)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self));
  return THCNumerics<accreal>::sqrt((THCTensor_(varall)(state, self, biased)));
}

accreal THCTensor_(varall)(THCState *state, THCTensor *self, int biased)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self));
  accreal mean = THCTensor_(meanall)(state, self);

  accreal val;
  if (!THC_reduceAll<scalar_t>(state, self,
                           SquareFunctor<accreal>(mean),
                           ReduceAdd<accreal>(),
                           scalar_cast<accreal>(0),
                           &val, 0)) {
    THArgCheck(false, 1, CUTORCH_DIM_WARNING);
  }

  val = THCNumerics<accreal>::div(
    val,
    scalar_cast<accreal>(std::max<int64_t>(0, THCTensor_(nElement)(state, self) - (biased ? 0 : 1)))
  );

  THCudaCheck(cudaGetLastError());
  return val;
}

void THCTensor_(norm)(THCState *state, THCTensor* self, THCTensor* src, scalar_t _value, int dimension, int keepdim)
{
  const accreal value = scalar_cast<accreal>(_value);
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self, src));
  if (THCNumerics<accreal>::eq(value, scalar_cast<accreal>(0))) {
    THC_reduceDim<scalar_t>(state, self, src,
                        TensorNonZeroOp<accreal>{},
                        ReduceAdd<accreal>{},
                        thrust::identity<accreal>{},
                        scalar_cast<accreal>(0),
                        dimension, keepdim);
  } else if (THCNumerics<accreal>::eq(value, scalar_cast<accreal>(1))) {
    THC_reduceDim<scalar_t>(state, self, src,
                        TensorNormOp<accreal, 1>{value},
                        ReduceAdd<accreal>{},
                        thrust::identity<accreal>{},
                        scalar_cast<accreal>(0),
                        dimension, keepdim);
  } else if (THCNumerics<accreal>::eq(value, scalar_cast<accreal>(2))) {
    THC_reduceDim<scalar_t>(state, self, src,
                        TensorNormOp<accreal, 2>{value},
                        ReduceAdd<accreal>{},
                        ReducePow<accreal>{scalar_cast<accreal>(.5)},
                        scalar_cast<accreal>(0),
                        dimension, keepdim);
  } else if (THCNumerics<accreal>::eq(value, scalar_cast<accreal>(INFINITY))) {
    THC_reduceDim<scalar_t>(state, self, src,
                        TensorNormOp<accreal, 1>{value},
                        ReduceMax<accreal>{},
                        thrust::identity<accreal>{},
                        scalar_cast<accreal>(0),
                        dimension, keepdim);
  } else if (THCNumerics<accreal>::eq(value, scalar_cast<accreal>(-INFINITY))) {
    THC_reduceDim<scalar_t>(state, self, src,
                        TensorNormOp<accreal, 1>{value},
                        ReduceMin<accreal>{},
                        thrust::identity<accreal>{},
                        scalar_cast<accreal>(INFINITY),
                        dimension, keepdim);
  } else {
    THC_reduceDim<scalar_t>(state, self, src,
                        TensorNormOp<accreal, -1>{value},
                        ReduceAdd<accreal>{},
                        ReducePow<accreal>{THCNumerics<accreal>::cinv(value)},
                        scalar_cast<accreal>(0),
                        dimension, keepdim);
  }

  THCudaCheck(cudaGetLastError());
}

accreal THCTensor_(normall)(THCState *state, THCTensor *self, scalar_t _value)
{
  const accreal value = scalar_cast<accreal>(_value);
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self));
  accreal result;

  if (THCNumerics<accreal>::eq(value, scalar_cast<accreal>(0))) {
    THC_reduceAll<scalar_t>(state, self,
                        TensorNonZeroOp<accreal>{},
                        ReduceAdd<accreal>{},
                        scalar_cast<accreal>(0),
                        &result, 0);
  } else if (THCNumerics<accreal>::eq(value, scalar_cast<accreal>(1))) {
    THC_reduceAll<scalar_t>(state, self,
                        TensorNormOp<accreal, 1>{value},
                        ReduceAdd<accreal>{},
                        scalar_cast<accreal>(0),
                        &result, 0);
  } else if (THCNumerics<accreal>::eq(value, scalar_cast<accreal>(2))) {
    THC_reduceAll<scalar_t>(state, self,
                        TensorNormOp<accreal, 2>{value},
                        ReduceAdd<accreal>{},
                        scalar_cast<accreal>(0),
                        &result, 0);
    result = THCNumerics<accreal>::sqrt(result);
  } else if (THCNumerics<accreal>::eq(value, scalar_cast<accreal>(INFINITY))) {
    THC_reduceAll<scalar_t>(state, self,
                        TensorNormOp<accreal, 1>{value},
                        ReduceMax<accreal>{},
                        scalar_cast<accreal>(0),
                        &result, 0);
  } else if (THCNumerics<accreal>::eq(value, scalar_cast<accreal>(-INFINITY))) {
    THC_reduceAll<scalar_t>(state, self,
                        TensorNormOp<accreal, 1>{value},
                        ReduceMin<accreal>{},
                        scalar_cast<accreal>(INFINITY),
                        &result, 0);
  } else {
    THC_reduceAll<scalar_t>(state, self,
                        TensorNormOp<accreal, -1>{value},
                        ReduceAdd<accreal>{},
                        scalar_cast<accreal>(0),
                        &result, 0);
    result = THCNumerics<accreal>::pow(result,
                                       THCNumerics<accreal>::cinv(value));
  }

  THCudaCheck(cudaGetLastError());
  return result;
}

accreal THCTensor_(dist)(THCState *state, THCTensor *self,
                         THCTensor *src, scalar_t _value)
{
  const accreal value = scalar_cast<accreal>(_value);
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self, src));
  self = THCTensor_(newContiguous)(state, self);
  ptrdiff_t size = THCTensor_(nElement)(state, self);
  src = THCTensor_(newContiguous)(state, src);
  thrust::device_ptr<scalar_t> self_data(THCTensor_(data)(state, self));
  thrust::device_ptr<scalar_t> src_data(THCTensor_(data)(state, src));

  THCThrustAllocator thrustAlloc(state);
  accreal result;

  if (THCNumerics<accreal>::eq(value, scalar_cast<accreal>(INFINITY))) {
    result = thrust::inner_product(
#if CUDA_VERSION >= 7000 || defined __HIP_PLATFORM_HCC__
      thrust::cuda::par(thrustAlloc).on(THCState_getCurrentStream(state)),
#endif
      self_data, self_data+size, src_data, scalar_cast<accreal>(0),
      ReduceMax<accreal>(),
      ThrustTensorDistOp<scalar_t, accreal>(scalar_cast<scalar_t>(1)));
  } else if (THCNumerics<accreal>::eq(value, scalar_cast<accreal>(-INFINITY))) {
    result = thrust::inner_product(
#if CUDA_VERSION >= 7000 || defined __HIP_PLATFORM_HCC__
      thrust::cuda::par(thrustAlloc).on(THCState_getCurrentStream(state)),
#endif
      self_data, self_data+size, src_data, scalar_cast<accreal>(INFINITY),
      ReduceMin<accreal>(),
      ThrustTensorDistOp<scalar_t, accreal>(scalar_cast<scalar_t>(1)));
  } else if (THCNumerics<accreal>::eq(value, scalar_cast<accreal>(0))) {
    result = thrust::inner_product(
#if CUDA_VERSION >= 7000 || defined __HIP_PLATFORM_HCC__
      thrust::cuda::par(thrustAlloc).on(THCState_getCurrentStream(state)),
#endif
      self_data, self_data+size, src_data, scalar_cast<accreal>(0),
      thrust::plus<accreal>(),
      ThrustTensorDistOp<scalar_t, accreal>(scalar_cast<scalar_t>(0)));
  } else {
    result = thrust::inner_product(
#if CUDA_VERSION >= 7000 || defined __HIP_PLATFORM_HCC__
      thrust::cuda::par(thrustAlloc).on(THCState_getCurrentStream(state)),
#endif
      self_data, self_data+size, src_data, scalar_cast<accreal>(0),
      thrust::plus<accreal>(),
      ThrustTensorDistOp<scalar_t, accreal>(value));

    result = THCNumerics<accreal>::pow(result, THCNumerics<accreal>::cinv(value));
  }
  THCTensor_(free)(state, src);
  THCTensor_(free)(state, self);

  return result;
}

#endif

accreal THCTensor_(sumall)(THCState *state, THCTensor *self) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self));
  accreal val;
  if (!THC_reduceAll<scalar_t>(state, self,
                           thrust::identity<accreal>{},
                           ReduceAdd<accreal>{},
                           scalar_cast<accreal>(0),
                           &val, 0)) {
    THArgCheck(false, 1, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(cudaGetLastError());
  return val;
}

accreal THCTensor_(meanall)(THCState *state, THCTensor *self)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self));
  return THCTensor_(sumall)(state, self)/THCTensor_(nElement)(state, self);
}

scalar_t THCTensor_(minall)(THCState *state, THCTensor *self) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self));
  accreal val;
  if (!THC_reduceAll<scalar_t>(state, self,
                           thrust::identity<accreal>{},
                           ReduceMin<accreal>{},
                           THCNumerics<accreal>::upper_bound(), &val, 0)) {
    THArgCheck(false, 1, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(cudaGetLastError());
  return scalar_cast<scalar_t>(val);
}

scalar_t THCTensor_(maxall)(THCState *state, THCTensor *self) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self));
  accreal val;
  if (!THC_reduceAll<scalar_t>(state, self,
                           thrust::identity<accreal>{},
                           ReduceMax<accreal>{},
                           THCNumerics<accreal>::lower_bound(), &val, 0)) {
    THArgCheck(false, 1, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(cudaGetLastError());
  return scalar_cast<scalar_t>(val);
}

void THCTensor_(max)(THCState *state,
                     THCTensor *values,
                     THCudaLongTensor *indices,
                     THCTensor *src,
                     int dimension,
                     int keepdim) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, values, indices, src));

  thrust::pair<scalar_t, int64_t>
    init =
    thrust::make_pair<scalar_t, int64_t>(
      THCNumerics<scalar_t>::lower_bound(), 0);

  return THC_reduceDimIndex<scalar_t, int64_t>(
    state, values, indices, src, dimension, keepdim, init,
    MaxValuePair<scalar_t, int64_t>());
}

void THCTensor_(min)(THCState *state,
                     THCTensor *values,
                     THCudaLongTensor *indices,
                     THCTensor *src,
                     int dimension,
                     int keepdim) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, values, indices, src));

  thrust::pair<scalar_t, int64_t>
    init =
    thrust::make_pair<scalar_t, int64_t>(
      THCNumerics<scalar_t>::upper_bound(), 0);

  return THC_reduceDimIndex<scalar_t, int64_t>(
    state, values, indices, src, dimension, keepdim, init,
    MinValuePair<scalar_t, int64_t>());
}

#endif


#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THC/generic/THCTensorRandom.cu"
#else

#include "ATen/cuda/CUDAContext.h"

#define NUM_BLOCKS min((int)THCCeilDiv(size, (ptrdiff_t) BLOCK_SIZE), MAX_NUM_BLOCKS)

#if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE) || defined(THC_REAL_IS_HALF)

void THCTensor_(uniform)(THCState* state, THCTensor *self_, double a, double b)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self_));
  ptrdiff_t size = THCTensor_(nElement)(state, self_);
  if (size == 0) return;
  THCGenerator* gen = THCRandom_getGenerator(state);
  THCTensor *self = THCTensor_(newContiguous)(state, self_);
  scalar_t *data = THCTensor_(data)(state, self);

  generate_uniform<<<NUM_BLOCKS, BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
      gen->state.gen_states, size, data, a, b);

  THCTensor_(freeCopyTo)(state, self, self_);
};

void THCTensor_(normal)(THCState* state, THCTensor *self_, double mean, double stdv)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self_));
  ptrdiff_t size = THCTensor_(nElement)(state, self_);
  if (size == 0) return;
  THCGenerator* gen = THCRandom_getGenerator(state);
  THCTensor *self = THCTensor_(newContiguous)(state, self_);
  scalar_t *data = THCTensor_(data)(state, self);

  generate_normal<<<NUM_BLOCKS, BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
      gen->state.gen_states, size, data, mean, stdv);

  THCTensor_(freeCopyTo)(state, self, self_);
};

void THCTensor_(normal_means)(THCState *state, THCTensor *self, THCTensor *means, double stddev) {
  THCTensor_(resizeAs)(state, self, means);
  THCTensor_(normal)(state, self, 0, stddev);
  THCTensor_(cadd)(state, self, self, ScalarConvert<int, scalar_t>::to(1), means);
}

void THCTensor_(normal_stddevs)(THCState *state, THCTensor *self, double mean, THCTensor *stddevs)
{
  THCTensor_(resizeAs)(state, self, stddevs);
  THCTensor_(normal)(state, self, 0, 1);
  THCTensor_(cmul)(state, self, self, stddevs);
  THCTensor_(add)(state, self, self, ScalarConvert<double, scalar_t>::to(mean));
}

void THCTensor_(normal_means_stddevs)(THCState *state, THCTensor *self, THCTensor *means, THCTensor *stddevs)
{
  THCTensor_(resizeAs)(state, self, means);
  THCTensor_(normal)(state, self, 0, 1);
  THCTensor_(cmul)(state, self, self, stddevs);
  THCTensor_(cadd)(state, self, self, ScalarConvert<int, scalar_t>::to(1), means);
}

void THCTensor_(logNormal)(THCState* state, THCTensor *self_, double mean, double stdv)
{

  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self_));
  ptrdiff_t size = THCTensor_(nElement)(state, self_);
  if (size == 0) return;
  THCGenerator* gen = THCRandom_getGenerator(state);

  THCTensor *self = THCTensor_(newContiguous)(state, self_);
  scalar_t *data = THCTensor_(data)(state, self);

  generateLogNormal<scalar_t><<<NUM_BLOCKS, BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
      gen->state.gen_states, size, data, mean, stdv);

  THCTensor_(freeCopyTo)(state, self, self_);
};

void THCTensor_(exponential)(THCState* state, THCTensor *self_, double lambda)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self_));
  ptrdiff_t size = THCTensor_(nElement)(state, self_);
  if (size == 0) return;
  THCGenerator* gen = THCRandom_getGenerator(state);

  THCTensor *self = THCTensor_(newContiguous)(state, self_);
  scalar_t *data = THCTensor_(data)(state, self);

  generate_exponential<<<NUM_BLOCKS, BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
      gen->state.gen_states, size, data, lambda);

  THCTensor_(freeCopyTo)(state, self, self_);
};

void THCTensor_(cauchy)(THCState* state, THCTensor *self_, double median, double sigma)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self_));
  ptrdiff_t size = THCTensor_(nElement)(state, self_);
  if (size == 0) return;
  THCGenerator* gen = THCRandom_getGenerator(state);

  THCTensor *self = THCTensor_(newContiguous)(state, self_);
  scalar_t *data = THCTensor_(data)(state, self);

  generate_cauchy<<<NUM_BLOCKS, BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
      gen->state.gen_states, size, data, median, sigma);

  THCTensor_(freeCopyTo)(state, self, self_);
};

void THCTensor_(renormRows)(struct THCState* state,
                             THCTensor* t) {
  THAssert(THCTensor_(nDimensionLegacyAll)(state, t) == 2);
  int64_t rows = THCTensor_(size)(state, t, 0);
  int64_t cols = THCTensor_(size)(state, t, 1);

  cudaDeviceProp* props = at::cuda::getCurrentDeviceProperties();
  THAssert(props != NULL);

  int numSM = props->multiProcessorCount;
  int maxThreads = props->maxThreadsPerBlock;

  dim3 grid(rows < numSM * 4 ? rows : numSM * 4);
  dim3 block(cols < maxThreads ? cols : maxThreads);

  renormRowsL1<scalar_t>
    <<<grid, block, block.x * sizeof(scalar_t),
    THCState_getCurrentStream(state)>>>(THCTensor_(data)(state, t),
                                        rows, cols);
}

void THCTensor_(multinomial)(struct THCState *state,
                              THCudaLongTensor *self,
                              THCTensor *prob_dist,
                              int n_sample,
                              int with_replacement)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self, prob_dist));
  THCGenerator* gen = THCRandom_getGenerator(state);

  int inputSize = THCTensor_(nDimensionLegacyAll)(state, prob_dist);
  THArgCheck(inputSize > 0 && inputSize <= 2, 2,
             "prob_dist must be 1 or 2 dim");

  // Categories are in the innermost dimension
  int64_t numDist =
    inputSize == 1 ? 1 : THCTensor_(sizeLegacyNoScalars)(state, prob_dist, 0);
  int64_t numCategoriesLong =
    inputSize == 1 ? THCTensor_(sizeLegacyNoScalars)(state, prob_dist, 0) :
    THCTensor_(sizeLegacyNoScalars)(state, prob_dist, 1);

  // Since the index tensor is float, numCategories cannot exceed max
  // float integer precision
  THArgCheck(numCategoriesLong <= FLOAT32_MAX_CONSECUTIVE_INT, 2,
             "number of categories cannot exceed 2^24");
  int numCategories = (int) numCategoriesLong;

  THArgCheck(n_sample > 0, 3, "cannot sample <= 0 samples");

  if (!with_replacement) {
    THArgCheck(n_sample <= numCategories, 2,
               "cannot sample n_sample > prob_dist:size(1) samples without "
               "replacement");
  }

  int free_prob_dist = 0;

  // Restructure data for 2d
  if (inputSize == 1) {
    THCTensor *temp = THCTensor_(new)(state);
    THCTensor_(unsqueeze1d)(state, temp, prob_dist, 0);
    prob_dist = temp;
    free_prob_dist = 1;
  }

  THCudaLongTensor_resize2d(state, self, numDist, n_sample);

  // get current device properties
  cudaDeviceProp* props = at::cuda::getCurrentDeviceProperties();
  THAssert(props != NULL);
  int numSM = props->multiProcessorCount;
  int maxThreads = props->maxThreadsPerBlock;
  int maxShared = props->sharedMemPerBlock;
  int requiredShared = (numCategories < maxThreads ? numCategories : maxThreads)
                                * (sizeof(scalar_t) + sizeof(accreal));

  if (n_sample == 1 && maxShared >= requiredShared) {
    // Optimized allocation-free implementation
    // To exploit greater parallelism for the sampling, generate the
    // Uniform random samples in a separate kernel launch, into
    // temporarily allocated memory. The device RNG is thread-limited
    THCTensor *sampled = THCTensor_(newWithSize2d)(state, numDist, n_sample);
    THCTensor_(uniform)(state, sampled, 0.0, 1.0);

    dim3 block(numCategories < maxThreads ? numCategories : maxThreads);
    dim3 grid(numDist < numSM * 4 ? numDist : numSM * 4);

    sampleMultinomialOnce<scalar_t, accreal>
      <<<grid, block,
         requiredShared,
         THCState_getCurrentStream(state)>>>(
      THCudaLongTensor_data(state, self),
      numDist,
      numCategories,
      THCTensor_(data)(state, sampled),
      THCTensor_(data)(state, prob_dist),
      THCTensor_(stride)(state, prob_dist, 0),
      THCTensor_(stride)(state, prob_dist, 1)
      );
    THCTensor_(free)(state, sampled);
  } else {
    // Generic, slow implementation with memory allocations

    // For sampling without replacement, we modify the distribution
    // for subsequent samples in this space
    THCTensor* origDist = THCTensor_(new)(state);
    THCTensor_(resizeAs)(state, origDist, prob_dist);
    THCTensor_(copy)(state, origDist, prob_dist);

    THCTensor* normDist = THCTensor_(new)(state);
    THCTensor_(resizeAs)(state, normDist, prob_dist);

    THCTensor* prefixSum = THCTensor_(new)(state);

    // Renorm along rows
    THCTensor_(copy)(state, normDist, origDist);
    THCTensor_(renormRows)(state, normDist);

    // Prefix sum along rows
    THCTensor_(cumsum)(state, prefixSum, normDist, 1);

    if (with_replacement) {
      // Sample with replacement

      // Binary search is warp divergent (so effectively we're running
      // with just a single thread), but for better utilization,
      // we need each block to have at least 4 warps.
      dim3 block(32, 4);

      // Each warp in a block will generate a sample from one
      // distribution concurrently.
      dim3 grid(numDist < MAX_NUM_BLOCKS ? numDist : MAX_NUM_BLOCKS);

      sampleMultinomialWithReplacement
        <<<grid, block, 0, THCState_getCurrentStream(state)>>>(
          gen->state.gen_states,
          n_sample,
          THCudaLongTensor_data(state, self),
          numDist, numCategories,
          THCTensor_(data)(state, prefixSum),
          THCTensor_(data)(state, normDist));
    } else {
      // Sample without replacement

      // Binary search is warp divergent (so effectively we're running
      // with just a single thread), but for better utilization,
      // we need each block to have at least 4 warps.
      dim3 block(32, 4);

      // Each warp in a block will generate a sample from a different
      // distribution concurrently.
      ptrdiff_t numBlocks = THCCeilDiv(numDist, (int64_t) 4);
      dim3 grid(numBlocks < MAX_NUM_BLOCKS ? numBlocks : MAX_NUM_BLOCKS);

      for (int sample = 0; sample < n_sample; ++sample) {
        if (sample > 0) {
          // Update probabilities
          // Renorm along rows
          THCTensor_(copy)(state, normDist, origDist);
          THCTensor_(renormRows)(state, normDist);

          // Prefix sum along rows
          THCTensor_(cumsum)(state, prefixSum, normDist, 1);
        }

        // The kernel can only draw one sample before we have to
        // recalculate our distribution
        sampleMultinomialWithoutReplacement
          <<<grid, block, 0, THCState_getCurrentStream(state)>>>(
            gen->state.gen_states,
            n_sample,
            sample,
            THCudaLongTensor_data(state, self),
            numDist, numCategories,
            THCTensor_(data)(state, origDist),
            THCTensor_(data)(state, prefixSum));
      }
    }

    THCTensor_(free)(state, prefixSum);
    THCTensor_(free)(state, normDist);
    THCTensor_(free)(state, origDist);
  }

  // Revert data restructuring based on input sizes
  if (inputSize == 1) {
    THCudaLongTensor_resize1d(state, self, n_sample);
  }
  if (free_prob_dist) {
    THCTensor_(free)(state, prob_dist);
  }
}

void THCTensor_(multinomialAliasSetup)(THCState *state, THCTensor *_probs, THCudaLongTensor *_J, THCTensor *_q){
  THArgCheck(_probs->dim() == 1, 1,
             "expected 1-D probability tensor, got %d-D probability tensor instead",
             _probs->dim());
  THAssert(THCTensor_(isContiguous)(state, _q));
  THAssert(THCudaLongTensor_isContiguous(state, _J));
  THCTensor *probs = THCTensor_(newContiguous)(state, _probs);
  THAssert(THCTensor_(isContiguous)(state, probs));
  int64_t inputsize = THCTensor_(nElement)(state, probs);
  THCudaLongTensor *smaller = THCudaLongTensor_newWithSize1d(state, inputsize);
  THCudaLongTensor *larger = THCudaLongTensor_newWithSize1d(state, inputsize);
  THCudaLongTensor *smaller_short = THCudaLongTensor_newWithSize1d(state, inputsize);
  THCudaLongTensor *larger_short = THCudaLongTensor_newWithSize1d(state, inputsize);

  THCudaLongTensor_resize1d(state, _J, inputsize);
  THCTensor_(resize1d)(state, _q, inputsize);

  scalar_t one = ScalarConvert<int64_t, scalar_t>::to(1);
  int inputBlockDim = THCCeilDiv((int)inputsize + BLOCK_SIZE - 1, BLOCK_SIZE);
  aliasMultinomialFilter
    <<<inputBlockDim, BLOCK_SIZE, 0, THCState_getCurrentStream(state) >>>(
                     THCTensor_(data)(state, _q),
                     THCTensor_(data)(state, probs),
                     THCudaLongTensor_data(state, smaller),
                     THCudaLongTensor_data(state, larger),
                     THCudaLongTensor_data(state, _J),
                     THCudaLongTensor_data(state, smaller_short),
                     THCudaLongTensor_data(state, larger_short),
                     one, inputsize
                     );

  THCudaLongTensor_nonzero(state, smaller_short, smaller);
  THCudaLongTensor_nonzero(state, larger_short, larger);
  int h_large_c = THCudaLongTensor_nElement(state, larger_short);
  THCudaLongTensor_resize1d(state, smaller_short, inputsize);
  THCudaLongTensor_resize1d(state, larger_short, inputsize);
  aliasMultinomialSetup
    <<<1, 1, 0, THCState_getCurrentStream(state)>>>(
                THCudaLongTensor_data(state, _J),
                THCTensor_(data)(state, _q),
                inputsize,
                THCudaLongTensor_data(state, smaller_short),
                THCudaLongTensor_data(state, larger_short),
                inputsize - h_large_c, h_large_c
                );
  scalar_t q_max = THCTensor_(maxall)(state, _q);
  condDiv<<<
    inputBlockDim, BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
                      THCTensor_(data)(state, _q),
                      THCudaLongTensor_data(state, _J),
                      inputsize, q_max
                      );

  THCudaLongTensor_free(state, smaller);
  THCudaLongTensor_free(state, larger);
  THCudaLongTensor_free(state, smaller_short);
  THCudaLongTensor_free(state, larger_short);
  THCTensor_free(state, probs);
}

void THCTensor_(multinomialAliasDraw)(THCState *state, THCudaLongTensor *self, THCTensor *_q, THCudaLongTensor *_J, int n_sample){
  THArgCheck(_q->dim() == 1, 1,
             "expected 1-D probability table, got %d-D probability table instead",
             _q->dim());
  THArgCheck(_J->dim() == 1, 2,
             "expected 1-D alias table, got %d-D alias table instead",
             _J->dim());
  THArgCheck(n_sample > 0, 3, "cannot sample <= 0 samples");
  THAssert(THCTensor_(isContiguous)(state, _q));
  THAssert(THCudaLongTensor_isContiguous(state, _J));
  THCGenerator* gen = THCRandom_getGenerator(state);
  int64_t K = THCudaLongTensor_nElement(state, _J);
  THCudaLongTensor_resize1d(state, self, n_sample);
  ptrdiff_t size = THCudaLongTensor_nElement(state, self);

  THCTensor *uniform = THCTensor_(newWithSize1d)(state, n_sample);
  THCTensor *bernoulli = THCTensor_(newWithSize1d)(state, n_sample);

  THCTensor_(uniform)(state, uniform, 0, K);
  THCTensor_(uniform)(state, bernoulli, 0, 1);

  multinomialAliasDrawKernel
    <<<THCCeilDiv((int)n_sample+BLOCK_SIZE-1, BLOCK_SIZE), BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
          size,
          THCudaLongTensor_data(state, self),
          THCudaLongTensor_data(state, _J),
          THCTensor_(data)(state, _q),
          K,
          THCTensor_(data)(state, uniform),
          THCTensor_(data)(state, bernoulli)
          );
  THCTensor_(free)(state, uniform);
  THCTensor_(free)(state, bernoulli);
}

#endif

#if defined(THC_REAL_IS_DOUBLE)
GENERATE_KERNEL1(generate_geometric, double, double p, double, curand_uniform_double, ceil(log(x) / log(1-p)))
#else
GENERATE_KERNEL1(generate_geometric, scalar_t, double p, float, curand_uniform, (ScalarConvert<float, scalar_t>::to(ceilf(logf(x) / log(1-p)))))
#endif

#if defined(THC_REAL_IS_LONG) || defined(THC_REAL_IS_DOUBLE) || defined(THC_REAL_IS_FLOAT)
#define CURAND64(STATE) (((uint64_t)curand(STATE)) << 32) | (uint64_t)curand(STATE)
GENERATE_KERNEL2(generate_random, scalar_t, int32_t base, uint32_t range, uint32_t, curand, \
    static_cast<scalar_t>(static_cast<int32_t>((x % range) + base)))
GENERATE_KERNEL2(generate_random_64, scalar_t, int64_t base, uint64_t range, uint64_t, CURAND64, \
    static_cast<scalar_t>(static_cast<int64_t>((x % range) + base)))
#elif defined(THC_REAL_IS_HALF)
GENERATE_KERNEL2(generate_random, scalar_t, int32_t base, uint32_t range, uint32_t, curand,
    (ScalarConvert<int32_t, scalar_t>::to(static_cast<int32_t>(x % range + base))))
#else
GENERATE_KERNEL2(generate_random, scalar_t, int32_t base, uint32_t range, uint32_t, curand,
    static_cast<scalar_t>(static_cast<int32_t>(x % range + base)))
#endif

void THCTensor_(geometric)(THCState* state, THCTensor *self_, double p)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self_));
  ptrdiff_t size = THCTensor_(nElement)(state, self_);
  if (size == 0) return;
  THCGenerator* gen = THCRandom_getGenerator(state);

  THCTensor *self = THCTensor_(newContiguous)(state, self_);
  scalar_t *data = THCTensor_(data)(state, self);

  generate_geometric<<<NUM_BLOCKS, BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
      gen->state.gen_states, size, data, p);

  THCTensor_(freeCopyTo)(state, self, self_);
};

void THCTensor_(clampedRandom)(THCState* state, THCTensor *self_, int64_t min_val, int64_t max_val)
{
  THArgCheck(min_val < max_val, 2,
             "max must be greater than min, but got: min = %lld, max = %lld", min_val, max_val);
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self_));
  ptrdiff_t size = THCTensor_(nElement)(state, self_);
  if (size == 0) return;
  THCGenerator* gen = THCRandom_getGenerator(state);
  THCTensor *self = THCTensor_(newContiguous)(state, self_);
  scalar_t *data = THCTensor_(data)(state, self);

  uint64_t range = max_val - min_val;

#if defined(THC_REAL_IS_LONG) || defined(THC_REAL_IS_DOUBLE) || defined(THC_REAL_IS_FLOAT)
  if (range > 1ULL << 32) {
    generate_random_64<<<NUM_BLOCKS, BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
        gen->state.gen_states, static_cast<int>(size), data, min_val, range);
  } else {
#endif
    generate_random<<<NUM_BLOCKS, BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
        gen->state.gen_states, static_cast<int>(size), data, static_cast<int32_t>(min_val), static_cast<uint32_t>(range));
#if defined(THC_REAL_IS_LONG) || defined(THC_REAL_IS_DOUBLE) || defined(THC_REAL_IS_FLOAT)
  }
#endif

  THCTensor_(freeCopyTo)(state, self, self_);
};

void THCTensor_(cappedRandom)(THCState* state, THCTensor *self_, int64_t max_val)
{
  THCTensor_(clampedRandom)(state, self_, 0LL, max_val);
};

#define HLF_MANT_DIG 11

void THCTensor_(random)(THCState* state, THCTensor *self_)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self_));
  ptrdiff_t size = THCTensor_(nElement)(state, self_);
  if (size == 0) return;
  THCGenerator* gen = THCRandom_getGenerator(state);
  THCTensor *self = THCTensor_(newContiguous)(state, self_);
  scalar_t *data = THCTensor_(data)(state, self);

#if defined(THC_REAL_IS_HALF)
  generate_random<<<NUM_BLOCKS, BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
      gen->state.gen_states, static_cast<int>(size), data, static_cast<int32_t>(0UL), static_cast<uint32_t>((1UL << HLF_MANT_DIG) + 1));
#elif defined(THC_REAL_IS_FLOAT)
  generate_random<<<NUM_BLOCKS, BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
      gen->state.gen_states, static_cast<int>(size), data, static_cast<int32_t>(0UL), static_cast<uint32_t>((1UL << FLT_MANT_DIG) + 1));
#elif defined(THC_REAL_IS_DOUBLE)
  generate_random_64<<<NUM_BLOCKS, BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
      gen->state.gen_states, static_cast<int>(size), data, static_cast<int64_t>(0ULL), static_cast<uint64_t>((1ULL << DBL_MANT_DIG) + 1));
#elif defined(THC_REAL_IS_LONG)
  generate_random_64<<<NUM_BLOCKS, BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
      gen->state.gen_states, static_cast<int>(size), data, static_cast<int64_t>(0ULL), static_cast<uint64_t>(std::numeric_limits<scalar_t>::max()) + 1);
#else
  generate_random<<<NUM_BLOCKS, BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
      gen->state.gen_states, static_cast<int>(size), data, static_cast<int32_t>(0UL), static_cast<uint32_t>(std::numeric_limits<scalar_t>::max()) + 1);
#endif

  THCTensor_(freeCopyTo)(state, self, self_);
};

#undef HLF_MANT_DIG
#undef CURAND64
#undef NUM_BLOCKS

#endif

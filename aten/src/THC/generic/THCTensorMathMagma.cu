
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THC/generic/THCTensorMathMagma.cu"
#else

#if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE)

#ifdef USE_MAGMA

static void THCTensor_(copyArray1d)(THCState *state, THCTensor *self, scalar_t *src, int k)
{
  int64_t size[1] = { k };
  int64_t stride[1] = { 1 };
  THCTensor_(resizeNd)(state, self, 1, size, stride);
  size_t len = k * sizeof(scalar_t);
  THCudaCheck(cudaMemcpy(THCStorage_(data)(state, THTensor_getStoragePtr(self)) + self->storage_offset(), src, len, cudaMemcpyHostToDevice));
}

static void THCTensor_(copyArray2d)(THCState *state, THCTensor *self, scalar_t *src, int m, int n)
{
  int64_t size[2] = { m, n };
  int64_t stride[2] = { 1, m };
  THCTensor_(resizeNd)(state, self, 2, size, stride);
  size_t len = m * n * sizeof(scalar_t);
  THCudaCheck(cudaMemcpy(THCStorage_(data)(state, THTensor_getStoragePtr(self)) + self->storage_offset(), src, len, cudaMemcpyHostToDevice));
}

static void THCTensor_(copyTensor2d)(THCState *state, scalar_t *dst, THCTensor *self)
{
  THAssert(self->dim() == 2);
  size_t len = THCTensor_(nElement)(state, self)*sizeof(scalar_t);
  THCTensor *temp = THCTensor_(newTranspose)(state, self, 0, 1);
  THCTensor *selfc = THCTensor_(newContiguous)(state, temp);
  THCudaCheck(cudaMemcpy(dst, THCStorage_(data)(state, THTensor_getStoragePtr(selfc)) + selfc->storage_offset(), len, cudaMemcpyDeviceToHost));
  THCTensor_(free)(state, temp);
  THCTensor_(free)(state, selfc);
}

#endif // USE_MAGMA

static THCTensor* THCTensor_(newColumnMajor)(THCState *state, THCTensor *self, THCTensor *src)
{
  THAssert(src->dim() == 2);
  if (self == src && self->stride(0) == 1 && self->stride(1) == self->size(0))
  {
    THCTensor_(retain)(state, self);
    return self;
  }

  if (self == src)
    self = THCTensor_(new)(state);
  else
    THCTensor_(retain)(state, self);

  int64_t size[2] = { src->size(0), src->size(1) };
  int64_t stride[2] = { 1, src->size(0) };

  THCTensor_(resizeNd)(state, self, 2, size, stride);
  THCTensor_(copy)(state, self, src);
  return self;
}

void THCTensor_(gels)(THCState *state, THCTensor *rb_, THCTensor *ra_, THCTensor *b_, THCTensor *a_)
{
#ifdef USE_MAGMA
  THArgCheck(!a_->is_empty() && a_->dim() == 2, 1, "A should be (non-empty) 2 dimensional");
  THArgCheck(!b_->is_empty() && b_->dim() == 2, 1, "b should be (non-empty) 2 dimensional");
  AT_CHECK(a_->size(0) == b_->size(0), "Expected A and b to have same size "
      "at dim 0, but A has ", a_->size(0), " rows and B has ", b_->size(0), " rows");
  THArgCheck(a_->size(0) >= a_->size(1), 2, "Expected A with shape (m x n) to have "
      "m >= n. The case for m < n is not implemented yet.");

  THCTensor *a = THCTensor_(newColumnMajor)(state, ra_, a_);
  THCTensor *b = THCTensor_(newColumnMajor)(state, rb_, b_);
  scalar_t *a_data = THCTensor_(data)(state, a);
  scalar_t *b_data = THCTensor_(data)(state, b);

  int64_t m = a->size(0);
  int64_t n = a->size(1);
  int64_t nrhs = b->size(1);
  scalar_t wkopt;

  int info;
#if defined(THC_REAL_IS_FLOAT)
  magma_sgels_gpu(MagmaNoTrans, m, n, nrhs, a_data, m, b_data, m, &wkopt, -1, &info);
#else
  magma_dgels_gpu(MagmaNoTrans, m, n, nrhs, a_data, m, b_data, m, &wkopt, -1, &info);
#endif

  scalar_t *hwork = th_magma_malloc_pinned<scalar_t>((size_t)wkopt);

#if defined(THC_REAL_IS_FLOAT)
  magma_sgels_gpu(MagmaNoTrans, m, n, nrhs, a_data, m, b_data, m, hwork, (int)wkopt, &info);
#else
  magma_dgels_gpu(MagmaNoTrans, m, n, nrhs, a_data, m, b_data, m, hwork, (int)wkopt, &info);
#endif

  magma_free_pinned(hwork);

  if (info != 0)
    THError("MAGMA gels : Argument %d : illegal value", -info);

  THCTensor_(freeCopyTo)(state, a, ra_);
  THCTensor_(freeCopyTo)(state, b, rb_);
#else
  THError(NoMagma(gels));
#endif
}

void THCTensor_(syev)(THCState *state, THCTensor *re_, THCTensor *rv_, THCTensor *a, const char *jobzs, const char *uplos)
{
#ifdef USE_MAGMA
  int64_t n = THTensor_sizeLegacyNoScalars(a, 0);
  int64_t lda = n;

  magma_uplo_t uplo = uplos[0] == 'U' ?  MagmaUpper : MagmaLower;
  magma_vec_t jobz = jobzs[0] == 'N' ? MagmaNoVec : MagmaVec;

  THCTensor *input = THCTensor_(newColumnMajor)(state, rv_, a);
  scalar_t *input_data = THCTensor_(data)(state, input);

  if (n > 0) {
    // eigen values and workspace
    scalar_t *w = th_magma_malloc_pinned<scalar_t>(n);
    scalar_t *wA = th_magma_malloc_pinned<scalar_t>(lda * n);

    // compute optimal size of work array
    int info;
    scalar_t lwork;
    int liwork;

#if defined(THC_REAL_IS_FLOAT)
    magma_ssyevd_gpu(jobz, uplo, n, input_data, lda, w, wA, n, &lwork, -1, &liwork, -1, &info);
#else
    magma_dsyevd_gpu(jobz, uplo, n, input_data, lda, w, wA, n, &lwork, -1, &liwork, -1, &info);
#endif

    scalar_t *work = th_magma_malloc_pinned<scalar_t>((size_t)lwork);
    int *iwork = th_magma_malloc_pinned<int>(liwork);

  // compute eigenvalues and, optionally, eigenvectors
#if defined(THC_REAL_IS_FLOAT)
    magma_ssyevd_gpu(jobz, uplo, n, input_data, lda, w, wA, n, work, (int) lwork, iwork, liwork, &info);
#else
    magma_dsyevd_gpu(jobz, uplo, n, input_data, lda, w, wA, n, work, (int) lwork, iwork, liwork, &info);
#endif

    // copy eigen values from w to re_
    if (info == 0)
      THCTensor_(copyArray1d)(state, re_, w, n);

    magma_free_pinned(iwork);
    magma_free_pinned(work);
    magma_free_pinned(wA);
    magma_free_pinned(w);

    // check error value
    if (info > 0)
      THError("MAGMA syev : Failed to converge. %d off-diagonal elements of an didn't converge to zero", info);
    else if (info < 0)
      THError("MAGMA syev : Argument %d : illegal value", -info);
  }
  if (jobzs[0] == 'N') {
    // If eigenvector is not needed, fill the result with zeros.
    THCTensor_(zero)(state, rv_);
    THCTensor_(free)(state, input);
  } else {
    THCTensor_(freeCopyTo)(state, input, rv_);
  }
#else
  THError(NoMagma(syev));
#endif
}

void THCTensor_(geev)(THCState *state, THCTensor *re_, THCTensor *rv_, THCTensor *a_, const char *jobvrs)
{
#ifdef USE_MAGMA
  THArgCheck(a_->dim() == 2, 3, "A should be 2 dimensional");
  THArgCheck(a_->size(0) == a_->size(1), 3, "A should be square");

  magma_vec_t jobvr = jobvrs[0] == 'N' ? MagmaNoVec : MagmaVec;
  int64_t n = a_->size(0);

  scalar_t *a_data = th_magma_malloc_pinned<scalar_t>(n * n);
  THCTensor_(copyTensor2d)(state, a_data, a_);

  scalar_t *wr = th_magma_malloc_pinned<scalar_t>(n);
  scalar_t *wi = th_magma_malloc_pinned<scalar_t>(n);

  scalar_t *vr_data = NULL;
  int64_t ldvr = 1;
  if (jobvr == MagmaVec)
  {
    vr_data = th_magma_malloc_pinned<scalar_t>(n * n);
    ldvr = n;
  }

  scalar_t *work_data = nullptr;

  if (n > 0) {
    int info;
    scalar_t wkopt;
#if defined(THC_REAL_IS_FLOAT)
    magma_sgeev(MagmaNoVec, jobvr, n, a_data, n, wr, wi, NULL, 1, vr_data, ldvr, &wkopt, -1, &info);
#else
    magma_dgeev(MagmaNoVec, jobvr, n, a_data, n, wr, wi, NULL, 1, vr_data, ldvr, &wkopt, -1, &info);
#endif

    int lwork = (int) wkopt;
    work_data = th_magma_malloc_pinned<scalar_t>(lwork);

#if defined(THC_REAL_IS_FLOAT)
    magma_sgeev(MagmaNoVec, jobvr, n, a_data, n, wr, wi, NULL, 1, vr_data, ldvr, work_data, lwork, &info);
#else
    magma_dgeev(MagmaNoVec, jobvr, n, a_data, n, wr, wi, NULL, 1, vr_data, ldvr, work_data, lwork, &info);
#endif

    if (info > 0)
      THError("MAGMA geev : Failed to converge. %d off-diagonal elements of an didn't converge to zero", info);
    else if (info < 0)
      THError("MAGMA geev : Argument %d : illegal value", -info);
  }

  {
    THCTensor_(resize2d)(state, re_, 2, n);
    THCTensor *re = THCTensor_(newContiguous)(state, re_);
    if (n > 0) {
      THCudaCheck(cudaMemcpy(THCStorage_(data)(state, THTensor_getStoragePtr(re)) + re->storage_offset(), wr, n*sizeof(scalar_t), cudaMemcpyHostToDevice));
      THCudaCheck(cudaMemcpy(THCStorage_(data)(state, THTensor_getStoragePtr(re)) + re->storage_offset() + n, wi, n*sizeof(scalar_t), cudaMemcpyHostToDevice));
    }
    THCTensor_(freeCopyTo)(state, re, re_);
    THCTensor_(transpose)(state, re_, NULL, 0, 1);
  }

  if (jobvr == MagmaVec)
    THCTensor_(copyArray2d)(state, rv_, vr_data, n, n);

  magma_free_pinned(work_data);
  magma_free_pinned(vr_data);
  magma_free_pinned(wi);
  magma_free_pinned(wr);
  magma_free_pinned(a_data);

#else
  THError(NoMagma(geev));
#endif
}

void THCTensor_(gesdd)(THCState *state, THCTensor *ru_, THCTensor *rs_, THCTensor *rv_, THCTensor *a,
                       const char *some, const char* compute_uv)
{
#ifdef USE_MAGMA
  THCTensor *ra_ = THCTensor_(new)(state);
  THCTensor_(gesdd2)(state, ru_, rs_, rv_,  ra_, a, some, compute_uv);
  THCTensor_(free)(state, ra_);
#else
  THError(NoMagma(gesdd));
#endif
}

void THCTensor_(gesdd2)(THCState *state, THCTensor *ru_, THCTensor *rs_, THCTensor *rv_, THCTensor *ra_, THCTensor *a,
                        const char *some, const char* compute_uv)
{
#ifdef USE_MAGMA
  THArgCheck(!a->is_empty() && a->dim() == 2, 2, "A should be non-empty 2 dimensional");

  char jobus = compute_uv[0] == 'N' ? 'N' : some[0];
  magma_vec_t jobz = jobus == 'A' ? MagmaAllVec : jobus == 'S' ? MagmaSomeVec : jobus == 'O' ? MagmaOverwriteVec : MagmaNoVec;

  int iunused[1];
  int64_t m = a->size(0);
  int64_t n = a->size(1);
  int64_t k = m < n ? m : n;
  int64_t j = (jobz == MagmaAllVec) ? m : k;
  int64_t jv = (jobz == MagmaAllVec) ? n : k;

  scalar_t *a_data = th_magma_malloc_pinned<scalar_t>(m * n);
  THCTensor_(copyTensor2d)(state, a_data, a);

  scalar_t *rs_data = th_magma_malloc_pinned<scalar_t>(k);
  scalar_t *ru_data = NULL;
  scalar_t *rv_data = NULL;
  if (jobz != MagmaNoVec) {
    ru_data = th_magma_malloc_pinned<scalar_t>(m * j);
    rv_data = th_magma_malloc_pinned<scalar_t>(n * n);
  }

  scalar_t wkopt;
  int info;

#if defined(THC_REAL_IS_FLOAT)
  magma_sgesdd(jobz, m, n, a_data, m, rs_data, ru_data, m, rv_data, n, &wkopt, -1, iunused, &info);
#else
  magma_dgesdd(jobz, m, n, a_data, m, rs_data, ru_data, m, rv_data, n, &wkopt, -1, iunused, &info);
#endif

  int lwork = (int) wkopt;
  scalar_t *work_data = th_magma_malloc_pinned<scalar_t>(lwork);
  int *iwork = th_magma_malloc_pinned<int>(8 * k);

#if defined(THC_REAL_IS_FLOAT)
  magma_sgesdd(jobz, m, n, a_data, m, rs_data, ru_data, m, rv_data, n, work_data, lwork, iwork, &info);
#else
  magma_dgesdd(jobz, m, n, a_data, m, rs_data, ru_data, m, rv_data, n, work_data, lwork, iwork, &info);
#endif

  if (info > 0)
    THError("MAGMA gesdd : the updating process of SBDSDC did not converge (error: %d)", info);
  else if (info < 0)
    THError("MAGMA gesdd : Argument %d : illegal value", -info);

  THCTensor_(copyArray1d)(state, rs_, rs_data, k);
  THCTensor_(copyArray2d)(state, ra_, a_data, m, n);
  if (jobz != MagmaNoVec) {
    THCTensor_(copyArray2d)(state, rv_, rv_data, n, n);
    THCTensor_(transpose)(state, rv_, NULL, 0, 1);
    if (jobz != MagmaAllVec)
      THCTensor_(narrow)(state, rv_, rv_, 1, 0, jv);
    THCTensor_(copyArray2d)(state, ru_, ru_data, m, j);
    magma_free_pinned(rv_data);
    magma_free_pinned(ru_data);
  } else {
    THCTensor_(resize2d)(state, rv_, n, n);
    THCTensor_(zero)(state, rv_);
    THCTensor_(resize2d)(state, ru_, m, m);
    THCTensor_(zero)(state, ru_);
  }

  magma_free_pinned(work_data);
  magma_free_pinned(iwork);
  magma_free_pinned(rs_data);
  magma_free_pinned(a_data);
#else
  THError(NoMagma(gesdd2));
#endif
}

void THCTensor_(getri)(THCState *state, THCTensor *ra_, THCTensor *a)
{
  THArgCheck(!a->is_empty() && a->dim() == 2, 2, "A should be non-empty 2 dimensional");
  THArgCheck(a->size(0) == a->size(1), 2, "A should be square");

#ifdef USE_MAGMA
  int info;
  int64_t n = a->size(0);
  int lwork = n * magma_get_sgetri_nb(n);

  THCTensor *input = THCTensor_(newColumnMajor)(state, ra_, a);
  scalar_t *input_data = THCTensor_(data)(state, input);

  int *ipiv = th_magma_malloc_pinned<int>(n);

  THCTensor *work = THCTensor_(newWithSize1d)(state, lwork);
  scalar_t *work_data = THCTensor_(data)(state, work);

  // Run LU
#if defined(THC_REAL_IS_FLOAT)
  magma_sgetrf_gpu(n, n, input_data, n, ipiv, &info);
#else
  magma_dgetrf_gpu(n, n, input_data, n, ipiv, &info);
#endif

  if (info > 0)
    THError("MAGMA getrf : U(%d,%d) is 0, U is singular", info, info);
  else if (info < 0)
    THError("MAGMA getrf : Argument %d : illegal value", -info);

  // Inverse
#if defined(THC_REAL_IS_FLOAT)
  magma_sgetri_gpu(n, input_data, n, ipiv, work_data, lwork, &info);
#else
  magma_dgetri_gpu(n, input_data, n, ipiv, work_data, lwork, &info);
#endif

  if (info > 0)
    THError("MAGMA getri : U(%d,%d) is 0, U is singular", info, info);
  else if (info < 0)
    THError("MAGMA getri : Argument %d : illegal value", -info);

  THCTensor_(free)(state, work);
  magma_free_pinned(ipiv);
  THCTensor_(freeCopyTo)(state, input, ra_);
#else
  int64_t n = a->size(0);

  // input
  THCTensor *input = THCTensor_(newColumnMajor)(state, a, a);
  THCTensor_(resizeNd)(state, ra_, 2, THTensor_getSizePtr(input), THTensor_getStridePtr(input));

  scalar_t *matrices1[1] = { THCTensor_(data)(state, input) };
  scalar_t *matrices2[1] = { THCTensor_(data)(state, ra_) };

  // Copy pointers to device.
  auto d_matrices1 = static_cast<scalar_t**>(THCudaMalloc(state, sizeof(scalar_t*)));
  auto d_matrices2 = static_cast<scalar_t**>(THCudaMalloc(state, sizeof(scalar_t*)));

  THCudaCheck(cudaMemcpyAsync(d_matrices1, matrices1, sizeof(scalar_t*),
                              cudaMemcpyHostToDevice, THCState_getCurrentStream(state)));
  THCudaCheck(cudaMemcpyAsync(d_matrices2, matrices2, sizeof(scalar_t*),
                              cudaMemcpyHostToDevice, THCState_getCurrentStream(state)));
  int info;
  auto info_gpu = static_cast<int*>(THCudaMalloc(state, sizeof(int)));

  auto ipiv_gpu = static_cast<int*>(THCudaMalloc(state, n * sizeof(int)));

  // Run LU
#if defined(THC_REAL_IS_FLOAT)
  THCudaBlas_Sgetrf(state, n, d_matrices1, n, ipiv_gpu, info_gpu, 1);
#else
  THCudaBlas_Dgetrf(state, n, d_matrices1, n, ipiv_gpu, info_gpu, 1);
#endif

  THCudaCheck(cudaMemcpy(&info, info_gpu, sizeof(int), cudaMemcpyDeviceToHost));

  if (info > 0)
    THError("CUBLAS getrf : U(%d,%d) is 0, U is singular", info, info);
  else if (info < 0)
    THError("CUBLAS getrf : Argument %d : illegal value", -info);

  // Inverse
#if defined(THC_REAL_IS_FLOAT)
  THCudaBlas_Sgetri(state, n, (const scalar_t**)d_matrices1, n, ipiv_gpu, d_matrices2, n, info_gpu, 1);
#else
  THCudaBlas_Dgetri(state, n, (const scalar_t**)d_matrices1, n, ipiv_gpu, d_matrices2, n, info_gpu, 1);
#endif

  THCudaCheck(cudaMemcpy(&info, info_gpu, sizeof(int), cudaMemcpyDeviceToHost));

  if (info > 0)
    THError("CUBLAS getri : U(%d,%d) is 0, U is singular", info, info);
  else if (info < 0)
    THError("CUBLAS getri : Argument %d : illegal value", -info);

  THCudaFree(state, ipiv_gpu);
  THCudaFree(state, info_gpu);

  THCudaFree(state, d_matrices1);
  THCudaFree(state, d_matrices2);

  THCTensor_(free)(state, input);
#endif
}

__global__ void THCTensor_(copyUpperSymmetric)(scalar_t *input, int n, int len)
{
  for (int idx = threadIdx.x + blockIdx.x * blockDim.x; idx < len; idx += 65535) {
    const int r = idx % n;
    const int c = idx / n;
    if (r > c) {
      input[idx] = input[r*n + c];
    }
  }
}

__global__ void THCTensor_(copyLowerSymmetric)(scalar_t *input, int n, int len)
{
  for (int idx = threadIdx.x + blockIdx.x * blockDim.x; idx < len; idx += 65535) {
    const int r = idx % n;
    const int c = idx / n;
    if (r < c) {
      input[idx] = input[r*n + c];
    }
  }
}

void THCTensor_(potri)(THCState *state, THCTensor *ra_, THCTensor *a, const char *uplo)
{
#ifdef USE_MAGMA
  THArgCheck(!a->is_empty() && a->dim() == 2, 2, "A should be non-empty 2 dimensional");
  THArgCheck(a->size(0) == a->size(1), 2, "A should be square");

  int64_t n = a->size(0);
  magma_uplo_t ul = uplo[0] == 'U' ?  MagmaUpper : MagmaLower;

  THCTensor *input = THCTensor_(newColumnMajor)(state, ra_, a);
  scalar_t *input_data = THCTensor_(data)(state, input);

  int info;
#if defined(THC_REAL_IS_FLOAT)
  magma_spotri_gpu(ul, n, input_data, n, &info);
#else
  magma_dpotri_gpu(ul, n, input_data, n, &info);
#endif

  if (info > 0)
    THError("MAGMA potri : A(%d,%d) is 0, A cannot be factorized", info, info);
  else if (info < 0)
    THError("MAGMA potri : Argument %d : illegal value", -info);

  cudaStream_t stream = THCState_getCurrentStream(state);
  const int len = n*n;
  dim3 blocks(std::min(DIVUP(len, 128), 65535));
  dim3 threads(128);
  if (uplo[0] == 'U') {
    THCTensor_(copyUpperSymmetric)<<<blocks, threads, 0, stream>>>(input_data, n, len);
  } else {
    THCTensor_(copyLowerSymmetric)<<<blocks, threads, 0, stream>>>(input_data, n, len);
  }

  THCTensor_(freeCopyTo)(state, input, ra_);
#else
  THError(NoMagma(potri));
#endif
}

void THCTensor_(geqrf)(THCState *state, THCTensor *ra_, THCTensor *rtau_, THCTensor *a_)
{
#ifdef USE_MAGMA
  THArgCheck(!a_->is_empty() && a_->dim() == 2, 2, "A should be non-empty 2 dimensional");

  THCTensor *a = THCTensor_(newColumnMajor)(state, ra_, a_);
  int64_t m = a->size(0);
  int64_t n = a->size(1);
  int64_t k = (m < n ? m : n);

#if defined(THC_REAL_IS_FLOAT)
  int64_t nb = magma_get_sgeqrf_nb(m, n);
#else
  int64_t nb = magma_get_dgeqrf_nb(m, n);
#endif

  scalar_t *rtau_data = th_magma_malloc_pinned<scalar_t>(k);
  scalar_t *a_data = THCTensor_(data)(state, a);

  int info;
#if defined(THC_REAL_IS_FLOAT)
  magma_sgeqrf2_gpu(m, n, a_data, m, rtau_data, &info);
#else
  magma_dgeqrf2_gpu(m, n, a_data, m, rtau_data, &info);
#endif

  if (info != 0)
    THError("MAGMA geqrf2 : Argument %d : illegal value.", -info);

  THCTensor_(freeCopyTo)(state, a, ra_);
  THCTensor_(copyArray1d)(state, rtau_, rtau_data, k);
  magma_free_pinned(rtau_data);
#else
  THError(NoMagma(geqrf));
#endif
}

void THCTensor_(qr)(THCState *state, THCTensor *rq_, THCTensor *rr_, THCTensor *a_)
{
#ifdef USE_MAGMA
  THArgCheck(!a_->is_empty() && a_->dim() == 2, 2, "A should be non-empty 2 dimensional");

  THCTensor *a = THCTensor_(newColumnMajor)(state, rr_, a_);
  int64_t m = a->size(0);
  int64_t n = a->size(1);
  int64_t k = (m < n ? m : n);

#if defined(THC_REAL_IS_FLOAT)
  int64_t nb = magma_get_sgeqrf_nb(m, n);
#else
  int64_t nb = magma_get_dgeqrf_nb(m, n);
#endif

  scalar_t *a_data = THCTensor_(data)(state, a);
  scalar_t *tau_data = th_magma_malloc_pinned<scalar_t>(k);
  THCTensor *work = THCTensor_(newWithSize1d)(state, (2*k + magma_roundup(n, 32))*nb);
  scalar_t *work_data = THCTensor_(data)(state, work);

  int info;
  // We need to call two different versions of ?geqrf:
  //   ?geqrf_gpu allows fast computation of Q via ?orqrf_gpu, but doesn't give
  //     R properly. Note that the MAGMA documentation for this method is wrong.
  //     http://icl.cs.utk.edu/magma/forum/viewtopic.php?f=2&t=1015&p=2800&hilit=geqrf_gpu#p2800
  //   ?geqrf2_gpu gives correct R, but doesn't allow computation of Q via ?orqrf_gpu
#if defined(THC_REAL_IS_FLOAT)
  magma_sgeqrf2_gpu(m, n, a_data, m, tau_data, &info);
#else
  magma_dgeqrf2_gpu(m, n, a_data, m, tau_data, &info);
#endif

  if (info != 0)
    THError("MAGMA geqrf2 : Argument %d : illegal value.", -info);

  THCTensor_(narrow)(state, a, a, 0, 0, k);
  THCTensor_(triu)(state, rr_, a, 0);
  THCTensor_(free)(state, a);

  a = THCTensor_(newColumnMajor)(state, rq_, a_);
  a_data = THCTensor_(data)(state, a);

#if defined(THC_REAL_IS_FLOAT)
  magma_sgeqrf_gpu(m, n, a_data, m, tau_data, work_data, &info);
#else
  magma_dgeqrf_gpu(m, n, a_data, m, tau_data, work_data, &info);
#endif

  if (info != 0)
    THError("MAGMA geqrf : Argument %d : illegal value.", -info);

  THCTensor *q = THCTensor_(newColumnMajor)(state, rq_, a);
  scalar_t *q_data = THCTensor_(data)(state, q);

#if defined(THC_REAL_IS_FLOAT)
  magma_sorgqr_gpu(m, k, k, q_data, m, tau_data, work_data, nb, &info);
#else
  magma_dorgqr_gpu(m, k, k, q_data, m, tau_data, work_data, nb, &info);
#endif

  if (info != 0)
    THError("MAGMA orgqr : Argument %d : illegal value.", -info);

  THCTensor_(free)(state, a);
  THCTensor_(free)(state, work);
  magma_free_pinned(tau_data);

  THCTensor_(narrow)(state, q, q, 1, 0, k);
  THCTensor_(freeCopyTo)(state, q, rq_);
#else
  THError(NoMagma(qr));
#endif
}

#endif

#endif


#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THC/generic/THCTensorMasked.cu"
#else


void THCTensor_(maskedFill)(THCState* state,
                            THCTensor *tensor, THCudaByteTensor *mask, scalar_t value)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, tensor, mask));
  THArgCheck(THCTensor_(nElement)(state, tensor) ==
             THCudaByteTensor_nElement(state, mask),
             2, "sizes do not match");

  if (!THC_pointwiseApply2<scalar_t, uint8_t>(state, tensor, mask,
                                          TensorMaskedFillOp<scalar_t, unsigned char>(value))) {
    THArgCheck(false, 2, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(cudaGetLastError());
}

void THCTensor_(maskedFillByte)(THCState* state,
                                THCTensor *tensor, THByteTensor *mask, scalar_t value)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, tensor));
  THCudaByteTensor* maskCuda = THCudaByteTensor_newWithSize(state, mask->sizes(), {});
  THCTensor_(copy)(state, maskCuda, mask);
  THCTensor_(maskedFill)(state, tensor, maskCuda, value);
  THCudaByteTensor_free(state, maskCuda);
}

void THCTensor_(maskedCopy)(THCState* state,
                            THCTensor *tensor, THCudaByteTensor *mask, THCTensor *src)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, tensor, src, mask));
  ptrdiff_t maskSize = THCudaByteTensor_nElement(state, mask);
  ptrdiff_t tensorSize = THCTensor_(nElement)(state, tensor);
  ptrdiff_t srcSize = THCTensor_(nElement)(state, src);

  // `mask` and `tensor` must have the same number of elements
  THArgCheck(maskSize == tensorSize, 2,
             "mask and tensor must have the same number of elements");

  // Determine our output size
  ptrdiff_t totalElements = THCudaByteTensor_sumall(state, mask);

  // The number of `1` elements present in the mask must be <= the
  // number of elements available in `src`
  if (totalElements > srcSize) {
    THArgCheck(false, 2, "source nElements must be == mask `1` elements");
  }

  // FIXME: there appears to be a bug in Thrust (CUDA 7.0) for mixed
  // iterator prefix sums? Convert `mask` to the same datatype as what
  // we're accumulating the prefix sum in (int64_t) to get around it
  THCudaLongTensor* maskLong = THCudaLongTensor_new(state);
  at::IntArrayRef maskSizes = mask->sizes();
  THCudaLongTensor_resize(state, maskLong, maskSizes, {});
  THCTensor_(copy)(state, maskLong, mask);

  // Use a prefix sum to determine the output locations of the masked elements
  THCudaLongTensor* maskPrefixSum = THCudaLongTensor_new(state);
  THCudaLongTensor_resize(state, maskPrefixSum, maskSizes, {});

  THCThrustAllocator thrustAlloc(state);
  thrust::device_ptr<int64_t>
    maskData(THCudaLongTensor_data(state, maskLong));
  thrust::device_ptr<int64_t>
    maskPrefixSumData(THCudaLongTensor_data(state, maskPrefixSum));

  thrust::exclusive_scan(
#if CUDA_VERSION >= 7000 || defined __HIP_PLATFORM_HCC__
    thrust::cuda::par(thrustAlloc).on(THCState_getCurrentStream(state)),
#endif
    maskData,
    maskData + THCudaLongTensor_nElement(state, maskLong),
    maskPrefixSumData);

  // We are getting elements from `src` based on an offset from
  // `maskPrefixSum`, so that should be made contiguous too
  THCTensor* contigSrc = THCTensor_(newContiguous)(state, src);

  // update `tensor` where `mask` == 1 but pull from `src` at
  // maskPrefixSum
  bool status = THC_pointwiseApply3<scalar_t, uint8_t, int64_t>(
    state, tensor, mask, maskPrefixSum,
    TensorMaskedCopyOp<scalar_t, unsigned char, int64_t>(
      THCTensor_(data)(state, contigSrc)));

  THCTensor_(free)(state, contigSrc);
  THCudaLongTensor_free(state, maskLong);
  THCudaLongTensor_free(state, maskPrefixSum);

  THArgCheck(status, 2, CUTORCH_DIM_WARNING);
  THCudaCheck(cudaGetLastError());
}

void THCTensor_(maskedCopyByte)(THCState* state,
                                THCTensor *tensor, THByteTensor *mask, THCTensor *src) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, tensor, src));
  THCudaByteTensor* maskCuda = THCudaByteTensor_newWithSize(state, mask->sizes(), {});
  THCTensor_(copy)(state, maskCuda, mask);
  THCTensor_(maskedCopy)(state, tensor, maskCuda, src);
  THCudaByteTensor_free(state, maskCuda);
}

void THCTensor_(maskedSelect)(THCState* state,
                              THCTensor* tensor, THCTensor* src, THCudaByteTensor* mask) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, tensor, src, mask));
  THArgCheck(THCudaByteTensor_nElement(state, mask) ==
             THCTensor_(nElement)(state, src),
             2, "sizes do not match");

  // Determine our output size
  ptrdiff_t totalElements = THCudaByteTensor_sumall(state, mask);
  THCTensor* tensorContig = THCTensor_(newContiguous)(state, tensor);

  THCTensor_(resize1d)(state, tensorContig, totalElements);
  if (tensor != tensorContig) {
    THCTensor_(resize1d)(state, tensor, totalElements);
  }

  // FIXME: there appears to be a bug in Thrust (CUDA 7.0) for mixed
  // iterator prefix sums? Convert `mask` to the same datatype as what
  // we're accumulating the prefix sum in (int64_t) to get around it
  THCudaLongTensor* maskLong = THCudaLongTensor_new(state);
  at::IntArrayRef maskSizes = mask->sizes();
  THCudaLongTensor_resize(state, maskLong, maskSizes, {});
  THCTensor_(copy)(state, maskLong, mask);

  // Use a prefix sum to determine the output locations of the masked elements
  THCudaLongTensor* maskPrefixSum = THCudaLongTensor_new(state);
  THCudaLongTensor_resize(state, maskPrefixSum, maskSizes, {});

  THCThrustAllocator thrustAlloc(state);
  thrust::device_ptr<int64_t>
    maskData(THCudaLongTensor_data(state, maskLong));
  thrust::device_ptr<int64_t>
    maskPrefixSumData(THCudaLongTensor_data(state, maskPrefixSum));

  thrust::exclusive_scan(
#if CUDA_VERSION >= 7000 || defined __HIP_PLATFORM_HCC__
    thrust::cuda::par(thrustAlloc).on(THCState_getCurrentStream(state)),
#endif
    maskData,
    maskData + THCudaLongTensor_nElement(state, maskLong),
    maskPrefixSumData);

  // Then copy over the masked elements at their desired output index
  bool status = THC_pointwiseApply3<uint8_t, int64_t, scalar_t>(
    state, mask, maskPrefixSum,
    src, TensorMaskedSelectOp<scalar_t, unsigned char, int64_t>(
      THCTensor_(data)(state, tensor)));

  THCudaLongTensor_free(state, maskLong);
  THCudaLongTensor_free(state, maskPrefixSum);

  if (tensor != tensorContig) {
    THCTensor_(freeCopyTo)(state, tensorContig, tensor);
  } else {
    THCTensor_(free)(state, tensorContig);
  }

  THArgCheck(status, 2, CUTORCH_DIM_WARNING);
  THCudaCheck(cudaGetLastError());
}

// FIXME: remove now that we have THCudaByteTensor?
void THCTensor_(maskedSelectByte)(THCState* state,
                                  THCTensor *tensor, THCTensor *src, THByteTensor *mask)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, tensor, src));
  THCudaByteTensor* maskCuda = THCudaByteTensor_newWithSize(state, mask->sizes(), {});
  THCTensor_(copy)(state, maskCuda, mask);
  THCTensor_(maskedSelect)(state, tensor, src, maskCuda);
  THCudaByteTensor_free(state, maskCuda);
}

#endif

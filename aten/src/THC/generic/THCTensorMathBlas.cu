
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THC/generic/THCTensorMathBlas.cu"
#else

#include "ATen/cuda/CUDAContext.h"

#define ERROR_ONLY_FP_TYPES(func) \
  THError("%s for CUDA tensors only supports floating-point types. Try converting the tensors with .float()", func);

accreal THCTensor_(dot)(THCState *state, THCTensor *self, THCTensor *src)
{
#if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE) || defined(THC_REAL_IS_HALF)
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self, src));
  THArgCheck(THCTensor_(nElement)(state, self) ==
             THCTensor_(nElement)(state, src), 2, "sizes do not match");

  self = THCTensor_(newContiguous)(state, self);
  src = THCTensor_(newContiguous)(state, src);

#ifdef THC_REAL_IS_FLOAT
  accreal result = THCudaBlas_Sdot(state,
                                THCTensor_(nElement)(state, self),
                                THCTensor_(data)(state, self), 1,
                                THCTensor_(data)(state, src), 1);
#elif defined(THC_REAL_IS_DOUBLE)
  accreal result = THCudaBlas_Ddot(state,
                                THCTensor_(nElement)(state, self),
                                THCTensor_(data)(state, self), 1,
                                THCTensor_(data)(state, src), 1);
#elif defined(THC_REAL_IS_HALF)
  accreal result = THCudaBlas_Hdot(state,
                                THCTensor_(nElement)(state, self),
                                THCTensor_(data)(state, self), 1,
                                THCTensor_(data)(state, src), 1);
#endif

  THCTensor_(free)(state, src);
  THCTensor_(free)(state, self);
  return result;

#else
  ERROR_ONLY_FP_TYPES("dot");
  return ScalarConvert<int, accreal>::to(0);
#endif
}

void THCTensor_(addmv)(THCState *state, THCTensor *r_, scalar_t beta, THCTensor *t, scalar_t alpha, THCTensor *mat, THCTensor *vec)
{
#if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE) || defined(THC_REAL_IS_HALF)
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 4, r_, t, mat, vec));
  if( (mat->dim() != 2) || (THTensor_nDimensionLegacyNoScalars(vec) != 1) )
    THError("2D tensor and 1D tensor expected, got %dD, %dD tensors",
       mat->dim(), THTensor_nDimensionLegacyNoScalars(vec));


  auto vec_size = THTensor_sizeLegacyNoScalars(vec, 0);
  auto vec_stride = THTensor_strideLegacyNoScalars(vec, 0);

  if( mat->size(1) != THTensor_sizeLegacyNoScalars(vec, 0) )
    THError("size mismatch");

  if(t->dim() != 1)
    THError("size mismatch");

  if(THTensor_sizeLegacyNoScalars(t, 0) != mat->size(0))
    THError("size mismatch");

#if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE)
  if(r_ != t)
  {
    THCTensor_(resizeAs)(state, r_, t);
    THCTensor_(copy)(state, r_, t);
  }

  auto r_stride = THTensor_strideLegacyNoScalars(r_, 0);

  if(mat->stride(0) == 1)
  {
#ifdef THC_REAL_IS_FLOAT
    THCudaBlas_Sgemv(state, 'n', mat->size(0), mat->size(1),
                    alpha, THCTensor_(data)(state, mat), mat->stride(1),
                    THCTensor_(data)(state, vec), vec_stride,
                    beta, THCTensor_(data)(state, r_), r_stride);
#elif defined(THC_REAL_IS_DOUBLE)
    THCudaBlas_Dgemv(state, 'n', mat->size(0), mat->size(1),
                    alpha, THCTensor_(data)(state, mat), mat->stride(1),
                    THCTensor_(data)(state, vec), vec_stride,
                    beta, THCTensor_(data)(state, r_), r_stride);
#endif
  }
  else if(mat->stride(1) == 1)
  {
#ifdef THC_REAL_IS_FLOAT
    THCudaBlas_Sgemv(state, 't',  mat->size(1), mat->size(0),
                    alpha, THCTensor_(data)(state, mat), mat->stride(0),
                    THCTensor_(data)(state, vec), vec_stride,
                    beta, THCTensor_(data)(state, r_), r_stride);
#elif defined(THC_REAL_IS_DOUBLE)
    THCudaBlas_Dgemv(state, 't',  mat->size(1), mat->size(0),
                     alpha, THCTensor_(data)(state, mat), mat->stride(0),
                     THCTensor_(data)(state, vec), vec_stride,
                     beta, THCTensor_(data)(state, r_), r_stride);
#endif
  }
  else
  {
    THCTensor *cmat = THCTensor_(newContiguous)(state, mat);

#ifdef THC_REAL_IS_FLOAT
    THCudaBlas_Sgemv(state, 't',  mat->size(1), mat->size(0),
                    alpha, THCTensor_(data)(state, cmat), cmat->stride(0),
                    THCTensor_(data)(state, vec), vec_stride,
                    beta, THCTensor_(data)(state, r_), r_stride);
#elif defined(THC_REAL_IS_DOUBLE)
    THCudaBlas_Dgemv(state, 't',  mat->size(1), mat->size(0),
                    alpha, THCTensor_(data)(state, cmat), cmat->stride(0),
                    THCTensor_(data)(state, vec), vec_stride,
                    beta, THCTensor_(data)(state, r_), r_stride);
#endif

    THCTensor_(free)(state, cmat);
  }

  // In cublasSgemv, cublasDgemv (x,0).mv(0) does not
  // handle beta, whereas cublasSgemm, cublasDgemm do for case where (x,0).mm(0,y).
  if (THTensor_sizeLegacyNoScalars(vec, 0) == 0 && mat->size(0) != 0) {
    if(THCNumerics<scalar_t>::eq(beta, ScalarConvert<int, scalar_t>::to(0))) {
      THCTensor_(zero)(state, r_);
    } else if(THCNumerics<scalar_t>::ne(beta, ScalarConvert<int, scalar_t>::to(1))) {
      THCTensor_(mul)(state, r_, r_, beta);
    }
  }

#elif defined(THC_REAL_IS_HALF)
    // Currently no Hgemv/SgemvEx in Cublas
    THCTensor *vecAsMatrix = THCTensor_(newWithTensor)(state, vec);
    THCTensor_(resize2d)(state, vecAsMatrix, vec_size, 1);

    THCTensor *tAsMatrix = THCTensor_(newWithTensor)(state, t);
    THCTensor_(resize2d)(state, tAsMatrix, THTensor_sizeLegacyNoScalars(tAsMatrix, 0), 1);

    THCTensor_(addmm)(state, r_, beta, tAsMatrix, alpha, mat, vecAsMatrix);

    // r_ will have answer as matrix, need to return a vector
    THCTensor_(resize1d)(state, r_, THTensor_sizeLegacyNoScalars(r_, 0));
    THCTensor_(free)(state, vecAsMatrix);
    THCTensor_(free)(state, tAsMatrix);
#endif
#else
  ERROR_ONLY_FP_TYPES("addmv");
#endif
}

void THCTensor_(addr)(THCState *state, THCTensor *r_, scalar_t beta, THCTensor *t, scalar_t alpha, THCTensor *vec1, THCTensor *vec2)
{
#if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE) || defined(THC_REAL_IS_HALF)
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 4, r_, t, vec1, vec2));
  if ( (THTensor_nDimensionLegacyNoScalars(vec1) != 1) || (THTensor_nDimensionLegacyNoScalars(vec2) != 1) ) {
    THError("1D tensors expected, got %dD, %dD tensors",
       THTensor_nDimensionLegacyNoScalars(vec1), THTensor_nDimensionLegacyNoScalars(vec2));
  }
  auto vec1_size = THTensor_sizeLegacyNoScalars(vec1, 0);
  auto vec2_size = THTensor_sizeLegacyNoScalars(vec2, 0);
  auto vec1_stride = THTensor_strideLegacyNoScalars(vec1, 0);
  auto vec2_stride = THTensor_strideLegacyNoScalars(vec2, 0);

  if (t->dim() != 2) {
    THError("size mismatch");
  }

  if ( (t->size(0) != vec1_size) || (t->size(1) != vec2_size) ) {
    THError("size mismatch");
  }

#if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE)
  if (r_ != t) {
    THCTensor_(resizeAs)(state, r_, t);
    THCTensor_(copy)(state, r_, t);
  }

  if(THCNumerics<scalar_t>::eq(beta, ScalarConvert<int, scalar_t>::to(0))) {
    THCTensor_(zero)(state, r_);
  } else if(THCNumerics<scalar_t>::ne(beta, ScalarConvert<int, scalar_t>::to(1))) {
    THCTensor_(mul)(state, r_, r_, beta);
  }

  if(r_->stride(0) == 1)
  {
#ifdef THC_REAL_IS_FLOAT
    THCudaBlas_Sger(state, vec1_size, vec2_size,
                   alpha, THCTensor_(data)(state, vec1), vec1_stride,
                   THCTensor_(data)(state, vec2), vec2_stride,
                   THCTensor_(data)(state, r_), r_->stride(1));
#elif defined(THC_REAL_IS_DOUBLE)
    THCudaBlas_Dger(state, vec1_size, vec2_size,
                   alpha, THCTensor_(data)(state, vec1), vec1_stride,
                   THCTensor_(data)(state, vec2), vec2_stride,
                   THCTensor_(data)(state, r_), r_->stride(1));
#endif
  }
  else if(r_->stride(1) == 1)
  {
#ifdef THC_REAL_IS_FLOAT
    THCudaBlas_Sger(state, vec2_size, vec1_size,
                   alpha, THCTensor_(data)(state, vec2), vec2_stride,
                   THCTensor_(data)(state, vec1), vec1_stride,
                   THCTensor_(data)(state, r_), r_->stride(0));
#elif defined(THC_REAL_IS_DOUBLE)
    THCudaBlas_Dger(state, vec2_size, vec1_size,
                   alpha, THCTensor_(data)(state, vec2), vec2_stride,
                   THCTensor_(data)(state, vec1), vec1_stride,
                   THCTensor_(data)(state, r_), r_->stride(0));
#endif
  }
  else
  {
    THCTensor *cr = THCTensor_(newClone)(state, r_);

#ifdef THC_REAL_IS_FLOAT
    THCudaBlas_Sger(state, vec2_size, vec1_size,
                   alpha, THCTensor_(data)(state, vec2), vec2_stride,
                   THCTensor_(data)(state, vec1), vec1_stride,
                   THCTensor_(data)(state, cr), cr->stride(0));
#elif defined(THC_REAL_IS_DOUBLE)
    THCudaBlas_Dger(state, vec2_size, vec1_size,
                   alpha, THCTensor_(data)(state, vec2), vec2_stride,
                   THCTensor_(data)(state, vec1), vec1_stride,
                   THCTensor_(data)(state, cr), cr->stride(0));
#endif

    THCTensor_(freeCopyTo)(state, cr, r_);
  }
#elif defined(THC_REAL_IS_HALF)
  // currently no Hger/SgerEx in Cublas.
  THCTensor *vec2T = THCTensor_(newWithTensor)(state, vec2);
  THCTensor_(resize2d)(state, vec2T, vec2_size, 1);
  THCTensor_(transpose)(state, vec2T, NULL, 0, 1);

  THCTensor *vec1M = THCTensor_(newWithTensor)(state, vec1);
  THCTensor_(resize2d)(state, vec1M, vec1_size, 1);

  THCTensor_(addmm)(state, r_, beta, t, alpha, vec1M, vec2T);
  THCTensor_(free)(state, vec2T);
  THCTensor_(free)(state, vec1M);
#endif
#else
  ERROR_ONLY_FP_TYPES("addr");
#endif
}

void THCTensor_(addmm)(THCState *state, THCTensor *r_, scalar_t beta, THCTensor *t, scalar_t alpha, THCTensor *m1, THCTensor *m2)
{
#if defined(THC_REAL_IS_HALF) || defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE)

  THCAssertSameGPU(THCTensor_(checkGPU)(state, 4, r_, t, m1, m2));
  char transpose_r, transpose_m1, transpose_m2;
  THCTensor *r__, *m1_, *m2_;

  if( (m1->dim() != 2) || (m2->dim() != 2) )
    THError("2D tensors expected, got %dD, %dD tensors", m1->dim(), m2->dim());

  if(t->dim() != 2)
    THError("2D tensor expected, got %dD tensor for t", t->dim());

  if(m1->size(1) != m2->size(0)) {
    THCDescBuff bm1 = THCTensor_(sizeDesc)(state, m1);
    THCDescBuff bm2 = THCTensor_(sizeDesc)(state, m2);
    THError("size mismatch, m1: %s, m2: %s", bm1.str, bm2.str);
  }

  if( (t->size(0) != m1->size(0)) || (t->size(1) != m2->size(1)) ) {
    THCDescBuff bt  = THCTensor_(sizeDesc)(state, t);
    THCDescBuff bm1 = THCTensor_(sizeDesc)(state, m1);
    THCDescBuff bm2 = THCTensor_(sizeDesc)(state, m2);
    THError("size mismatch, t: %s, m1: %s, m2: %s", bt.str, bm1.str, bm2.str);
  }

  if(t != r_)
  {
    THCTensor_(resizeAs)(state, r_, t);
    if (ScalarConvert<scalar_t, double>::to(beta) != 0.0) {
      THCTensor_(copy)(state, r_, t);
    }
  }

  /* r_ */
  if(r_->stride(0) == 1 &&
     r_->stride(1) != 0)
  {
    transpose_r = 'n';
    r__ = r_;
  }
  else if(r_->stride(1) == 1 &&
          r_->stride(0) != 0)
  {
    THCTensor *swap = m2;
    m2 = m1;
    m1 = swap;
    transpose_r = 't';
    r__ = r_;
  }
  else
  {
    transpose_r = 'n';

    THCTensor *transp_r_ = THCTensor_(newTranspose)(state, r_, 0, 1);
    r__ = THCTensor_(newClone)(state, transp_r_);
    THCTensor_(free)(state, transp_r_);
    THCTensor_(transpose)(state, r__, NULL, 0, 1);
  }

  /* m1 */
  if(m1->stride((transpose_r == 'n' ? 0 : 1)) == 1 &&
     m1->stride((transpose_r == 'n' ? 1 : 0)) != 0)
  {
    transpose_m1 = 'n';
    m1_ = m1;
  }
  else if(m1->stride((transpose_r == 'n' ? 1 : 0)) == 1 &&
          m1->stride((transpose_r == 'n' ? 0 : 1)) != 0)
  {
    transpose_m1 = 't';
    m1_ = m1;
  }
  else
  {
    transpose_m1 = (transpose_r == 'n' ? 't' : 'n');
    m1_ = THCTensor_(newContiguous)(state, m1);
  }

  /* m2 */
  if(m2->stride((transpose_r == 'n' ? 0 : 1)) == 1 &&
     m2->stride((transpose_r == 'n' ? 1 : 0)) != 0)
  {
    transpose_m2 = 'n';
    m2_ = m2;
  }
  else if(m2->stride((transpose_r == 'n' ? 1 : 0)) == 1 &&
          m2->stride((transpose_r == 'n' ? 0 : 1)) != 0)
  {
    transpose_m2 = 't';
    m2_ = m2;
  }
  else
  {
    transpose_m2 = (transpose_r == 'n' ? 't' : 'n');
    m2_ = THCTensor_(newContiguous)(state, m2);
  }

#ifdef THC_REAL_IS_HALF
  THCudaBlas_Hgemm(state,
                   transpose_m1,
                   transpose_m2,
                   r__->size((transpose_r == 'n' ? 0 : 1)),
                   r__->size((transpose_r == 'n' ? 1 : 0)),
                   m1_->size((transpose_r == 'n' ? 1 : 0)),
                   alpha,
                   THCTensor_(data)(state, m1_),
                   (transpose_m1 == 'n' ? m1_->stride((transpose_r == 'n' ? 1 : 0)) : m1_->stride((transpose_r == 'n' ? 0 : 1))),
                   THCTensor_(data)(state, m2_),
                   (transpose_m2 == 'n' ? m2_->stride((transpose_r == 'n' ? 1 : 0)) : m2_->stride((transpose_r == 'n' ? 0 : 1))),
                   beta,
                   THCTensor_(data)(state, r__),
                   r__->stride((transpose_r == 'n' ? 1 : 0)));
#elif defined(THC_REAL_IS_FLOAT)
  THCudaBlas_Sgemm(state,
                   transpose_m1,
                   transpose_m2,
                   r__->size((transpose_r == 'n' ? 0 : 1)),
                   r__->size((transpose_r == 'n' ? 1 : 0)),
                   m1_->size((transpose_r == 'n' ? 1 : 0)),
                   alpha,
                   THCTensor_(data)(state, m1_),
                   (transpose_m1 == 'n' ? m1_->stride((transpose_r == 'n' ? 1 : 0)) : m1_->stride((transpose_r == 'n' ? 0 : 1))),
                   THCTensor_(data)(state, m2_),
                   (transpose_m2 == 'n' ? m2_->stride((transpose_r == 'n' ? 1 : 0)) : m2_->stride((transpose_r == 'n' ? 0 : 1))),
                   beta,
                   THCTensor_(data)(state, r__),
                   r__->stride((transpose_r == 'n' ? 1 : 0)));
#elif defined(THC_REAL_IS_DOUBLE)
  THCudaBlas_Dgemm(state,
                   transpose_m1,
                   transpose_m2,
                   r__->size((transpose_r == 'n' ? 0 : 1)),
                   r__->size((transpose_r == 'n' ? 1 : 0)),
                   m1_->size((transpose_r == 'n' ? 1 : 0)),
                   alpha,
                   THCTensor_(data)(state, m1_),
                   (transpose_m1 == 'n' ? m1_->stride((transpose_r == 'n' ? 1 : 0)) : m1_->stride((transpose_r == 'n' ? 0 : 1))),
                   THCTensor_(data)(state, m2_),
                   (transpose_m2 == 'n' ? m2_->stride((transpose_r == 'n' ? 1 : 0)) : m2_->stride((transpose_r == 'n' ? 0 : 1))),
                   beta,
                   THCTensor_(data)(state, r__),
                   r__->stride((transpose_r == 'n' ? 1 : 0)));
#endif

  /* free intermediate variables */
  if(m1_ != m1) {
    THCTensor_(free)(state, m1_);
  }

  if(m2_ != m2) {
    THCTensor_(free)(state, m2_);
  }

  if(r__ != r_) {
    THCTensor_(freeCopyTo)(state, r__, r_);
  }
#else
  ERROR_ONLY_FP_TYPES("addmm");
#endif
}

void THCTensor_(addbmm)(THCState *state, THCTensor *result, scalar_t beta, THCTensor *t,
                        scalar_t alpha, THCTensor *batch1, THCTensor *batch2) {
#if defined(THC_REAL_IS_HALF) || defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE)
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 4, result, t, batch1, batch2));
  THArgCheck(THCTensor_(nDimensionLegacyNoScalars)(state, t) == 2, 4, "expected 2D tensor");
  THArgCheck(THCTensor_(nDimensionLegacyNoScalars)(state, batch1) == 3, 6, "expected 3D tensor");
  THArgCheck(THCTensor_(nDimensionLegacyNoScalars)(state, batch2) == 3, 7, "expected 3D tensor");

  int64_t batchnum = THCTensor_(size)(state, batch1, 0);
  int64_t m1d1 = THCTensor_(size)(state, batch1, 1);
  int64_t innerdim = THCTensor_(size)(state, batch1, 2);
  int64_t m2d2 = THCTensor_(size)(state, batch2, 2);

  THArgCheck(batchnum == THCTensor_(size)(state, batch2, 0), 7,
      "equal number of batches expected");
  // M is t, as listed in the docs under addbmm
  THArgCheck(m1d1 == THCTensor_(size)(state, t, 0), 6,
      "first dimension must match first dimension of M");
  THArgCheck(m2d2 == THCTensor_(size)(state, t, 1), 7,
      "second dimension must match second dimension of M");
  THArgCheck(innerdim == THCTensor_(size)(state, batch2, 1), 6,
      "second dimension must match first dimension of batch2");

  if (t != result) {
    THCTensor_(resizeAs)(state, result, t);
    if (ScalarConvert<scalar_t, double>::to(beta) != 0.0) {
      THCTensor_(copy)(state, result, t);
    }
  }

  THCTensor *slice1 = THCTensor_(new)(state);
  THCTensor *slice2 = THCTensor_(new)(state);
  for (int64_t i=0; i<batchnum; i++) {
    THCTensor_(select)(state, slice1, batch1, 0, i);
    THCTensor_(select)(state, slice2, batch2, 0, i);

    THCTensor_(addmm)(state, result, beta, result, alpha, slice1, slice2);
    beta = ScalarConvert<int, scalar_t>::to(1);
  }
  THCTensor_(free)(state, slice1);
  THCTensor_(free)(state, slice2);
#else
  ERROR_ONLY_FP_TYPES("addbmm");
#endif
}

__global__ void createBatchGemmBuffer(const scalar_t** buffer, scalar_t* data,
                                      int64_t stride, int64_t num_batches) {
  const int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_batches) {
    buffer[idx] = data + idx * stride;
  }
}

__global__ void createBatchGemmBuffer3(const scalar_t** buffer1, const scalar_t ** buffer2, const scalar_t ** buffer3, scalar_t* data1,
                                       scalar_t * data2, scalar_t * data3, int64_t stride1, int64_t stride2, int64_t stride3, int64_t num_batches) {
  const int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_batches) {
    buffer1[idx] = data1 + idx * stride1;
    buffer2[idx] = data2 + idx * stride2;
    buffer3[idx] = data3 + idx * stride3;
  }
}

void THCTensor_(baddbmm)(THCState *state, THCTensor *result, scalar_t beta, THCTensor *t,
                         scalar_t alpha, THCTensor *batch1, THCTensor *batch2) {
#if defined(THC_REAL_IS_HALF) || defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE)
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 4, result, t, batch1, batch2));
  THArgCheck(THCTensor_(nDimensionLegacyNoScalars)(state, t) == 3, 4, "expected 3D tensor");
  THArgCheck(THCTensor_(nDimensionLegacyNoScalars)(state, batch1) == 3, 6, "expected 3D tensor");
  THArgCheck(THCTensor_(nDimensionLegacyNoScalars)(state, batch2) == 3, 7, "expected 3D tensor");
  THArgCheck(THCTensor_(size)(state, t, 0) == THCTensor_(size)(state, batch1, 0), 6,
             "equal number of batches expected");
  THArgCheck(THCTensor_(size)(state, t, 0) == THCTensor_(size)(state, batch2, 0), 7,
             "equal number of batches expected");
  THArgCheck(THCTensor_(size)(state, t, 1) == THCTensor_(size)(state, batch1, 1), 6,
             "wrong matrix size");
  THArgCheck(THCTensor_(size)(state, t, 2) == THCTensor_(size)(state, batch2, 2), 7,
             "wrong matrix size");
  THArgCheck(THCTensor_(size)(state, batch1, 2) == THCTensor_(size)(state, batch2, 1), 6,
             "wrong matrix size");

  if (t != result) {
    THCTensor_(resizeAs)(state, result, t);
    if (ScalarConvert<scalar_t, double>::to(beta) != 0.0) {
      THCTensor_(copy)(state, result, t);
    }
  }

  bool transpose_result;
  char transpose_batch1, transpose_batch2;
  int64_t lda, ldb, ldc;
  THCTensor *result_, *batch1_, *batch2_;
  if (result->stride(1) == 1)
  {
    transpose_result = false;
    result_ = result;
    ldc = result_->stride(2);
  }
  else if (result->stride(2) == 1)
  {
    transpose_result = true;

    THCTensor *swap = batch2;
    batch2 = batch1;
    batch1 = swap;

    result_ = result;
    ldc = result_->stride(1);
  }
  else
  {
    transpose_result = false;

    THCTensor *transp_r_ = THCTensor_(newTranspose)(state, result, 1, 2);
    result_ = THCTensor_(newClone)(state, transp_r_);
    THCTensor_(free)(state, transp_r_);
    THCTensor_(transpose)(state, result_, NULL, 1, 2);

    ldc = result_->stride(2);
  }

  if (batch1->stride(transpose_result ? 2 : 1) == 1 &&
   batch1->stride(transpose_result ? 1 : 2) != 0)
  {
    transpose_batch1 = 'n';
    batch1_ = batch1;
    lda = batch1_->stride(transpose_result ? 1 : 2);
  }
  else if (batch1->stride(transpose_result ? 1 : 2) == 1 &&
   batch1->stride(transpose_result ? 2 : 1) != 0)
  {
    transpose_batch1 = 't';
    batch1_ = batch1;
    lda = batch1_->stride(transpose_result ? 2 : 1);
  }
  else
  {
    transpose_batch1 = transpose_result ? 'n' : 't';
    // batch1_ is later freed if batch1_ != batch1
    if (THCTensor_(isContiguous)(state, batch1)) {
      batch1_ = batch1;
    } else {
      batch1_ = THCTensor_(newContiguous)(state, batch1);
    }
    lda = batch1_->stride(1);
  }

  if (batch2->stride(transpose_result ? 2 : 1) == 1 &&
   batch2->stride(transpose_result ? 1 : 2) != 0)
  {
    transpose_batch2 = 'n';
    batch2_ = batch2;
    ldb = batch2_->stride(transpose_result ? 1 : 2);
  }
  else if (batch2->stride(transpose_result ? 1 : 2) == 1 &&
   batch2->stride(transpose_result ? 2 : 1) != 0)
  {
    transpose_batch2 = 't';
    batch2_ = batch2;
    ldb = batch2_->stride(transpose_result ? 2 : 1);
  }
  else
  {
    transpose_batch2 = transpose_result ? 'n' : 't';
    // batch2_ is later freed if batch2_ != batch2
    if (THCTensor_(isContiguous)(state, batch2)) {
      batch2_ = batch2;
    } else {
      batch2_ = THCTensor_(newContiguous)(state, batch2);
    }
    ldb = batch2_->stride(1);
  }
  int64_t num_batches = result_->size(0);

#if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE)
  // Compute pointers to matrices in each batch.
#if CUDA_VERSION < 8000 && !defined __HIP_PLATFORM_HCC__
  size_t matrices_size = num_batches * sizeof(scalar_t*);

//   Copy pointers to device.
  auto d_matrices1 = static_cast<const scalar_t**>(THCudaMalloc(state, matrices_size));
  auto d_matrices2 = static_cast<const scalar_t**>(THCudaMalloc(state, matrices_size));
  auto d_result_matrices = static_cast<scalar_t**>(THCudaMalloc(state, matrices_size));

  const int64_t block = 512;
  const int64_t grid = (num_batches + block - 1) / block;

  createBatchGemmBuffer3<<<grid, block, 0, THCState_getCurrentStream(state)>>>(
    d_matrices1, d_matrices2, (const scalar_t**)d_result_matrices, THCTensor_(data)(state, batch1_),
    THCTensor_(data)(state, batch2_), THCTensor_(data)(state, result_),
    batch1_->stride(0), batch2_->stride(0), result_->stride(0), num_batches);

#ifdef THC_REAL_IS_FLOAT
  THCudaBlas_SgemmBatched(
      state,
      transpose_batch1,
      transpose_batch2,
      result_->size(transpose_result ? 2 : 1),
      result_->size(transpose_result ? 1 : 2),
      batch1_->size(transpose_result ? 1 : 2),
      alpha,
      d_matrices1, lda,
      d_matrices2, ldb,
      beta,
      d_result_matrices, ldc,
      num_batches);
#elif defined(THC_REAL_IS_DOUBLE)
  THCudaBlas_DgemmBatched(
      state,
      transpose_batch1,
      transpose_batch2,
      result_->size(transpose_result ? 2 : 1),
      result_->size(transpose_result ? 1 : 2),
      batch1_->size(transpose_result ? 1 : 2),
      alpha,
      d_matrices1, lda,
      d_matrices2, ldb,
      beta,
      d_result_matrices, ldc,
      num_batches);
#endif //THC_REAL

  THCudaFree(state, d_matrices1);
  THCudaFree(state, d_matrices2);
  THCudaFree(state, d_result_matrices);

#else
#ifdef THC_REAL_IS_FLOAT
  THCudaBlas_SgemmStridedBatched(
      state,
      transpose_batch1,
      transpose_batch2,
      result_->size(transpose_result ? 2 : 1),
      result_->size(transpose_result ? 1 : 2),
      batch1_->size(transpose_result ? 1 : 2),
      alpha,
      THCTensor_(data)(state, batch1_), lda, batch1_->stride(0),
      THCTensor_(data)(state, batch2_), ldb, batch2_->stride(0),
      beta,
      THCTensor_(data)(state, result_), ldc, result_->stride(0),
      num_batches);
#elif defined(THC_REAL_IS_DOUBLE)
  THCudaBlas_DgemmStridedBatched(
      state,
      transpose_batch1,
      transpose_batch2,
      result_->size(transpose_result ? 2 : 1),
      result_->size(transpose_result ? 1 : 2),
      batch1_->size(transpose_result ? 1 : 2),
      alpha,
      THCTensor_(data)(state, batch1_), lda, batch1_->stride(0),
      THCTensor_(data)(state, batch2_), ldb, batch2_->stride(0),
      beta,
      THCTensor_(data)(state, result_), ldc, result_->stride(0),
      num_batches);
#endif //THC_REAL
#endif //CUDA_VERSION

#elif defined(THC_REAL_IS_HALF)

#if CUDA_VERSION < 9010
  // Currently no HgemmBatched in Cublas
  for (int64_t i = 0; i < num_batches; ++i) {
    THCudaBlas_Hgemm(
        state,
        transpose_batch1,
        transpose_batch2,
        result_->size(transpose_result ? 2 : 1),
        result_->size(transpose_result ? 1 : 2),
        batch1_->size(transpose_result ? 1 : 2),
        alpha,
        THCTensor_(data)(state, batch1_) + i * batch1_->stride(0), lda,
        THCTensor_(data)(state, batch2_) + i * batch2_->stride(0), ldb,
        beta,
        THCTensor_(data)(state, result_) + i * result_->stride(0), ldc);
  }
#else
#ifndef __HIP_PLATFORM_HCC__
  cudaDeviceProp* prop = at::cuda::getCurrentDeviceProperties();
  if (prop->major >= 5){
#endif

  THCudaBlas_HgemmStridedBatched(
      state,
      transpose_batch1,
      transpose_batch2,
      result_->size(transpose_result ? 2 : 1),
      result_->size(transpose_result ? 1 : 2),
      batch1_->size(transpose_result ? 1 : 2),
      alpha,
      THCTensor_(data)(state, batch1_), lda, batch1_->stride(0),
      THCTensor_(data)(state, batch2_), ldb, batch2_->stride(0),
      beta,
      THCTensor_(data)(state, result_), ldc, result_->stride(0),
      num_batches);
#ifndef __HIP_PLATFORM_HCC__
   } else {
      for (int64_t i = 0; i < num_batches; ++i) {
        THCudaBlas_Hgemm(
        state,
        transpose_batch1,
        transpose_batch2,
        result_->size(transpose_result ? 2 : 1),
        result_->size(transpose_result ? 1 : 2),
        batch1_->size(transpose_result ? 1 : 2),
        alpha,
        THCTensor_(data)(state, batch1_) + i * batch1_->stride(0), lda,
        THCTensor_(data)(state, batch2_) + i * batch2_->stride(0), ldb,
        beta,
        THCTensor_(data)(state, result_) + i * result_->stride(0), ldc);
      }
   }
#endif

#endif
#endif
  if (batch1_ != batch1) {
    THCTensor_(free)(state, batch1_);
  }

  if (batch2_ != batch2) {
    THCTensor_(free)(state, batch2_);
  }

  if (result_ != result) {
    THCTensor_(freeCopyTo)(state, result_, result);
  }

#else
  ERROR_ONLY_FP_TYPES("baddbmm");
#endif
}

void THCTensor_(btrisolve)(THCState *state, THCTensor *rb_, THCTensor *b,
                           THCTensor *atf, THCudaIntTensor *pivots)
{
#if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE)
  THAssert(THCTensor_(checkGPU)(state, 3, rb_, atf, b));
  THArgCheck(THCTensor_(nDimensionLegacyAll)(state, atf) == 3, 3, "expected 3D tensor");
  THArgCheck(THCTensor_(nDimensionLegacyAll)(state, b) == 3 ||
             THCTensor_(nDimensionLegacyAll)(state, b) == 2, 4, "expected 2D or 3D tensor");
  THArgCheck(THCTensor_(size)(state, atf, 0) ==
             THCTensor_(size)(state, b, 0), 3, "number of batches must be equal");
  THArgCheck(THCTensor_(size)(state, atf, 1) ==
             THCTensor_(size)(state, atf, 2), 3, "A matrices must be square");
  THArgCheck(THCTensor_(size)(state, atf, 1) ==
             THCTensor_(size)(state, b, 1), 3, "dimensions of A and b must be equal");

  if (rb_ != b) {
    THCTensor_(resizeAs)(state, rb_, b);
    THCTensor_(copy)(state, rb_, b);
  }


  int n = atf->size(1);
  int nrhs = THTensor_nDimensionLegacyAll(rb_) > 2 ? rb_->size(2) : 1;
  THCTensor *atf_;
  THCTensor *rb__;
  int lda, ldb;

  // correct ordering of A_tf
  if (atf->stride(1) == 1) {
    // column ordered, what BLAS wants
    lda = atf->stride(2);
    atf_ = atf;
  } else {
    // not column ordered, need to make it such (requires copy)
    // it would be nice if we could use the op(A) flags to automatically
    // transpose A if needed, but this leads to unpredictable behavior if the
    // user clones A_tf later with a different ordering
    THCTensor *transp_r_ = THCTensor_(newTranspose)(state, atf, 1, 2);
    atf_ = THCTensor_(newClone)(state, transp_r_);
    THCTensor_(free)(state, transp_r_);
    THCTensor_(transpose)(state, atf_, NULL, 1, 2);
    lda = atf_->stride(2);
  }

  // correct ordering of B
  if (rb_->stride(1) == 1) {
    // column ordered
    if (THTensor_nDimensionLegacyAll(rb_) == 2 || rb_->size(2) == 1) {
      ldb = n;
    } else {
      ldb = rb_->stride(2);
    }
    rb__ = rb_;
  } else {
    // make column ordered
    if (THTensor_nDimensionLegacyAll(rb_) > 2) {
      THCTensor *transp_r_ = THCTensor_(newTranspose)(state, rb_, 1, 2);
      rb__ = THCTensor_(newClone)(state, transp_r_);
      THCTensor_(free)(state, transp_r_);
      THCTensor_(transpose)(state, rb__, NULL, 1, 2);
      ldb = rb__->stride(2);
    } else {
      rb__ = THCTensor_(newClone)(state, rb_);
      ldb = n;
    }
  }

  int64_t num_batches = rb_->size(0);
  size_t matrices_size = num_batches * sizeof(scalar_t*);

  // Copy pointers to device.
  auto d_result = static_cast<scalar_t**>(THCudaMalloc(state, matrices_size));
  auto d_atf = static_cast<const scalar_t**>(THCudaMalloc(state, matrices_size));

  const int64_t block = 512;
  const int64_t grid = (num_batches + block - 1) / block;
  createBatchGemmBuffer<<<grid, block, 0, THCState_getCurrentStream(state)>>>(
    (const scalar_t**)d_result, THCTensor_(data)(state, rb__),
    rb__->stride(0), num_batches);
  createBatchGemmBuffer<<<grid, block, 0, THCState_getCurrentStream(state)>>>(
    d_atf, THCTensor_(data)(state, atf_),
    atf_->stride(0), num_batches);

  if (!THCudaIntTensor_isContiguous(state, pivots)) {
      THError("Error: pivots is not contiguous.");
  }

  int *pivots_data = THCudaIntTensor_data(state, pivots);
  int info;

#ifdef THC_REAL_IS_FLOAT
  THCudaBlas_Sgetrs(state, 'n', n, nrhs, d_atf, lda, pivots_data, d_result, ldb, &info, num_batches);
#elif defined(THC_REAL_IS_DOUBLE)
  THCudaBlas_Dgetrs(state, 'n', n, nrhs, d_atf, lda, pivots_data, d_result, ldb, &info, num_batches);
#endif

  if (info < 0) {
    THError("Illegal arg %d", -info);
  }

  THCudaFree(state, d_result);
  THCudaFree(state, d_atf);

  if (atf_ != atf) {
    THCTensor_(free)(state, atf_);
  }

  if (rb__ != rb_) {
    THCTensor_(freeCopyTo)(state, rb__, rb_);
  }

#else
  THError("btrisolve for CUDA tensors is only supported for floats and doubles");
#endif
}

#endif

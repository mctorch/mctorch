
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THC/generic/THCTensorMathPointwise.cu"
#else

#include <ATen/MemoryOverlap.h>

#define IMPLEMENT_CUDA_TENSOR_BASIC_FUNC_(NAME, CFUNC, REAL)             \
  struct Tensor_##NAME##_##REAL##_Op {                                  \
    __device__ __forceinline__ void operator()(scalar_t* out, scalar_t* in) const { \
      *out = CFUNC(*in);                                                \
    }                                                                   \
                                                                        \
    __device__ __forceinline__ void operator()(scalar_t* v) const {         \
      *v = CFUNC(*v);                                                   \
    }                                                                   \
  };                                                                    \
                                                                        \
  void THCTensor_(NAME)(THCState* state, THCTensor* self_, THCTensor* src) { \
    THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, src));               \
    at::assert_no_internal_overlap(self_, #NAME);                       \
    if (self_ == src) {                                                 \
      if (!THC_pointwiseApply1<scalar_t>(state, self_, Tensor_##NAME##_##REAL##_Op())) { \
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);                      \
      }                                                                 \
    } else {                                                            \
      THCTensor_(resizeAs)(state, self_, src);                          \
                                                                        \
      if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self_, src, Tensor_##NAME##_##REAL##_Op())) { \
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);                      \
      }                                                                 \
    }                                                                   \
                                                                        \
    THCudaCheck(cudaGetLastError());                                    \
  }

#define IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(NAME, CFUNC, REAL) \
  IMPLEMENT_CUDA_TENSOR_BASIC_FUNC_(NAME, CFUNC, REAL)

#if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE) || defined(THC_REAL_IS_HALF)

IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  log, THCNumerics<scalar_t>::log,   Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(lgamma, THCNumerics<scalar_t>::lgamma, Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(log10, THCNumerics<scalar_t>::log10, Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(log1p, THCNumerics<scalar_t>::log1p, Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC( log2, THCNumerics<scalar_t>::log2,  Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  exp, THCNumerics<scalar_t>::exp,   Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(expm1, THCNumerics<scalar_t>::expm1, Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  cos, THCNumerics<scalar_t>::cos,   Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  sin, THCNumerics<scalar_t>::sin,   Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC( sqrt, THCNumerics<scalar_t>::sqrt,  Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(rsqrt, THCNumerics<scalar_t>::rsqrt, Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC( ceil, THCNumerics<scalar_t>::ceil,  Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(floor, THCNumerics<scalar_t>::floor, Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(trunc, THCNumerics<scalar_t>::trunc, Real)

IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  acos, THCNumerics<scalar_t>::acos,  Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  cosh, THCNumerics<scalar_t>::cosh,  Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  asin, THCNumerics<scalar_t>::asin,  Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  sinh, THCNumerics<scalar_t>::sinh,  Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(   tan, THCNumerics<scalar_t>::tan,   Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  atan, THCNumerics<scalar_t>::atan,  Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  tanh, THCNumerics<scalar_t>::tanh,  Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(   erf, THCNumerics<scalar_t>::erf,   Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  erfc, THCNumerics<scalar_t>::erfc,  Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(erfinv, THCNumerics<scalar_t>::erfinv,Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC( round, THCNumerics<scalar_t>::round, Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  frac, THCNumerics<scalar_t>::frac,  Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  cinv, THCNumerics<scalar_t>::cinv,  Real)

#endif

IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  neg, THCNumerics<scalar_t>::neg,   Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  abs, THCNumerics<scalar_t>::abs,   Real)

#undef IMPLEMENT_CUDA_TENSOR_BASIC_FUNC_
#undef IMPLEMENT_CUDA_TENSOR_BASIC_FUNC

void THCTensor_(sign)(THCState* state, THCTensor* self_, THCTensor* src) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, src));
  if (self_ == src) {
    if (!THC_pointwiseApply1<scalar_t>(state, self_, TensorSignOp<scalar_t>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src);

    if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self_, src, TensorSignOp<scalar_t>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

void THCTensor_(clamp)(THCState *state, THCTensor *self_, THCTensor *src, scalar_t min_value,
  scalar_t max_value)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, src));
  if (self_ == src) {
    if (!THC_pointwiseApply1<scalar_t>(state, self_, TensorClampOp<scalar_t>(min_value, max_value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src);

    if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self_, src, TensorClampOp<scalar_t>(min_value, max_value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

void THCTensor_(crossKernel)(THCState *state, THCTensor *self, THCTensor *x, THCTensor *y, int dimension)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, self, x, y));

  int64_t sx = THCTensor_(stride)(state, x, dimension);
  int64_t sy = THCTensor_(stride)(state, y, dimension);
  int64_t so = THCTensor_(stride)(state, self, dimension);
  THCTensor *nx = THCTensor_(newNarrow)(state, x, dimension, 0, 1);
  THCTensor *ny = THCTensor_(newNarrow)(state, y, dimension, 0, 1);
  THCTensor *nself = THCTensor_(newNarrow)(state, self, dimension, 0, 1);
  if (!THC_pointwiseApply3<scalar_t, scalar_t, scalar_t>(state, nself, nx, ny, TensorCrossOp<scalar_t>(sx, sy, so))) {
    THArgCheck(false, 2, CUTORCH_DIM_WARNING);
  }
  THCTensor_(free)(state, nx);
  THCTensor_(free)(state, ny);
  THCTensor_(free)(state, nself);
}

#if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE) || defined(THC_REAL_IS_HALF)

void THCTensor_(atan2)(THCState *state, THCTensor *self_, THCTensor *tx, THCTensor *ty)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, self_, tx, ty));
  THArgCheck(THCTensor_(nElement)(state, tx) ==
             THCTensor_(nElement)(state, ty), 3, "sizes do not match");
  THCTensor_(resizeAs)(state, self_, tx);

  if (!THC_pointwiseApply3<scalar_t, scalar_t, scalar_t>(state, self_, tx, ty, TensorATan2Op<scalar_t>())) {
    THArgCheck(false, 2, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(cudaGetLastError());
}

void THCTensor_(sigmoid)(THCState* state, THCTensor* self_, THCTensor* src) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, src));
  if (self_ == src) {
    if (!THC_pointwiseApply1<scalar_t>(state, self_, TensorSigmoidOp<scalar_t>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src);

    if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self_, src, TensorSigmoidOp<scalar_t>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

void THCTensor_(digamma)(THCState* state, THCTensor* self_, THCTensor* src) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, src));
  if (self_ != src) {
    THCTensor_(resizeAs)(state, self_, src);
  }
  if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self_, src, TensorDigammaOp<scalar_t, accreal>())) {
    THArgCheck(false, 2, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(cudaGetLastError());
}

void THCTensor_(polygamma)(THCState* state, THCTensor* self_, int64_t n, THCTensor* src) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, src));
  if (self_ != src) {
    THCTensor_(resizeAs)(state, self_, src);
  }
  switch (n) {
    case 0:
      if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self_, src, TensorDigammaOp<scalar_t, accreal>())) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
      break;
    case 1:
      if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self_, src, TensorTrigammaOp<scalar_t, accreal>())) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
      break;
    default:
      THError("polygamma(n,x) is not implemented for n>=2");
  }

  THCudaCheck(cudaGetLastError());
}

#endif

namespace {
c10::intrusive_ptr<at::TensorImpl, at::UndefinedTensorImpl> retainTensorImpl(THCTensor* self) {
  c10::raw::intrusive_ptr::incref(self);
  return c10::intrusive_ptr<at::TensorImpl, at::UndefinedTensorImpl>::reclaim(self);
}
}

void THCTensor_(cadd)(THCState *state, THCTensor *self_, THCTensor* src1, scalar_t value, THCTensor *src2)
{
  auto out = at::Tensor(retainTensorImpl(self_));
#ifdef THC_REAL_IS_HALF
  auto alpha = at::Half(value);
#else
  auto alpha = value;
#endif
  at::add_out(out, at::Tensor(retainTensorImpl(src1)), at::Tensor(retainTensorImpl(src2)), alpha);
}

void THCTensor_(csub)(THCState *state, THCTensor *self_, THCTensor* src1, scalar_t value, THCTensor *src2)
{
  auto out = at::Tensor(retainTensorImpl(self_));
#ifdef THC_REAL_IS_HALF
  auto alpha = at::Half(value);
#else
  auto alpha = value;
#endif
  at::sub_out(out, at::Tensor(retainTensorImpl(src1)), at::Tensor(retainTensorImpl(src2)), alpha);
}

void THCTensor_(cmul)(THCState *state, THCTensor *self_, THCTensor *src1, THCTensor *src2)
{
  auto out = at::Tensor(retainTensorImpl(self_));
  at::mul_out(out, at::Tensor(retainTensorImpl(src1)), at::Tensor(retainTensorImpl(src2)));
}

void THCTensor_(cpow)(THCState *state, THCTensor *self_, THCTensor *src1, THCTensor *src2)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, self_, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 3, "sizes do not match");

  if (self_ == src1) {
    // self = pow(self, src2)
    if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self_, src2, TensorCPowOp<scalar_t>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src1);

    // self = pow(src1, src2)
    if (!THC_pointwiseApply3<scalar_t, scalar_t, scalar_t>(state, self_, src1, src2, TensorCPowOp<scalar_t>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

void THCTensor_(pow)(THCState *state, THCTensor *self_, THCTensor *src, scalar_t value) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, src));
  if (self_ == src) {
    if (THCNumerics<scalar_t>::eq(value, ScalarConvert<int, scalar_t>::to(1))) {
      if (!THC_pointwiseApply1<scalar_t>(state, self_, TensorPowOp<scalar_t, 1>(value))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    } else if (THCNumerics<scalar_t>::eq(value, ScalarConvert<int, scalar_t>::to(2))) {
      if (!THC_pointwiseApply1<scalar_t>(state, self_, TensorPowOp<scalar_t, 2>(value))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    } else if (THCNumerics<scalar_t>::eq(value, ScalarConvert<int, scalar_t>::to(3))) {
      if (!THC_pointwiseApply1<scalar_t>(state, self_, TensorPowOp<scalar_t, 3>(value))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
#if defined(THC_REAL_IS_HALF) || defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE)
    } else if (THCNumerics<scalar_t>::eq(value, ScalarConvert<int, scalar_t>::to(-1))) {
      if (!THC_pointwiseApply1<scalar_t>(state, self_, TensorPowOp<scalar_t, -1>(value))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    } else if (THCNumerics<scalar_t>::eq(value, ScalarConvert<int, scalar_t>::to(-2))) {
      if (!THC_pointwiseApply1<scalar_t>(state, self_, TensorPowOp<scalar_t, -2>(value))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
#endif
    } else {
      // fallback implementation using pow
      if (!THC_pointwiseApply1<scalar_t>(state, self_, TensorPowOp<scalar_t, -3>(value))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src);

    if (THCNumerics<scalar_t>::eq(value, ScalarConvert<int, scalar_t>::to(1))) {
      if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self_, src, TensorPowOp<scalar_t, 1>(value))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    } else if (THCNumerics<scalar_t>::eq(value, ScalarConvert<int, scalar_t>::to(2))) {
      if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self_, src, TensorPowOp<scalar_t, 2>(value))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    } else if (THCNumerics<scalar_t>::eq(value, ScalarConvert<int, scalar_t>::to(3))) {
      if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self_, src, TensorPowOp<scalar_t, 3>(value))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
#if defined(THC_REAL_IS_HALF) || defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE)
    } else if (THCNumerics<scalar_t>::eq(value, ScalarConvert<int, scalar_t>::to(-1))) {
      if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self_, src, TensorPowOp<scalar_t, -1>(value))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    } else if (THCNumerics<scalar_t>::eq(value, ScalarConvert<int, scalar_t>::to(-2))) {
      if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self_, src, TensorPowOp<scalar_t, -2>(value))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
#endif
    } else {
      // fallback implementation using pow
      if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self_, src, TensorPowOp<scalar_t, -3>(value))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    }
  }

  THCudaCheck(cudaGetLastError());
}

void THCTensor_(tpow)(THCState *state, THCTensor *self_, scalar_t value, THCTensor *src)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, src));
  if (self_ == src) {
    if (!THC_pointwiseApply1<scalar_t>(state, self_, TensorTPowOp<scalar_t>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src);

    if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self_, src, TensorTPowOp<scalar_t>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}
void THCTensor_(cdiv)(THCState* state, THCTensor *self_, THCTensor *src1, THCTensor *src2)
{
  auto out = at::Tensor(retainTensorImpl(self_));
  at::div_out(out, at::Tensor(retainTensorImpl(src1)), at::Tensor(retainTensorImpl(src2)));
}

void THCTensor_(clshift)(THCState* state, THCTensor *self_, THCTensor *src1, THCTensor *src2)
{
#if defined(THC_REAL_IS_HALF)
  return THError("clshift not supported for torch.CudaHalfTensor");
#else
  THAssert(THCTensor_(checkGPU)(state, 3, self_, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 3, "sizes do not match");

  if (self_ == src1) {
    // self /= src2
    if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self_, src2, TensorLShiftOp<scalar_t>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src1);

    // self = src1 / src2
    if (!THC_pointwiseApply3<scalar_t, scalar_t, scalar_t>(state, self_, src1, src2, TensorLShiftOp<scalar_t>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
#endif
}

void THCTensor_(crshift)(THCState* state, THCTensor *self_, THCTensor *src1, THCTensor *src2)
{
#if defined(THC_REAL_IS_HALF)
  return THError("crshift not supported for torch.CudaHalfTensor");
#else
  THAssert(THCTensor_(checkGPU)(state, 3, self_, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 3, "sizes do not match");

  if (self_ == src1) {
    // self /= src2
    if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self_, src2, TensorRShiftOp<scalar_t>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src1);

    // self = src1 / src2
    if (!THC_pointwiseApply3<scalar_t, scalar_t, scalar_t>(state, self_, src1, src2, TensorRShiftOp<scalar_t>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
#endif
}

void THCTensor_(cmax)(THCState *state, THCTensor *self, THCTensor *src1, THCTensor *src2)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, self, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 2, "sizes do not match");

  if (self == src1) {
    if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self, src2, TensorMaxOp<scalar_t>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self, src1);
    if (!THC_pointwiseApply3<scalar_t, scalar_t, scalar_t>(state, self, src1, src2, TensorMaxOp<scalar_t>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }
}

void THCTensor_(cmin)(THCState *state, THCTensor *self, THCTensor *src1, THCTensor *src2)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, self, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 2, "sizes do not match");

  if (self == src1) {
    if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self, src2, TensorMinOp<scalar_t>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self, src1);
    if (!THC_pointwiseApply3<scalar_t, scalar_t, scalar_t>(state, self, src1, src2, TensorMinOp<scalar_t>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }
}

void THCTensor_(cremainder)(THCState *state, THCTensor *self, THCTensor *src1, THCTensor *src2)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, self, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 2, "sizes do not match");

  if (self == src1) {
    if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self, src2, TensorCRemainderOp<scalar_t>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self, src1);
    if (!THC_pointwiseApply3<scalar_t, scalar_t, scalar_t>(state, self, src1, src2, TensorCRemainderOp<scalar_t>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }
}

void THCTensor_(cfmod)(THCState *state, THCTensor *self, THCTensor *src1, THCTensor *src2)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, self, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 2, "sizes do not match");

  if (self == src1) {
    if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self, src2, TensorCFmodOp<scalar_t>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self, src1);
    if (!THC_pointwiseApply3<scalar_t, scalar_t, scalar_t>(state, self, src1, src2, TensorCFmodOp<scalar_t>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }
}

void THCTensor_(cmaxValue)(THCState *state, THCTensor *self, THCTensor *src, scalar_t value)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self, src));

  if (self == src) {
    if (!THC_pointwiseApply1<scalar_t>(state, self, TensorMaxValueOp<scalar_t>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self, src);
    if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self, src, TensorMaxValueOp<scalar_t>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }
}

void THCTensor_(cminValue)(THCState *state, THCTensor *self, THCTensor *src, scalar_t value)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self, src));

  if (self == src) {
    if (!THC_pointwiseApply1<scalar_t>(state, self, TensorMinValueOp<scalar_t>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self, src);
    if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self, src, TensorMinValueOp<scalar_t>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }
}

void THCTensor_(addcmul)(THCState *state, THCTensor *self_, THCTensor *t, scalar_t value, THCTensor *src1, THCTensor *src2)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 4, self_, t, src1, src2));
  if(self_ != t)
  {
    THCTensor_(resizeAs)(state, self_, t);
    THCTensor_(copy)(state, self_, t);
  }
  else
  {
    THArgCheck(THCTensor_(nElement)(state, self_) == THCTensor_(nElement)(state, src1),
               1, "sizes do not match");
  }

  THArgCheck(THCTensor_(nElement)(state, src1) == THCTensor_(nElement)(state, src2),
             3, "sizes do not match");

  if (!THC_pointwiseApply3<scalar_t, scalar_t, scalar_t>(state, self_, src1, src2, TensorAddCMulOp<scalar_t>(value))) {
    THArgCheck(false, 2, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(cudaGetLastError());
}

void THCTensor_(addcdiv)(THCState *state, THCTensor *self_, THCTensor *t, scalar_t value, THCTensor *src1, THCTensor *src2)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 4, self_, t, src1, src2));
  if(self_ != t)
  {
    THCTensor_(resizeAs)(state, self_, t);
    THCTensor_(copy)(state, self_, t);
  }
  else
  {
    THArgCheck(THCTensor_(nElement)(state, self_) == THCTensor_(nElement)(state, src1),
               1, "sizes do not match");
  }
  THArgCheck(THCTensor_(nElement)(state, src1) == THCTensor_(nElement)(state, src2),
             3, "sizes do not match");

  if (!THC_pointwiseApply3<scalar_t, scalar_t, scalar_t>(state, self_, src1, src2, TensorAddCDivOp<scalar_t>(value))) {
    THArgCheck(false, 2, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(cudaGetLastError());
}

void THCTensor_(cbitand)(THCState* state, THCTensor *self_, THCTensor *src1, THCTensor *src2)
{
#if defined(THC_REAL_IS_HALF) || defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE)
  return THError("cbitand is only supported for integer type tensors");
#else
  THAssert(THCTensor_(checkGPU)(state, 3, self_, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 3, "sizes do not match");

  if (self_ == src1) {
    // self /= src2
    if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self_, src2, TensorBitAndOp<scalar_t>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src1);

    // self = src1 / src2
    if (!THC_pointwiseApply3<scalar_t, scalar_t, scalar_t>(state, self_, src1, src2, TensorBitAndOp<scalar_t>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
#endif
}

void THCTensor_(cbitor)(THCState* state, THCTensor *self_, THCTensor *src1, THCTensor *src2)
{
#if defined(THC_REAL_IS_HALF) || defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE)
  return THError("cbitor is only supported for integer type tensors");
#else
  THAssert(THCTensor_(checkGPU)(state, 3, self_, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 3, "sizes do not match");

  if (self_ == src1) {
    // self /= src2
    if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self_, src2, TensorBitOrOp<scalar_t>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src1);

    // self = src1 / src2
    if (!THC_pointwiseApply3<scalar_t, scalar_t, scalar_t>(state, self_, src1, src2, TensorBitOrOp<scalar_t>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
#endif
}

void THCTensor_(cbitxor)(THCState* state, THCTensor *self_, THCTensor *src1, THCTensor *src2)
{
#if defined(THC_REAL_IS_HALF) || defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE)
  return THError("cbitor is only supported for integer type tensors");
#else
  THAssert(THCTensor_(checkGPU)(state, 3, self_, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 3, "sizes do not match");

  if (self_ == src1) {
    // self /= src2
    if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self_, src2, TensorBitXorOp<scalar_t>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src1);

    // self = src1 / src2
    if (!THC_pointwiseApply3<scalar_t, scalar_t, scalar_t>(state, self_, src1, src2, TensorBitXorOp<scalar_t>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
#endif
}
#endif

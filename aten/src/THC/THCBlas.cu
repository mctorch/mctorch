#include "hip/hip_runtime.h"
#include <THC/THCBlas.h>
#include <THC/THCGeneral.h>
#include <TH/THHalf.h>
#include <ATen/cuda/HIPContext.h>

#include <algorithm>

float THCudaBlas_Sdot(THCState *state, int64_t n, float *x, int64_t incx, float *y, int64_t incy)
{
  if (n == 1) {
    incx = 1;
    incy = 1;
  }

  if ((n <= INT_MAX) && (incx <= INT_MAX) && (incy <= INT_MAX)) {
    int i_n = (int)n;
    int i_incx = (int)incx;
    int i_incy = (int)incy;
    float result;
    hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
    hipblasSetStream(handle, THCState_getCurrentStream(state));
    THCublasCheck(hipblasSdot(handle, i_n, x, i_incx, y, i_incy, &result));
    return result;
  }

  THError("Cublas_Sdot only supports n, incx and incy "
          "up to signed integer limits: %d", INT_MAX);
  return 0;
}

double THCudaBlas_Ddot(THCState *state, int64_t n, double *x, int64_t incx, double *y, int64_t incy)
{
  if (n == 1) {
    incx = 1;
    incy = 1;
  }

  if ((n <= INT_MAX) && (incx <= INT_MAX) && (incy <= INT_MAX)) {
    int i_n = (int)n;
    int i_incx = (int)incx;
    int i_incy = (int)incy;
    double result;
    hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
    hipblasSetStream(handle, THCState_getCurrentStream(state));
    THCublasCheck(hipblasDdot(handle, i_n, x, i_incx, y, i_incy, &result));
    return result;
  }

  THError("Cublas_Ddot only supports n, incx and incy "
          "up to signed integer limits: %d", INT_MAX);
  return 0;
}

at::Half THCudaBlas_Hdot(THCState *state, int64_t n, at::Half *x, int64_t incx, at::Half *y, int64_t incy)
{
#if CUDA_VERSION >= 8000
  if (n == 1) {
    incx = 1;
    incy = 1;
  }

  if ((n <= INT_MAX) && (incx <= INT_MAX) && (incy <= INT_MAX)) {
    at::Half result;
    hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
    hipblasSetStream(handle, THCState_getCurrentStream(state));
    THCublasCheck(hipblasDotEx(handle, n,
                              x, HIP_R_16F, incx,
                              y, HIP_R_16F, incy,
                              &result, HIP_R_16F,
                              HIP_R_32F));
    return result;
  }

  THError("Cublas_Hdot only supports n, incx and incy "
          "up to signed integer limits: %d", INT_MAX);
  return 0.0;
#else
  THError("Cublas_Hdot requires CUDA 8.0+");
  return 0.0;
#endif
}

/* Level 2 */

void adjustLdLevel2(int64_t m, int64_t n, int64_t *lda)
{
  // Note: leading dimensions generally are checked that they are > 0 and at least as big the result
  // requires (even if the value won't be used).
  // TODO: why does Level3 check trans but this doesn't?
  if (n <= 1)
    *lda = std::max<int64_t>(m, 1);
}

void THCudaBlas_Sgemv(THCState *state, char trans, int64_t m, int64_t n, float alpha, float *a, int64_t lda, float *x, int64_t incx, float beta, float *y, int64_t incy)
{
  adjustLdLevel2(m, n, &lda);

  hipblasOperation_t op;
  if (trans == 't') op = HIPBLAS_OP_T;
  else if (trans == 'n') op = HIPBLAS_OP_N;
  else if (trans == 'c') op = HIPBLAS_OP_C;
  else THError("Cublas_Sgemv parameter trans should be 't', 'n' or 'c'.");

  if( (m <= INT_MAX) && (n <= INT_MAX) &&
      (lda > 0) && (lda <= INT_MAX) &&
      (incx > 0) && (incx <= INT_MAX) &&
      (incy > 0) && (incy <= INT_MAX) )
  {
    int i_m = (int)m;
    int i_n = (int)n;
    int i_lda = (int)lda;
    int i_incx = (int)incx;
    int i_incy = (int)incy;

    hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
    hipblasSetStream(handle, THCState_getCurrentStream(state));
    THCublasCheck(hipblasSgemv(handle, op, i_m, i_n, &alpha, a, i_lda, x, i_incx, &beta, y, i_incy));
    return;
  }
  THError("Cublas_Sgemv only supports m, n, lda, incx, incy"
          "in the range 0 < [val] <= %d", INT_MAX);
}

void THCudaBlas_Dgemv(THCState *state, char trans, int64_t m, int64_t n, double alpha, double *a, int64_t lda, double *x, int64_t incx, double beta, double *y, int64_t incy)
{
  adjustLdLevel2(m, n, &lda);

  hipblasOperation_t op;
  if (trans == 't') op = HIPBLAS_OP_T;
  else if (trans == 'n') op = HIPBLAS_OP_N;
  else if (trans == 'c') op = HIPBLAS_OP_C;
  else THError("Cublas_Sgemv parameter trans should be 't', 'n' or 'c'.");

  if( (m <= INT_MAX) && (n <= INT_MAX) &&
      (lda > 0) && (lda <= INT_MAX) &&
      (incx > 0) && (incx <= INT_MAX) &&
      (incy > 0) && (incy <= INT_MAX) )
  {
    int i_m = (int)m;
    int i_n = (int)n;
    int i_lda = (int)lda;
    int i_incx = (int)incx;
    int i_incy = (int)incy;

    hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
    hipblasSetStream(handle, THCState_getCurrentStream(state));
    THCublasCheck(hipblasDgemv(handle, op, i_m, i_n, &alpha, a, i_lda, x, i_incx, &beta, y, i_incy));
    return;
  }
  THError("Cublas_Dgemv only supports m, n, lda, incx, incy"
          "in the range 0 < [val] <= %d", INT_MAX);
}

void THCudaBlas_Sger(THCState *state, int64_t m, int64_t n, float alpha, float *x, int64_t incx, float *y, int64_t incy, float *a, int64_t lda)
{
  adjustLdLevel2(m, n, &lda);

  if( (m <= INT_MAX) && (n <= INT_MAX) && (lda <= INT_MAX)  && (incx <= INT_MAX) && (incy <= INT_MAX) )
    {
      int i_m = (int)m;
      int i_n = (int)n;
      int i_lda = (int)lda;
      int i_incx = (int)incx;
      int i_incy = (int)incy;

      hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
      hipblasSetStream(handle, THCState_getCurrentStream(state));
      THCublasCheck(hipblasSger(handle, i_m, i_n, &alpha, x, i_incx, y, i_incy, a, i_lda));
      return;
    }
  THError("Cublas_Sger only supports m, n, lda, incx, incy"
          "with the bound [val] <= %d", INT_MAX);
}

void THCudaBlas_Dger(THCState *state, int64_t m, int64_t n, double alpha, double *x, int64_t incx, double *y, int64_t incy, double *a, int64_t lda)
{
  adjustLdLevel2(m, n, &lda);

  if( (m <= INT_MAX) && (n <= INT_MAX) && (lda <= INT_MAX)  && (incx <= INT_MAX) && (incy <= INT_MAX) )
    {
      int i_m = (int)m;
      int i_n = (int)n;
      int i_lda = (int)lda;
      int i_incx = (int)incx;
      int i_incy = (int)incy;

      hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
      hipblasSetStream(handle, THCState_getCurrentStream(state));
      THCublasCheck(hipblasDger(handle, i_m, i_n, &alpha, x, i_incx, y, i_incy, a, i_lda));
      return;
    }
  THError("Cublas_Dger only supports m, n, lda, incx, incy"
          "with the bound [val] <= %d", INT_MAX);
}


hipblasOperation_t convertTransToCublasOperation(char trans) {
  if (trans == 't') return HIPBLAS_OP_T;
  else if (trans == 'n') return HIPBLAS_OP_N;
  else if (trans == 'c') return HIPBLAS_OP_C;
  else {
    THError("trans must be one of: t, n, c");
    return HIPBLAS_OP_T;
  }
}

void adjustLdLevel3(char transa, char transb, int64_t m, int64_t n, int64_t k, int64_t *lda, int64_t *ldb, int64_t *ldc)
{
  int transa_ = ((transa == 't') || (transa == 'T'));
  int transb_ = ((transb == 't') || (transb == 'T'));

  // Note: leading dimensions generally are checked that they are > 0 and at least as big the result
  // requires (even if the value won't be used).
  if(n <= 1)
    *ldc = std::max<int64_t>(m, 1);

  if(transa_)
  {
    if(m <= 1)
      *lda = std::max<int64_t>(k, 1);
  }
  else
  {
    if(k <= 1)
      *lda = std::max<int64_t>(m, 1);
  }

  if(transb_)
  {
    if(k <= 1)
      *ldb = std::max<int64_t>(n, 1);
  }
  else
  {
    if(n <= 1)
      *ldb = std::max<int64_t>(k, 1);
  }

}

/* Level 3 */
void THCudaBlas_Sgemm(THCState *state, char transa, char transb, int64_t m, int64_t n, int64_t k, float alpha, float *a, int64_t lda, float *b, int64_t ldb, float beta, float *c, int64_t ldc)
{
  adjustLdLevel3(transa, transb, m, n, k, &lda, &ldb, &ldc);
  hipblasOperation_t opa = convertTransToCublasOperation(transa);
  hipblasOperation_t opb = convertTransToCublasOperation(transb);

  if( (m <= INT_MAX) && (n <= INT_MAX) && (k <= INT_MAX) && (lda <= INT_MAX)  && (ldb <= INT_MAX) && (ldc <= INT_MAX) )
  {
    int i_m = (int)m;
    int i_n = (int)n;
    int i_k = (int)k;
    int i_lda = (int)lda;
    int i_ldb = (int)ldb;
    int i_ldc = (int)ldc;

    hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
    hipblasSetStream(handle, THCState_getCurrentStream(state));
    THCublasCheck(hipblasSgemm(handle, opa, opb, i_m, i_n, i_k, &alpha, a, i_lda, b, i_ldb, &beta, c, i_ldc));
    return;
  }
  THError("Cublas_Sgemm only supports m, n, k, lda, ldb, ldc"
          "with the bound [val] <= %d", INT_MAX);
}

// In CUDA 8.0, definition of data types for sgemmex changed
#if CUDA_VERSION < 8000
#  define HIP_R_16F CUBLAS_DATA_HALF
#endif

void THCudaBlas_Hgemm(THCState *state, char transa, char transb, int64_t m, int64_t n, int64_t k, at::Half alpha, at::Half *a, int64_t lda, at::Half *b, int64_t ldb, at::Half beta, at::Half *c, int64_t ldc)
{
  adjustLdLevel3(transa, transb, m, n, k, &lda, &ldb, &ldc);
  hipblasOperation_t opa = convertTransToCublasOperation(transa);
  hipblasOperation_t opb = convertTransToCublasOperation(transb);

  if( (m <= INT_MAX) && (n <= INT_MAX) && (k <= INT_MAX) && (lda <= INT_MAX)  && (ldb <= INT_MAX) && (ldc <= INT_MAX) )
    {
      int i_m = (int)m;
      int i_n = (int)n;
      int i_k = (int)k;
      int i_lda = (int)lda;
      int i_ldb = (int)ldb;
      int i_ldc = (int)ldc;

      hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
      hipblasSetStream(handle, THCState_getCurrentStream(state));

#ifdef __HIP_PLATFORM_HCC__
     float fAlpha = alpha;
     float fBeta = beta;
     THCublasCheck(rocblas_gemm_ex(handle, opa, opb, i_m, i_n, i_k,
                   &fAlpha, a, rocblas_datatype_f16_r, i_lda, b, rocblas_datatype_f16_r,
                   i_ldb, &fBeta, c, rocblas_datatype_f16_r, i_ldc, c, rocblas_datatype_f16_r,
                   i_ldc, rocblas_datatype_f32_r, rocblas_gemm_algo_standard, 0, 0, NULL, NULL));
#else

      // Simulated Hgemm
      float fAlpha = alpha;
      float fBeta = beta;

#if CUDA_VERSION < 9000
      THCublasCheck(cublasSgemmEx(handle, opa, opb,
                                  i_m, i_n, i_k, &fAlpha,
                                  a, HIP_R_16F, i_lda, b, HIP_R_16F,
                                  i_ldb, &fBeta, c, HIP_R_16F, i_ldc));
#else
      hipDeviceProp_t* prop = at::cuda::getCurrentDeviceProperties();
      if (prop->major >= 5){
        THCublasCheck(hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH));
        THCublasCheck(hipblasGemmEx(handle, opa, opb,
                                   i_m, i_n, i_k, &fAlpha,
                                   a, HIP_R_16F, i_lda, b, HIP_R_16F,
                                   i_ldb, &fBeta, c, HIP_R_16F, i_ldc,
                                   HIP_R_32F, CUBLAS_GEMM_DFALT_TENSOR_OP));
        THCublasCheck(hipblasSetMathMode(handle, HIPBLAS_DEFAULT_MATH));
      }else{
        THCublasCheck(cublasSgemmEx(handle, opa, opb,
                                    i_m, i_n, i_k, &fAlpha,
                                    a, HIP_R_16F, i_lda, b, HIP_R_16F,
                                    i_ldb, &fBeta, c, HIP_R_16F, i_ldc));
      }
#endif
#endif
      return;
    }
  THError("Cublas_Hgemm only supports m, n, k, lda, ldb, ldc"
          "with th bound [val] <= %d", INT_MAX);
}

void THCudaBlas_Dgemm(THCState *state, char transa, char transb, int64_t m, int64_t n, int64_t k, double alpha, double *a, int64_t lda, double *b, int64_t ldb, double beta, double *c, int64_t ldc)
{
  adjustLdLevel3(transa, transb, m, n, k, &lda, &ldb, &ldc);
  hipblasOperation_t opa = convertTransToCublasOperation(transa);
  hipblasOperation_t opb = convertTransToCublasOperation(transb);

  if( (m <= INT_MAX) && (n <= INT_MAX) && (k <= INT_MAX) && (lda <= INT_MAX)  && (ldb <= INT_MAX) && (ldc <= INT_MAX) )
  {
    int i_m = (int)m;
    int i_n = (int)n;
    int i_k = (int)k;
    int i_lda = (int)lda;
    int i_ldb = (int)ldb;
    int i_ldc = (int)ldc;

    hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
    hipblasSetStream(handle, THCState_getCurrentStream(state));
    THCublasCheck(hipblasDgemm(handle, opa, opb, i_m, i_n, i_k, &alpha, a, i_lda, b, i_ldb, &beta, c, i_ldc));
    return;
  }
  THError("Cublas_Dgemm only supports m, n, k, lda, ldb, ldc"
          "with the bound [val] <= %d", INT_MAX);
}

#if CUDA_VERSION >= 9010  || defined __HIP_PLATFORM_HCC__
void THCudaBlas_HgemmStridedBatched(THCState *state, char transa, char transb, int64_t m, int64_t n, int64_t k,
                             at::Half alpha, const at::Half *a, int64_t lda, int64_t strideA, const at::Half *b, int64_t ldb, int64_t strideB,
                             at::Half beta, at::Half *c, int64_t ldc, int64_t strideC, int64_t batchCount)
{
  if( (m >= INT_MAX) || (n >= INT_MAX) || (k >= INT_MAX) || (lda >= INT_MAX)  || (ldb >= INT_MAX) || (ldc >= INT_MAX) || (batchCount >= INT_MAX) )

  {
    THError("Cublas_SgemmStridedBatched only supports m, n, k, lda, ldb, ldc, batchCount"
            "with the bound [val] <= %d", INT_MAX);
  }

  adjustLdLevel3(transa, transb, m, n, k, &lda, &ldb, &ldc);
  hipblasOperation_t opa = convertTransToCublasOperation(transa);
  hipblasOperation_t opb = convertTransToCublasOperation(transb);

  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  float fAlpha = alpha;
  float fBeta = beta;
#ifdef __HIP_PLATFORM_HCC__
  THCublasCheck(rocblas_gemm_strided_batched_ex(handle, opa, opb, (int)m, (int)n, (int)k,
                                   (void*)&fAlpha, a, rocblas_datatype_f16_r, (int)lda, strideA,
                                   b, rocblas_datatype_f16_r, (int)ldb, strideB,
                                   (void*)&fBeta, c, rocblas_datatype_f16_r, (int)ldc, strideC,
                                   c, rocblas_datatype_f16_r, (int)ldc, strideC,
                                   (int) batchCount, rocblas_datatype_f32_r, rocblas_gemm_algo_standard,
                                   0, 0, NULL, NULL));
#else
  THCublasCheck(hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH));
  THCublasCheck(hipblasGemmStridedBatchedEx(handle,
                                   opa, opb, (int)m, (int)n, (int)k,
                                   (void*)&fAlpha, a, HIP_R_16F, (int)lda, strideA,
                                   b, HIP_R_16F, (int)ldb, strideB,
                                   (void*)&fBeta, c, HIP_R_16F, (int)ldc, strideC,
                                   (int)batchCount, HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP));
  THCublasCheck(hipblasSetMathMode(handle, HIPBLAS_DEFAULT_MATH));
#endif
}
#endif

void THCudaBlas_SgemmBatched(THCState *state, char transa, char transb, int64_t m, int64_t n, int64_t k,
                             float alpha, const float *a[], int64_t lda, const float *b[], int64_t ldb,
                             float beta, float *c[], int64_t ldc, int64_t batchCount)
{
  if( (m >= INT_MAX) || (n >= INT_MAX) || (k >= INT_MAX) || (lda >= INT_MAX)  || (ldb >= INT_MAX) || (ldc >= INT_MAX) || (batchCount >= INT_MAX) )
  {
    THError("Cublas_SgemmBatched only supports m, n, k, lda, ldb, ldc, batchCount"
            "with the bound [val] <= %d", INT_MAX);
  }

#ifdef __HIP_PLATFORM_HCC__

  const int64_t stridea = (transa == 'N' || transa == 'n') ? lda*k : lda*n;
  const int64_t strideb = (transb == 'N' || transb == 'n') ? ldb*n : ldb*k;
  const int64_t stridec = ldc*n;

  THCudaBlas_SgemmStridedBatched(state, transa, transb, m, n, k, alpha, *a, lda, stridea, *b, ldb, strideb, beta, *c, ldc, stridec, batchCount);

#else

  adjustLdLevel3(transa, transb, m, n, k, &lda, &ldb, &ldc);
  hipblasOperation_t opa = convertTransToCublasOperation(transa);
  hipblasOperation_t opb = convertTransToCublasOperation(transb);

  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  THCublasCheck(hipblasSgemmBatched(handle,
                                   opa, opb, (int)m, (int)n, (int)k,
                                   &alpha, a, (int)lda, b, (int)ldb, &beta, c, (int)ldc,
                                   (int)batchCount));
#endif
}

#if CUDA_VERSION >= 8000 || defined __HIP_PLATFORM_HCC__
void THCudaBlas_SgemmStridedBatched(THCState *state, char transa, char transb, int64_t m, int64_t n, int64_t k,
                             float alpha, const float *a, int64_t lda, int64_t strideA, const float *b, int64_t ldb, int64_t strideB,
                             float beta, float *c, int64_t ldc, int64_t strideC, int64_t batchCount)
{
  if( (m >= INT_MAX) || (n >= INT_MAX) || (k >= INT_MAX) || (lda >= INT_MAX)  || (ldb >= INT_MAX) || (ldc >= INT_MAX) || (batchCount >= INT_MAX) )

  {
    THError("Cublas_SgemmStridedBatched only supports m, n, k, lda, ldb, ldc, batchCount"
            "with the bound [val] <= %d", INT_MAX);
  }

  adjustLdLevel3(transa, transb, m, n, k, &lda, &ldb, &ldc);
  hipblasOperation_t opa = convertTransToCublasOperation(transa);
  hipblasOperation_t opb = convertTransToCublasOperation(transb);

  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  THCublasCheck(hipblasSgemmStridedBatched(handle,
                                   opa, opb, (int)m, (int)n, (int)k,
                                   &alpha, a, (int)lda, strideA, b, (int)ldb, strideB, &beta, c, (int)ldc, strideC,
                                   (int)batchCount));
}
#endif

void THCudaBlas_DgemmBatched(THCState *state, char transa, char transb, int64_t m, int64_t n, int64_t k,
                             double alpha, const double *a[], int64_t lda, const double *b[], int64_t ldb,
                             double beta, double *c[], int64_t ldc, int64_t batchCount)
{
  if( (m >= INT_MAX) || (n >= INT_MAX) || (k >= INT_MAX) || (lda >= INT_MAX)  || (ldb >= INT_MAX) || (ldc >= INT_MAX) || (batchCount >= INT_MAX) )
  {
    THError("Cublas_DgemmBatched only supports m, n, k, lda, ldb, ldc, batchCount"
            "with the bound [val] <= %d", INT_MAX);
  }

#ifdef __HIP_PLATFORM_HCC__

  const int64_t stridea = (transa == 'N' || transa == 'n') ? lda*k : lda*n;
  const int64_t strideb = (transb == 'N' || transb == 'n') ? ldb*n : ldb*k;
  const int64_t stridec = ldc*n;

  THCudaBlas_DgemmStridedBatched(state, transa, transb, m, n, k, alpha, *a, lda, stridea, *b, ldb, strideb, beta, *c, ldc, stridec, batchCount);

#else

  adjustLdLevel3(transa, transb, m, n, k, &lda, &ldb, &ldc);
  hipblasOperation_t opa = convertTransToCublasOperation(transa);
  hipblasOperation_t opb = convertTransToCublasOperation(transb);

  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  THCublasCheck(hipblasDgemmBatched(handle,
                                   opa, opb, (int)m, (int)n, (int)k,
                                   &alpha, a, (int)lda, b, (int)ldb, &beta, c, (int)ldc,
                                   (int)batchCount));
#endif
}

#if CUDA_VERSION >= 8000 || defined __HIP_PLATFORM_HCC__
void THCudaBlas_DgemmStridedBatched(THCState *state, char transa, char transb, int64_t m, int64_t n, int64_t k,
                             double alpha, const double *a, int64_t lda, int64_t strideA, const double *b, int64_t ldb, int64_t strideB,
                             double beta, double *c, int64_t ldc, int64_t strideC, int64_t batchCount)
{
  if( (m >= INT_MAX) || (n >= INT_MAX) || (k >= INT_MAX) || (lda >= INT_MAX)  || (ldb >= INT_MAX) || (ldc >= INT_MAX) || (batchCount >= INT_MAX) )
  {
    THError("Cublas_DgemmBatched only supports m, n, k, lda, ldb, ldc, batchCount"
            "with the bound [val] <= %d", INT_MAX);
  }

  adjustLdLevel3(transa, transb, m, n, k, &lda, &ldb, &ldc);
  hipblasOperation_t opa = convertTransToCublasOperation(transa);
  hipblasOperation_t opb = convertTransToCublasOperation(transb);

  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  THCublasCheck(hipblasDgemmStridedBatched(handle,
                                   opa, opb, (int)m, (int)n, (int)k,
                                   &alpha, a, (int)lda, strideA, b, (int)ldb, strideB, &beta, c, (int)ldc, strideC,
                                   (int)batchCount));
}
#endif

/* Inverse */
void THCudaBlas_Sgetrf(THCState *state, int n, float **a, int lda, int *pivot, int *info, int batchSize) {
#ifndef __HIP_PLATFORM_HCC__
  if( (n >= INT_MAX) || (lda >= INT_MAX) || (batchSize >= INT_MAX) )
  {
    THError("Cublas_Sgetrf only supports n, lda, batchSize"
            "with the bound [val] <= %d", INT_MAX);
  }
  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  THCublasCheck(hipblasSgetrfBatched(handle, n, a, lda, pivot, info, batchSize));
#else
  THError("THCudaBlas_Sgetrf not supported in ROCM.");
#endif
}

void THCudaBlas_Dgetrf(THCState *state, int n, double **a, int lda, int *pivot, int *info, int batchSize) {
#ifndef __HIP_PLATFORM_HCC__
  if( (n >= INT_MAX) || (lda >= INT_MAX) || (batchSize >= INT_MAX) )
  {
    THError("Cublas_Dgetrf only supports n, lda, batchSize"
            "with the bound [val] <= %d", INT_MAX);
  }
  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  THCublasCheck(hipblasDgetrfBatched(handle, n, a, lda, pivot, info, batchSize));
#else
  THError("THCudaBlas_Dgetrf not supported in ROCM.");
#endif
}

void THCudaBlas_Sgetrs(THCState *state, char transa, int n, int nrhs, const float **a, int lda, int *pivot, float **b, int ldb, int *info, int batchSize)
{
#ifndef __HIP_PLATFORM_HCC__
  if( (n >= INT_MAX) || (nrhs >= INT_MAX) || (lda >= INT_MAX) || (ldb >= INT_MAX) || (batchSize >= INT_MAX) )
  {
    THError("Cublas_Dgetrs only supports n, nrhs, lda, ldb, batchSize"
            "with the bound [val] <= %d", INT_MAX);
  }

  // no need to adjust leading dimensions, since matrices are square
  hipblasOperation_t opa = convertTransToCublasOperation(transa);

  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  THCublasCheck(hipblasSgetrsBatched(handle, opa, n, nrhs, a, lda, pivot, b, ldb, info, batchSize));
#else
  THError("THCudaBlas_Sgetrs not supported in ROCM.");
#endif
}


void THCudaBlas_Dgetrs(THCState *state, char transa, int n, int nrhs, const double **a, int lda, int *pivot, double **b, int ldb, int *info, int batchSize)
{
#ifndef __HIP_PLATFORM_HCC__
  if( (n >= INT_MAX) || (nrhs >= INT_MAX) || (lda >= INT_MAX) || (ldb >= INT_MAX) || (batchSize >= INT_MAX) )
  {
    THError("Cublas_Dgetrs only supports n, nrhs, lda, ldb, batchSize"
            "with the bound [val] <= %d", INT_MAX);
  }

  // no need to adjust leading dimensions, since matrices are square
  hipblasOperation_t opa = convertTransToCublasOperation(transa);

  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  THCublasCheck(hipblasDgetrsBatched(handle, opa, n, nrhs, a, lda, pivot, b, ldb, info, batchSize));
#else
  THError("THCudaBlas_Dgetrs not supported in ROCM.");
#endif
}

void THCudaBlas_Sgetri(THCState *state, int n, const float **a, int lda, int *pivot, float **c, int ldc, int *info, int batchSize) {
#ifndef __HIP_PLATFORM_HCC__
  if( (n >= INT_MAX) || (lda >= INT_MAX)|| (ldc >= INT_MAX) || (batchSize >= INT_MAX) )
  {
    THError("Cublas_Sgetri only supports n, lda, ldc, batchSize"
            "with the bound [val] <= %d", INT_MAX);
  }
  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  THCublasCheck(hipblasSgetriBatched(handle, n, a, lda, pivot, c, ldc, info, batchSize));
#else
  THError("THCudaBlas_Sgetri not supported in ROCM.");
#endif
}

void THCudaBlas_Dgetri(THCState *state, int n, const double **a, int lda, int *pivot, double **c, int ldc, int *info, int batchSize) {
#ifndef __HIP_PLATFORM_HCC__
  if( (n >= INT_MAX) || (lda >= INT_MAX)|| (ldc >= INT_MAX) || (batchSize >= INT_MAX) )
  {
    THError("Cublas_Dgetri only supports n, lda, ldc, batchSize"
            "with the bound [val] <= %d", INT_MAX);
  }
  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  THCublasCheck(hipblasDgetriBatched(handle, n, a, lda, pivot, c, ldc, info, batchSize));
#else
  THError("THCudaBlas_Dgetri not supported in ROCM.");
#endif
}

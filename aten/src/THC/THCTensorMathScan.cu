#include "hip/hip_runtime.h"
#include <THC/THCTensorMath.h>
#include <THC/THCGeneral.h>
#include <THC/THCBlas.h>
#include <THC/THCTensorCopy.h>
#include <THC/THCApply.cuh>
#include <THC/THCReduce.cuh>
#include <THC/THCNumerics.cuh>
#include <THC/THCTensorMathReduce.cuh>
#include <thrust/scan.h>
#include <thrust/execution_policy.h>

/* Perform an inclusive scan along an outer dimension of a tensor.
 *
 * - num_orows is the size of the flattened outer dimensions;
 * - num_irows is the size of the flattened inner dimensions;
 * - row_size is the size of the dimension along which to compute the variance;
 *
 * The dimensions to the outside and inside of the specified dimension are considered as flattened.
 * Thread blocks with the same blockIdx.y process an "outer row" (i.e. an element of the flattened
 * outer dimensions, which contains several "inner rows").
 * Each thread processes a single inner row at a time.
 */
template<typename T, class BinaryOp>
__global__ void THCTensor_kernel_scanOuterDim(T *tgt_, T *src_,
                                              unsigned num_orows, unsigned num_irows, unsigned row_size,
                                              T init, BinaryOp binary_op)
{
  for (unsigned orow = blockIdx.x; orow < num_orows; orow += gridDim.x) {
    for (unsigned irow = blockIdx.y * blockDim.x + threadIdx.x; irow < num_irows; irow += gridDim.y * blockDim.x) {
      T *src = src_ + orow * row_size * num_irows + irow;
      T *tgt = tgt_ + orow * row_size * num_irows + irow;
      T acc = init;

      for (unsigned col = 0; col < row_size; ++col) {
        acc = binary_op(acc, *src);
        *tgt = acc;

        src += num_irows;
        tgt += num_irows;
      }
    }
  }
}

/* Perform an inclusive scan along the innermost dimension of a tensor.
 *
 * - num_rows is the size of the flattened outer dimensions;
 * - row_size is the size of the innermost dimension;
 *
 * The outer dimensions of the tensor are considered as a single dimension, i.e. the tensor is
 * considered as having 'num_rows' rows of size 'row_size'.
 * Each thread block processes one or more sets of contiguous rows (processing multiple rows
 * per thread block is quicker than processing a single row, especially for short rows).
 */
template<typename T, int num_threads_x, int num_threads_y, class BinaryFunction>
__global__ void THCTensor_kernel_scanInnermostDim(T *tgt_, T *src_,
                                                  unsigned num_rows, unsigned row_size,
                                                  T init, BinaryFunction binary_op)
{
  __shared__ T sbuf[num_threads_y][2 * num_threads_x];

  T* row_buf = sbuf[threadIdx.y];

  for (unsigned block_row = blockIdx.x * blockDim.y;
       block_row < num_rows;
       block_row += blockDim.y * gridDim.x) {
    unsigned row = block_row + threadIdx.y;
    T block_total = init;

    T *row_src = src_ + row * row_size;
    T *row_tgt = tgt_ + row * row_size;

    // Perform scan on one block at a time, keeping track of the total value of
    // all blocks processed so far.
    for (unsigned block_col = 0; block_col < row_size; block_col += 2 * num_threads_x) {
      // Load data into shared memory (two values per thread).
      unsigned col1 = block_col + threadIdx.x;
      unsigned col2 = block_col + num_threads_x + threadIdx.x;
      if (row < num_rows) {
        if (col1 < row_size) {
          row_buf[threadIdx.x] = row_src[col1];
        } else {
          row_buf[threadIdx.x] = init;
        }

        if (col2 < row_size) {
          row_buf[num_threads_x + threadIdx.x] = row_src[col2];
        } else {
          row_buf[num_threads_x + threadIdx.x] = init;
        }

        // Add the total value of all previous blocks to the first value of this block.
        if (threadIdx.x == 0) {
          row_buf[0] = binary_op(row_buf[0], block_total);
        }
      }
      __syncthreads();

      // Parallel reduction (up-sweep).
      for (unsigned s = num_threads_x, d = 1; s >= 1; s >>= 1, d <<= 1) {
        if (row < num_rows && threadIdx.x < s) {
          unsigned offset = (2 * threadIdx.x + 1) * d - 1;
          row_buf[offset + d] = binary_op(row_buf[offset], row_buf[offset + d]);
        }
        __syncthreads();
      }

      // Down-sweep.
      for (unsigned s = 2, d = num_threads_x / 2; d >= 1; s <<= 1, d >>= 1) {
        if (row < num_rows && threadIdx.x < s - 1) {
          unsigned offset = 2 * (threadIdx.x + 1) * d - 1;
          row_buf[offset + d] = binary_op(row_buf[offset], row_buf[offset + d]);
        }
        __syncthreads();
      }

      // Write back to output.
      if (row < num_rows) {
        if (col1 < row_size) row_tgt[col1] = row_buf[threadIdx.x];
        if (col2 < row_size) row_tgt[col2] = row_buf[num_threads_x + threadIdx.x];
      }
      block_total = row_buf[2 * num_threads_x - 1];
      __syncthreads();
    }
  }
}

#include <THC/generic/THCTensorMathScan.cu>
#include <THC/THCGenerateAllTypes.h>

#include "hip/hip_runtime.h"
#include <THC/THCTensorRandom.h>
#include <THC/THCDeviceUtils.cuh>
#include <THC/THCGeneral.h>
#include <THC/THCTensorCopy.h>
#include <THC/THCTensorMath.h>
#include <THC/THCReduceApplyUtils.cuh>
#include <THC/THCTensorRandom.cuh>
#include <THC/THCGenerator.hpp>
#include <ATen/Config.h>

#include <ATen/cuda/_curand_mtgp32_host.h>

#include <thrust/functional.h>

#define MAX_NUM_BLOCKS 200
#define BLOCK_SIZE 256


THCGenerator* THCRandom_getGenerator(THCState* state);

/* Sets up generator. Allocates but does not create the generator states. Not thread-safe. */
__host__ void initializeGenerator(THCState *state, THCGenerator* gen)
{
  gen->state.gen_states = static_cast<hiprandStateMtgp32*>(THCudaMalloc(state, MAX_NUM_BLOCKS * sizeof(hiprandStateMtgp32)));
  gen->state.kernel_params = static_cast<mtgp32_kernel_params*>(THCudaMalloc(state, sizeof(mtgp32_kernel_params)));
}

/* Creates a new generator state given the seed. Not thread-safe. */
__host__ void createGeneratorState(THCGenerator* gen, uint64_t seed)
{
  if (hiprandMakeMTGP32Constants(mtgp32dc_params_fast_11213, gen->state.kernel_params) != HIPRAND_STATUS_SUCCESS)
  {
    THError("Creating MTGP constants failed.");
  }
  if (hiprandMakeMTGP32KernelState(gen->state.gen_states, mtgp32dc_params_fast_11213,
                                  gen->state.kernel_params, MAX_NUM_BLOCKS, seed) != HIPRAND_STATUS_SUCCESS)
  {
    THError("Creating MTGP kernel state failed.");
  }
  // seed and offset for philox
  gen->state.initial_seed = seed;
  gen->state.philox_seed_offset = 0;
}

THC_API __host__ void THCRandom_getRNGState(THCState* state, THByteTensor *rng_state)
{
  THCGenerator* gen = THCRandom_getGenerator(state);
  std::lock_guard<std::mutex> lock(gen->mutex);

  // The RNG state comprises the MTPG32 states, the seed, and an offset used for Philox
  static const size_t states_size = MAX_NUM_BLOCKS * sizeof(hiprandStateMtgp32);
  static const size_t seed_size = sizeof(gen->state.initial_seed);
  static const size_t offset_size = sizeof(gen->state.philox_seed_offset);
  static const size_t total_size = states_size + seed_size + offset_size;
  THByteTensor_resize1d(rng_state, total_size);
  THArgCheck(THByteTensor_nElement(rng_state) == total_size, 1, "RNG state is wrong size");
  THArgCheck(THByteTensor_isContiguous(rng_state), 1, "RNG state must be contiguous");
  THCudaCheck(hipMemcpy(THByteTensor_data(rng_state), gen->state.gen_states,
                         states_size, hipMemcpyDeviceToHost));
  memcpy(THByteTensor_data(rng_state) + states_size, &gen->state.initial_seed, seed_size);
  memcpy(THByteTensor_data(rng_state) + states_size + seed_size, &gen->state.philox_seed_offset, offset_size);
}

__global__ void set_rngstate_kernel(hiprandStateMtgp32 *state, mtgp32_kernel_params *kernel)
{
#ifndef __HIP_PLATFORM_HCC__
  state[threadIdx.x].k = kernel;
#else
  state[threadIdx.x].set_params(kernel);
#endif
}

THC_API __host__ void THCRandom_setRNGState(THCState* state, THByteTensor *rng_state)
{
  THCGenerator* gen = THCRandom_getGenerator(state);
  std::lock_guard<std::mutex> lock(gen->mutex);

  static const size_t states_size = MAX_NUM_BLOCKS * sizeof(hiprandStateMtgp32);
  static const size_t seed_size = sizeof(gen->state.initial_seed);
  static const size_t offset_size = sizeof(gen->state.philox_seed_offset);
  static const size_t total_size = states_size + seed_size + offset_size;
  bool no_philox_seed = false;
  if (THByteTensor_nElement(rng_state) == total_size - offset_size) {
    no_philox_seed = true;
  }
  else {
    THArgCheck(THByteTensor_nElement(rng_state) == total_size, 1, "RNG state is wrong size");
  }
  THArgCheck(THByteTensor_isContiguous(rng_state), 1, "RNG state must be contiguous");

  THCudaCheck(hipMemcpy(gen->state.gen_states, THByteTensor_data(rng_state),
                         states_size, hipMemcpyHostToDevice));
  set_rngstate_kernel<<<1, MAX_NUM_BLOCKS, 0, THCState_getCurrentStream(state)>>>(
      gen->state.gen_states, gen->state.kernel_params);
  memcpy(&gen->state.initial_seed, THByteTensor_data(rng_state) + states_size, seed_size);
  if (!no_philox_seed) {
    memcpy(&gen->state.philox_seed_offset, THByteTensor_data(rng_state) + states_size + seed_size, offset_size);
  }
  else {
    gen->state.philox_seed_offset = 0;
  }
}

// Goes from (0, 1] to [0, 1). Note 1-x is not sufficient since for some floats
// eps near 0, 1-eps will round to 1.
template <typename T>
__device__ inline T reverse_bounds(T value) {
  if (THCNumerics<T>::eq(value, ScalarConvert<int, T>::to(1))) {
    return ScalarConvert<int, T>::to(0);
  }
  return value;
}


__device__ inline at::Half half_uniform_scale_and_shift(float x, double a, double b) {
  at::Half width = ScalarConvert<double, at::Half>::to(b - a);
  at::Half start = ScalarConvert<double, at::Half>::to(a);
  at::Half scaled = THCNumerics<at::Half>::mul(reverse_bounds(ScalarConvert<float, at::Half>::to(x)), width);
  return THCNumerics<at::Half>::add(scaled, start);
}

#define GENERATE_KERNEL1(NAME, T, ARG1, CURAND_T, CURAND_FUNC, TRANSFORM)      \
__global__ void NAME(hiprandStateMtgp32 *state, int size, T *result, ARG1)    \
{                                                                              \
  int idx = blockIdx.x * BLOCK_SIZE + threadIdx.x;                             \
  int rounded_size = THCCeilDiv(size, BLOCK_SIZE) * BLOCK_SIZE;                \
  for (int i = idx; i < rounded_size; i += BLOCK_SIZE * MAX_NUM_BLOCKS) {      \
    CURAND_T x = CURAND_FUNC(&state[blockIdx.x]);                              \
    if (i < size) {                                                            \
      T y = TRANSFORM;                                                         \
      result[i] = y;                                                           \
    }                                                                          \
  }                                                                            \
}

#define GENERATE_KERNEL2(NAME, T, ARG1, ARG2, CURAND_T, CURAND_FUNC, TRANSFORM)      \
__global__ void NAME(hiprandStateMtgp32 *state, int size, T *result, ARG1, ARG2)    \
{                                                                                    \
  int idx = blockIdx.x * BLOCK_SIZE + threadIdx.x;                                   \
  int rounded_size = THCCeilDiv(size, BLOCK_SIZE) * BLOCK_SIZE;                      \
  for (int i = idx; i < rounded_size; i += BLOCK_SIZE * MAX_NUM_BLOCKS) {            \
    CURAND_T x = CURAND_FUNC(&state[blockIdx.x]);                                    \
    if (i < size) {                                                                  \
      T y = TRANSFORM;                                                               \
      result[i] = y;                                                                 \
    }                                                                                \
  }                                                                                  \
}

// NOTE: hiprand_uniform is (0, 1] and we want [a, b)
GENERATE_KERNEL2(generate_uniform, float, float a, float b, float, hiprand_uniform, reverse_bounds(x) * (b-a) + a)
GENERATE_KERNEL2(generate_uniform, float, double a, double b, float, hiprand_uniform, reverse_bounds(x) * (b-a) + a)
GENERATE_KERNEL2(generate_uniform, double, double a, double b, double, hiprand_uniform_double, reverse_bounds(x) * (b-a) + a)

GENERATE_KERNEL2(generate_normal, float, double mean, double stdv, float, hiprand_normal, (x * stdv) + mean)
GENERATE_KERNEL2(generate_normal, double, double mean, double stdv, double, hiprand_normal_double, (x * stdv) + mean)

GENERATE_KERNEL1(generate_exponential, float, double lambda, float, hiprand_uniform, (float)(-1. / lambda * log(x)))
GENERATE_KERNEL1(generate_exponential, double, double lambda, double, hiprand_uniform_double, (double)(-1. / lambda * log(x)))

GENERATE_KERNEL2(generate_cauchy, float, double median, double sigma, float, hiprand_uniform, (float)(median + sigma * tan(M_PI*(x-0.5))))
GENERATE_KERNEL2(generate_cauchy, double, double median, double sigma, double, hiprand_uniform_double, (double)(median + sigma * tan(M_PI*(x-0.5))))

GENERATE_KERNEL2(generate_uniform, at::Half, double a, double b, float, hiprand_uniform, (half_uniform_scale_and_shift(x, a, b)))
GENERATE_KERNEL2(generate_normal, at::Half, double mean, double stdv, float, hiprand_normal, (ScalarConvert<float, at::Half>::to((x * stdv) + mean)))
GENERATE_KERNEL1(generate_exponential, at::Half, double lambda, float, hiprand_uniform, (ScalarConvert<float, at::Half>::to((float)(-1. / lambda * log(x)))))
GENERATE_KERNEL2(generate_cauchy, at::Half, double median, double sigma, float, hiprand_uniform, (ScalarConvert<float, at::Half>::to((float)(median + sigma * tan(M_PI*(x-0.5))))))

#include <THC/generic/THCTensorRandom.cu>
#include <THC/THCGenerateAllTypes.h>

#include <THC/generic/THCTensorRandom.cu>
#include <THC/THCGenerateBoolType.h>

#undef GENERATE_KERNEL1
#undef GENERATE_KERNEL2

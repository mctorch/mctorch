#include "hip/hip_runtime.h"
#include <THC/THCTensorMath.h>
#include <THC/THCGeneral.h>
#include <TH/THHalf.h>
#include <THC/THCTensorCopy.h>
#include <THC/THCApply.cuh>
#include <THC/THCNumerics.cuh>
#include <THC/THCTensorMathCompareT.cuh>
#include <THC/THCTensor.hpp>

template <typename T>
struct TensorAddConstantOp {
  TensorAddConstantOp(T v) : val(v) {}
  __device__ __forceinline__ void operator()(T* out, T* in) {
    *out = *in + val;
  }

  __device__ __forceinline__ void operator()(T* v) {
    *v += val;
  }

  const T val;
};

template <typename T>
struct TensorSubConstantOp {
  TensorSubConstantOp(T v) : val(v) {}
  __device__ __forceinline__ void operator()(T* out, T* in) {
    *out = *in - val;
  }

  __device__ __forceinline__ void operator()(T* v) {
    *v -= val;
  }

  const T val;
};

template <typename T>
struct TensorMulConstantOp {
  TensorMulConstantOp(T v) : val(v) {}
  __device__ __forceinline__ void operator()(T* out, T* in) {
    *out = *in * val;
  }

  __device__ __forceinline__ void operator()(T* v) {
    *v *= val;
  }

  const T val;
};

template <typename T>
struct TensorDivConstantOp {
  TensorDivConstantOp(T v) : val(v) {}
  __device__ __forceinline__ void operator()(T* out, T* in) {
    *out = *in / val;
  }

  __device__ __forceinline__ void operator()(T* v) {
    *v /= val;
  }

  const T val;
};

template <>
struct TensorDivConstantOp<float> {
  TensorDivConstantOp(float v) : val(1.f / v) {}
  __device__ __forceinline__ void operator()(float* out, float* in) {
    *out = *in * val;
  }

  __device__ __forceinline__ void operator()(float* v) {
    *v *= val;
  }

  const float val;
};

template <>
struct TensorDivConstantOp<double> {
  TensorDivConstantOp(double v) : val(1. / v) {}
  __device__ __forceinline__ void operator()(double* out, double* in) {
    *out = *in * val;
  }

  __device__ __forceinline__ void operator()(double* v) {
    *v *= val;
  }

  const double val;
};

template<typename T>
static __device__ __forceinline__
typename std::enable_if<std::is_signed<T>::value, bool>::type
modulo_wrap(T a, T b) {
  return (a != 0) && (a < 0) != (b < 0);
}

template<typename T>
static __device__ __forceinline__
typename std::enable_if<std::is_unsigned<T>::value, bool>::type
modulo_wrap(T a, T b) {
  return false;
}

template <typename T>
struct TensorRemainderOp {
  TensorRemainderOp(T v) : val(v) {}
  __device__ __forceinline__ void operator()(T* out, T* in) {
    *out = *in % val;
    if (modulo_wrap<T>(*out, val)) {
      *out += val;
    }
  }

  __device__ __forceinline__ void operator()(T* v) {
    *v = *v % val;
    if (modulo_wrap<T>(*v, val)) {
      *v += val;
    }
  }

  const T val;
};

template <>
struct TensorRemainderOp<float> {
  TensorRemainderOp(float v) : val(v) {}
  __device__ __forceinline__ void operator()(float* out, float* in) {
    *out = *in - val * floorf(*in / val);
  }

  __device__ __forceinline__ void operator()(float* v) {
    *v = *v - val * floorf(*v / val);
  }

  const float val;
};

template <>
struct TensorRemainderOp<double> {
  TensorRemainderOp(double v) : val(v) {}
  __device__ __forceinline__ void operator()(double* out, double* in) {
    *out = *in - val * floor(*in / val);
  }

  __device__ __forceinline__ void operator()(double* v) {
    *v = *v - val * floor(*v / val);
  }

  const double val;
};

template <>
struct TensorRemainderOp<at::Half> {
  TensorRemainderOp(at::Half v): val(v) {}

  __device__ __forceinline__ void operator()(at::Half* out, at::Half* in) {
    *out = *in - val * floorf(*in / val);
  }

  __device__ __forceinline__ void operator()(at::Half* v) {
    *v = *v - val * floorf(*v / val);
  }

  const at::Half val;
};

template <typename T>
struct TensorFmodOp {
  TensorFmodOp(T v) : val((float)v) {}
  __device__ __forceinline__ void operator()(T* out, T* in) {
    *out = (T) fmodf((float) *in, val);
  }

  __device__ __forceinline__ void operator()(T* v) {
    *v = (T) fmodf((float) *v, val);
  }

  const float val;
};

template <>
struct TensorFmodOp<double> {
  TensorFmodOp(double v) : val(v) {}
  __device__ __forceinline__ void operator()(double* out, double* in) {
    *out = fmod(*in, val);
  }

  __device__ __forceinline__ void operator()(double* v) {
    *v = fmod(*v, val);
  }

  const double val;
};

template <typename T, int Upper>
struct TensorTriOp {
  TensorTriOp(T *start_, int64_t stride0_, int64_t stride1_, int64_t k_)
    : start(start_), stride0(stride0_), stride1(stride1_), k(k_) {}

  __device__ __forceinline__ int mask(T *out) {
    ptrdiff_t n = out - start;
    int64_t row, col;
    if (stride0 > stride1)
    {
      row = (int64_t) (n / stride0);
      col = (int64_t) ((n % stride0) / stride1);
    }
    else
    {
      row = (int64_t) ((n % stride1) / stride0);
      col = (int64_t) (n / stride1);
    }

    return Upper ? (col - row >= k) : (col - row <= k);
  }

  __device__ __forceinline__ void operator()(T* out, T* in) {
    *out = mask(out) ? *in : ScalarConvert<int, T>::to(0);
  }

  __device__ __forceinline__ void operator()(T* v) {
    if (!mask(v))
      *v = ScalarConvert<int, T>::to(0);
  }

  const T *start;
  const int64_t stride0, stride1, k;
};

template <typename T>
struct TensorLShiftConstantOp {
  TensorLShiftConstantOp(T v) : val(v) {}
  __device__ __forceinline__ void operator()(T* out, T* in) {
    *out = *in << val;
  }

  __device__ __forceinline__ void operator()(T* v) {
    *v <<= val;
  }

  const T val;
};

template <typename T>
struct TensorRShiftConstantOp {
  TensorRShiftConstantOp(T v) : val(v) {}
  __device__ __forceinline__ void operator()(T* out, T* in) {
    *out = *in >> val;
  }

  __device__ __forceinline__ void operator()(T* v) {
    *v >>= val;
  }

  const T val;
};

template <typename T>
struct TensorBitAndConstantOp {
  TensorBitAndConstantOp(T v) : val(v) {}
  __device__ __forceinline__ void operator()(T* out, T* in) {
    *out = *in & val;
  }

  __device__ __forceinline__ void operator()(T* v) {
    *v &= val;
  }

  const T val;
};

template <typename T>
struct TensorBitOrConstantOp {
  TensorBitOrConstantOp(T v) : val(v) {}
  __device__ __forceinline__ void operator()(T* out, T* in) {
    *out = *in | val;
  }

  __device__ __forceinline__ void operator()(T* v) {
    *v |= val;
  }

  const T val;
};

template <typename T>
struct TensorBitXorConstantOp {
  TensorBitXorConstantOp(T v) : val(v) {}
  __device__ __forceinline__ void operator()(T* out, T* in) {
    *out = *in ^ val;
  }

  __device__ __forceinline__ void operator()(T* v) {
    *v ^= val;
  }

  const T val;
};

#include <THC/generic/THCTensorMathPairwise.cu>
#include <THC/THCGenerateAllTypes.h>

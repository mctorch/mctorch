#include "hip/hip_runtime.h"
#include <THCUNN/THCUNN.h>
#include <TH/THHalf.h>
#include <THCUNN/THCHalfAutoNumerics.cuh>
#include <THC/THCAtomics.cuh>
#include <THC/THCTensor.hpp>
#include <THC/THCStorage.hpp>

#define divup(a, b) ((a) + (b) - 1) / (b)
const int THREADS_PER_BLOCK = 256;
const int THREADS_X = 32;
const int THREADS_Y = THREADS_PER_BLOCK / THREADS_X;
const int REPEAT = 32;
const int64_t NNZ_PER_BLOCK_MAX = 1024;

/* sign MACRO */
#ifndef clamp
#define clamp(a, low, high) max(min((a), (high)), (low))
#endif

__device__ double atomicExch(double *address, double val) {
  unsigned long long int* address_as_ull = (unsigned long long int*)address;
  unsigned long long res = atomicExch(address_as_ull, __double_as_longlong(val));
  return __longlong_as_double(res);
}

template<typename Ty, bool train>
__global__ static
void updateOutput(
    Ty *output,
    Ty *normalizedValues,
    const Ty *values,
    const int64_t *cumSumSizes,
    const int64_t *keys,
    const int64_t batchSize,
    const int64_t outDim,
    Ty *weight,
    const Ty *bias,
    const int64_t weightStride,
    const int64_t keysOffset,
    const int maxNormalize,
    const int nnzPerBlock)
{
    /*******************************************************
     * Adapted from the following file in arrayfire
     * https://github.com/arrayfire/arrayfire/blob/v3.4.1/src/backend/opencl/kernel/csrmm.cl
     *
     *******************************************************
     * Original copyright notice can be seen below:
     *
     * Copyright (c) 2016, ArrayFire
     * All rights reserved.
     *
     * This file is distributed under 3-clause BSD license.
     * The complete license agreement can be obtained at:
     * http://arrayfire.com/licenses/BSD-3-Clause
     ********************************************************/

    const int64_t tidx = threadIdx.x;
    const int64_t tidy = threadIdx.y;
    const int64_t tid  = tidy * blockDim.x + tidx;
    const int64_t gidx = blockIdx.x * blockDim.x + tidx;


    Ty *nWeight = weight;
     // Offset the number of elements specified by  maxNormalize
    weight += gidx + maxNormalize;
    output += gidx;

    bool within_N = (gidx < outDim);

    __shared__ Ty s_values[THREADS_PER_BLOCK];
    __shared__ int64_t s_keys[THREADS_PER_BLOCK];

    const int64_t rowId = blockIdx.y;
    // if (rowId >= batchSize) return;

    // Load the nonzero column offsets for current row
    const int64_t batchStart = (rowId == 0 ? 0 : cumSumSizes[rowId - 1]) + blockIdx.z * nnzPerBlock;
    const int64_t batchEnd   = min(batchStart + nnzPerBlock, cumSumSizes[rowId]);
    const int64_t batchStride = blockDim.x * blockDim.y;

    Ty outVal = 0;
    // Since the number of nonzero elements might be greater than local memory available,
    // Load only part of the row into local memory, perform partial dot, repeat until done.
    for (int64_t id = batchStart; id < batchEnd; id += batchStride) {
        // Load the current chunk of the row into local memory
        int64_t lim = min(batchEnd - id, (int64_t)batchStride);

        int64_t key = tid < lim ? keys[id + tid] + keysOffset : -1;
        Ty val = tid < lim ? values[id + tid] : 0;
        int64_t nWeightOffset = key * weightStride;

        if (tid < lim && maxNormalize) {
            Ty *nWeightCurr = nWeight + nWeightOffset;
            if (train) {
                Ty absVal = fabs(val);
                Ty maxVal = nWeightCurr[0];
                if (absVal > maxVal) {
                    // Updating maxVal and invMaxVal. Go hogwild!
                    Ty invAbsVal = 1.0 / absVal;
                    atomicExch(nWeightCurr + 0, absVal);
                    atomicExch(nWeightCurr + 1, invAbsVal);
                }
                val = clamp(val * nWeightCurr[1], -1.0, 1.0) + nWeightCurr[3];
                normalizedValues[id + tid] = val;
                nWeightCurr[2] = 1;
            } else {
                val = clamp(val * nWeightCurr[1], -1.0, 1.0) + nWeightCurr[3];
            }
        }

        s_keys[tid] = key;
        s_values[tid] = val;
        __syncthreads();

        // Perform a single "dot" operation for each thread
        for (int64_t idy = tidy; within_N && idy < lim; idy += blockDim.y) {
            outVal += s_values[idy] * weight[weightStride * s_keys[idy]];
        }
        __syncthreads();
    }

    // s_values is no longer used at this point. Reuse it for reducing outVal.
    // A reduction along the y dimension now gives a single output value along x.
    s_values[tid] = outVal;
    for (int64_t y = blockDim.y / 2; y >= 1; y /= 2) {
        __syncthreads();
        if (tidy < y) s_values[tid] = s_values[tid] + s_values[tid + y * blockDim.x];
    }

    if (within_N && tidy == 0) {
        Ty val = s_values[tid] + (blockIdx.z == 0 ? bias[gidx] : 0);
        if (gridDim.z == 1) {
            output[rowId * outDim] = val;
        } else {
            atomicAdd(output + rowId * outDim, val);
        }
    }
}

// This kernel takes in the following inputs:
// values of size [keysSize x 1] and gradOutput of size [batchSize x outDim],
// to generate gradWeight of size [keysSize x outDim]
// nth block along y dimension computes on the non zero elements from the nth batch.
template<typename Ty>
__global__ static
void accGradWeight(
    Ty *gradWeight,
    const Ty *gradOutput,
    const Ty *values,
    const int64_t  *cumSumSizes,
    const int64_t  outDim,
    const int64_t  gradWeightStride,
    const Ty scale,
    const Ty weightDecay,
    const int maxNormalize)
{
    const int64_t bidy = blockIdx.y;
    const int64_t tidx = threadIdx.x;
    const int64_t tidy = threadIdx.y;
    const int64_t tid  = tidy * blockDim.x + tidx;
    const int64_t ntid = blockDim.x * blockDim.y;
    const int64_t gidx = blockIdx.x * blockDim.x + tidx;

    // All the y threads in the block will use the same gradOutput value
    gradOutput += bidy * outDim;
    Ty gradOutVal = scale * (gidx < outDim ? gradOutput[gidx] : 0);

    // Calculate the amount of work for the current block / batch.
    const int64_t batchStart = bidy == 0 ? 0 : cumSumSizes[bidy - 1];
    const int64_t batchEnd   = cumSumSizes[bidy];
    const int64_t batchLimit = batchEnd - batchStart;

    // Number of iterations required to finish the work for the current batch.
    const int64_t iters    = divup(batchLimit, ntid);

    // Offset the values to the current batch.
    values += batchStart;

    // When maxNormalize is enabled, gradWeight will be twice the size.
    // The first half will contain the gradients required for maxNormalization.
    // The second half will contain the gradients required for updating weights.
    // if maxNormalize is false, both will evaluate to the same pointer.
    Ty *gradWeight0 = gradWeight + batchStart * gradWeightStride + gidx;
    Ty *gradWeight1 = gradWeight0 + (maxNormalize ? outDim : 0);

    __shared__ Ty s_values[THREADS_PER_BLOCK];

    // Using iters to avoid divergence + synchtreads
    for (int64_t n = 0; n < iters; n++) {
        int64_t off = n * ntid;
        int64_t id = off + tid;
        int64_t lim = min(ntid, batchLimit - off);

        // Read the values required for the current iteration.
        s_values[tid] = id < batchLimit ? values[id] : 0;
        __syncthreads();

        if (gidx < outDim) {
            if (maxNormalize) {
                for (int64_t idy = tidy; idy < lim; idy += blockDim.y) {
                    // gradOutVal is already scaled
                    gradWeight0[(off + idy) * gradWeightStride] = gradOutVal;
                }
            }

            for (int64_t idy = tidy; idy < lim; idy += blockDim.y) {
                gradWeight1[(off + idy) * gradWeightStride] = s_values[idy] * gradOutVal;
            }
        }
        __syncthreads();
    }
}

// The gradBias is just a reduction of gradOutput along the batches.
// There is only one block along y dimension performing the reduction.
template<typename Ty, bool update>
__global__ static
void accGradBias(
    Ty *buffer,
    const Ty *gradOutput,
    const int64_t  outDim,
    const int64_t  batchSize,
    const Ty scale,
    const Ty weightDecay)
{
    const int tidx = threadIdx.x;
    const int tidy = threadIdx.y;
    const int tid = tidy * blockDim.x + tidx;
    const int64_t idx = blockIdx.x * blockDim.x + tidx;


    Ty gradBiasVal = 0;
    gradOutput += idx;
    __shared__ Ty s_gradBiasVals[THREADS_PER_BLOCK];

    // Each thread along y calculates the partial sum.
    if (idx < outDim) {
        for (int64_t idy = tidy; idy < batchSize; idy += blockDim.y) {
            gradBiasVal += gradOutput[idy * outDim];
        }
    }
    s_gradBiasVals[tid] = gradBiasVal * scale;
    __syncthreads();

    // Perform reduction is performed along y.
    for (int y = blockDim.y / 2; y >= 1; y /= 2) {
        if (tidy < y) {
            s_gradBiasVals[tid] += s_gradBiasVals[tid + y * blockDim.x];
        }
        __syncthreads();
    }

    // Write the output only from the first lane.
    if (tidy == 0 && idx < outDim) {
        if (update) {
            // If performing inplace update, subtract from bias.
            Ty *bias = buffer;
            bias[idx] = (bias[idx] - s_gradBiasVals[tid]);
        } else {
            // If just accumulating gradients, write to gradBias.
            Ty *gradBias = buffer;
            gradBias[idx] = s_gradBiasVals[tid];
        }
    }
}

// Use gradWeight from accGradWeight to update the weight.
// This kernel is launched batchSize number of times.
// At each step in the iteration, the weights are updated in a sparse manner.
template<typename Ty>
__global__ static
void updateWeight(
    Ty *weight,
    const Ty *gradWeight,
    const int64_t *keys,
    const int64_t *cumSumSizes,
    const int64_t outDim,
    const int64_t gradWeightStride,
    const int64_t weightStride,
    const int64_t keysOffset,
    const Ty learningRate,
    const Ty weightDecay,
    const int maxNormalize,
    const int64_t batchId)
{
    int64_t gidx = blockIdx.x * blockDim.x + threadIdx.x;
    int64_t gidy = blockIdx.y * blockDim.y + threadIdx.y;

    // Find the limits of the work to be done
    const int64_t batchStart = batchId == 0 ? 0 : cumSumSizes[batchId - 1];
    const int64_t batchEnd = cumSumSizes[batchId];

    // When maxNormalize is turned on, the weight tensor will contain
    // an extra "maxNormalize" number of terms per output at the beginning.
    // When maxNormalize is false, both will evaluate to same pointer.
    // when maxNormalize is true,
    // - nWeight[2] will contain the individual scaling factor.
    // - nWeight[3] will contain the individual bias for the normalized input.
    Ty *nWeight = weight;
    weight += maxNormalize + gidx;

    // When maxNormalize is enabled, gradWeight will be twice the size.
    // The first half will contain the gradients required for maxNormalization.
    // The second half will contain the gradients required for updating weights.
    // if maxNormalize is false, both will evaluate to the same pointer.
    const Ty *gradWeight0 = gradWeight + gidx;
    const Ty *gradWeight1 = gradWeight0 + (maxNormalize ? outDim : 0);

    if (gidx >= outDim) return;
    for (int64_t id = batchStart + gidy; id < batchEnd; id += blockDim.y * gridDim.y) {
        Ty lr = learningRate;
        Ty wd = weightDecay;
        int64_t weightOffset = (keys[id] + keysOffset) * weightStride;
        Ty weightVal = weight[weightOffset];

        if (maxNormalize) {
            Ty scale = nWeight[weightOffset + 2];
            lr *= scale;
            wd *= scale;
            // nWeight[3] needs to be updated in the following manner for a given input.
            // nWeight[3] = nWeight[3] - sum(gradWeight0[gidx] * weight[gidx]);
            // Since problem is parallelized along gidx, use atomicAdd for the update.
            Ty gradNormBias = lr * weightVal * gradWeight0[id * gradWeightStride];
            atomicAdd(nWeight + weightOffset + 3, -gradNormBias);
        }

        // Perform the regular update
        Ty gradWeightVal = lr * gradWeight1[id * gradWeightStride];
        if (weightDecay == 0) {
            weight[weightOffset] = weightVal - gradWeightVal;
        } else {
            weight[weightOffset] = weightVal * (1 - wd) - gradWeightVal;
        }
    }
}

// This kernel is launched batchSize number of times.
// At each step in the iteration, the weights are updated in place in a sparse manner.
template<typename Ty>
__global__ static
void accUpdateWeight(
    Ty *weight,
    const int64_t weightStride,
    const Ty *gradOutput,
    const int64_t outDim,
    const Ty *values,
    const int64_t *cumSumSizes,
    const int64_t *keys,
    const int64_t keysOffset,
    const Ty scale,
    const Ty weightDecay,
    const int maxNormalize,
    const int64_t batchId)
{
    // Parallel along outDim.
    int64_t gidx = blockIdx.x * blockDim.x + threadIdx.x;
    // Parallel along the sparse input size for current batch.
    int64_t gidy = blockIdx.y * blockDim.y + threadIdx.y;

    if (gidx >= outDim) return;

    // Find the limits of the work to be done.
    const int64_t batchStart = batchId == 0 ? 0 : cumSumSizes[batchId - 1];
    const int64_t batchEnd = cumSumSizes[batchId];

    gradOutput += batchId * outDim;
    Ty gradOutVal = scale * (gidx < outDim ? gradOutput[gidx] : 0);

    // When maxNormalize is turned on, the weight tensor will contain
    // an extra "maxNormalize" number of terms per output at the beginning.
    // When maxNormalize is false, both will evaluate to same pointer.
    // when maxNormalize is true,
    // - nWeight[2] will contain the individual scaling factor.
    // - nWeight[3] will contain the individual bias for the normalized input.
    Ty *nWeight = weight;
    weight += maxNormalize + gidx;

    for (int64_t id = batchStart + gidy; id < batchEnd; id += blockDim.y * gridDim.y) {
        Ty wd = weightDecay;
        int64_t weightOffset = (keys[id] + keysOffset) * weightStride;
        Ty gradWeightVal = gradOutVal * values[id];
        Ty weightVal = weight[weightOffset];

        if (maxNormalize) {
            Ty nScale = nWeight[weightOffset + 2];
            gradWeightVal *= nScale;
            wd *= nScale;
            // nWeight[3] needs to be updated in the following manner for a given input.
            // nWeight[3] = nWeight[3] - sum(gradOut[gidx] * weight[gidx]);
            // Since problem is parallelized along gidx, use atomicAdd for the update.
            Ty gradNormBias = nScale * weightVal * gradOutVal;
            atomicAdd(nWeight + weightOffset + 3, -gradNormBias);
        }

        // Perform the regular update
        if (weightDecay == 0) {
            weight[weightOffset] = weightVal - gradWeightVal;
        } else {
            weight[weightOffset] = weightVal * (1 - wd) - gradWeightVal;
        }
    }
}


void THNN_CudaHalfIndexLinear_updateOutput(
                  THCState *state,
                  THCudaLongTensor *keys,
                  int64_t keysOffset,
                  THCudaHalfTensor *values,
                  THCudaLongTensor *sizes,
                  THCudaLongTensor *cumSumSizes,
                  THCudaHalfTensor *output,
                  THCudaHalfTensor *weight,
                  THCudaHalfTensor *bias,
                  THCudaHalfTensor *normalizedValues,
                  int   train) {
    THError("THCudaHalfTensor not supported with IndexLinear");
}

void THNN_CudaHalfIndexLinear_accGradParameters(
                  THCState *state,
                  THCudaLongTensor *keys,
                  int64_t keysOffset,
                  THCudaHalfTensor *values,
                  THCudaLongTensor *sizes,
                  THCudaLongTensor *cumSumSizes,
                  THCudaHalfTensor *gradOutput,
                  THCudaHalfTensor *gradWeight,
                  THCudaHalfTensor *gradBias,
                  THCudaHalfTensor *weight,
                  THCudaHalfTensor *bias,
                  THCudaHalfTensor* valuesBuffer,
                  float weightDecay,
                  float scale) {
    THError("THCudaHalfTensor not supported with IndexLinear");
}

void THNN_CudaHalfIndexLinear_accUpdateGradParameters(
                  THCState *state,
                  THCudaLongTensor *keys,
                  int64_t keysOffset,
                  THCudaHalfTensor *values,
                  THCudaLongTensor *sizes,
                  THCudaLongTensor *cumSumSizes,
                  THCudaHalfTensor *gradOutput,
                  THCudaHalfTensor *weight,
                  THCudaHalfTensor *bias,
                  float weightDecay,
                  float scale) {
    THError("THCudaHalfTensor not supported with IndexLinear");
}

void THNN_CudaHalfIndexLinear_updateParameters(
                  THCState *state,
                  THCudaHalfTensor *gradWeight,
                  THCudaHalfTensor *gradBias,
                  THCudaHalfTensor *weight,
                  THCudaHalfTensor *bias,
                  THCudaLongTensor *runningKeys,
                  THCudaLongTensor *cumSumSizes,
                  int64_t keysOffset,
                  float weightDecay,
                  float learningRate) {
    THError("THCudaHalfTensor not supported with IndexLinear");
}

#include <THCUNN/generic/IndexLinear.cu>
#include <THC/THCGenerateFloatType.h>
#include <THCUNN/generic/IndexLinear.cu>
#include <THC/THCGenerateDoubleType.h>

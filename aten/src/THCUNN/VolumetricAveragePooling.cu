#include "hip/hip_runtime.h"
#include <THCUNN/THCUNN.h>
#include <THC/THCTensor.hpp>
#include <THCUNN/common.h>
#include <THC/THCDeviceTensor.cuh>
#include <THC/THCDeviceTensorUtils.cuh>
#include <THC/THCDeviceUtils.cuh>
#include <TH/THHalf.h>
#include <THCUNN/THCHalfAutoNumerics.cuh>
#include <THC/THCAtomics.cuh>

template <typename Dtype, typename Acctype>
__global__ void cuda_VolumetricAveragePooling_updateOutput(
  THCDeviceTensor<Dtype, 4> input,
  THCDeviceTensor<Dtype, 4> output,
  int kT, int kH, int kW,
  int dT, int dH, int dW,
  int padT, int padH, int padW,
  bool count_include_pad, int offsetZ)
{
  int oCol   = blockIdx.x * blockDim.x + threadIdx.x;
  int oRow   = blockIdx.y * blockDim.y + threadIdx.y;
  int oFrame = (blockIdx.z + offsetZ) % output.getSize(1); // output frame/time
  int slice  = (blockIdx.z + offsetZ) / output.getSize(1); // output slice/feature

  if (oRow < output.getSize(2) && oCol < output.getSize(3))
  {
    Acctype sum = 0.0;

    int tstart = oFrame * dT - padT;
    int hstart = oRow   * dH - padH;
    int wstart = oCol   * dW - padW;
    int tend = min(tstart + kT, input.getSize(1) + padT);
    int hend = min(hstart + kH, input.getSize(2) + padH);
    int wend = min(wstart + kW, input.getSize(3) + padW);
    int pool_size = (tend - tstart) * (hend - hstart) * (wend - wstart);
    tstart = max(tstart, 0);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    tend = min(tend, input.getSize(1));
    hend = min(hend, input.getSize(2));
    wend = min(wend, input.getSize(3));

    Acctype divide_factor;
    if (count_include_pad)
      divide_factor = static_cast<Acctype>(pool_size);
    else
      divide_factor = static_cast<Acctype>((tend - tstart) * (hend - hstart) * (wend - wstart));

    int ti, hi, wi;
    for (ti = tstart; ti < tend; ++ti)
    {
      for (hi = hstart; hi < hend; ++hi)
      {
        for (wi = wstart; wi < wend; ++wi)
        {
          Dtype val = input[slice][ti][hi][wi];
          sum += val;
        }
      }
    }

    output[slice][oFrame][oRow][oCol] = ScalarConvert<Acctype, Dtype>::to(sum / divide_factor);
  }
}

// Inner-most loop size (kW) passed as template parameter for
// performance reasons.
//
template<int KERNEL_WIDTH, typename Dtype, typename Acctype>
__global__ void cuda_VolumetricAveragePooling_updateOutput_fixedKW(
  THCDeviceTensor<Dtype, 4> input,
  THCDeviceTensor<Dtype, 4> output,
  int kT, int kH,
  int dT, int dH, int dW,
  int padT, int padH, int padW,
  bool count_include_pad, int offsetZ)
{
  int oCol   = blockIdx.x * blockDim.x + threadIdx.x;
  int oRow   = blockIdx.y * blockDim.y + threadIdx.y;
  int oFrame = (blockIdx.z + offsetZ) % output.getSize(1); // output frame/time
  int slice  = (blockIdx.z + offsetZ) / output.getSize(1); // output slice/feature

  if (oRow < output.getSize(2) && oCol < output.getSize(3))
  {
    Acctype sum = 0.0;

    int tstart = oFrame * dT - padT;
    int hstart = oRow   * dH - padH;
    int wstart = oCol   * dW - padW;
    int tend = min(tstart + kT, input.getSize(1) + padT);
    int hend = min(hstart + kH, input.getSize(2) + padH);
    int wend = min(wstart + KERNEL_WIDTH, input.getSize(3) + padW);
    int pool_size = (tend - tstart) * (hend - hstart) * (wend - wstart);
    tstart = max(tstart, 0);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    tend = min(tend, input.getSize(1));
    hend = min(hend, input.getSize(2));
    wend = min(wend, input.getSize(3));

    Acctype divide_factor;
    if (count_include_pad)
      divide_factor = static_cast<Acctype>(pool_size);
    else
      divide_factor = static_cast<Acctype>((tend - tstart) * (hend - hstart) * (wend - wstart));

    int ti, hi, wi;
    for (ti = tstart; ti < tend; ++ti)
    {
      for (hi = hstart; hi < hend; ++hi)
      {
        for (wi = wstart; wi < wend; ++wi)
        {
          Dtype val = input[slice][ti][hi][wi];
          sum += val;
        }
      }
    }

    output[slice][oFrame][oRow][oCol] = ScalarConvert<Acctype, Dtype>::to(sum / divide_factor);
  }
}

#define LAUNCH_UPDATE_OUTPUT_KERNEL_WIDTH(KW) case KW: \
  cuda_VolumetricAveragePooling_updateOutput_fixedKW<KW, scalar_t, accreal> \
    <<<grid, block, 0, THCState_getCurrentStream(state)>>>( \
      cudaInput, cudaOutput, kT, kH, dT, dH, dW, padT, padH, padW, count_include_pad, offsetZ); \
  break

template <typename Dtype, typename Acctype>
__global__ void cuda_VolumetricAveragePooling_updateGradInput_Stride1(
  THCDeviceTensor<Dtype, 4> gradOutput,
  THCDeviceTensor<Dtype, 4> gradInput,
  int kT, int kH, int kW,
  Acctype normFactor, int offsetZ)
{
  int iCol   = blockIdx.x * blockDim.x + threadIdx.x;
  int iRow   = blockIdx.y * blockDim.y + threadIdx.y;
  int iFrame = (blockIdx.z + offsetZ) % gradInput.getSize(1); // input frame/time
  int slice  = (blockIdx.z + offsetZ) / gradInput.getSize(1); // input slice/feature

  // guard against over-tiled threads
  if (iRow < gradInput.getSize(2) && iCol < gradInput.getSize(3))
  {
    Acctype sum = 0.0;
    Dtype *gOut = &gradOutput[slice][max(0, iFrame - kT + 1)]
      [max(0, iRow - kH + 1)][max(0, iCol - kW + 1)];
    int frameOffset = 0;
    for (int oFrame  = max(0, iFrame - kT + 1);
         oFrame < min(iFrame + 1, gradOutput.getSize(1));
         ++oFrame)
    {
      int rowOffset = frameOffset;
      for (int oRow = max(0, iRow - kH + 1);
           oRow < min(iRow + 1, gradOutput.getSize(2));
           ++oRow)
      {
        int colOffset = rowOffset;
        for (int oCol = max(0, iCol - kW + 1);
             oCol < min(iCol + 1, gradOutput.getSize(3));
             ++oCol)
        {
          sum += gOut[colOffset];
          ++colOffset;
        }
        rowOffset += gradOutput.getSize(3);
      }
      frameOffset += gradOutput.getSize(2) * gradOutput.getSize(3);
    }
    gradInput[slice][iFrame][iRow][iCol] = ScalarConvert<Acctype, Dtype>::to(sum * normFactor);
  }
}

template <typename Dtype, typename Acctype>
__global__ void cuda_VolumetricAveragePooling_updateGradInput_atomicAdd(
  THCDeviceTensor<Dtype, 4> gradOutput,
  THCDeviceTensor<Dtype, 4> gradInput,
  int kT, int kH, int kW,
  int dT, int dH, int dW,
  int padT, int padH, int padW,
  bool count_include_pad, int offsetZ)
{
  int oCol   = blockIdx.x * blockDim.x + threadIdx.x;
  int oRow   = blockIdx.y * blockDim.y + threadIdx.y;
  int oFrame = (blockIdx.z + offsetZ) % gradOutput.getSize(1); // gradOutput frame/time
  int slice  = (blockIdx.z + offsetZ) / gradOutput.getSize(1); // gradOutput slice/feature

  // guard against over-tiled threads
  if (oRow < gradOutput.getSize(2) && oCol < gradOutput.getSize(3))
  {
    int tstart = oFrame * dT - padT;
    int hstart = oRow   * dH - padH;
    int wstart = oCol   * dW - padW;
    int tend = min(tstart + kT, gradInput.getSize(1) + padT);
    int hend = min(hstart + kH, gradInput.getSize(2) + padH);
    int wend = min(wstart + kW, gradInput.getSize(3) + padW);
    int pool_size = (tend - tstart) * (hend - hstart) * (wend - wstart);
    tstart = max(tstart, 0);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    tend = min(tend, gradInput.getSize(1));
    hend = min(hend, gradInput.getSize(2));
    wend = min(wend, gradInput.getSize(3));

    Acctype divide_factor;
    if (count_include_pad)
      divide_factor = static_cast<Acctype>(pool_size);
    else
      divide_factor = static_cast<Acctype>((tend - tstart) * (hend - hstart) * (wend - wstart));

    Dtype val = ScalarConvert<Acctype, Dtype>::to(
      ScalarConvert<Dtype, Acctype>::to(gradOutput[slice][oFrame][oRow][oCol]) / divide_factor);
    for (int iFrame = tstart; iFrame < tend; ++iFrame)
    {
      for (int iRow = hstart; iRow < hend; ++iRow)
      {
        for (int iCol = wstart; iCol < wend; ++iCol)
        {
          atomicAdd(&gradInput[slice][iFrame][iRow][iCol], val);
        }
      }
    }
  }
}

template <typename Dtype, typename Acctype>
__global__ void cuda_VolumetricAveragePooling_updateGradInput(
  THCDeviceTensor<Dtype, 4> gradOutput,
  THCDeviceTensor<Dtype, 4> gradInput,
  int kT, int kH, int kW,
  int dT, int dH, int dW,
  int padT, int padH, int padW,
  bool count_include_pad, int offsetZ)
{
  int oCol   = blockIdx.x * blockDim.x + threadIdx.x;
  int oRow   = blockIdx.y * blockDim.y + threadIdx.y;
  int oFrame = (blockIdx.z + offsetZ) % gradOutput.getSize(1); // gradOutput frame/time
  int slice  = (blockIdx.z + offsetZ) / gradOutput.getSize(1); // gradOutput slice/feature

  // guard against over-tiled threads
  if (oRow < gradOutput.getSize(2) && oCol < gradOutput.getSize(3))
  {
    int tstart = oFrame * dT - padT;
    int hstart = oRow   * dH - padH;
    int wstart = oCol   * dW - padW;
    int tend = min(tstart + kT, gradInput.getSize(1) + padT);
    int hend = min(hstart + kH, gradInput.getSize(2) + padH);
    int wend = min(wstart + kW, gradInput.getSize(3) + padW);
    int pool_size = (tend - tstart) * (hend - hstart) * (wend - wstart);
    tstart = max(tstart, 0);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    tend = min(tend, gradInput.getSize(1));
    hend = min(hend, gradInput.getSize(2));
    wend = min(wend, gradInput.getSize(3));

    Acctype divide_factor;
    if (count_include_pad)
      divide_factor = static_cast<Acctype>(pool_size);
    else
      divide_factor = static_cast<Acctype>((tend - tstart) * (hend - hstart) * (wend - wstart));

    Dtype val = ScalarConvert<Acctype, Dtype>::to(
      ScalarConvert<Dtype, Acctype>::to(gradOutput[slice][oFrame][oRow][oCol]) / divide_factor);
    for (int iFrame = tstart; iFrame < tend; ++iFrame)
    {
      for (int iRow = hstart; iRow < hend; ++iRow)
      {
        for (int iCol = wstart; iCol < wend; ++iCol)
        {
          gradInput[slice][iFrame][iRow][iCol] = val;
        }
      }
    }
  }
}

#include <THCUNN/generic/VolumetricAveragePooling.cu>
#include <THC/THCGenerateFloatTypes.h>

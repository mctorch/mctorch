#include "hip/hip_runtime.h"
#include <THCUNN/THCUNN.h>
#include <TH/THHalf.h>
#include <THCUNN/THCHalfAutoNumerics.cuh>
#include <THC/THCApply.cuh>

#if defined(_MSC_VER) || defined(__HIP_PLATFORM_HCC__)
#define ZERO_MACRO zero<T>()
template <typename T>
inline __device__ typename std::enable_if<std::is_same<T, double>::value, T>::type zero() {
        return 0.;
}

template <typename T>
inline __device__ typename std::enable_if<!std::is_same<T, double>::value, T>::type zero() {
        return 0.f;
}
#else
#define ZERO_MACRO 0.f
#endif

template <typename T>
struct logSigmoid_updateOutput_functor
{
  __device__ void operator()(T *output, const T *input) const {
    const T max = fmaxType(ZERO_MACRO, -*input);
    const T z = THCNumerics<T>::exp(-max) + THCNumerics<T>::exp(-*input -max);
    *output = -(max + THCNumerics<T>::log(z));
  }
};


template <typename T>
struct logSigmoid_updateGradInput_functor
{
  __device__ void operator()(T *gradInput, const T *input, const T *gradOutput) const {
    const T max = fmaxType(ZERO_MACRO, -*input);
    const T z = THCNumerics<T>::exp(-max) + THCNumerics<T>::exp(-*input -max);
    T max_deriv = 0.f;
    T sign = -1.f;
    if (*input < 0.f){
        max_deriv = -1.f;
        sign = 1.f;
    }
    *gradInput = *gradOutput * (-max_deriv - sign*((z - 1.f)/z));
  }
};

template <>
struct logSigmoid_updateOutput_functor<half> {
  __device__ __forceinline__ void operator()(half* output, const half *input) const {
    float in = __half2float(*input);
    float max = fmaxType(0.f, -in);
    float z = THCNumerics<float>::exp(-max) + THCNumerics<float>::exp(-in - max);
    *output = __float2half(-(max + THCNumerics<float>::log(z)));
  }
};

template <>
struct logSigmoid_updateGradInput_functor<half> {
  __device__ __forceinline__ void operator()(half* gradInput, const half *input, const half *gradOutput) const {
    const float in = __half2float(*input);
    const float max = fmaxType(0.f, -in);
    const float z = THCNumerics<float>::exp(-max) + THCNumerics<float>::exp(-in - max);
    const float go = __half2float(*gradOutput);
    float max_deriv = 0.f;
    float sign = -1.f;
    if(in < 0.f){
        max_deriv = -1.f;
        sign = 1.f;
    }
    *gradInput = __float2half(go * (-max_deriv - sign*((z - 1.f)/z)));
  }
};

#include <THCUNN/generic/LogSigmoid.cu>
#include <THC/THCGenerateFloatTypes.h>

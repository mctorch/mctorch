#include "hip/hip_runtime.h"
#include <THCUNN/THCUNN.h>
#include <TH/THHalf.h>
#include <THCUNN/THCHalfAutoNumerics.cuh>
#include <THC/THCApply.cuh>

template <typename T>
struct absupdateOutput_functor
{
  __device__ void operator()(T* output, const T* input) const
  {
    *output = THCNumerics<T>::abs(*input);
  }
};

template <typename T>
struct absupdateGradInput_functor
{
  __device__ void operator()(T* gradInput, const T* input, const T* gradOutput) const
  {
    *gradInput = *input < 0 ? - *gradOutput : *gradOutput;
  }
};

#include <THCUNN/generic/Abs.cu>
#include <THC/THCGenerateFloatTypes.h>

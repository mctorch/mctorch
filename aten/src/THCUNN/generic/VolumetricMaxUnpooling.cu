
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THCUNN/generic/VolumetricMaxUnpooling.cu"
#else

static inline void THNN_(VolumetricMaxUnpooling_shapeCheck)(
                         THCState *state,
                         THCTensor *input,
                         THCTensor *gradOutput,
                         THCIndexTensor *indices,
                         int oT,
                         int oW,
                         int oH,
                         int dT,
                         int dW,
                         int dH,
                         int pT,
                         int pW,
                         int pH) {
  int inputSlices = 0;

  THCUNN_check_shape_indices(state, indices, input);

  THArgCheck(dT > 0 && dW > 0 && dH > 0, 10,
             "stride should be greater than zero, but got dT: %d dH: %d dW: %d",
             dT, dH, dW);

  if (THCTensor_(nDimensionLegacyNoScalars)(state, input) == 4)
  {
    inputSlices = THCTensor_(size)(state, input, 0);
  }
  else if (THCTensor_(nDimensionLegacyNoScalars)(state, input) == 5)
  {
    inputSlices = THCTensor_(size)(state, input, 1);
  }
  else
  {
    AT_ERROR("non-empty 4D or 5D tensor expected, got size: ",
             input->sizes());
  }

  int dimw = 3;
  int dimh = 2;
  int dimt = 1;
  int dimn = 0;
  if (input->dim() == 5)
  {
    dimt++;
    dimw++;
    dimh++;
    dimn++;
  }

  if (gradOutput != NULL) {
    if (oT != gradOutput->size(dimt) || oW != gradOutput->size(dimw) || oH != gradOutput->size(dimh))
    {
      THError(
        "Inconsistent gradOutput size. oT= %d, oH= %d, oW= %d, gradOutput: %dx%dx%d",
        oT, oH, oW, gradOutput->size(dimt), gradOutput->size(dimh), gradOutput->size(dimw));
    }

    THCUNN_check_dim_size(state, gradOutput, input->dim(), dimn, inputSlices);
  }
}

void THNN_(VolumetricMaxUnpooling_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           THCIndexTensor *indices,
           int outputTime, int outputWidth, int outputHeight,
           int dT, int dW, int dH,
           int padT, int padW, int padH)
{
  int batchSize = 0;
  int inputSlices = 0;
  int inputTime = 0;
  int inputHeight = 0;
  int inputWidth = 0;

  THNN_(VolumetricMaxUnpooling_shapeCheck)(
        state, input, NULL, indices,
        outputTime, outputWidth, outputHeight,
        dT, dW, dH, padT, padW, padH);
  THCUNN_assertSameGPU(state, 3, input, indices, output);

  int fiveDimensionalInput = THCTensor_(nDimensionLegacyNoScalars)(state, input) == 5;
  if (THCTensor_(nDimensionLegacyNoScalars)(state, input) == 4)
  {
    /* sizes */
    batchSize   = 1;
    inputSlices = THCTensor_(size)(state, input, 0);
    inputTime   = THCTensor_(size)(state, input, 1);
    inputHeight = THCTensor_(size)(state, input, 2);
    inputWidth  = THCTensor_(size)(state, input, 3);
  }
  else if (fiveDimensionalInput)
  {
    /* sizes */
    batchSize   = THCTensor_(size)(state, input, 0);
    inputSlices = THCTensor_(size)(state, input, 1);
    inputTime   = THCTensor_(size)(state, input, 2);
    inputHeight = THCTensor_(size)(state, input, 3);
    inputWidth  = THCTensor_(size)(state, input, 4);
  }

  if (!fiveDimensionalInput) /* 4D */
  {
    /* resize output */
    THCTensor_(resize4d)(state, output, inputSlices,
                          outputTime, outputHeight, outputWidth);
  }
  else
  { /* 5D */
    THCTensor_(resize5d)(state, output, batchSize, inputSlices,
                          outputTime, outputHeight, outputWidth);
  }

  input = THCTensor_(newContiguous)(state, input);
  indices = THCIndexTensor_(newContiguous)(state, indices);
  output = THCTensor_(newContiguous)(state, output);
  THCTensor_(zero)(state, output);

  if (fiveDimensionalInput) {
    // Collapse batch and feature dimensions
    // newFoldBatchDim assumes contiguity so the newContiguous calls must
    // preceed this
    THCTensor *old_output = output;
    output = THCTensor_(newFoldBatchDim)(state, output);
    THCTensor_(free)(state, old_output);

    THCTensor *old_input = input;
    input = THCTensor_(newFoldBatchDim)(state, input);
    THCTensor_(free)(state, old_input);

    THCIndexTensor *old_indices = indices;
    indices = THCIndexTensor_(newFoldBatchDim)(state, indices);
    THCIndexTensor_(free)(state, old_indices);
  }

  scalar_t* outputData = THCTensor_(data)(state, output);

  THCDeviceTensor<scalar_t, 4> cudaInput;
  THCDeviceTensor<THCIndex_t, 4> cudaIndices;

  cudaInput  = toDeviceTensor<scalar_t, 4>(state, input);
  cudaIndices = toDeviceTensor<THCIndex_t, 4>(state, indices);

  int totalZ = inputTime * inputSlices * batchSize;
  int offsetZ = 0;
  dim3 block(32, 8);

  while (totalZ > 0) {
    dim3 grid(THCCeilDiv(inputWidth, static_cast<int>(block.x)),
              THCCeilDiv(inputHeight, static_cast<int>(block.y)),
              totalZ > 65535 ? 65535 : totalZ);

    cuda_VolumetricMaxUnpooling_updateOutput<<<grid, block,
          0, THCState_getCurrentStream(state)>>>(
                             cudaInput, cudaIndices, outputData,
                             outputTime, outputHeight, outputWidth,
                             dT, dH, dW,
                             padT, padH, padW, offsetZ);
    THCudaCheck(cudaGetLastError());
    totalZ -= 65535;
    offsetZ += 65535;
  }

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, output);
  THCIndexTensor_(free)(state, indices);
}

void THNN_(VolumetricMaxUnpooling_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           THCIndexTensor *indices,
           int outputTime, int outputWidth, int outputHeight,
           int dT, int dW, int dH,
           int padT, int padW, int padH)
{
  int batchSize = 0;
  int inputSlices = 0;
  int inputTime = 0;
  int inputHeight = 0;
  int inputWidth = 0;

  THNN_(VolumetricMaxUnpooling_shapeCheck)(
        state, input, gradOutput, indices,
        outputTime, outputWidth, outputHeight,
        dT, dW, dH, padT, padW, padH);
  THCUNN_assertSameGPU(state, 4, input, indices, gradOutput, gradInput);

  int fiveDimensionalInput = THCTensor_(nDimensionLegacyNoScalars)(state, input) == 5;
  if (!fiveDimensionalInput) /* 4D */
  {
    batchSize = 1;
    inputSlices  = THCTensor_(size)(state, input, 0);
    inputTime   = THCTensor_(size)(state, input, 1);
    inputHeight = THCTensor_(size)(state, input, 2);
    inputWidth  = THCTensor_(size)(state, input, 3);
  }
  else
  {
    batchSize    = THCTensor_(size)(state, input, 0);
    inputSlices  = THCTensor_(size)(state, input, 1);
    inputTime   = THCTensor_(size)(state, input, 2);
    inputHeight = THCTensor_(size)(state, input, 3);
    inputWidth  = THCTensor_(size)(state, input, 4);
  }

  input = THCTensor_(newContiguous)(state, input);
  THCTensor_(resizeAs)(state, gradInput, input);
  THCTensor_(zero)(state, gradInput);
  indices = THCIndexTensor_(newContiguous)(state, indices);
  gradOutput = THCTensor_(newContiguous)(state, gradOutput);

  // Collapse batch and feature dimensions
  if (fiveDimensionalInput) {
    gradInput = THCTensor_(newFoldBatchDim)(state, gradInput);

    THCIndexTensor *old_indices = indices;
    indices = THCIndexTensor_(newFoldBatchDim)(state, indices);
    THCIndexTensor_(free)(state, old_indices);

    THCTensor *old_gradOutput = gradOutput;
    gradOutput = THCTensor_(newFoldBatchDim)(state, gradOutput);
    THCTensor_(free)(state, old_gradOutput);
  } else {
    THCTensor_(retain)(state, gradInput);
  }

  scalar_t* gradOutputData = THCTensor_(data)(state, gradOutput);

  THCDeviceTensor<scalar_t, 4> cudaGradInput;
  THCDeviceTensor<THCIndex_t, 4> cudaIndices;

  cudaGradInput  = toDeviceTensor<scalar_t, 4>(state, gradInput);
  cudaIndices = toDeviceTensor<THCIndex_t, 4>(state, indices);

  int totalZ = inputTime * inputSlices * batchSize;
  int offsetZ = 0;
  dim3 block(32, 8);

  while (totalZ > 0) {
    dim3 grid(THCCeilDiv(inputWidth, static_cast<int>(block.x)),
              THCCeilDiv(inputHeight, static_cast<int>(block.y)),
              totalZ > 65535 ? 65535 : totalZ);

    cuda_VolumetricMaxUnpooling_updateGradInput<<<grid, block,
      0, THCState_getCurrentStream(state)>>>(
                                             gradOutputData,
                                             outputTime, outputHeight, outputWidth,
                                             cudaIndices,
                                             cudaGradInput,
                                             dT, dH, dW,
                                             padT, padH, padW, offsetZ);
    THCudaCheck(cudaGetLastError());
    totalZ -= 65535;
    offsetZ += 65535;
  }

  // cleanup
  THCTensor_(free)(state, gradOutput);
  THCTensor_(free)(state, gradInput);
  THCIndexTensor_(free)(state, indices);
  THCTensor_(free)(state, input);
}

#endif


#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THCUNN/generic/TemporalConvolution.cu"
#else

static inline void THNN_(TemporalConvolution_shapeCheck)(
                         THCState *state,
                         THCTensor *input,
                         int kW,
                         int dW,
                         int *inputFrameSize) {

  THArgCheck(kW > 0, 9,
             "kernel size should be greater than zero, but got kW: %d", kW);
  THArgCheck(dW > 0, 11,
             "stride should be greater than zero, but got dW: %d", dW);

  int dimS = 0; // sequence dimension
  int dimF = 1; // feature dimension

  if (input->dim() == 3)
  {
    dimS = 1;
    dimF = 2;
  }
  THCUNN_argCheck(state, !input->is_empty() && (input->dim() == 2 || input->dim() == 3), 2, input,
                  "non-empty 2D or 3D (batch mode) tensor expected for input, but got: %s");
  if (inputFrameSize != NULL) {
    THArgCheck(input->size(dimF) == *inputFrameSize, 2,
               "invalid input frame size. Got: %d, Expected: %d",
               input->size(dimF), *inputFrameSize);
  }
  THArgCheck(input->size(dimS) >= kW, 2,
             "input sequence smaller than kernel size. Got: %d, Expected: %d",
             input->size(dimS), kW);
}

void THNN_(TemporalConvolution_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           THCTensor *weight,
           THCTensor *bias,
           int kW, int dW,
           int inputFrameSize,
           int outputFrameSize) {

  THCTensor *outputWindow, *inputWindow;
  int nInputFrame, nOutputFrame;
  int64_t k, i;

  int dimS = 0; // sequence dimension

  THCUNN_assertSameGPU(state, 4, input, output, weight, bias);
  THNN_(TemporalConvolution_shapeCheck)
       (state, input, kW, dW, &inputFrameSize);
  THArgCheck(THCTensor_(isContiguous)(state, weight), 4, "weight must be contiguous");
  THArgCheck(!bias || THCTensor_(isContiguous)(state, bias), 5, "bias must be contiguous");

  if (input->dim() == 3)
  {
    dimS = 1;
  }

  input = THCTensor_(newContiguous)(state, input);
  outputWindow = THCTensor_(new)(state);
  inputWindow = THCTensor_(new)(state);

  nInputFrame = input->size(dimS);
  nOutputFrame = (nInputFrame - kW) / dW + 1;

  if (input->dim() == 2)
  {
    THCTensor_(resize2d)(state, output,
                          nOutputFrame,
                          outputFrameSize);

    /* bias first */
    for(k = 0; k < nOutputFrame; k++)
    {
      THCTensor_(select)(state, outputWindow, output, 0, k);
      THCTensor_(copy)(state, outputWindow, bias);
    }


    /* ouch */
    for(k = 0; nOutputFrame > 0; k++)
    {
      int64_t outputFrameStride = (kW-1)/dW+1;
      int64_t inputFrameStride = outputFrameStride*dW;
      int64_t nFrame = (nInputFrame-k*dW-kW)/inputFrameStride + 1;
      nOutputFrame -= nFrame;

      THCTensor_(setStorage2d)(state, inputWindow, THTensor_getStoragePtr(input),
                              input->storage_offset()+k*dW*input->size(1),
                              nFrame, inputFrameStride*input->size(1),
                              kW*input->size(1), 1);

      THCTensor_(setStorage2d)(state, outputWindow, THTensor_getStoragePtr(output),
                              output->storage_offset() + k*output->size(1),
                              nFrame, outputFrameStride*output->size(1),
                              output->size(1), 1);

      THCTensor *tweight = THCTensor_(new)(state);
      THCTensor_(transpose)(state, tweight, weight, 0, 1);
      THCTensor_(addmm)(state, outputWindow, ScalarConvert<int, scalar_t>::to(1), outputWindow, ScalarConvert<int, scalar_t>::to(1), inputWindow, tweight);
      THCTensor_(free)(state, tweight);
    }
  }
  else
  {
    THCTensor *outputSample = THCTensor_(new)(state);
    THCTensor *inputSample = THCTensor_(new)(state);
    int nBatchFrame = input->size(0);

    THCTensor_(resize3d)(state, output,
                          nBatchFrame,
                          nOutputFrame,
                          outputFrameSize);

    for(i = 0; i < nBatchFrame; i++)
    {
      THCTensor_(select)(state, outputSample, output, 0, i);
      THCTensor_(select)(state, inputSample, input, 0, i);
      int64_t nOutputSampleFrame = nOutputFrame;

      /* bias first */
      for(k = 0; k < nOutputFrame; k++)
      {
        THCTensor_(select)(state, outputWindow, outputSample, 0, k);
        THCTensor_(copy)(state, outputWindow, bias);
      }

      /* ouch */
      for(k = 0; nOutputSampleFrame > 0; k++)
      {
        int64_t outputFrameStride = (kW-1)/dW+1;
        int64_t inputFrameStride = outputFrameStride*dW;
        int64_t nFrame = (nInputFrame-k*dW-kW)/inputFrameStride + 1;
        nOutputSampleFrame -= nFrame;

        THCTensor_(setStorage2d)(state, inputWindow, THTensor_getStoragePtr(inputSample),
                                inputSample->storage_offset()+k*dW*inputSample->size(1),
                                nFrame, inputFrameStride*inputSample->size(1),
                                kW*inputSample->size(1), 1);

        THCTensor_(setStorage2d)(state, outputWindow, THTensor_getStoragePtr(outputSample),
                                outputSample->storage_offset() + k*outputSample->size(1),
                                nFrame, outputFrameStride*outputSample->size(1),
                                outputSample->size(1), 1);

        THCTensor *tweight = THCTensor_(new)(state);
        THCTensor_(transpose)(state, tweight, weight, 0, 1);
        THCTensor_(addmm)(state, outputWindow, ScalarConvert<int, scalar_t>::to(1), outputWindow, ScalarConvert<int, scalar_t>::to(1), inputWindow, tweight);
        THCTensor_(free)(state, tweight);
      }
    }
    THCTensor_(free)(state, outputSample);
    THCTensor_(free)(state, inputSample);
  }

  THCTensor_(free)(state, outputWindow);
  THCTensor_(free)(state, inputWindow);
  THCTensor_(free)(state, input);

}

void THNN_(TemporalConvolution_updateGradInput)(
           THCState* state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           THCTensor *weight,
           int kW, int dW) {

  int64_t nInputFrame;
  int64_t nOutputFrame;

  THCTensor *gradOutputWindow;
  THCTensor *gradInputWindow;
  int64_t k, i;

  int dimS = 0; // sequence dimension

  THCUNN_assertSameGPU(state, 4, input, gradOutput, weight, gradInput);
  THArgCheck(THCTensor_(isContiguous)(state, weight), 4, "weight must be contiguous");
  input = THCTensor_(newContiguous)(state, input);
  gradOutput = THCTensor_(newContiguous)(state, gradOutput);

  THNN_(TemporalConvolution_shapeCheck)
       (state, input, kW, dW, NULL);

  if (gradOutput->dim() == 3)
  {
    dimS = 1;
  }

  nInputFrame = input->size(dimS);
  nOutputFrame = gradOutput->size(dimS);


  /* Not necessary with partial backprop: */
  gradOutputWindow = THCTensor_(new)(state);
  gradInputWindow = THCTensor_(new)(state);

  THCTensor_(resizeAs)(state, gradInput, input);
  THCTensor_(zero)(state, gradInput);

  if (gradOutput->dim() == 2)
  {
    /* ouch */
    for(k = 0; nOutputFrame > 0; k++)
    {
      int64_t outputFrameStride = (kW-1)/dW+1;
      int64_t inputFrameStride = outputFrameStride*dW;
      int64_t nFrame = (nInputFrame-k*dW-kW)/inputFrameStride + 1;
      nOutputFrame -= nFrame;

      THCTensor_(setStorage2d)(state, gradOutputWindow, THTensor_getStoragePtr(gradOutput),
                              gradOutput->storage_offset() + k*gradOutput->size(1),
                              nFrame, outputFrameStride*gradOutput->size(1),
                              gradOutput->size(1), 1);

      THCTensor_(setStorage2d)(state, gradInputWindow, THTensor_getStoragePtr(gradInput),
                              gradInput->storage_offset()+k*dW*gradInput->size(1),
                              nFrame, inputFrameStride*gradInput->size(1),
                              kW*gradInput->size(1), 1);

      THCTensor_(addmm)(state, gradInputWindow, ScalarConvert<int, scalar_t>::to(1), gradInputWindow, ScalarConvert<int, scalar_t>::to(1), gradOutputWindow, weight);
    }
  }
  else
  {
    THCTensor *gradOutputSample = THCTensor_(new)(state);
    THCTensor *gradInputSample = THCTensor_(new)(state);
    int64_t nBatchFrame = input->size(0);
    for(i = 0; i < nBatchFrame; i++)
    {
      THCTensor_(select)(state, gradOutputSample, gradOutput, 0, i);
      THCTensor_(select)(state, gradInputSample, gradInput, 0, i);
      int64_t nOutputSampleFrame = nOutputFrame;

      /* ouch */
      for(k = 0; nOutputSampleFrame > 0; k++)
      {
        int64_t outputFrameStride = (kW-1)/dW+1;
        int64_t inputFrameStride = outputFrameStride*dW;
        int64_t nFrame = (nInputFrame-k*dW-kW)/inputFrameStride + 1;
        nOutputSampleFrame -= nFrame;

        THCTensor_(setStorage2d)(state, gradOutputWindow, THTensor_getStoragePtr(gradOutputSample),
                                gradOutputSample->storage_offset() + k*gradOutputSample->size(1),
                                nFrame, outputFrameStride*gradOutputSample->size(1),
                                gradOutputSample->size(1), 1);

        THCTensor_(setStorage2d)(state, gradInputWindow, THTensor_getStoragePtr(gradInputSample),
                                gradInputSample->storage_offset()+k*dW*gradInputSample->size(1),
                                nFrame, inputFrameStride*gradInputSample->size(1),
                                kW*gradInputSample->size(1), 1);

        THCTensor_(addmm)(state, gradInputWindow, ScalarConvert<int, scalar_t>::to(1), gradInputWindow, ScalarConvert<int, scalar_t>::to(1), gradOutputWindow, weight);
      }
    }
    THCTensor_(free)(state, gradOutputSample);
    THCTensor_(free)(state, gradInputSample);
  }

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, gradOutput);
  THCTensor_(free)(state, gradOutputWindow);
  THCTensor_(free)(state, gradInputWindow);

}

void THNN_(TemporalConvolution_accGradParameters)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradWeight,
           THCTensor *gradBias,
           int kW, int dW,
           accreal scale_) {

  scalar_t scale = ScalarConvert<accreal, scalar_t>::to(scale_);
  int64_t nInputFrame;
  int64_t nOutputFrame;

  THCTensor *gradOutputWindow;
  THCTensor *inputWindow;
  int64_t k, i;

  THNN_(TemporalConvolution_shapeCheck)
       (state, input, kW, dW, NULL);

  int dimS = 0; // sequence dimension

  if (gradOutput->dim() == 3)
  {
    dimS = 1;
  }

  nInputFrame = input->size(dimS);
  nOutputFrame = gradOutput->size(dimS);

  /* Not necessary with partial backprop: */
  input = THCTensor_(newContiguous)(state, input);
  gradOutput = THCTensor_(newContiguous)(state, gradOutput);
  gradOutputWindow = THCTensor_(new)(state);
  inputWindow = THCTensor_(new)(state);

  if (input->dim() == 2)
  {
    /* bias first */
    for(k = 0; k < nOutputFrame; k++)
    {
      THCTensor_(select)(state, gradOutputWindow, gradOutput, 0, k);
      THCTensor_(cadd)(state, gradBias, gradBias, scale, gradOutputWindow);
    }

    /* ouch */
    for(k = 0; nOutputFrame > 0; k++)
    {
      int64_t outputFrameStride = (kW-1)/dW+1;
      int64_t inputFrameStride = outputFrameStride*dW;
      int64_t nFrame = (nInputFrame-k*dW-kW)/inputFrameStride + 1;
      nOutputFrame -= nFrame;

      THCTensor_(setStorage2d)(state, inputWindow, THTensor_getStoragePtr(input),
                              input->storage_offset()+k*dW*input->size(1),
                              nFrame, inputFrameStride*input->size(1),
                              kW*input->size(1), 1);

      THCTensor_(setStorage2d)(state, gradOutputWindow, THTensor_getStoragePtr(gradOutput),
                              gradOutput->storage_offset() + k*gradOutput->size(1),
                              nFrame, outputFrameStride*gradOutput->size(1),
                              gradOutput->size(1), 1);

      THCTensor *tgradOutputWindow = THCTensor_(new)(state);
      THCTensor_(transpose)(state, tgradOutputWindow, gradOutputWindow, 0, 1);
      THCTensor_(addmm)(state, gradWeight, ScalarConvert<int, scalar_t>::to(1), gradWeight, scale, tgradOutputWindow, inputWindow);
      THCTensor_(free)(state, tgradOutputWindow);
    }
  }
  else
  {
    THCTensor *gradOutputSample = THCTensor_(new)(state);
    THCTensor *inputSample = THCTensor_(new)(state);
    int64_t nBatchFrame = input->size(0);

    for(i = 0; i < nBatchFrame; i++)
    {
      THCTensor_(select)(state, gradOutputSample, gradOutput, 0, i);
      THCTensor_(select)(state, inputSample, input, 0, i);
      int64_t nOutputSampleFrame = nOutputFrame;

      /* bias first */
      for(k = 0; k < nOutputFrame; k++)
      {
        THCTensor_(select)(state, gradOutputWindow, gradOutputSample, 0, k);
        THCTensor_(cadd)(state, gradBias, gradBias, scale, gradOutputWindow);
      }

      /* ouch */
      for(k = 0; nOutputSampleFrame > 0; k++)
      {
        int64_t outputFrameStride = (kW-1)/dW+1;
        int64_t inputFrameStride = outputFrameStride*dW;
        int64_t nFrame = (nInputFrame-k*dW-kW)/inputFrameStride + 1;
        nOutputSampleFrame -= nFrame;

        THCTensor_(setStorage2d)(state, inputWindow, THTensor_getStoragePtr(inputSample),
                                inputSample->storage_offset()+k*dW*inputSample->size(1),
                                nFrame, inputFrameStride*inputSample->size(1),
                                kW*inputSample->size(1), 1);

        THCTensor_(setStorage2d)(state, gradOutputWindow, THTensor_getStoragePtr(gradOutputSample),
                                gradOutputSample->storage_offset() + k*gradOutputSample->size(1),
                                nFrame, outputFrameStride*gradOutputSample->size(1),
                                gradOutputSample->size(1), 1);

        THCTensor *tgradOutputWindow = THCTensor_(new)(state);
        THCTensor_(transpose)(state, tgradOutputWindow, gradOutputWindow, 0, 1);
        THCTensor_(addmm)(state, gradWeight, ScalarConvert<int, scalar_t>::to(1), gradWeight, scale, tgradOutputWindow, inputWindow);
        THCTensor_(free)(state, tgradOutputWindow);
      }
    }
    THCTensor_(free)(state, gradOutputSample);
    THCTensor_(free)(state, inputSample);
  }

  THCTensor_(free)(state, gradOutputWindow);
  THCTensor_(free)(state, inputWindow);
  THCTensor_(free)(state, gradOutput);
  THCTensor_(free)(state, input);

}

#endif

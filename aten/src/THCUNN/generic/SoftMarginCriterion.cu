
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THCUNN/generic/SoftMarginCriterion.cu"
#else

void THNN_(SoftMarginCriterion_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *target,
           THCTensor *output,
           int64_t reduction)
{
  THCUNN_check_shape(state, input, target);
  THCUNN_assertSameGPU(state, 3, input, target, output);

  if (reduction == Reduction::None) {
    THCTensor_(resizeAs)(state, output, input);
    THC_pointwiseApply3<scalar_t, scalar_t, scalar_t>(state, input, target, output,
        softmargin_no_reduce_functor<scalar_t, accreal>());
    return;
  }

  accreal sum;
  ptrdiff_t size = THCTensor_(nElement)(state, input);

  input = THCTensor_(newContiguous)(state, input);
  target = THCTensor_(newContiguous)(state, target);
  THCTensor_(resize0d)(state, output);

  thrust::device_ptr<scalar_t> input_data(THCTensor_(data)(state, input));
  thrust::device_ptr<scalar_t> target_data(THCTensor_(data)(state, target));
  sum = thrust::inner_product(input_data, input_data+size, target_data, (accreal) 0, thrust::plus<accreal>(), softmargin_functor<scalar_t, accreal>());

  if (reduction == Reduction::Mean)
    sum /= size;

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, target);

  THCTensor_(set0d)(state, output, ScalarConvert<accreal, scalar_t>::to(sum));
}

void THNN_(SoftMarginCriterion_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *target,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           int64_t reduction)
{
  THCUNN_check_shape(state, input, target);
  THCUNN_assertSameGPU(state, 4, input, target, gradInput, gradOutput);

  THCTensor_(resizeAs)(state, gradInput, input);

  if (reduction == Reduction::None) {
    THCUNN_check_shape(state, gradOutput, input);
    THC_pointwiseApply3<scalar_t, scalar_t, scalar_t>(state, input, target, gradInput,
        softmargin_updateGradInput_no_reduce_functor<scalar_t, accreal>());
    THCTensor_(cmul)(state, gradInput, gradInput, gradOutput);
    return;
  }

  ptrdiff_t size = THCTensor_(nElement)(state, input);
  accreal norm = (reduction == Reduction::Mean ? 1./size : 1.);

  input = THCTensor_(newContiguous)(state, input);
  target = THCTensor_(newContiguous)(state, target);


  thrust::device_ptr<scalar_t> input_data(THCTensor_(data)(state, input));
  thrust::device_ptr<scalar_t> target_data(THCTensor_(data)(state, target));
  thrust::device_ptr<scalar_t> gradInput_data(THCTensor_(data)(state, gradInput));

  thrust::transform(input_data, input_data+size, target_data, gradInput_data,
                    softmargin_updateGradInput_functor<scalar_t, accreal>(norm, THCTensor_(get0d)(state, gradOutput)));

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, target);
}

#endif

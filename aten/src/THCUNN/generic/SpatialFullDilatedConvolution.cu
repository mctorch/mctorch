
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THCUNN/generic/SpatialFullDilatedConvolution.cu"
#else

static inline void THNN_(SpatialFullDilatedConvolution_shapeCheck)(
                         THCState *state,
                         THCTensor *input, THCTensor *gradOutput,
                         THCTensor *weight, THCTensor *bias,
                         int kH, int kW, int dH, int dW, int padH, int padW,
                         int dilationH, int dilationW,
                         int adjH, int adjW, int weight_nullable) {
  THArgCheck(kW > 0 && kH > 0, 9,
             "kernel size should be greater than zero, but got kH: %d kW: %d", kH, kW);
  THArgCheck(dW > 0 && dH > 0, 11,
             "stride should be greater than zero, but got dH: %d dW: %d", dH, dW);
  THArgCheck(dilationW > 0 && dilationH > 0, 15,
             "dilation should be greater than zero, but got dilationH: %d, dilationW: %d",
             dilationH, dilationW);
  THArgCheck((adjW < dW || adjW < dilationW) && (adjH < dH || adjH < dilationH), 15,
             "output padding must be smaller than either stride or dilation, but got adjH: %d adjW: %d dH: %d dW: %d dilationH: %d dilationW: %d",
             adjH, adjW, dH, dW, dilationH, dilationW);

  if (weight != NULL) {
    THCUNN_argCheck(state, !weight->is_empty() && (weight->dim() == 2 || weight->dim() == 4), 5, weight,
                    "non-empty 2D or 4D weight tensor expected, but got: %s");
    if (bias != NULL) {
      THCUNN_check_dim_size(state, bias, 1, 0, weight->size(1));
    }
  } else if (!weight_nullable) {
    THError("weight tensor is expected to be non-nullable");
  }

  int ndim = input->dim();
  int dimf = 0;
  int dimh = 1;
  int dimw = 2;

  if (ndim == 4) {
    dimf++;
    dimh++;
    dimw++;
  }

  THCUNN_argCheck(state, !input->is_empty() && (ndim == 3 || ndim == 4), 2, input,
                  "non-empty 3D or 4D input tensor expected but got: %s");

  int64_t inputHeight  = input->size(dimh);
  int64_t inputWidth   = input->size(dimw);
  int64_t outputHeight = (inputHeight - 1) * dH - 2*padH + (dilationH * (kH - 1) + 1) + adjH;
  int64_t outputWidth  = (inputWidth - 1) * dW - 2*padW + (dilationW * (kW - 1) + 1) + adjW;

  if (outputWidth < 1 || outputHeight < 1) {
    THError("Given input size per channel: (%ld x %ld). "
      "Calculated output spatial size per channel: (%ld x %ld). Output size is too small",
      inputHeight, inputWidth, outputHeight, outputWidth);
  }

  if (weight != NULL) {
    int64_t nInputPlane = weight->size(0);
    THCUNN_check_dim_size(state, input, ndim, dimf, nInputPlane);
  }

  if (gradOutput != NULL) {
    if (weight != NULL) {
      int64_t nOutputPlane = weight->size(1);
      THCUNN_check_dim_size(state, gradOutput, ndim, dimf, nOutputPlane);
    } else if (bias != NULL) {
      int64_t nOutputPlane = THTensor_sizeLegacyNoScalars(bias, 0);
      THCUNN_check_dim_size(state, gradOutput, ndim, dimf, nOutputPlane);
    }
    THCUNN_check_dim_size(state, gradOutput, ndim, dimh, outputHeight);
    THCUNN_check_dim_size(state, gradOutput, ndim, dimw, outputWidth);
  }
}

void THNN_(SpatialFullDilatedConvolution_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           THCTensor *weight,
           THCTensor *bias,
           THCTensor *columns,
           THCTensor *ones,
           int kW, int kH,
           int dW, int dH,
           int padW, int padH,
           int dilationW, int dilationH,
           int adjW, int adjH)
{

  int nInputPlane = THCTensor_(size)(state, weight, 0);
  int nOutputPlane = THCTensor_(size)(state, weight, 1);

  THCUNN_assertSameGPU(state, 6, input, output, weight,
                       bias, columns, ones);
  THNN_(SpatialFullDilatedConvolution_shapeCheck)
       (state, input, NULL, weight, bias, kH, kW, dH, dW, padH, padW, dilationH, dilationW, adjH, adjW, 0);

  THArgCheck(!bias || THCTensor_(isContiguous)(state, bias), 5,
             "bias tensor has to be contiguous");
  input = THCTensor_(newContiguous)(state, input);
  weight = THCTensor_(newContiguous)(state, weight);

  int is_batch = 1;
  if (input->dim() == 3) {
    // Force batch
    is_batch = 0;
    THCTensor_(resize4d)(state, input, 1, input->size(0), input->size(1), input->size(2));
  }

  int64_t inputWidth   = input->size(3);
  int64_t inputHeight  = input->size(2);
  int64_t outputHeight = (inputHeight - 1) * dH - 2*padH + (dilationH * (kH - 1) + 1) + adjH;
  int64_t outputWidth  = (inputWidth - 1) * dW - 2*padW + (dilationW * (kW - 1) + 1) + adjW;

  // Batch size + input planes
  int64_t batchSize = input->size(0);

  // Resize output
  THCTensor_(resize4d)(state, output, batchSize, nOutputPlane, outputHeight, outputWidth);

  // Resize temporary columns
  THCTensor_(resize2d)(state, columns, nOutputPlane*kW*kH, inputHeight*inputWidth);

  // Define a buffer of ones, for bias accumulation
  // Note: this buffer can be shared with other modules, it only ever gets increased,
  // and always contains ones.
  if (ones->dim() != 2 || ones->size(0)*ones->size(1) < outputHeight*outputWidth) {
    // Resize plane and fill with ones...
    THCTensor_(resize2d)(state, ones, outputHeight, outputWidth);
    THCTensor_(fill)(state, ones, ScalarConvert<int, scalar_t>::to(1));
  }

  // Helpers
  THCTensor *input_n = THCTensor_(new)(state);
  THCTensor *output_n = THCTensor_(new)(state);

  // For each elt in batch, do:
  for (int elt = 0; elt < batchSize; elt ++) {
    // Matrix mulitply per output:
    THCTensor_(select)(state, input_n, input, 0, elt);
    THCTensor_(select)(state, output_n, output, 0, elt);

    // M,N,K are dims of matrix A and B
    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
    int64_t m = weight->size(1) * weight->size(2) * weight->size(3);
    int64_t n = columns->size(1);
    int64_t k = weight->size(0);

    // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
    #ifdef THC_REAL_IS_FLOAT
    THCudaBlas_Sgemm(
    #elif defined(THC_REAL_IS_HALF)
    THCudaBlas_Hgemm(
    #elif defined(THC_REAL_IS_DOUBLE)
    THCudaBlas_Dgemm(
    #endif
        state,
        'n', 't',
        n, m, k,
        ScalarConvert<int, scalar_t>::to(1),
        THCTensor_(data)(state, input_n), n,
        THCTensor_(data)(state, weight), m,
        ScalarConvert<int, scalar_t>::to(0),
        THCTensor_(data)(state, columns), n
    );

    // Unpack columns back into input:
    col2im<scalar_t, accreal>(
      THCState_getCurrentStream(state),
      THCTensor_(data)(state, columns),
      nOutputPlane, outputHeight, outputWidth, inputHeight, inputWidth, kH, kW, padH, padW, dH, dW,
      dilationH, dilationW, THCTensor_(data)(state, output_n)
    );

    // Do Bias after:
    // M,N,K are dims of matrix A and B
    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
    int64_t m_ = nOutputPlane;
    int64_t n_ = outputHeight * outputWidth;
    int64_t k_ = 1;

    // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
    if (bias) {
      #ifdef THC_REAL_IS_FLOAT
      THCudaBlas_Sgemm(
      #elif defined(THC_REAL_IS_HALF)
      THCudaBlas_Hgemm(
      #elif defined(THC_REAL_IS_DOUBLE)
      THCudaBlas_Dgemm(
      #endif
          state,
          't', 'n',
          n_, m_, k_,
          ScalarConvert<int, scalar_t>::to(1),
          THCTensor_(data)(state, ones), k_,
          THCTensor_(data)(state, bias), k_,
          ScalarConvert<int, scalar_t>::to(1),
          THCTensor_(data)(state, output_n), n_
      );
    }
  }

  // Free
  THCTensor_(free)(state, input_n);
  THCTensor_(free)(state, output_n);

  // Resize output
  if (is_batch == 0) {
    THCTensor_(resize3d)(state, output, nOutputPlane, outputHeight, outputWidth);
    THCTensor_(resize3d)(state, input, nInputPlane, inputHeight, inputWidth);
  }

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, weight);
}

void THNN_(SpatialFullDilatedConvolution_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           THCTensor *weight,
           THCTensor *gradColumns,
           int kW, int kH,
           int dW, int dH,
           int padW, int padH,
           int dilationW, int dilationH,
           int adjW, int adjH)
{
  int nInputPlane = THCTensor_(size)(state, weight, 0);
  int nOutputPlane = THCTensor_(size)(state, weight, 1);

  THCUNN_assertSameGPU(state, 5, input, gradOutput, weight,
                       gradColumns, gradInput);
  THNN_(SpatialFullDilatedConvolution_shapeCheck)
       (state, input, gradOutput, weight, NULL, kH, kW, dH, dW, padH, padW, dilationH, dilationW, adjH, adjW, 0);

  input = THCTensor_(newContiguous)(state, input);
  gradOutput = THCTensor_(newContiguous)(state, gradOutput);
  weight = THCTensor_(newContiguous)(state, weight);

  int is_batch = 1;
  if (input->dim() == 3) {
    // Force batch
    is_batch = 0;
    THCTensor_(resize4d)(state, input, 1, input->size(0), input->size(1), input->size(2));
    THCTensor_(resize4d)(state, gradOutput, 1, gradOutput->size(0), gradOutput->size(1), gradOutput->size(2));
  }

  int64_t inputWidth   = input->size(3);
  int64_t inputHeight  = input->size(2);
  int64_t outputHeight = (inputHeight - 1) * dH - 2*padH + (dilationH * (kH - 1) + 1) + adjH;
  int64_t outputWidth  = (inputWidth - 1) * dW - 2*padW + (dilationW * (kW - 1) + 1) + adjW;

  // Batch size + input planes
  int64_t batchSize = input->size(0);

  // Resize output
  THCTensor_(resize4d)(state, gradInput, batchSize, nInputPlane, inputHeight, inputWidth);

  // Resize temporary columns
  THCTensor_(resize2d)(state, gradColumns, nOutputPlane*kW*kH, inputHeight*inputWidth);

  // Helpers
  THCTensor *gradInput_n = THCTensor_(new)(state);
  THCTensor *gradOutput_n = THCTensor_(new)(state);

  // For each elt in batch, do:
  for (int elt = 0; elt < batchSize; elt ++) {
    // Matrix mulitply per sample:
    THCTensor_(select)(state, gradInput_n, gradInput, 0, elt);
    THCTensor_(select)(state, gradOutput_n, gradOutput, 0, elt);

    // Extract columns:
    im2col(
      THCState_getCurrentStream(state),
      THCTensor_(data)(state, gradOutput_n),
      nOutputPlane, outputHeight, outputWidth,
      inputHeight, inputWidth,
      kH, kW, padH, padW, dH, dW,
      dilationH, dilationW, THCTensor_(data)(state, gradColumns)
    );


    // M,N,K are dims of matrix A and B
    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
    int64_t m = weight->size(0);
    int64_t n = gradColumns->size(1);
    int64_t k = weight->size(1) * weight->size(2) * weight->size(3);

    // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
    #ifdef THC_REAL_IS_FLOAT
    THCudaBlas_Sgemm(
    #elif defined(THC_REAL_IS_HALF)
    THCudaBlas_Hgemm(
    #elif defined(THC_REAL_IS_DOUBLE)
    THCudaBlas_Dgemm(
    #endif
        state,
        'n', 'n',
        n, m, k,
        ScalarConvert<int, scalar_t>::to(1),
        THCTensor_(data)(state, gradColumns), n,
        THCTensor_(data)(state, weight), k,
        ScalarConvert<int, scalar_t>::to(0),
        THCTensor_(data)(state, gradInput_n), n
    );
  }

  // Free
  THCTensor_(free)(state, gradInput_n);
  THCTensor_(free)(state, gradOutput_n);

  // Resize output
  if (is_batch == 0) {
    THCTensor_(resize3d)(state, gradOutput, nOutputPlane, outputHeight, outputWidth);
    THCTensor_(resize3d)(state, input, nInputPlane, inputHeight, inputWidth);
    THCTensor_(resize3d)(state, gradInput, nInputPlane, inputHeight, inputWidth);
  }

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, gradOutput);
  THCTensor_(free)(state, weight);
}


void THNN_(SpatialFullDilatedConvolution_accGradParameters)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradWeight,
           THCTensor *gradBias,
           THCTensor *columns,
           THCTensor *ones,
           int kW, int kH,
           int dW, int dH,
           int padW, int padH,
           int dilationW, int dilationH,
           int adjW, int adjH,
           accreal scale_)
{
  scalar_t scale = ScalarConvert<accreal, scalar_t>::to(scale_);
  THCUNN_assertSameGPU(state, 6, input, gradOutput, gradWeight,
                       gradBias, columns, ones);
  THNN_(SpatialFullDilatedConvolution_shapeCheck)
       (state, input, gradOutput, gradWeight, gradBias, kH, kW, dH, dW,
        padH, padW, dilationH, dilationW, adjH, adjW, 1);

  int nOutputPlane;
  if (gradWeight != NULL) {
    nOutputPlane = THCTensor_(size)(state, gradWeight, 1);
  } else if (gradBias != NULL) {
    nOutputPlane = THCTensor_(sizeLegacyNoScalars)(state, gradBias, 0);
  } else {
    return;
  }

  if (gradWeight) {
    THArgCheck(THCTensor_(isContiguous)(state, gradWeight), 4, "gradWeight needs to be contiguous");
  }
  THArgCheck(THCTensor_(isContiguous)(state, columns), 6, "columns needs to be contiguous");
  if (gradBias) {
    THArgCheck(THCTensor_(isContiguous)(state, gradBias), 5, "gradBias needs to be contiguous");
    THArgCheck(THCTensor_(isContiguous)(state, ones), 7, "ones needs to be contiguous");
  }
  input = THCTensor_(newContiguous)(state, input);
  gradOutput = THCTensor_(newContiguous)(state, gradOutput);

  int is_batch = 1;
  if (input->dim() == 3) {
    // Force batch
    is_batch = 0;
    THCTensor_(resize4d)(state, input, 1, input->size(0), input->size(1), input->size(2));
    THCTensor_(resize4d)(state, gradOutput, 1, gradOutput->size(0), gradOutput->size(1), gradOutput->size(2));
  }

  int64_t inputWidth   = input->size(3);
  int64_t inputHeight  = input->size(2);
  int64_t outputHeight = (inputHeight - 1) * dH - 2*padH + (dilationH * (kH - 1) + 1) + adjH;
  int64_t outputWidth  = (inputWidth - 1) * dW - 2*padW + (dilationW * (kW - 1) + 1) + adjW;

  // Batch size + input planes
  int64_t batchSize = input->size(0);

  // Define a buffer of ones, for bias accumulation
  if (ones->dim() != 2 || ones->size(0)*ones->size(1) < outputHeight*outputWidth) {
    // Resize plane and fill with ones...
    THCTensor_(resize2d)(state, ones, outputHeight, outputWidth);
    THCTensor_(fill)(state, ones, ScalarConvert<int, scalar_t>::to(1));
  }

  // Resize temporary columns
  THCTensor_(resize2d)(state, columns, nOutputPlane*kW*kH, inputHeight*inputWidth);

  // Helpers
  THCTensor *input_n = THCTensor_(new)(state);
  THCTensor *gradOutput_n = THCTensor_(new)(state);

  // For each elt in batch, do:
  for (int elt = 0; elt < batchSize; elt ++) {
    // Matrix mulitply per output:
    THCTensor_(select)(state, gradOutput_n, gradOutput, 0, elt);

    // Do Weight:
    if (gradWeight) {
      // Matrix mulitply per output:
      THCTensor_(select)(state, input_n, input, 0, elt);

      // Extract columns:
      im2col(
        THCState_getCurrentStream(state),
        THCTensor_(data)(state, gradOutput_n),
        nOutputPlane, outputHeight, outputWidth,
        inputHeight, inputWidth,
        kH, kW, padH, padW, dH, dW,
        dilationH, dilationW, THCTensor_(data)(state, columns)
      );

      // M,N,K are dims of matrix A and B
      // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
      int64_t n = columns->size(0);   // nOutputPlane * kh * kw
      int64_t m = input_n->size(0);   // nInputPlane
      int64_t k = columns->size(1);   // inputHeight * inputWidth

      // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
      #ifdef THC_REAL_IS_FLOAT
      THCudaBlas_Sgemm(
      #elif defined(THC_REAL_IS_HALF)
      THCudaBlas_Hgemm(
      #elif defined(THC_REAL_IS_DOUBLE)
      THCudaBlas_Dgemm(
      #endif
          state,
          't', 'n',
          n, m, k,
          scale,
          THCTensor_(data)(state, columns), k,
          THCTensor_(data)(state, input_n), k,
          ScalarConvert<int, scalar_t>::to(1),
          THCTensor_(data)(state, gradWeight), n
      );
    }

    // Do Bias:
    if (gradBias) {
      // M,N,K are dims of matrix A and B
      // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
      int64_t m_ = nOutputPlane;
      int64_t k_ = outputHeight * outputWidth;

      // Do GEMV (note: this is a bit confusing because gemv assumes column-major matrices)
      #if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE)
      #ifdef THC_REAL_IS_FLOAT
      THCudaBlas_Sgemv(
      #elif defined(THC_REAL_IS_DOUBLE)
      THCudaBlas_Dgemv(
      #endif
          state,
          't',
          k_, m_,
          scale,
          THCTensor_(data)(state, gradOutput_n), k_,
          THCTensor_(data)(state, ones), 1,
          ScalarConvert<int, scalar_t>::to(1),
          THCTensor_(data)(state, gradBias), 1
      );
      #endif
      #ifdef THC_REAL_IS_HALF
      THCudaBlas_Hgemm(
          state,
          't', 'n',
          m_, 1, k_,
          scale,
          THCTensor_(data)(state, gradOutput_n), k_,
          THCTensor_(data)(state, ones), k_,
          ScalarConvert<int, scalar_t>::to(1),
          THCTensor_(data)(state, gradBias), m_
      );
      #endif
    }
  }

  // Free
  THCTensor_(free)(state, input_n);
  THCTensor_(free)(state, gradOutput_n);

  // Resize
  if (is_batch == 0) {
    THCTensor_(resize3d)(state, gradOutput, nOutputPlane, outputHeight, outputWidth);
    THCTensor_(resize3d)(state, input, input->size(1), inputHeight, inputWidth);
  }

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, gradOutput);
}

#endif

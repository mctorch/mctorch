
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THCUNN/generic/VolumetricFullConvolution.cu"
#else

void THNN_(VolumetricFullConvolution_updateOutput)(
       THCState *state,
       THCTensor  *input,
       THCTensor  *output,
       THCTensor  *weight,
       THCTensor  *bias,
       THCTensor  *finput,
       THCTensor  *fgradInput,
       int kT, int kW, int kH,
       int dT, int dW, int dH,
       int padT, int padW, int padH,
       int adjT, int adjW, int adjH)
{
  THNN_(VolumetricFullDilatedConvolution_updateOutput)(
       state, input, output, weight, bias, finput, fgradInput,
       kT, kW, kH, dT, dW, dH, padT, padW, padH, 1, 1, 1, adjT, adjW, adjH);
}

void THNN_(VolumetricFullConvolution_updateGradInput)(
       THCState *state,
       THCTensor  *input,
       THCTensor  *gradOutput,
       THCTensor  *gradInput,
       THCTensor  *weight,
       THCTensor  *finput,
       THCTensor  *fgradInput,
       int kT, int kW, int kH,
       int dT, int dW, int dH,
       int padT, int padW, int padH,
       int adjT, int adjW, int adjH)
{
  THNN_(VolumetricFullDilatedConvolution_updateGradInput)(
       state, input, gradOutput, gradInput, weight, finput, fgradInput,
       kT, kW, kH, dT, dW, dH, padT, padW, padH, 1, 1, 1, adjT, adjW, adjH);
}


void THNN_(VolumetricFullConvolution_accGradParameters)(
           THCState *state,
           THCTensor  *input,
           THCTensor  *gradOutput,
           THCTensor  *gradWeight,
           THCTensor  *gradBias,
           THCTensor  *finput,
           THCTensor  *fgradInput,
           int kT, int kW, int kH,
           int dT, int dW, int dH,
           int padT, int padW, int padH,
           int adjT, int adjW, int adjH,
           accreal scale_)
{
  THNN_(VolumetricFullDilatedConvolution_accGradParameters)(
       state, input, gradOutput, gradWeight, gradBias, finput, fgradInput,
       kT, kW, kH, dT, dW, dH, padT, padW, padH, 1, 1, 1, adjT, adjW, adjH, scale_);
}

#endif


#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THCUNN/generic/BCECriterion.cu"
#else

void THNN_(BCECriterion_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *target,
           THCTensor *output,
           int64_t reduction,
           THCTensor *weights)
{
  THCUNN_check_nElement(state, input, target);
  THCUNN_check_nElement(state, input, weights);
  THCUNN_assertSameGPU(state, 3, input, target, weights);

  if (reduction == Reduction::None) {
    THCTensor_(resizeAs)(state, output, input);
    THC_pointwiseApply3<scalar_t, scalar_t, scalar_t>(state, input, target, output,
        bce_updateOutput_no_reduce_functor<scalar_t, accreal>());
    if (weights) {
      THCTensor_(cmul)(state, output, output, weights);
    }
    return;
  }

  THCTensor_(resize0d)(state, output);
  ptrdiff_t size = THCTensor_(nElement)(state, input);

  input = THCTensor_(newContiguous)(state, input);
  target = THCTensor_(newContiguous)(state, target);
  THCThrustAllocator thrustAlloc(state);
  thrust::device_ptr<scalar_t> input_data(THCTensor_(data)(state, input));
  thrust::device_ptr<scalar_t> target_data(THCTensor_(data)(state, target));

  accreal sum;
  if (weights) {
    weights = THCTensor_(newContiguous)(state, weights);
    thrust::device_ptr<scalar_t> weights_data(THCTensor_(data)(state, weights));
    sum = thrust::transform_reduce(
      thrust::cuda::par(thrustAlloc).on(THCState_getCurrentStream(state)),
      thrust::make_zip_iterator(thrust::make_tuple(input_data, target_data, weights_data)),
      thrust::make_zip_iterator(thrust::make_tuple(input_data+size, target_data+size, weights_data+size)),
      bce_functor_weights<scalar_t, accreal>(),
      (accreal) 0,
      thrust::plus<accreal>()
    );
    THCTensor_(free)(state, weights);
  } else {
    sum = thrust::transform_reduce(
      thrust::cuda::par(thrustAlloc).on(THCState_getCurrentStream(state)),
      thrust::make_zip_iterator(thrust::make_tuple(input_data, target_data)),
      thrust::make_zip_iterator(thrust::make_tuple(input_data+size, target_data+size)),
      bce_functor<scalar_t, accreal>(),
      (accreal) 0,
      thrust::plus<accreal>()
    );
  }

  if (reduction == Reduction::Mean)
    sum /= size;

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, target);

  THCTensor_(set0d)(state, output, ScalarConvert<accreal, scalar_t>::to(sum));
}

void THNN_(BCECriterion_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *target,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           int64_t reduction,
           THCTensor *weights)
{
  THCUNN_check_nElement(state, input, target);
  THCUNN_check_nElement(state, input, weights);
  THCUNN_assertSameGPU(state, 4, input, target, gradInput, weights);

  THCTensor_(resizeAs)(state, gradInput, input);

  if (reduction == Reduction::None) {
    THCUNN_check_nElement(state, gradOutput, input);
    THC_pointwiseApply3<scalar_t, scalar_t, scalar_t>(state, input, target, gradInput,
        bce_updateGradInput_no_reduce_functor<scalar_t, accreal>());
    THCTensor_(cmul)(state, gradInput, gradInput, gradOutput);
    if (weights) {
      THCTensor_(cmul)(state, gradInput, gradInput, weights);
    }
    return;
  }

  THCUNN_check_dim_size(state, gradOutput, 1, 0, 1);

  ptrdiff_t size = THCTensor_(nElement)(state, input);
  scalar_t norm = ScalarConvert<accreal, scalar_t>::to((reduction == Reduction::Mean ? accreal(1)/size : accreal(1)) * THCTensor_(get0d)(state, gradOutput));

  input = THCTensor_(newContiguous)(state, input);
  target = THCTensor_(newContiguous)(state, target);

  thrust::device_ptr<scalar_t> input_data(THCTensor_(data)(state, input));
  thrust::device_ptr<scalar_t> target_data(THCTensor_(data)(state, target));
  thrust::device_ptr<scalar_t> gradInput_data(THCTensor_(data)(state, gradInput));

  if (weights) {
    weights = THCTensor_(newContiguous)(state, weights);
    thrust::device_ptr<scalar_t> weights_data(THCTensor_(data)(state, weights));
    thrust::transform(
      thrust::make_zip_iterator(thrust::make_tuple(input_data, target_data, weights_data)),
      thrust::make_zip_iterator(thrust::make_tuple(input_data+size, target_data+size, weights_data+size)),
      gradInput_data,
      bce_updateGradInput_functor_weights<scalar_t, accreal>(norm)
    );
    THCTensor_(free)(state, weights);
  } else {
    thrust::transform(
      thrust::make_zip_iterator(thrust::make_tuple(input_data, target_data)),
      thrust::make_zip_iterator(thrust::make_tuple(input_data+size, target_data+size)),
      gradInput_data,
      bce_updateGradInput_functor<scalar_t, accreal>(norm)
    );
  }

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, target);
}

#endif

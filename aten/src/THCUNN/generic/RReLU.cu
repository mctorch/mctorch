
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THCUNN/generic/RReLU.cu"
#else

#include <THCUNN/common.h>

void THNN_(RReLU_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           THCTensor *noise,
           double lower,
           double upper,
           bool train,
           bool inplace,
           void *generator)
{
  THCUNN_assertSameGPU(state, 3, input, output, noise);
  curandStateMtgp32* gen_states = THCRandom_generatorStates(state);

  if (train)
  {
    input = THCTensor_(newContiguous)(state, input);
    THCTensor_(resizeAs)(state, noise, input);
    scalar_t *input_data = THCTensor_(data)(state, input);
    scalar_t *noise_data = THCTensor_(data)(state, noise);
    ptrdiff_t n = THCTensor_(nElement)(state, input);
    if (inplace)
    {
      rreluUpdateOutputTrain<<<NUM_BLOCKS(n), BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
        n, gen_states, input_data, noise_data, input_data, lower, upper);
      THCTensor_(set)(state, output, input);
    }
    else
    {
      THCTensor_(resizeAs)(state, output, input);
      scalar_t *output_data = THCTensor_(data)(state, output);
      rreluUpdateOutputTrain<<<NUM_BLOCKS(n), BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
        n, gen_states, input_data, noise_data, output_data, lower, upper);
    }
    THCudaCheck(cudaGetLastError());
    THCTensor_(free)(state, input);
  }
  else
  {
    const scalar_t negSlope = ScalarConvert<double, scalar_t>::to((lower + upper) / 2);
    if (inplace)
    {
      THC_pointwiseApply1<scalar_t>(state, input, RReLUUpdateOutputEvalIP_functor<scalar_t>(negSlope));
      THCTensor_(set)(state, output, input);
    }
    else
    {
      THCTensor_(resizeAs)(state, output, input);
      THC_pointwiseApply2<scalar_t, scalar_t>(state, output, input, RReLUUpdateOutputEval_functor<scalar_t>(negSlope));
    }
  }
}

void THNN_(RReLU_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           THCTensor *noise,
           double lower,
           double upper,
           bool train,
           bool inplace)
{
  THCUNN_check_nElement(state, input, gradOutput);
  THCUNN_assertSameGPU(state, 4, input, gradOutput, gradInput, noise);

  gradOutput = THCTensor_(newContiguous)(state, gradOutput);

  if (train && upper - lower > 1E-6)    // e.g. if upper == lower, RReLU behaves like LeakyReLU
  {
    // multiply the gradient by the noise tensor
    if (inplace)
    {
      THCTensor_(cmul)(state, gradOutput, gradOutput, noise);
      THCTensor_(set)(state, gradInput, gradOutput);
    }
    else
    {
      THCTensor_(resizeAs)(state, gradInput, input);
      THCTensor_(cmul)(state, gradInput, gradOutput, noise);
    }
  }
  else
  {
    // use constant factor for negative input values
    const scalar_t negSlope = ScalarConvert<double, scalar_t>::to((lower + upper) / 2);
    if (inplace)
    {
      THC_pointwiseApply2<scalar_t, scalar_t>(state, gradOutput, input, RReLUupdateGradInputEvalIP_functor<scalar_t>(negSlope));
      THCTensor_(set)(state, gradInput, gradOutput);
    }
    else
    {
      THCTensor_(resizeAs)(state, gradInput, input);
      THC_pointwiseApply3<scalar_t, scalar_t, scalar_t>(state, gradInput, gradOutput, input, RReLUupdateGradInputEval_functor<scalar_t>(negSlope));
    }
  }

  THCTensor_(free)(state, gradOutput);
}

#endif

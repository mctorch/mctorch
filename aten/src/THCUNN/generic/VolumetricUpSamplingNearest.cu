
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THCUNN/generic/VolumetricUpSamplingNearest.cu"
#else

#include <THCUNN/common.h>
#include "ATen/cuda/CUDAContext.h"

static inline void THNN_(VolumetricUpSamplingNearest_shapeCheck)
                        (THCState *state,
                         THCTensor *input, THCTensor *gradOutput,
                         int nBatch, int nChannels,
                         int inputDepth, int inputHeight, int inputWidth,
                         int outputDepth, int outputHeight, int outputWidth) {
  THArgCheck(inputDepth > 0 && inputHeight > 0 && inputWidth > 0
             && outputDepth && outputHeight > 0 && outputWidth > 0, 2,
             "input and output sizes should be greater than 0,"
             " but got input (D: %d, H: %d, W: %d) output (D: %d, H: %d, W: %d)",
             inputDepth, inputHeight, inputWidth, outputDepth, outputHeight, outputWidth);
  if (input != NULL) {
     THCUNN_argCheck(state, THTensor_nDimensionLegacyAll(input) == 5, 2, input,
                     "5D input tensor expected but got: %s");
  }

  if (gradOutput != NULL) {
    THCUNN_check_dim_size(state, gradOutput, 5, 0, nBatch);
    THCUNN_check_dim_size(state, gradOutput, 5, 1, nChannels);
    THCUNN_check_dim_size(state, gradOutput, 5, 2, outputDepth);
    THCUNN_check_dim_size(state, gradOutput, 5, 3, outputHeight);
    THCUNN_check_dim_size(state, gradOutput, 5, 4, outputWidth);
  }
}


void THNN_(VolumetricUpSamplingNearest_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           int outputDepth,
           int outputHeight,
           int outputWidth)
{
  THCUNN_assertSameGPU(state, 2, input, output);
  int nbatch = THCTensor_(size)(state, input, 0);
  int channels = THCTensor_(size)(state, input, 1);
  int inputDepth = THCTensor_(size)(state, input, 2);
  int inputHeight = THCTensor_(size)(state, input, 3);
  int inputWidth  = THCTensor_(size)(state, input, 4);

  THNN_(VolumetricUpSamplingNearest_shapeCheck)(state, input, NULL, nbatch, channels,
                  inputDepth, inputHeight, inputWidth,
                  outputDepth, outputHeight, outputWidth);
  THAssert(inputDepth > 0 && inputHeight > 0 && inputWidth > 0 &&
                  outputDepth > 0 && outputHeight > 0 && outputWidth > 0);

  THCTensor_(resize5d)(state, output,
                       THCTensor_(size)(state, input, 0),
                       THCTensor_(size)(state, input, 1),
                       outputDepth,
                       outputHeight,
                       outputWidth);
  THCTensor_(zero)(state, output);

  THCDeviceTensor<scalar_t, 5> idata = toDeviceTensor<scalar_t, 5>(state, input);
  THCDeviceTensor<scalar_t, 5> odata = toDeviceTensor<scalar_t, 5>(state, output);

  const int num_kernels = outputDepth * outputHeight * outputWidth;
  const int num_threads = at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock;
  cudaStream_t stream = THCState_getCurrentStream(state);
  nearest_neighbor_5d_kernel<scalar_t, accreal> <<<THCCeilDiv(num_kernels, num_threads), num_threads,
         0, stream>>>(num_kernels, idata, odata);
  THCudaCheck(cudaGetLastError());
}



void THNN_(VolumetricUpSamplingNearest_updateGradInput)(
           THCState *state,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           int nbatch,
           int nchannels,
           int inputDepth,
           int inputHeight,
           int inputWidth,
           int outputDepth,
           int outputHeight,
           int outputWidth)
{
  THCUNN_assertSameGPU(state, 2, gradOutput, gradInput);
  THNN_(VolumetricUpSamplingNearest_shapeCheck)(state, NULL, gradOutput, nbatch, nchannels,
                  inputDepth, inputHeight, inputWidth,
                  outputDepth, outputHeight, outputWidth);
  gradOutput = THCTensor_(newContiguous)(state, gradOutput);
  THCTensor_(resize5d)(state, gradInput, nbatch, nchannels, inputDepth, inputHeight, inputWidth);

  THCTensor_(zero)(state, gradInput);
  THCDeviceTensor<scalar_t, 5> data1 = toDeviceTensor<scalar_t, 5>(state, gradInput);
  THCDeviceTensor<scalar_t, 5> data2 = toDeviceTensor<scalar_t, 5>(state, gradOutput);
  const int num_kernels = outputDepth * outputHeight * outputWidth;
  const int num_threads = at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock;
  cudaStream_t stream = THCState_getCurrentStream(state);
  nearest_neighbor_5d_kernel_backward<scalar_t, accreal> <<<THCCeilDiv(num_kernels, num_threads),
          num_threads, 0, stream>>>(num_kernels, data1, data2);
  THCudaCheck(cudaGetLastError());
  THCTensor_(free)(state, gradOutput);
}

#endif

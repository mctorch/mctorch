
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THCUNN/generic/LookupTable.cu"
#else

void THNN_(LookupTable_accGradParameters)(
           THCState *state,
           THCIndexTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradWeight,
           THCIndexTensor *count,
           THCIndexTensor *sortedIndices,
           THCIndexTensor *origIndices,
           bool scaleGradByFreq,
           int paddingValue,
           accreal scale_)
{
  scalar_t scale = ScalarConvert<accreal, scalar_t>::to(scale_);
  THCUNN_assertSameGPU(state, 5, input, gradOutput, gradWeight, sortedIndices, origIndices);
  gradOutput = THCTensor_(newContiguous)(state, gradOutput);
  if (!(THCIndexTensor_(isContiguous)(state, input) &&
        THCTensor_(isContiguous)(state, gradWeight))) {
    THError("Tensors must be contiguous");
  }

  int nDim = THCIndexTensor_(nDimensionLegacyAll)(state, input);
  if (THCIndexTensor_(nDimensionLegacyAll)(state, input) != 1 && THCIndexTensor_(nDimensionLegacyAll)(state, input) != 2) {
    THCDescBuff s1 = THCIndexTensor_(sizeDesc)(state, input);
    THError("input must be a vector or matrix, but is of shape: %s", s1.str);
  }

  ptrdiff_t numel = THCIndexTensor_(nElement)(state, input);
  int64_t stride = THCTensor_(stride)(state, gradWeight, 0);

  cudaStream_t stream = THCState_getCurrentStream(state);

  if (numel <= 768 && !scaleGradByFreq) {
    const int WARP_SIZE = 32;
    const int BLOCKDIMY = 32;
    dim3 grid(THCCeilDiv(stride, (int64_t)WARP_SIZE));
    dim3 block(WARP_SIZE, BLOCKDIMY);

    cunn_LookupTable_accGradParametersKernelByFeature<scalar_t, accreal>
    <<<grid,
       block,
       sizeof(accreal)*WARP_SIZE*BLOCKDIMY + sizeof(int)*WARP_SIZE*BLOCKDIMY,
       stream>>>
      (THCIndexTensor_(data)(state, input),
       THCTensor_(data)(state, gradOutput),
       THCTensor_(data)(state, gradWeight),
       scale,
       numel,
       stride,
       paddingValue);
    THCTensor_(free)(state, gradOutput);
    THCudaCheck(cudaGetLastError());
    return;
  }

  THCIndexTensor_(resize)(state, sortedIndices, input->sizes(), {});
  THCIndexTensor_(resize)(state, origIndices, input->sizes(), {});

  // Sort the inputs into sorted with the corresponding indices; we
  // don't need a stable or multidimensional sort, so just use Thrust
  // directly
  {
    THCIndexTensor_(copy)(state, sortedIndices, input);

    THCThrustAllocator thrustAlloc(state);

    thrust::device_ptr<THCIndex_t>
      sortedIndicesIter(THCIndexTensor_(data)(state, sortedIndices));
    thrust::device_ptr<THCIndex_t>
      origIndicesIter(THCIndexTensor_(data)(state, origIndices));

    // Fill sortedOrigIndices with sequential indices
    thrust::counting_iterator<THCIndex_t> countIter(0);

    thrust::copy(
#if CUDA_VERSION >= 7000 || defined __HIP_PLATFORM_HCC__
      thrust::cuda::par(thrustAlloc).on(THCState_getCurrentStream(state)),
#endif
      countIter, countIter + numel, origIndicesIter);

    // Sort; a stable sort is not required
    thrust::sort_by_key(
#if CUDA_VERSION >= 7000 || defined __HIP_PLATFORM_HCC__
      thrust::cuda::par(thrustAlloc).on(THCState_getCurrentStream(state)),
#endif
      sortedIndicesIter, sortedIndicesIter + numel,
      origIndicesIter, ThrustLTOp<int64_t>());
  }

  THCIndex_t *sortedIndices_data = THCIndexTensor_(data)(state, sortedIndices);
  THCIndex_t *origIndices_data = THCIndexTensor_(data)(state, origIndices);
  THCIndex_t *count_data = NULL;

  if (scaleGradByFreq) {
    THCIndexTensor_(resizeAs)(state, count, input);
    count_data = THCIndexTensor_(data)(state, count);

    THCThrustAllocator thrustAlloc(state);
    thrust::device_ptr<THCIndex_t> sortedIndices_ptr(sortedIndices_data);
    thrust::device_ptr<THCIndex_t> count_ptr(count_data);

    // Compute an increasing sequence per unique item in sortedIndices:
    // sorted: 2 5 5 5 7 7 8 9 9
    //  count: 1 1 2 3 1 2 1 1 2
    thrust::inclusive_scan_by_key(
#if CUDA_VERSION >= 7000 || defined __HIP_PLATFORM_HCC__
      thrust::cuda::par(thrustAlloc).on(THCState_getCurrentStream(state)),
#endif
      sortedIndices_ptr,
      sortedIndices_ptr + numel,
      thrust::make_constant_iterator(1),
      count_ptr
    );

    // Take the maximum of each count per unique key in reverse:
    // sorted: 2 5 5 5 7 7 8 9 9
    //  count: 1 3 3 3 2 2 1 2 2
    thrust::inclusive_scan_by_key(
#if CUDA_VERSION >= 7000 || defined __HIP_PLATFORM_HCC__
      thrust::cuda::par(thrustAlloc).on(THCState_getCurrentStream(state)),
#endif
      thrust::make_reverse_iterator(sortedIndices_ptr + numel),
      thrust::make_reverse_iterator(sortedIndices_ptr),
      thrust::make_reverse_iterator(count_ptr + numel),
      thrust::make_reverse_iterator(count_ptr + numel),
      thrust::equal_to<int64_t>(),
      thrust::maximum<int64_t>()
    );
  }

  dim3 grid(THCCeilDiv(numel, (ptrdiff_t) 4), THCCeilDiv(stride, (int64_t) 128));
  dim3 block(32, 4);
  cunn_LookupTable_accGradParametersKernel<scalar_t, accreal><<<grid, block, 0, stream>>>(
    sortedIndices_data,
    origIndices_data,
    THCTensor_(data)(state, gradOutput),
    THCTensor_(data)(state, gradWeight),
    count_data,
    scale,
    numel,
    stride,
    paddingValue
  );

  THCTensor_(free)(state, gradOutput);
  THCudaCheck(cudaGetLastError());
}

#define THREADS 256
#define RUN(NORM, IDXTYPE) \
  calculate_norms_and_renorm<scalar_t, accreal, IDXTYPE, NORM> \
    <<<numel, THREADS/2, THREADS * sizeof(accreal), THCState_getCurrentStream(state)>>> \
    (weightsRaw, idxRaw, normType, maxNorm, THCTensor_(stride)(state, weight, 0))

void THNN_(LookupTable_renorm)(
           THCState *state,
           THCIndexTensor *idx,
           THCTensor *weight,
           accreal maxNorm,
           accreal normType)
{
  THCUNN_assertSameGPU(state, 2, idx, weight);
  if (!(THCIndexTensor_(isContiguous)(state, idx) &&
        THCTensor_(isContiguous)(state, weight))) {
    THError("Tensors must be contiguous");
  }

  if (THCIndexTensor_(nDimensionLegacyAll)(state, idx) != 1) {
    THError("idx must be a vector");
  }

  if (normType <= 0) {
    THError("non-positive-norm not supported");
  }

  THCIndex_t numel = THCIndexTensor_(nElement)(state, idx);

  scalar_t * weightsRaw = THCTensor_(data)(state, weight);
  THCIndex_t * idxRaw = THCIndexTensor_(data)(state, idx);

  // get the unique indices
  thrust::device_ptr<THCIndex_t> idxThrust(idxRaw);
  thrust::device_ptr<THCIndex_t> endIdxThrust(thrust::unique(idxThrust, idxThrust+numel));
  numel = endIdxThrust - idxThrust;

  // At launch time figure out what the index type is and norm type
  int Norm = ScalarConvert<accreal, int>::to(normType);
  if (THCTensor_canUse32BitIndexMath(state, idx)) {
    if (Norm == 1) {
      RUN(1, unsigned int);
    } else if (Norm == 2) {
      RUN(2, unsigned int);
    } else {
      RUN(-1, unsigned int);
    }
  } else {
    if (Norm == 1) {
      RUN(1, unsigned long);
    } else if (Norm == 2) {
      RUN(2, unsigned long);
    } else {
      RUN(-1, unsigned long);
    }
  }
}

#endif

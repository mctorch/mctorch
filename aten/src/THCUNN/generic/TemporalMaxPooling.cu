
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THCUNN/generic/TemporalMaxPooling.cu"
#else

static inline void THNN_(TemporalMaxPooling_shapeCheck)(
                         THCState *state,
                         THCTensor *input,
                         THCTensor *gradOutput,
                         THCIndexTensor *indices,
                         int kW, int dW) {
  int dimT = 0; // Temporal dimension
  int dimF = 1; // Feature dimension
  int input_w;
  int input_n;
  int output_w;
  int ndims = input->dim();

  if (ndims == 3)
  {
    dimT = 1;
    dimF = 2;
  }
  THArgCheck(kW > 0, 5,
             "kernel size should be greater than zero, but got kW: %d", kW);
  THArgCheck(dW > 0, 6,
             "stride should be greater than zero, but got dW: %d", dW);

  THCUNN_argCheck(state, !input->is_empty() && (input->dim() == 2 || input->dim() == 3), 2, input,
                  "non-empty 2D or 3D (batch mode) tensor expected for input, but got: %s");
  THArgCheck(input->size(dimT) >= kW, 2,
             "input sequence smaller than kernel size. Got: %d, Expected: %d",
             input->size(dimT), kW);

  input_w = input->size(dimT);
  input_n = input->size(dimF);
  output_w = (input_w - kW) / dW + 1;

  if (gradOutput != NULL) {
    THCUNN_check_dim_size(state, gradOutput, ndims, dimT, output_w);
    THCUNN_check_dim_size(state, gradOutput, ndims, dimF, input_n)
  }
  if (indices != NULL) {
    THCUNN_check_dim_size_indices(state, indices, ndims, dimT, output_w);
    THCUNN_check_dim_size_indices(state, indices, ndims, dimF, input_n);
  }
}

void THNN_(TemporalMaxPooling_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           THCIndexTensor *indices,
           int kW, int dW) {

  int dimT = 0; // Temporal dimension
  int dimF = 1; // Feature dimension

  int batch = 1;
  int input_w;
  int input_n;
  int output_w;
  int nthreads;

  scalar_t *input_data;
  scalar_t *output_data;
  THCIndex_t *indices_data;

  THCUNN_assertSameGPU(state, 3, input, output, indices);
  THNN_(TemporalMaxPooling_shapeCheck)(state, input, NULL, NULL, kW, dW);
  if (input->dim() == 3)
  {
    dimT = 1;
    dimF = 2;
    batch = input->size(0);
  }
  input = THCTensor_(newContiguous)(state, input);

  input_w = input->size(dimT);
  input_n = input->size(dimF);
  output_w = (input_w - kW) / dW + 1;

  if (input->dim() == 2)
  {
    THCTensor_(resize2d)(state, output, output_w, input->size(dimF));
    THCIndexTensor_(resize2d)(state, indices, output_w, input->size(dimF));
  }
  else
  {
    THCTensor_(resize3d)(state, output, batch, output_w, input->size(dimF));
    THCIndexTensor_(resize3d)(state, indices, batch, output_w, input->size(dimF));
  }

  input_data = THCTensor_(data)(state, input);
  output_data = THCTensor_(data)(state, output);
  indices_data = THCIndexTensor_(data)(state, indices);

  dim3 blocks(batch);
  nthreads = (output_w / 32) * 32;
  if (output_w % 32 > 0) {
    nthreads += 32;
  }

  if (nthreads > TEMPORAL_MAX_POOLING_THREADS) {
    blocks.y = nthreads / TEMPORAL_MAX_POOLING_THREADS;
    if (nthreads % TEMPORAL_MAX_POOLING_THREADS > 0) {
      blocks.y += 1;
    }
    nthreads = TEMPORAL_MAX_POOLING_THREADS;
  }

  dim3 threads(nthreads);
  cunn_TemporalMaxPooling_updateOutputKernel <<< blocks, threads, 0, THCState_getCurrentStream(state) >>>(
      input_data, output_data, indices_data, input_w, input_n, output_w, kW, dW);
  THCudaCheck(cudaGetLastError());
  THCTensor_(free)(state, input);

}

void THNN_(TemporalMaxPooling_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           THCIndexTensor *indices,
           int kW, int dW) {

  int dimT = 0; // Temporal dimension
  int dimF = 1; // Feature dimension

  int batch = 1;
  int input_w;
  int input_n;
  int output_w;
  int nthreads;

  scalar_t *gradInput_data;
  scalar_t *gradOutput_data;
  THCIndex_t *indices_data;

  THCUNN_assertSameGPU(state, 4, input, gradOutput, gradInput, indices);
  THNN_(TemporalMaxPooling_shapeCheck)(state, input, gradOutput, indices, kW, dW);
  THCTensor_(resizeAs)(state, gradInput, input);
  THCTensor_(zero)(state, gradInput);

  if (input->dim() == 3)
  {
    dimT = 1;
    dimF = 2;
    batch = input->size(0);
  }
  gradOutput = THCTensor_(newContiguous)(state, gradOutput);

  input_w = input->size(dimT);
  input_n = input->size(dimF);
  output_w = (input_w - kW) / dW + 1;

  gradInput_data = THCTensor_(data)(state, gradInput);
  gradOutput_data = THCTensor_(data)(state, gradOutput);
  indices_data = THCIndexTensor_(data)(state, indices);

  dim3 blocks(batch);
  nthreads = (output_w / 32) * 32;
  if (output_w % 32 > 0) {
    nthreads += 32;
  }

  if (nthreads > TEMPORAL_MAX_POOLING_THREADS) {
    blocks.y = nthreads / TEMPORAL_MAX_POOLING_THREADS;
    if (nthreads % TEMPORAL_MAX_POOLING_THREADS > 0) {
      blocks.y += 1;
    }
    nthreads = TEMPORAL_MAX_POOLING_THREADS;
  }

  dim3 threads(nthreads);
  if (kW <= dW) {
    cunn_TemporalMaxPooling_updateGradInputKernel <<< blocks, threads, 0, THCState_getCurrentStream(state) >>>(
        gradInput_data, gradOutput_data, indices_data, input_w, input_n, output_w, kW, dW);
  } else {
    cunn_TemporalMaxPooling_updateGradInputKernelAtomic <<< blocks, threads, 0, THCState_getCurrentStream(state) >>>(
        gradInput_data, gradOutput_data, indices_data, input_w, input_n, output_w, kW, dW);
  }
  THCudaCheck(cudaGetLastError());
  THCTensor_(free)(state, gradOutput);

}

#endif

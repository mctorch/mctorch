
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THCUNN/generic/L1Cost.cu"
#else

void THNN_(L1Cost_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output)
{
  THCUNN_check_dim_size(state, output, 1, 0, 1);
  THCUNN_assertSameGPU(state, 1, input);
  accreal sum;
  ptrdiff_t size = THCTensor_(nElement)(state, input);
  input = THCTensor_(newContiguous)(state, input);
  thrust::device_ptr<scalar_t> input_data(THCTensor_(data)(state, input));
  sum = thrust::transform_reduce(input_data, input_data+size, l1cost_functor<scalar_t, accreal>(), accreal(0), thrust::plus<accreal>());

  THCTensor_(free)(state, input);

  THCTensor_(set1d)(state, output, 0, ScalarConvert<accreal, scalar_t>::to(sum));
}

void THNN_(L1Cost_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput)
{
  THCUNN_check_nElement(state, input, gradOutput);
  THCUNN_assertSameGPU(state, 2, input, gradInput);
  ptrdiff_t size = THCTensor_(nElement)(state, input);

  input = THCTensor_(newContiguous)(state, input);
  THCTensor_(resizeAs)(state, gradInput, input);

  thrust::device_ptr<scalar_t> input_data(THCTensor_(data)(state, input));
  thrust::device_ptr<scalar_t> gradInput_data(THCTensor_(data)(state, gradInput));

  thrust::transform(input_data, input_data+size, gradInput_data, l1cost_updateGradInput_functor<scalar_t>());

  THCTensor_(free)(state, input);
}

#endif


#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THCUNN/generic/VolumetricFullDilatedConvolution.cu"
#else

static inline void THNN_(VolumetricFullDilatedConvolution_shapeCheck)(
               THCState *state,
               THCTensor *input,
               THCTensor *gradOutput,
               THCTensor *weight,
               THCTensor *bias,
               int kT, int kW, int kH,
               int dT, int dW, int dH,
               int padT, int padW, int padH,
               int dilationT, int dilationW, int dilationH,
               int adjT, int adjW, int adjH, int weight_nullable) {
  THCUNN_argCheck(state, !input->is_empty() && (input->dim() == 4 || input->dim() == 5), 2, input,
            "non-empty 4D or 5D (batch mode) tensor expected for input, but got: %s");
  THArgCheck(dT > 0 && dW > 0 && dH > 0, 8,
         "stride should be greater than zero, but got dT: %d dH: %d dW: %d", dT, dH, dW);
  THArgCheck(dilationT > 0 && dilationW > 0 && dilationH > 0, 15,
             "dilation should be greater than zero, but got dilationT: %d, dilationH: %d, dilationW: %d",
             dilationT, dilationH, dilationW);
  THArgCheck((adjT < dT || adjT < dilationT)
             && (adjW < dW || adjW < dilationW)
             && (adjH < dH || adjH < dilationH), 15,
             "output padding must be smaller than either stride or dilation,"
             " but got adjT: %d adjH: %d adjW: %d dT: %d dH: %d dW: %d "
             "dilationT: %d dilationH: %d dilationW: %d",
             adjT, adjH, adjW, dT, dH, dW, dilationT, dilationH, dilationW);

   // number of input & output planes and kernel size is indirectly defined by the weight tensor
  if (weight != NULL) {
    THCUNN_argCheck(state, !weight->is_empty() && weight->dim() == 5, 4, weight,
                  "non-empty 5D (nOutputPlane x nInputPlane x kT x kH x kW) tensor "
                  "expected for weight, but got: %s");
    if (bias != NULL) {
      THCUNN_check_dim_size(state, bias, 1, 0, weight->size(1));
    }
  } else if (!weight_nullable) {
    THError("weight tensor is expected to be non-nullable");
  }

  int ndim = input->dim();
  int dimf = 0;
  int dimd = 1;
  int dimh = 2;
  int dimw = 3;

  if (ndim == 5) {
    dimf++;
    dimd++;
    dimh++;
    dimw++;
  }

  if (weight != NULL) {
    const int64_t nInputPlane = THCTensor_(size)(state, weight, 0);
    THCUNN_check_dim_size(state, input, ndim, dimf, nInputPlane);
  }

  int64_t inputWidth   = input->size(dimw);
  int64_t inputHeight  = input->size(dimh);
  int64_t inputDepth  = input->size(dimd);
  int64_t outputDepth  = (inputDepth - 1) * dT - 2*padT + (dilationT * (kT - 1) + 1) + adjT;
  int64_t outputHeight = (inputHeight - 1) * dH - 2*padH + (dilationH * (kH - 1) + 1) + adjH;
  int64_t outputWidth  = (inputWidth - 1) * dW - 2*padW + (dilationW * (kW - 1) + 1) + adjW;

  if (outputDepth < 1 || outputWidth < 1 || outputHeight < 1) {
    THError("Given input size per channel: (%ld x %ld x %ld). "
      "Calculated output size per channel: (%ld x %ld x %ld). Output size is too small",
      inputDepth, inputHeight, inputWidth, outputDepth, outputHeight, outputWidth);
  }

  if (gradOutput != NULL) {
    if (weight != NULL) {
      const int64_t nOutputPlane = THCTensor_(size)(state, weight, 1);
      THCUNN_check_dim_size(state, gradOutput, ndim, dimf, nOutputPlane);
    } else if (bias != NULL) {
      const int64_t nOutputPlane = THCTensor_(size)(state, bias, 0);
      THCUNN_check_dim_size(state, gradOutput, ndim, dimf, nOutputPlane);
    }
    THCUNN_check_dim_size(state, gradOutput, ndim, dimd, outputDepth);
    THCUNN_check_dim_size(state, gradOutput, ndim, dimh, outputHeight);
    THCUNN_check_dim_size(state, gradOutput, ndim, dimw, outputWidth);
  }
}

void THNN_(VolumetricFullDilatedConvolution_updateOutput)(
       THCState *state,
       THCTensor  *input,
       THCTensor  *output,
       THCTensor  *weight,
       THCTensor  *bias,
       THCTensor  *finput,
       THCTensor  *fgradInput,
       int kT, int kW, int kH,
       int dT, int dW, int dH,
       int padT, int padW, int padH,
       int dilationT, int dilationW, int dilationH,
       int adjT, int adjW, int adjH)
{

  THCTensor  *columns = finput;
  THCTensor  *ones    = fgradInput;

  int nInputPlane = THCTensor_(size)(state, weight, 0);
  int nOutputPlane = THCTensor_(size)(state, weight, 1);

  THCUNN_assertSameGPU(state, 6, input, output, weight,
               bias, columns, ones);
  THNN_(VolumetricFullDilatedConvolution_shapeCheck)(
      state, input, NULL, weight, bias, kT, kW, kH,
      dT, dW, dH, padT, padW, padH, dilationT, dilationW, dilationH,
      adjT, adjW, adjH, 0);

  THArgCheck(!bias || THCTensor_(isContiguous)(state, bias), 5,
         "bias tensor has to be contiguous");
  input = THCTensor_(newContiguous)(state, input);
  weight = THCTensor_(newContiguous)(state, weight);

  int is_batch = 1;
  if (input->dim() == 4) {
    // Force batch
    is_batch = 0;
    THCTensor_(resize5d)(state, input, 1, input->size(0), input->size(1), input->size(2), input->size(3));
  }

  int64_t inputWidth   = input->size(4);
  int64_t inputHeight  = input->size(3);
  int64_t inputDepth  = input->size(2);
  int64_t outputDepth  = (inputDepth - 1) * dT - 2*padT + (dilationT * (kT - 1) + 1) + adjT;
  int64_t outputHeight = (inputHeight - 1) * dH - 2*padH + (dilationH * (kH - 1) + 1) + adjH;
  int64_t outputWidth  = (inputWidth - 1) * dW - 2*padW + (dilationW * (kW - 1) + 1) + adjW;

  // Batch size + input planes
  int64_t batchSize = input->size(0);

  // Resize output
  THCTensor_(resize5d)(state, output, batchSize, nOutputPlane, outputDepth, outputHeight, outputWidth);

  // Resize temporary columns
  THCTensor_(resize2d)(state, columns, nOutputPlane*kW*kH*kT, inputDepth*inputHeight*inputWidth);

  // Define a buffer of ones, for bias accumulation
  // Note: this buffer can be shared with other modules, it only ever gets increased,
  // and always contains ones.
  if (ones->dim() != 3 || ones->size(0)*ones->size(1)*ones->size(2) < outputDepth*outputHeight*outputWidth) {
    // Resize plane and fill with ones...
    THCTensor_(resize3d)(state, ones, outputDepth, outputHeight, outputWidth);
    THCTensor_(fill)(state, ones, ScalarConvert<int, scalar_t>::to(1));
  }

  // Helpers
  THCTensor  *input_n = THCTensor_(new)(state);
  THCTensor  *output_n = THCTensor_(new)(state);

  // For each elt in batch, do:
  for (int elt = 0; elt < batchSize; elt ++) {
    // Matrix mulitply per output:
    THCTensor_(select)(state, input_n, input, 0, elt);
    THCTensor_(select)(state, output_n, output, 0, elt);

    // M,N,K are dims of matrix A and B
    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
    int64_t m = weight->size(1) * weight->size(2) * weight->size(3) * weight->size(4);
    int64_t n = columns->size(1);
    int64_t k = weight->size(0);

    // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
    #ifdef THC_REAL_IS_FLOAT
    THCudaBlas_Sgemm(
    #elif defined(THC_REAL_IS_HALF)
    THCudaBlas_Hgemm(
    #elif defined(THC_REAL_IS_DOUBLE)
    THCudaBlas_Dgemm(
    #endif
      state,
      'n', 't',
      n, m, k,
      ScalarConvert<int, scalar_t>::to(1),
      THCTensor_(data)(state, input_n), n,
      THCTensor_(data)(state, weight), m,
      ScalarConvert<int, scalar_t>::to(0),
      THCTensor_(data)(state, columns), n
    );

    // Unpack columns back into input:
    col2vol<scalar_t, accreal>(
      THCState_getCurrentStream(state),
      THCTensor_(data)(state, columns),
      nOutputPlane, outputDepth, outputHeight, outputWidth,
      inputDepth, inputHeight, inputWidth,
      kT, kH, kW, padT, padH, padW, dT, dH, dW,
      dilationT, dilationH, dilationW,
      THCTensor_(data)(state, output_n)
    );

    // Do Bias after:
    // M,N,K are dims of matrix A and B
    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
    int64_t m_ = nOutputPlane;
    int64_t n_ = outputDepth * outputHeight * outputWidth;
    int64_t k_ = 1;

    // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
    if (bias) {
      #ifdef THC_REAL_IS_FLOAT
      THCudaBlas_Sgemm(
      #elif defined(THC_REAL_IS_HALF)
      THCudaBlas_Hgemm(
      #elif defined(THC_REAL_IS_DOUBLE)
      THCudaBlas_Dgemm(
      #endif
        state,
        't', 'n',
        n_, m_, k_,
        ScalarConvert<int, scalar_t>::to(1),
        THCTensor_(data)(state, ones), k_,
        THCTensor_(data)(state, bias), k_,
        ScalarConvert<int, scalar_t>::to(1),
        THCTensor_(data)(state, output_n), n_
      );
    }
  }

  // Free
  THCTensor_(free)(state, input_n);
  THCTensor_(free)(state, output_n);

  // Resize output
  if (is_batch == 0) {
    THCTensor_(resize4d)(state, output, nOutputPlane, outputDepth, outputHeight, outputWidth);
    THCTensor_(resize4d)(state, input, nInputPlane, inputDepth, inputHeight, inputWidth);
  }

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, weight);

}

void THNN_(VolumetricFullDilatedConvolution_updateGradInput)(
       THCState *state,
       THCTensor  *input,
       THCTensor  *gradOutput,
       THCTensor  *gradInput,
       THCTensor  *weight,
       THCTensor  *finput,
       THCTensor  *fgradInput,
       int kT, int kW, int kH,
       int dT, int dW, int dH,
       int padT, int padW, int padH,
       int dilationT, int dilationW, int dilationH,
       int adjT, int adjW, int adjH)
{
  THCTensor  *gradColumns = finput;

  int nInputPlane = THCTensor_(size)(state, weight, 0);
  int nOutputPlane = THCTensor_(size)(state, weight, 1);

  THCUNN_assertSameGPU(state, 5, input, gradOutput, weight,
               gradColumns, gradInput);
  THNN_(VolumetricFullDilatedConvolution_shapeCheck)(
      state, input, gradOutput, weight, NULL, kT, kW, kH,
      dT, dW, dH, padT, padW, padH, dilationT, dilationW, dilationH,
      adjT, adjW, adjH, 0);

  input = THCTensor_(newContiguous)(state, input);
  gradOutput = THCTensor_(newContiguous)(state, gradOutput);
  weight = THCTensor_(newContiguous)(state, weight);

  int is_batch = 1;
  if (input->dim() == 4) {
    // Force batch
    is_batch = 0;
    THCTensor_(resize5d)(state, input, 1, input->size(0), input->size(1), input->size(2), input->size(3));
    THCTensor_(resize5d)(state, gradOutput, 1, gradOutput->size(0), gradOutput->size(1), gradOutput->size(2), gradOutput->size(3));
  }

  int64_t inputWidth   = input->size(4);
  int64_t inputHeight  = input->size(3);
  int64_t inputDepth   = input->size(2);
  int64_t outputDepth  = (inputDepth - 1) * dT - 2*padT + (dilationT * (kT - 1) + 1) + adjT;
  int64_t outputHeight = (inputHeight - 1) * dH - 2*padH + (dilationH * (kH - 1) + 1) + adjH;
  int64_t outputWidth  = (inputWidth - 1) * dW - 2*padW + (dilationW * (kW - 1) + 1) + adjW;

  // Batch size + input planes
  int64_t batchSize = input->size(0);

  // Resize output
  THCTensor_(resize5d)(state, gradInput, batchSize, nInputPlane, inputDepth, inputHeight, inputWidth);

  // Resize temporary columns
  THCTensor_(resize2d)(state, gradColumns, nOutputPlane*kW*kH*kT, inputDepth*inputHeight*inputWidth);

  // Helpers
  THCTensor  *gradInput_n = THCTensor_(new)(state);
  THCTensor  *gradOutput_n = THCTensor_(new)(state);

  // For each elt in batch, do:
  for (int elt = 0; elt < batchSize; elt ++) {
    // Matrix mulitply per sample:
    THCTensor_(select)(state, gradInput_n, gradInput, 0, elt);
    THCTensor_(select)(state, gradOutput_n, gradOutput, 0, elt);

    // Extract columns:
    vol2col(
      THCState_getCurrentStream(state),
      THCTensor_(data)(state, gradOutput_n),
      nOutputPlane, outputDepth, outputHeight, outputWidth,
      inputDepth, inputHeight, inputWidth,
      kT, kH, kW, padT, padH, padW, dT, dH, dW,
      dilationT, dilationH, dilationW,
      THCTensor_(data)(state, gradColumns)
    );


    // M,N,K are dims of matrix A and B
    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
    int64_t m = weight->size(0);
    int64_t n = gradColumns->size(1);
    int64_t k = weight->size(1) * weight->size(2) * weight->size(3) * weight->size(4);

    // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
    #ifdef THC_REAL_IS_FLOAT
    THCudaBlas_Sgemm(
    #elif defined(THC_REAL_IS_HALF)
    THCudaBlas_Hgemm(
    #elif defined(THC_REAL_IS_DOUBLE)
    THCudaBlas_Dgemm(
    #endif
      state,
      'n', 'n',
      n, m, k,
      ScalarConvert<int, scalar_t>::to(1),
      THCTensor_(data)(state, gradColumns), n,
      THCTensor_(data)(state, weight), k,
      ScalarConvert<int, scalar_t>::to(0),
      THCTensor_(data)(state, gradInput_n), n
    );
  }


  // Free
  THCTensor_(free)(state, gradInput_n);
  THCTensor_(free)(state, gradOutput_n);

  // Resize output
  if (is_batch == 0) {
    THCTensor_(resize4d)(state, gradOutput, nOutputPlane, outputDepth, outputHeight, outputWidth);
    THCTensor_(resize4d)(state, input, nInputPlane, inputDepth, inputHeight, inputWidth);
    THCTensor_(resize4d)(state, gradInput, nInputPlane, inputDepth, inputHeight, inputWidth);
  }

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, gradOutput);
  THCTensor_(free)(state, weight);
}


void THNN_(VolumetricFullDilatedConvolution_accGradParameters)(
           THCState *state,
           THCTensor  *input,
           THCTensor  *gradOutput,
           THCTensor  *gradWeight,
           THCTensor  *gradBias,
           THCTensor  *finput,
           THCTensor  *fgradInput,
           int kT, int kW, int kH,
           int dT, int dW, int dH,
           int padT, int padW, int padH,
           int dilationT, int dilationW, int dilationH,
           int adjT, int adjW, int adjH,
           accreal scale_)
{
  THCTensor  *columns = finput;
  THCTensor  *ones = fgradInput;

  scalar_t scale = ScalarConvert<accreal, scalar_t>::to(scale_);
  THCUNN_assertSameGPU(state, 6, input, gradOutput, gradWeight,
               gradBias, columns, ones);
  THNN_(VolumetricFullDilatedConvolution_shapeCheck)(
      state, input, gradOutput, gradWeight, gradBias, kT, kW, kH,
      dT, dW, dH, padT, padW, padH, dilationT, dilationW, dilationH,
      adjT, adjW, adjH, 1);

  int nOutputPlane;
  if (gradWeight) {
    nOutputPlane = THCTensor_(size)(state, gradWeight, 1);
  } else if (gradBias) {
    nOutputPlane = THCTensor_(sizeLegacyNoScalars)(state, gradBias, 0);
  } else {
    return;
  }

  if (gradWeight) {
    THArgCheck(THCTensor_(isContiguous)(state, gradWeight), 4, "gradWeight needs to be contiguous");
  }
  if (gradBias) {
    THArgCheck(THCTensor_(isContiguous)(state, gradBias), 5, "gradBias needs to be contiguous");
    THArgCheck(THCTensor_(isContiguous)(state, ones), 7, "ones needs to be contiguous");
  }

  input = THCTensor_(newContiguous)(state, input);
  gradOutput = THCTensor_(newContiguous)(state, gradOutput);

  int is_batch = 1;
  if (input->dim() == 4) {
    // Force batch
    is_batch = 0;
    THCTensor_(resize5d)(state, input, 1, input->size(0), input->size(1), input->size(2), input->size(3));
    THCTensor_(resize5d)(state, gradOutput, 1, gradOutput->size(0), gradOutput->size(1), gradOutput->size(2), gradOutput->size(3));
  }

  int64_t inputWidth   = input->size(4);
  int64_t inputHeight  = input->size(3);
  int64_t inputDepth   = input->size(2);
  int64_t outputDepth  = (inputDepth - 1) * dT - 2*padT + (dilationT * (kT - 1) + 1) + adjT;
  int64_t outputHeight = (inputHeight - 1) * dH - 2*padH + (dilationH * (kH - 1) + 1) + adjH;
  int64_t outputWidth  = (inputWidth - 1) * dW - 2*padW + (dilationW * (kW - 1) + 1) + adjW;

  // Batch size + input planes
  int64_t batchSize = input->size(0);

  // Define a buffer of ones, for bias accumulation
  if (ones->dim() != 3 || ones->size(0)*ones->size(1)*ones->size(2) < outputDepth*outputHeight*outputWidth) {
    // Resize plane and fill with ones...
    THCTensor_(resize3d)(state, ones, outputDepth, outputHeight, outputWidth);
    THCTensor_(fill)(state, ones, ScalarConvert<int, scalar_t>::to(1));
  }

  // Resize temporary columns
  THCTensor_(resize2d)(state, columns, nOutputPlane*kW*kH*kT, inputDepth*inputHeight*inputWidth);

  // Helpers
  THCTensor  *input_n = THCTensor_(new)(state);
  THCTensor  *gradOutput_n = THCTensor_(new)(state);

  // For each elt in batch, do:
  for (int elt = 0; elt < batchSize; elt ++) {
    // Matrix mulitply per output:
    THCTensor_(select)(state, gradOutput_n, gradOutput, 0, elt);

    // Do Weight:
    if (gradWeight) {
      // Matrix mulitply per output:
      THCTensor_(select)(state, input_n, input, 0, elt);

      // Extract columns:
      vol2col(
        THCState_getCurrentStream(state),
        THCTensor_(data)(state, gradOutput_n),
        nOutputPlane, outputDepth, outputHeight, outputWidth,
        inputDepth, inputHeight, inputWidth,
        kT, kH, kW, padT, padH, padW, dT, dH, dW,
        dilationT, dilationH, dilationW,
        THCTensor_(data)(state, columns)
      );

      // M,N,K are dims of matrix A and B
      // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
      int64_t n = columns->size(0);   // nOutputPlane * kt * kh * kw
      int64_t m = input_n->size(0);   // nInputPlane
      int64_t k = columns->size(1);   // inputHeight * inputWidth

      // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
      #ifdef THC_REAL_IS_FLOAT
      THCudaBlas_Sgemm(
      #elif defined(THC_REAL_IS_HALF)
      THCudaBlas_Hgemm(
      #elif defined(THC_REAL_IS_DOUBLE)
      THCudaBlas_Dgemm(
      #endif
        state,
        't', 'n',
        n, m, k,
        scale,
        THCTensor_(data)(state, columns), k,
        THCTensor_(data)(state, input_n), k,
        ScalarConvert<int, scalar_t>::to(1),
        THCTensor_(data)(state, gradWeight), n
      );
    }

    // Do Bias:
    if (gradBias) {
      // M,N,K are dims of matrix A and B
      // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
      int64_t m_ = nOutputPlane;
      int64_t k_ = outputDepth * outputHeight * outputWidth;

      // Do GEMV (note: this is a bit confusing because gemv assumes column-major matrices)
      #if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE)
      #ifdef THC_REAL_IS_FLOAT
      THCudaBlas_Sgemv(
      #elif defined(THC_REAL_IS_DOUBLE)
      THCudaBlas_Dgemv(
      #endif
        state,
        't',
        k_, m_,
        scale,
        THCTensor_(data)(state, gradOutput_n), k_,
        THCTensor_(data)(state, ones), 1,
        ScalarConvert<int, scalar_t>::to(1),
        THCTensor_(data)(state, gradBias), 1
      );
      #endif
      #ifdef THC_REAL_IS_HALF
      THCudaBlas_Hgemm(
        state,
        't', 'n',
        m_, 1, k_,
        scale,
        THCTensor_(data)(state, gradOutput_n), k_,
        THCTensor_(data)(state, ones), k_,
        ScalarConvert<int, scalar_t>::to(1),
        THCTensor_(data)(state, gradBias), m_
      );
      #endif
    }
  }

  // Free
  THCTensor_(free)(state, input_n);
  THCTensor_(free)(state, gradOutput_n);

  // Resize
  if (is_batch == 0) {
    THCTensor_(resize4d)(state, gradOutput, nOutputPlane, outputDepth, outputHeight, outputWidth);
    THCTensor_(resize4d)(state, input, input->size(1), inputDepth, inputHeight, inputWidth);
  }

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, gradOutput);
}

#endif

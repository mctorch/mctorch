
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THCUNN/generic/SpatialClassNLLCriterion.cu"
#else

void THNN_(SpatialClassNLLCriterion_shapeCheck)(
           THCState *state,
           THCTensor *input,
           THCIndexTensor *target,
           THCTensor *weights)
{
  AT_CHECK(!target->is_empty() && target->dim() == 3, 1,
           "only batches of spatial targets supported (non-empty 3D tensors)" \
           " but got targets of size: : ", target->sizes());
  AT_CHECK(!input->is_empty() && input->dim() == 4, 2,
           "only batches of spatial inputs supported (non-empty 4D tensors), "      \
           "but got input of size: ", input->sizes());
  if (THCTensor_(size)(state, input, 0) != THCIndexTensor_(size)(state, target, 0) ||
      THCTensor_(size)(state, input, 2) != THCIndexTensor_(size)(state, target, 1) ||
      THCTensor_(size)(state, input, 3) != THCIndexTensor_(size)(state, target, 2)) {
    THCDescBuff input_size = THCTensor_(sizeDesc)(state, input);
    THCDescBuff target_size = THCIndexTensor_(sizeDesc)(state, target);
    THError("input and target batch or spatial sizes don't match: target %s, input %s",
            target_size.str, input_size.str);
  }

  if (weights && THCTensor_(nElement)(state, weights) != THCTensor_(size)(state, input, 1)) {
    THError("weight tensor should be defined either for all or no classes");
  }
}

static void THNN_(SpatialClassNLLCriterion_gradOutput_no_reduce_shapeCheck)(
           THCState *state,
           THCTensor *gradOutput,
           THCIndexTensor *target)
{
  AT_CHECK(!gradOutput->is_empty() && THCTensor_(nDimensionLegacyNoScalars)(state, gradOutput) == 3, 2,
           "Expected non-empty dimension 3 but got gradOutput of size: ", gradOutput->sizes());
  if (THCTensor_(size)(state, gradOutput, 0) != THCIndexTensor_(size)(state, target, 0) ||
      THCTensor_(size)(state, gradOutput, 1) != THCIndexTensor_(size)(state, target, 1) ||
      THCTensor_(size)(state, gradOutput, 2) != THCIndexTensor_(size)(state, target, 2)) {
    THCDescBuff gradOutput_size = THCTensor_(sizeDesc)(state, gradOutput);
    THCDescBuff target_size = THCIndexTensor_(sizeDesc)(state, target);
    THError("gradOutput sizes don't match target sizes: target %s, gradOutput %s",
            target_size.str, gradOutput_size.str);
  }
}

void THNN_(SpatialClassNLLCriterion_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCIndexTensor *target,
           THCTensor *output,
           int64_t reduction,
           THCTensor *weights,
           THCTensor *total_weight,
           int64_t ignore_index)
{
  THNN_(SpatialClassNLLCriterion_shapeCheck)(state, input, target, weights);
  THCTensor_(resize1d)(state, output, 1);
  THCTensor_(resize1d)(state, total_weight, 1);

  if (weights)
    THCUNN_assertSameGPU(state, 5, input, target, weights, output, total_weight);
  else
    THCUNN_assertSameGPU(state, 4, input, target, output, total_weight);

  if (reduction == Reduction::None) {
    int64_t batch_size = THCTensor_(size)(state, input, 0);
    int64_t H = THCTensor_(size)(state, input, 2);
    int64_t W = THCTensor_(size)(state, input, 3);

    THCTensor_(resize3d)(state, output, batch_size, H, W);

    if (weights) {
      weights = THCTensor_(newContiguous)(state, weights);
    }

    int64_t count = batch_size * H * W;
    SpatialClassNLLCriterion_updateOutput_no_reduce_kernel<scalar_t>
      <<<GET_BLOCKS(count), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state)>>>(
        count,
        toDeviceTensor<scalar_t, 4>(state, input),
        toDeviceTensor<THCIndex_t, 3>(state, target),
        toDeviceTensor<scalar_t, 3>(state, output),
        weights ? THCTensor_(data)(state, weights) : NULL,
        ignore_index);

    if (weights) {
      THCTensor_(free)(state, weights);
    }
    return;
  }

  input = THCTensor_(newContiguous)(state, input);
  weights = weights ? THCTensor_(newContiguous)(state, weights) : NULL;
  target = THCIndexTensor_(newContiguous)(state, target);

  scalar_t *input_data = THCTensor_(data)(state, input);
  scalar_t *weights_data = weights ? THCTensor_(data)(state, weights) : NULL;
  THCIndex_t  *target_data = THCIndexTensor_(data)(state, target);
  scalar_t *output_data = THCTensor_(data)(state, output);
  scalar_t *total_weight_data = THCTensor_(data)(state, total_weight);

  THCIndex_t batch_size = THCIndexTensor_(size)(state, target, 0);
  THCIndex_t map_nelem = THCIndexTensor_(nElement)(state, target) / batch_size;
  int blocks_per_sample = GET_BLOCKS(map_nelem) / 128;
  blocks_per_sample = (blocks_per_sample == 0) ? 1 : blocks_per_sample;
  int total_blocks = blocks_per_sample * batch_size;

  THCTensor_(fill)(state, output, ScalarConvert<int, scalar_t>::to(0));
  THCTensor_(fill)(state, total_weight, ScalarConvert<int, scalar_t>::to(0));

  cunn_SpatialClassNLLCriterion_updateOutput_kernel<scalar_t, accreal>
    <<<total_blocks, CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state)>>>(
      output_data,
      total_weight_data,
      input_data,
      target_data,
      weights_data,
      reduction == Reduction::Mean,
      THCTensor_(size)(state, input, 0),
      THCTensor_(size)(state, input, 1),
      THCTensor_(size)(state, input, 2) * THCTensor_(size)(state, input, 3),
      blocks_per_sample,
      ignore_index
  );
  THCudaCheck(cudaGetLastError());
  if (reduction == Reduction::Mean) {
    cunn_SpatialClassNLLCriterion_sizeAverage_kernel<<<1, 1, 0, THCState_getCurrentStream(state)>>>(
      output_data, total_weight_data
    );
    THCudaCheck(cudaGetLastError());
  }

  if (weights)
    THCTensor_(free)(state, weights);
  THCIndexTensor_(free)(state, target);
  THCTensor_(free)(state, input);
}

void THNN_(SpatialClassNLLCriterion_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCIndexTensor *target,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           int64_t reduction,
           THCTensor *weights,
           THCTensor *total_weight,
           int64_t ignore_index)
{
  THNN_(SpatialClassNLLCriterion_shapeCheck)(state, input, target, weights);
  THCTensor_(resizeAs)(state, gradInput, input);
  THCTensor_(zero)(state, gradInput);
  THArgCheck(THCTensor_(isContiguous)(state, gradInput), 4,
             "gradInput must be contiguous");

  if (weights)
    THCUNN_assertSameGPU(state, 5, weights, input, target, gradInput, total_weight);
  else
    THCUNN_assertSameGPU(state, 4, input, target, gradInput, total_weight);

  if (reduction == Reduction::None) {
    THNN_(SpatialClassNLLCriterion_gradOutput_no_reduce_shapeCheck)(
        state,
        gradOutput,
        target);

    int64_t batch_size = THCTensor_(size)(state, input, 0);
    int64_t H = THCTensor_(size)(state, input, 2);
    int64_t W = THCTensor_(size)(state, input, 3);

    if (weights) {
      weights = THCTensor_(newContiguous)(state, weights);
    }

    int64_t count = batch_size * H * W;
    SpatialClassNLLCriterion_updateGradInput_no_reduce_kernel<scalar_t>
      <<<GET_BLOCKS(count), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state)>>>(
        count,
        toDeviceTensor<THCIndex_t, 3>(state, target),
        toDeviceTensor<scalar_t, 3>(state, gradOutput),
        toDeviceTensor<scalar_t, 4>(state, gradInput),
        weights ? THCTensor_(data)(state, weights) : NULL,
        ignore_index);

    if (weights) {
      THCTensor_(free)(state, weights);
    }
    return;
  }

  input = THCTensor_(newContiguous)(state, input);
  weights = weights ? THCTensor_(newContiguous)(state, weights) : NULL;
  target = THCIndexTensor_(newContiguous)(state, target);

  scalar_t *gradOutput_data = THCTensor_(data)(state, gradOutput);
  scalar_t *weights_data = weights ? THCTensor_(data)(state, weights) : NULL;
  scalar_t *gradInput_data = THCTensor_(data)(state, gradInput);
  THCIndex_t *target_data = THCIndexTensor_(data)(state, target);
  scalar_t *total_weight_data = THCTensor_(data)(state, total_weight);

  THCIndex_t batch_size = THCIndexTensor_(size)(state, target, 0);
  THCIndex_t map_nelem = THCIndexTensor_(nElement)(state, target) / batch_size;
  int blocks_per_sample = GET_BLOCKS(map_nelem) / 128;
  blocks_per_sample = (blocks_per_sample == 0) ? 1 : blocks_per_sample;
  int total_blocks = blocks_per_sample * batch_size;

  cunn_SpatialClassNLLCriterion_updateGradInput_kernel
    <<<total_blocks, CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state)>>>(
      gradInput_data,
      gradOutput_data,
      target_data,
      weights_data,
      total_weight_data,
      reduction == Reduction::Mean,
      THCTensor_(size)(state, input, 0),
      THCTensor_(size)(state, input, 1),
      THCTensor_(size)(state, input, 2) *THCTensor_(size)(state, input, 3),
      blocks_per_sample,
      ignore_index
  );
  THCudaCheck(cudaGetLastError());

  if (weights)
    THCTensor_(free)(state, weights);
  THCIndexTensor_(free)(state, target);
  THCTensor_(free)(state, input);
}

#endif


#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THCUNN/generic/VolumetricAveragePooling.cu"
#else

#include <THCUNN/generic/pooling_shape.h>

static inline void THNN_(VolumetricAveragePooling_shapeCheck)(
                         THCState *state,
                         THCTensor *input,
                         THCTensor *gradOutput,
                         int kT, int kW, int kH,
                         int dT, int dW, int dH,
                         int padT, int padW, int padH,
                         bool ceil_mode)
{
  int inputSlices;
  int inputTime;
  int inputHeight;
  int inputWidth;

  int ndim = input->dim();
  int dimN = 0;
  int dimt = 1;
  int dimh = 2;
  int dimw = 3;

  if (input->dim() == 5)
  {
    dimN++;
    dimt++;
    dimh++;
    dimw++;
  }

  if (!input->is_empty() && THCTensor_(nDimensionLegacyNoScalars)(state, input) == 4)
  {
    THArgCheck(input->size(dimw) >= kW && input->size(dimh) >= kH
               && input->size(dimt) >= kT, 2,
               "input image (T: %d H: %d W: %d) smaller than "
               "kernel size (kT: %d kH: %d kW: %d)",
               input->size(dimt), input->size(dimh), input->size(dimw),
               kT, kH, kW);

    /* sizes */
    inputSlices = THCTensor_(size)(state, input, 0);
    inputTime   = THCTensor_(size)(state, input, 1);
    inputHeight = THCTensor_(size)(state, input, 2);
    inputWidth  = THCTensor_(size)(state, input, 3);
  }
  else if (!input->is_empty() && THCTensor_(nDimensionLegacyNoScalars)(state, input) == 5)
  {
    THArgCheck(input->size(dimw) >= kW && input->size(dimh) >= kH
               && input->size(dimt) >= kT, 2,
               "input image (T: %d H: %d W: %d) smaller than "
               "kernel size (kT: %d kH: %d kW: %d)",
               input->size(dimt), input->size(dimh), input->size(dimw),
               kT, kH, kW);

    /* sizes */
    inputSlices = THCTensor_(size)(state, input, 1);
    inputTime   = THCTensor_(size)(state, input, 2);
    inputHeight = THCTensor_(size)(state, input, 3);
    inputWidth  = THCTensor_(size)(state, input, 4);
  }
  else
  {
    AT_ERROR("non-empty 4D or 5D tensor expected, but got size: ", input->sizes());
  }

  // The second argument is the index of padH.
  THArgCheck(kT/2 >= padT && kW/2 >= padW && kH/2 >= padH, 11,
             "pad should not be greater than half of kernel size, but got "
             "padT = %d, padW = %d, padH = %d, kT = %d, kW = %d, kH = %d",
             padT, padW, padH, kT, kW, kH);

  int outputTime = pooling_output_shape<int>(inputTime, kT, padT, dT, 1, ceil_mode);
  int outputHeight = pooling_output_shape<int>(inputHeight, kH, padH, dH, 1, ceil_mode);
  int outputWidth = pooling_output_shape<int>(inputWidth, kW, padW, dW, 1, ceil_mode);

  if (gradOutput != NULL)
  {
     THCUNN_check_dim_size(state, gradOutput, ndim, dimN, inputSlices);
     THCUNN_check_dim_size(state, gradOutput, ndim, dimt, outputTime);
     THCUNN_check_dim_size(state, gradOutput, ndim, dimh, outputHeight);
     THCUNN_check_dim_size(state, gradOutput, ndim, dimw, outputWidth);
  }
}

void THNN_(VolumetricAveragePooling_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           int kT, int kW, int kH,
           int dT, int dW, int dH,
           int padT, int padW, int padH,
           bool ceil_mode,
           bool count_include_pad)
{
  int batchSize;
  int inputSlices;
  int inputTime;
  int inputHeight;
  int inputWidth;

  int dimt = 1;
  int dimh = 2;
  int dimw = 3;

  int fiveDimensionalInput = THCTensor_(nDimensionLegacyNoScalars)(state, input) == 5;
  if (fiveDimensionalInput)
  {
    dimt++;
    dimh++;
    dimw++;
  }

  THNN_(VolumetricAveragePooling_shapeCheck)
       (state, input, NULL, kT, kW, kH, dT, dW, dH,
        padT, padW, padH, ceil_mode);

  if (!fiveDimensionalInput) /* 4D */
  {
    /* sizes */
    batchSize   = 1;
    inputSlices = THCTensor_(size)(state, input, 0);
    inputTime   = THCTensor_(size)(state, input, 1);
    inputHeight = THCTensor_(size)(state, input, 2);
    inputWidth  = THCTensor_(size)(state, input, 3);
  }
  else /* 5D */
  {
    /* sizes */
    batchSize   = THCTensor_(size)(state, input, 0);
    inputSlices = THCTensor_(size)(state, input, 1);
    inputTime   = THCTensor_(size)(state, input, 2);
    inputHeight = THCTensor_(size)(state, input, 3);
    inputWidth  = THCTensor_(size)(state, input, 4);
  }

  int outputTime = pooling_output_shape<int>(inputTime, kT, padT, dT, 1, ceil_mode);
  int outputHeight = pooling_output_shape<int>(inputHeight, kH, padH, dH, 1, ceil_mode);
  int outputWidth = pooling_output_shape<int>(inputWidth, kW, padW, dW, 1, ceil_mode);

  if (!fiveDimensionalInput) /* 4D */
  {
    /* resize output */
    THCTensor_(resize4d)(state, output, inputSlices,
                         outputTime, outputHeight, outputWidth);
  }
  else /* 5D */
  {
    THCTensor_(resize5d)(state, output, batchSize, inputSlices,
                         outputTime, outputHeight, outputWidth);
  }

  input = THCTensor_(newContiguous)(state, input);
  if (fiveDimensionalInput) {
    // Collapse batch and feature dimensions
    output = THCTensor_(newFoldBatchDim)(state, output);

    THCTensor *old_input = input;
    input = THCTensor_(newFoldBatchDim)(state, input);
    THCTensor_(free)(state, old_input);
  } else {
    THCTensor_(retain)(state, output);
  }

  THCDeviceTensor<scalar_t, 4> cudaInput;
  THCDeviceTensor<scalar_t, 4> cudaOutput;
  cudaInput  = toDeviceTensor<scalar_t, 4>(state, input);
  cudaOutput = toDeviceTensor<scalar_t, 4>(state, output);

  int totalZ = outputTime * inputSlices * batchSize;
  int offsetZ = 0;
  dim3 block(32, 8);
  while (totalZ > 0) {
    dim3 grid(THCCeilDiv(outputWidth, static_cast<int>(block.x)),
              THCCeilDiv(outputHeight, static_cast<int>(block.y)),
              totalZ > 65535 ? 65535 : totalZ);

    switch (kW)
      {
        LAUNCH_UPDATE_OUTPUT_KERNEL_WIDTH(1);
        LAUNCH_UPDATE_OUTPUT_KERNEL_WIDTH(2);
        LAUNCH_UPDATE_OUTPUT_KERNEL_WIDTH(3);
        LAUNCH_UPDATE_OUTPUT_KERNEL_WIDTH(4);
        LAUNCH_UPDATE_OUTPUT_KERNEL_WIDTH(5);
        LAUNCH_UPDATE_OUTPUT_KERNEL_WIDTH(6);
        LAUNCH_UPDATE_OUTPUT_KERNEL_WIDTH(7);
      default:
        cuda_VolumetricAveragePooling_updateOutput<scalar_t, accreal>
          <<<grid, block, 0, THCState_getCurrentStream(state)>>>(
            cudaInput,
            cudaOutput,
            kT, kH, kW,
            dT, dH, dW,
            padT, padH, padW,
            count_include_pad,
            offsetZ);
        break;
      }
    totalZ -= 65535;
    offsetZ += 65535;
    THCudaCheck(cudaGetLastError());
  }

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, output);
}

void THNN_(VolumetricAveragePooling_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           int kT, int kW, int kH,
           int dT, int dW, int dH,
           int padT, int padW, int padH,
           bool ceil_mode,
           bool count_include_pad)
{
  THNN_(VolumetricAveragePooling_shapeCheck)
       (state, input, gradOutput, kT, kW, kH, dT, dW, dH,
        padT, padW, padH, ceil_mode);
  bool kernelsOverlap = (dT < kT) || (dH < kH) || (dW < kW);

  // Resize and initialize result tensor.
  THCTensor_(resizeAs)(state, gradInput, input);
  THCTensor_(zero)(state, gradInput);

  int batchSize;
  int inputSlices;
  int inputTime;
  int inputHeight;
  int inputWidth;

  int outputTime;
  int outputHeight;
  int outputWidth;

  int fiveDimensionalInput = THCTensor_(nDimensionLegacyNoScalars)(state, input) == 5;
  if (!fiveDimensionalInput) /* 4D */
  {
    batchSize = 1;
    inputSlices  = THCTensor_(size)(state, input, 0);
    inputTime    = THCTensor_(size)(state, input, 1);
    inputHeight  = THCTensor_(size)(state, input, 2);
    inputWidth   = THCTensor_(size)(state, input, 3);

    outputTime   = THCTensor_(size)(state, gradOutput, 1);
    outputHeight = THCTensor_(size)(state, gradOutput, 2);
    outputWidth  = THCTensor_(size)(state, gradOutput, 3);
  }
  else
  {
    batchSize    = THCTensor_(size)(state, input, 0);
    inputSlices  = THCTensor_(size)(state, input, 1);
    inputTime    = THCTensor_(size)(state, input, 2);
    inputHeight  = THCTensor_(size)(state, input, 3);
    inputWidth   = THCTensor_(size)(state, input, 4);

    outputTime   = THCTensor_(size)(state, gradOutput, 2);
    outputHeight = THCTensor_(size)(state, gradOutput, 3);
    outputWidth  = THCTensor_(size)(state, gradOutput, 4);
  }

  gradOutput = THCTensor_(newContiguous)(state, gradOutput);
  if (fiveDimensionalInput) {
    // Collapse batch and feature dimensions
    gradInput = THCTensor_(newFoldBatchDim)(state, gradInput);

    THCTensor *old_gradOutput = gradOutput;
    gradOutput = THCTensor_(newFoldBatchDim)(state, gradOutput);
    THCTensor_(free)(state, old_gradOutput);
  } else {
    THCTensor_(retain)(state, gradInput);
  }

  THCDeviceTensor<scalar_t, 4> cudaGradInput;
  THCDeviceTensor<scalar_t, 4> cudaGradOutput;
  cudaGradInput  = toDeviceTensor<scalar_t, 4>(state, gradInput);
  cudaGradOutput = toDeviceTensor<scalar_t, 4>(state, gradOutput);

  dim3 block(32, 8);

  // Optimizing for stride 1 is probably only of limited value, but this
  // specialization yields 3x speedup over the atomicAdd implementation.
  // Padding must be 0, otherwise, pool size may change.
  if (dT == 1 && dH == 1 && dW == 1 && padT == 0 && padH == 0 && padW == 0)
  {
    int totalZ = inputTime * inputSlices * batchSize;
    int offsetZ = 0;
    while (totalZ > 0) {
      dim3 grid(THCCeilDiv(inputWidth, static_cast<int>(block.x)),
                THCCeilDiv(inputHeight, static_cast<int>(block.y)),
                totalZ > 65535 ? 65535 : totalZ);
      cuda_VolumetricAveragePooling_updateGradInput_Stride1<scalar_t, accreal>
        <<<grid, block, 0, THCState_getCurrentStream(state)>>>(
          cudaGradOutput, cudaGradInput, kT, kH, kW, 1.0f/(kT * kH * kW), offsetZ);
      THCudaCheck(cudaGetLastError());
      totalZ -= 65535;
      offsetZ += 65535;
    }
  }
  else
  {
    int totalZ = outputTime * inputSlices * batchSize;
    int offsetZ = 0;
    while (totalZ > 0) {
      dim3 grid(THCCeilDiv(outputWidth, static_cast<int>(block.x)),
                THCCeilDiv(outputHeight, static_cast<int>(block.y)),
                totalZ > 65535 ? 65535 : totalZ);
      if (kernelsOverlap)
      {
        cuda_VolumetricAveragePooling_updateGradInput_atomicAdd<scalar_t, accreal>
          <<<grid, block, 0, THCState_getCurrentStream(state)>>>(
            cudaGradOutput, cudaGradInput, kT, kH, kW, dT, dH, dW,
            padT, padH, padW, count_include_pad, offsetZ);
      }
      else
      {
        cuda_VolumetricAveragePooling_updateGradInput<scalar_t, accreal>
          <<<grid, block, 0, THCState_getCurrentStream(state)>>>(
            cudaGradOutput, cudaGradInput, kT, kH, kW, dT, dH, dW,
            padT, padH, padW, count_include_pad, offsetZ);
      }
      THCudaCheck(cudaGetLastError());
      totalZ -= 65535;
      offsetZ += 65535;
    }
  }

  THCTensor_(free)(state, gradInput);
  THCTensor_(free)(state, gradOutput);
}

#endif

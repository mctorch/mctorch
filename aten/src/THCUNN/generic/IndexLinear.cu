
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THCUNN/generic/IndexLinear.cu"
#else

static bool THNN_(checkKeysValues)(THCState *state, THCudaLongTensor* keys,
                                   THCTensor* values)
{
    return THCudaLongTensor_size(state, keys, 0) == THCTensor_(nElement)(state, values)
        && THCTensor_(nDimensionLegacyAll)(state, values) == 1
        && THCudaLongTensor_nDimensionLegacyAll(state, keys) == 1;
}

void THNN_(IndexLinear_updateOutput)(
    THCState *state,
    THCudaLongTensor *keys,
    int64_t keysOffset,
    THCTensor *values,
    THCudaLongTensor *sizes,
    THCudaLongTensor *cumSumSizes,
    THCTensor *output,
    THCTensor *weight,
    THCTensor *bias,
    THCTensor *normalizedValues,
    int   train)
{
    // Make sure these inputs are contiguous to accelerate computations
    THArgCheck(THCudaLongTensor_isContiguous(state, keys), 1,
               "keys vector must be contiguous");
    THArgCheck(THCTensor_(isContiguous)(state, values), 3,
               "values vector must be contiguous");
    THArgCheck(THCudaLongTensor_isContiguous(state, sizes), 4,
               "sizes vector must be contiguous");
    THArgCheck(THCudaLongTensor_isContiguous(state, cumSumSizes), 5,
               "cumSumSizes vector must be contiguous");
    THArgCheck(THCTensor_(isContiguous)(state, output), 6,
               "output vector must be contiguous");
    THArgCheck(THCTensor_(isContiguous)(state, weight), 7,
               "weight matrix must be contiguous");
    THArgCheck(THCTensor_(isContiguous)(state, bias), 8,
               "bias vector must be contiguous");
    THArgCheck(THNN_(checkKeysValues)(state, keys, values), 1,
               "Keys and values should have the same number of elements");

    int64_t batchSize = sizes->size(0);
    int64_t outDim = bias->size(0);
    int64_t wDim = weight->size(1);
    int64_t weightStride = weight->stride(0);
    int maxNormalize = wDim - outDim;
    int64_t keysSize = keys->size(0);
    int64_t nnzPerRow = divup(keysSize, batchSize);

    THCTensor_(resize2d)(state, output, batchSize, outDim);
    int64_t *keysData        = THCudaLongTensor_data (state, keys);
    scalar_t *valuesData      = THCTensor_(data)      (state, values);
    int64_t *cumSumSizesData = THCudaLongTensor_data (state, cumSumSizes);
    scalar_t *biasData        = THCTensor_(data)      (state, bias);
    scalar_t *weightData      = THCTensor_(data)      (state, weight);
    scalar_t *outData         = THCTensor_(data)      (state, output);

    cudaStream_t stream = THCState_getCurrentStream(state);
    dim3 threads(THREADS_X, THREADS_Y);
    int blocks_x = divup(outDim, threads.x);
    int blocks_y = batchSize;
    int nnzPerBlock = ((outDim == 1 || batchSize == 1) ? THREADS_X : NNZ_PER_BLOCK_MAX);
    int blocks_z = divup(nnzPerRow, nnzPerBlock);

    dim3 blocks(blocks_x, blocks_y, blocks_z);

    if (blocks_z > 1) {
        THCudaCheck(cudaMemsetAsync(outData, 0, outDim * batchSize * sizeof(scalar_t), stream));
    }

    scalar_t *normalizedValuesData = NULL;
    if (maxNormalize && train) {
        THCTensor_(resize1d)(state, normalizedValues, keysSize);
        normalizedValuesData = THCTensor_(data)(state, normalizedValues);
        updateOutput<scalar_t, true><<<blocks, threads, 0, stream>>>
            (outData, normalizedValuesData, valuesData, cumSumSizesData, keysData,
             batchSize, outDim, weightData, biasData, weightStride, keysOffset, maxNormalize, nnzPerBlock);
    } else {
        updateOutput<scalar_t, false><<<blocks, threads, 0, stream>>>
            (outData, normalizedValuesData, valuesData, cumSumSizesData, keysData,
             batchSize, outDim, weightData, biasData, weightStride, keysOffset, maxNormalize, nnzPerBlock);
    }
}

void THNN_(IndexLinear_accGradParameters)(
    THCState *state,
    THCudaLongTensor *keys,
    int64_t keysOffset,
    THCTensor *values,
    THCudaLongTensor *sizes,
    THCudaLongTensor *cumSumSizes,
    THCTensor *gradOutput,
    THCTensor *gradWeight,
    THCTensor *gradBias,
    THCTensor *weight,
    THCTensor *bias,
    THCTensor* valuesBuffer,
    accreal weightDecay,
    accreal scale)
{
    int64_t keysSize = keys->size(0);
    int64_t batchSize = sizes->size(0);
    int64_t outDim = bias->size(0);
    int64_t wDim = weight->size(1);
    int maxNormalize = wDim - outDim;

    // Make sure these inputs are contiguous to accelerate computations
    THArgCheck(THCudaLongTensor_isContiguous(state, keys), 1,
               "keys vector must be contiguous");
    THArgCheck(THCTensor_(isContiguous)(state, values), 3,
               "values vector must be contiguous");
    THArgCheck(THCudaLongTensor_isContiguous(state, sizes), 4,
               "sizes vector must be contiguous");
    THArgCheck(THCudaLongTensor_isContiguous(state, cumSumSizes), 5,
               "cumSumSizes vector must be contiguous");
    THArgCheck(THCTensor_(isContiguous)(state, gradOutput), 6,
               "gradOutput vector must be contiguous");
    THArgCheck(THCTensor_(isContiguous)(state, gradWeight), 7,
               "gradWeight matrix must be contiguous");
    THArgCheck(THCTensor_(isContiguous)(state, gradBias), 8,
               "gradBias vector must be contiguous");
    THArgCheck(THCTensor_(isContiguous)(state, weight), 9,
               "weight matrix must be contiguous");
    THArgCheck(THCTensor_(isContiguous)(state, bias), 10,
               "bias vector must be contiguous");
    THArgCheck(THCTensor_(isContiguous)(state, valuesBuffer), 11,
               "valuesBuffer vector must be contiguous");
    THArgCheck(THNN_(checkKeysValues)(state, keys, values), 1,
               "Keys and values should have the same number of elements");

    THCTensor_(resize2d)(state, gradWeight, keysSize, outDim * (maxNormalize > 0 ? 2 : 1));

    scalar_t *valuesData      = THCTensor_(data)      (state, values);
    int64_t *cumSumSizesData = THCudaLongTensor_data (state, cumSumSizes);
    scalar_t *gradOutputData  = THCTensor_(data)      (state, gradOutput);
    scalar_t *gradBiasData    = THCTensor_(data)      (state, gradBias);
    scalar_t *gradWeightData  = THCTensor_(data)      (state, gradWeight);
    int64_t gradWeightStride = gradWeight->stride(0);

    cudaStream_t stream = THCState_getCurrentStream(state);
    dim3 threads(THREADS_X, THREADS_Y);
    int blocks_x = divup(outDim, threads.x);
    accGradBias<scalar_t, false><<<blocks_x, threads, 0, stream>>>
        (gradBiasData, gradOutputData, outDim, batchSize, scale, weightDecay);

    dim3 blocks(blocks_x, batchSize);
    accGradWeight<scalar_t><<<blocks, threads, 0, stream>>>
        (gradWeightData, gradOutputData, valuesData, cumSumSizesData, outDim,
         gradWeightStride, scale, weightDecay, maxNormalize);
}

void THNN_(IndexLinear_accUpdateGradParameters)(
    THCState *state,
    THCudaLongTensor *keys,
    int64_t keysOffset,
    THCTensor *values,
    THCudaLongTensor *sizes,
    THCudaLongTensor *cumSumSizes,
    THCTensor *gradOutput,
    THCTensor *weight,
    THCTensor *bias,
    accreal weightDecay,
    accreal scale)
{
    // Make sure these inputs are contiguous to accelerate computations
    THArgCheck(THCudaLongTensor_isContiguous(state, keys), 1,
               "keys vector must be contiguous");
    THArgCheck(THCTensor_(isContiguous)(state, values), 3,
               "values vector must be contiguous");
    THArgCheck(THCudaLongTensor_isContiguous(state, sizes), 4,
               "sizes vector must be contiguous");
    THArgCheck(THCudaLongTensor_isContiguous(state, cumSumSizes), 5,
               "cumSumSizes vector must be contiguous");
    THArgCheck(THCTensor_(isContiguous)(state, gradOutput), 6,
               "gradOutput vector must be contiguous");
    THArgCheck(THCTensor_(isContiguous)(state, weight), 7,
               "weight matrix must be contiguous");
    THArgCheck(THCTensor_(isContiguous)(state, bias), 8,
               "bias vector must be contiguous");
    THArgCheck(THNN_(checkKeysValues)(state, keys, values), 1,
               "Keys and values should have the same number of elements");

    int64_t batchSize = sizes->size(0);
    int64_t outDim = bias->size(0);
    int64_t keysSize = keys->size(0);
    int64_t wDim = weight->size(1);
    int maxNormalize = wDim - outDim;

    scalar_t *biasData         = THCTensor_(data)      (state, bias);
    scalar_t *weightData       = THCTensor_(data)      (state, weight);
    scalar_t *gradOutputData   = THCTensor_(data)      (state, gradOutput);
    scalar_t *valuesData       = THCTensor_(data)      (state, values);
    int64_t *keysData         = THCudaLongTensor_data (state, keys);
    int64_t *cumSumSizesData  = THCudaLongTensor_data (state, cumSumSizes);
    int64_t weightStride = weight->stride(0);

    cudaStream_t stream = THCState_getCurrentStream(state);
    dim3 threads(THREADS_X, THREADS_Y);
    int blocks_x = divup(outDim, threads.x);

    accGradBias<scalar_t, true><<<blocks_x, threads, 0, stream>>>
        (biasData, gradOutputData, outDim, batchSize, scale, weightDecay);

    int64_t nnzPerRow = divup(keysSize, batchSize);
    int blocks_y = divup(nnzPerRow, REPEAT * threads.y);
    dim3 blocks(blocks_x, blocks_y);

    for (int64_t batchId = 0; batchId < batchSize; batchId++) {
        accUpdateWeight<scalar_t><<<blocks, threads, 0, stream>>>
            (weightData, weightStride, gradOutputData, outDim, valuesData,
             cumSumSizesData, keysData, keysOffset, scale, weightDecay, maxNormalize,
             batchId);
    }
}

void THNN_(IndexLinear_updateParameters)(
    THCState *state,
    THCTensor *gradWeight,
    THCTensor *gradBias,
    THCTensor *weight,
    THCTensor *bias,
    THCudaLongTensor *runningKeys,
    THCudaLongTensor *cumSumSizes,
    int64_t keysOffset,
    accreal weightDecay,
    accreal learningRate)
{
    // Make sure these inputs are contiguous to accelerate computations
    THArgCheck(THCTensor_(isContiguous)(state, gradWeight), 1,
               "gradWeight matrix must be contiguous");
    THArgCheck(THCTensor_(isContiguous)(state, gradBias), 2,
               "gradBias vector must be contiguous");
    THArgCheck(THCTensor_(isContiguous)(state, weight), 3,
               "weight matrix must be contiguous");
    THArgCheck(THCTensor_(isContiguous)(state, bias), 4,
               "bias vector must be contiguous");
    THArgCheck(THCudaLongTensor_isContiguous(state, runningKeys), 5,
               "runningKeys vector must be contiguous");
    THArgCheck(THCudaLongTensor_isContiguous(state, cumSumSizes), 6,
               "cumSumSizes vector must be contiguous");

    int64_t outDim = bias->size(0);
    int64_t wDim = weight->size(1);
    int maxNormalize = wDim - outDim;
    int64_t keysSize = runningKeys->size(0);
    int64_t batchSize = cumSumSizes->size(0);

    THCTensor_(cadd)(state, bias, bias, -learningRate, gradBias);
    int64_t gradWeightStride = gradWeight->stride(0);
    int64_t weightStride = weight->stride(0);

    int64_t *keysData        = THCudaLongTensor_data (state, runningKeys);
    int64_t *cumSumSizesData = THCudaLongTensor_data (state, cumSumSizes);
    scalar_t *gradWeightData  = THCTensor_(data)      (state, gradWeight);
    scalar_t *weightData      = THCTensor_(data)      (state, weight);

    dim3 threads(THREADS_X, THREADS_Y);
    int64_t nnzPerRow = divup(keysSize, batchSize);
    int blocks_x = divup(outDim, threads.x);
    int blocks_y = divup(nnzPerRow, REPEAT * threads.y);
    dim3 blocks(blocks_x, blocks_y);
    cudaStream_t stream = THCState_getCurrentStream(state);

    for (int64_t batchId = 0; batchId < batchSize; batchId++) {
        updateWeight<scalar_t><<<blocks, threads, 0, stream>>>
            (weightData, gradWeightData, keysData, cumSumSizesData, outDim,
             gradWeightStride, weightStride, keysOffset, learningRate, weightDecay,
             maxNormalize, batchId);
    }
}
#endif


#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THCUNN/generic/SpatialSubSampling.cu"
#else

#include <THCUNN/common.h>

static inline void THNN_(SpatialSubSampling_shapeCheck)(
                         THCState *state,
                         THCTensor *input,
                         THCTensor *gradOutput,
                         THCTensor *weight,
                         int kW, int kH) {
  THCUNN_argCheck(state, !input->is_empty() && (input->dim() == 3 || input->dim() == 4), 2, input,
                  "non-empty 3D or 4D input tensor expected but got: %s");

  int nInputPlane = THCTensor_(size)(state, weight, 0);

  int dimc = 2;
  int dimr = 1;
  int dimp = 0;

  if (input->dim() == 4) {
    dimc++;
    dimr++;
    dimp++;
  }

  int64_t nInputCols = input->size(dimc);
  int64_t nInputRows = input->size(dimr);
  THArgCheck(input->size(dimp) == nInputPlane, 2, "invalid number of input planes");
  THArgCheck(nInputCols >= kW && nInputRows >= kH, 2, "input image smaller than kernel size");
}

void THNN_(SpatialSubSampling_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           THCTensor *weight,
           THCTensor *bias,
           int kW, int kH,
           int dW, int dH)
{
  scalar_t *weight_data = THCTensor_(data)(state, weight);
  scalar_t *bias_data = THCTensor_(data)(state, bias);
  scalar_t *output_data;
  scalar_t *input_data;

  int nInputPlane = THCTensor_(size)(state, weight, 0);

  THCUNN_assertSameGPU(state, 4, input, output, weight, bias);
  THNN_(SpatialSubSampling_shapeCheck)(state, input, NULL, weight, kW, kH);

  if (input->dim() == 3) {
    int64_t nInputCols = input->size(2);
    int64_t nInputRows = input->size(1);
    int64_t nOutputCols = (nInputCols - kW) / dW + 1;
    int64_t nOutputRows = (nInputRows - kH) / dH + 1;

    input = THCTensor_(newContiguous)(state, input);
    input_data = THCTensor_(data)(state, input);

    THCTensor_(resize3d)(state, output, nInputPlane, nOutputRows, nOutputCols);
    output_data = THCTensor_(data)(state, output);

    // cuda blocks & threads:
    int yblocks = (int)(16L / nInputPlane);
    yblocks = yblocks < 1 ? 1 : yblocks;
    dim3 blocks(nInputPlane,yblocks);
    dim3 threads(32,8);

    // run subsample kernel
    subsample<scalar_t, accreal> <<<blocks, threads, 0, THCState_getCurrentStream(state)>>> (
      input_data, output_data, weight_data, bias_data,
      nInputPlane, nInputRows, nInputCols, kH, kW, dH, dW);
    THCudaCheck(cudaGetLastError());
  } else {
    int64_t nInputCols = input->size(3);
    int64_t nInputRows = input->size(2);
    int64_t nbatch = input->size(0);
    int64_t nOutputCols = (nInputCols - kW) / dW + 1;
    int64_t nOutputRows = (nInputRows - kH) / dH + 1;

    input = THCTensor_(newContiguous)(state, input);
    input_data = THCTensor_(data)(state, input);

    THCTensor_(resize4d)(state, output, nbatch, nInputPlane, nOutputRows, nOutputCols);
    output_data = THCTensor_(data)(state, output);

    // cuda blocks & threads:
    int yblocks = (int)(16L / nInputPlane);
    yblocks = yblocks < 1 ? 1 : yblocks;
    dim3 blocks(nInputPlane*nbatch,yblocks);
    dim3 threads(32,8);

    // run subsample kernel
    subsample<scalar_t, accreal> <<<blocks, threads, 0, THCState_getCurrentStream(state)>>> (
      input_data, output_data, weight_data, bias_data,
      nInputPlane, nInputRows, nInputCols, kH, kW, dH, dW);
    THCudaCheck(cudaGetLastError());
  }

  // clean
  THCTensor_(free)(state, input);

}

void THNN_(SpatialSubSampling_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           THCTensor *weight,
           int kW, int kH,
           int dW, int dH)
{
  THCUNN_assertSameGPU(state, 4, input, gradOutput, weight, gradInput);
  THNN_(SpatialSubSampling_shapeCheck)(state, input, gradOutput, weight, kW, kH);

  int nInputPlane = THCTensor_(size)(state, weight, 0);

  if (input->dim() == 3) {
    int64_t nInputCols = input->size(2);
    int64_t nInputRows = input->size(1);

    scalar_t *weight_data = THCTensor_(data)(state, weight);
    gradOutput = THCTensor_(newContiguous)(state, gradOutput);
    scalar_t *gradOutput_data = THCTensor_(data)(state, gradOutput);
    scalar_t *gradInput_data;

    THCTensor_(resizeAs)(state, gradInput, input);
    THCTensor_(zero)(state, gradInput);
    gradInput_data = THCTensor_(data)(state, gradInput);

    // cuda blocks & threads:
    int yblocks = (int)(16L / nInputPlane);
    yblocks = yblocks < 1 ? 1 : yblocks;
    dim3 blocks(nInputPlane,yblocks);
    dim3 threads(32,8);

    // run updateGradInput kernel
    if (kH <= dH && kW <= dW) {
      subgradinput <<<blocks, threads, 0, THCState_getCurrentStream(state)>>> (
        gradInput_data, gradOutput_data, weight_data,
        nInputPlane, nInputRows, nInputCols, kH, kW, dH, dW);
    } else {
      subgradinputAtomic <<<blocks, threads, 0, THCState_getCurrentStream(state)>>> (
        gradInput_data, gradOutput_data, weight_data,
        nInputPlane, nInputRows, nInputCols, kH, kW, dH, dW);
    }
    THCudaCheck(cudaGetLastError());
  } else {
    int64_t nInputCols = input->size(3);
    int64_t nInputRows = input->size(2);
    int64_t nbatch = input->size(0);

    scalar_t *weight_data = THCTensor_(data)(state, weight);
    gradOutput = THCTensor_(newContiguous)(state, gradOutput);
    scalar_t *gradOutput_data = THCTensor_(data)(state, gradOutput);
    scalar_t *gradInput_data;

    THCTensor_(resizeAs)(state, gradInput, input);
    THCTensor_(zero)(state, gradInput);
    gradInput_data = THCTensor_(data)(state, gradInput);

    // cuda blocks & threads:
    int yblocks = (int)(16L / nInputPlane);
    yblocks = yblocks < 1 ? 1 : yblocks;
    dim3 blocks(nInputPlane*nbatch,yblocks);
    dim3 threads(32,8);

    // run updateGradInput kernel
    if (kH <= dH && kW <= dW) {
      subgradinput <<<blocks, threads, 0, THCState_getCurrentStream(state)>>> (
        gradInput_data, gradOutput_data, weight_data,
        nInputPlane, nInputRows, nInputCols, kH, kW, dH, dW);
    } else {
      subgradinputAtomic <<<blocks, threads, 0, THCState_getCurrentStream(state)>>> (
        gradInput_data, gradOutput_data, weight_data,
        nInputPlane, nInputRows, nInputCols, kH, kW, dH, dW);
    }
    THCudaCheck(cudaGetLastError());
  }
  THCTensor_(free)(state, gradOutput);
}

void THNN_(SpatialSubSampling_accGradParameters)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradWeight,
           THCTensor *gradBias,
           int kW, int kH,
           int dW, int dH,
           accreal scale)
{
  THCUNN_assertSameGPU(state, 4, input, gradOutput, gradWeight, gradBias);
  THNN_(SpatialSubSampling_shapeCheck)(state, input, gradOutput, gradWeight, kW, kH);

  int nInputPlane = THCTensor_(size)(state, gradWeight, 0);

  if (input->dim() == 3) {
    int64_t nInputCols = input->size(2);
    int64_t nInputRows = input->size(1);

    scalar_t *gradWeight_data = THCTensor_(data)(state, gradWeight);
    scalar_t *gradBias_data = THCTensor_(data)(state, gradBias);
    gradOutput = THCTensor_(newContiguous)(state, gradOutput);
    scalar_t *gradOutput_data = THCTensor_(data)(state, gradOutput);
    scalar_t *input_data;

    input = THCTensor_(newContiguous)(state, input);
    input_data = THCTensor_(data)(state, input);

    // cuda blocks & threads:
    dim3 blocks(nInputPlane);
    dim3 threads(32,8);

    // run gradweight kernel
    subgradweight<scalar_t, accreal> <<<blocks, threads, 0, THCState_getCurrentStream(state)>>> (
      input_data, gradOutput_data, gradWeight_data, gradBias_data,
      nInputPlane, nInputRows, nInputCols, kH, kW, dH, dW, scale);
    THCudaCheck(cudaGetLastError());
  } else {
    int64_t nInputCols = input->size(3);
    int64_t nInputRows = input->size(2);
    int64_t nbatch = input->size(0);

    scalar_t *gradWeight_data = THCTensor_(data)(state, gradWeight);
    scalar_t *gradBias_data = THCTensor_(data)(state, gradBias);
    gradOutput = THCTensor_(newContiguous)(state, gradOutput);
    scalar_t *gradOutput_data = THCTensor_(data)(state, gradOutput);
    scalar_t *input_data;

    input = THCTensor_(newContiguous)(state, input);
    input_data = THCTensor_(data)(state, input);

    // cuda blocks & threads:
    dim3 blocks(nInputPlane);
    dim3 threads(32,8);

    // run gradweight kernel
    int64_t sl;
    for (sl=0; sl<nbatch; sl++) {
      subgradweight<scalar_t, accreal> <<<blocks, threads, 0, THCState_getCurrentStream(state)>>> (
        input_data + sl*input->stride(0),
        gradOutput_data + sl*gradOutput->stride(0),
        gradWeight_data, gradBias_data,
        nInputPlane, nInputRows, nInputCols, kH, kW, dH, dW, scale);
    }
    THCudaCheck(cudaGetLastError());
  }

  // clean
  THCTensor_(free)(state, input);
  THCTensor_(free)(state, gradOutput);

}

#endif

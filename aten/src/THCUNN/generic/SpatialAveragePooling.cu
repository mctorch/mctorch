
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THCUNN/generic/SpatialAveragePooling.cu"
#else

#include <THCUNN/common.h>
#include <THCUNN/generic/pooling_shape.h>

static inline void THNN_(SpatialAveragePooling_shapeCheck)(
  THCState *state,
  THCTensor *input, THCTensor *gradOutput,
  int kH, int kW, int dH, int dW, int padH, int padW, bool ceil_mode) {

  THArgCheck(kW > 0 && kH > 0, 5,
             "kernel size should be greater than zero, but got kH: %d kW: %d", kH, kW);
  THArgCheck(dW > 0 && dH > 0, 8,
             "stride should be greater than zero, but got dH: %d dW: %d", dH, dW);

  int ndim = input->dim();
  int dimf = 0;
  int dimh = 1;
  int dimw = 2;

  if (ndim == 4) {
    dimf++;
    dimh++;
    dimw++;
  }

  THCUNN_argCheck(state, !input->is_empty() && (ndim == 3 || ndim == 4), 2, input,
                  "non-empty 3D or 4D input tensor expected but got: %s");
  THArgCheck(kW/2 >= padW && kH/2 >= padH, 2,
             "pad should be smaller than half of kernel size, but got "
             "padW = %d, padH = %d, kW = %d, kH = %d",
             padW, padH, kW, kH);

  int64_t nInputPlane = input->size(dimh-1);
  int64_t nInputRows = input->size(dimh);
  int64_t nInputCols = input->size(dimw);
  int64_t nOutputPlane = nInputPlane;

  int64_t nOutputCols = pooling_output_shape<int64_t>(nInputCols, kW, padW, dW, 1, ceil_mode);
  int64_t nOutputRows = pooling_output_shape<int64_t>(nInputRows, kH, padH, dH, 1, ceil_mode);

  if (nOutputCols < 1 || nOutputRows < 1)
    THError("Given input size: (%dx%dx%d). "
            "Calculated output size: (%dx%dx%d). Output size is too small",
            nInputPlane,nInputRows,nInputCols,nInputPlane,nOutputRows,nOutputCols);

  if (gradOutput != NULL) {
    THCUNN_check_dim_size(state, gradOutput, ndim, dimf, nOutputPlane);
    THCUNN_check_dim_size(state, gradOutput, ndim, dimh, nOutputRows);
    THCUNN_check_dim_size(state, gradOutput, ndim, dimw, nOutputCols);
  }
}

void THNN_(SpatialAveragePooling_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           int kW, int kH,
           int dW, int dH,
           int padW, int padH,
           bool ceil_mode,
           bool count_include_pad)
{
  THCUNN_assertSameGPU(state, 2, input, output);
  THNN_(SpatialAveragePooling_shapeCheck)
       (state, input, NULL, kH, kW, dH, dW,
        padH, padW, ceil_mode);

  int64_t nInputCols, nInputRows, nInputPlane, batchSize;
  int64_t nOutputCols, nOutputRows;

  if (input->dim() == 3) {
    nInputCols = input->size(2);
    nInputRows = input->size(1);
    nInputPlane = input->size(0);
    batchSize = 1;
  }
  else
  {
    nInputCols = input->size(3);
    nInputRows = input->size(2);
    nInputPlane = input->size(1);
    batchSize = input->size(0);
  }

  nOutputCols = pooling_output_shape<int64_t>(nInputCols, kW, padW, dW, 1, ceil_mode);
  nOutputRows = pooling_output_shape<int64_t>(nInputRows, kH, padH, dH, 1, ceil_mode);

  input = THCTensor_(newContiguous)(state, input);
  scalar_t* input_data = THCTensor_(data)(state, input);

  THCTensor_(resize4d)(state, output, batchSize, nInputPlane, nOutputRows, nOutputCols);

  scalar_t* output_data = THCTensor_(data)(state, output);

  int count = THCTensor_(nElement)(state, output);

  if(count_include_pad)
    AvePoolForward<scalar_t, accreal, true>
      <<<GET_BLOCKS(count), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state) >>>(
        count, input_data,
        batchSize, nInputPlane, nInputRows, nInputCols, nOutputRows, nOutputCols,
        kH, kW, dH, dW, padH, padW, output_data);
  else
    AvePoolForward<scalar_t, accreal, false>
      <<<GET_BLOCKS(count), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state) >>>(
        count, input_data,
        batchSize, nInputPlane, nInputRows, nInputCols, nOutputRows, nOutputCols,
        kH, kW, dH, dW, padH, padW, output_data);
  THCudaCheck(cudaGetLastError());

  if(input->dim() == 3)
    THCTensor_(resize3d)(state, output, nInputPlane, nOutputRows, nOutputCols);

  THCTensor_(free)(state, input);

}

void THNN_(SpatialAveragePooling_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           int kW, int kH,
           int dW, int dH,
           int padW, int padH,
           bool ceil_mode,
           bool count_include_pad)
{
  THCUNN_assertSameGPU(state, 3, input, gradOutput, gradInput);
  THNN_(SpatialAveragePooling_shapeCheck)
       (state, input, gradOutput, kH, kW, dH, dW,
        padH, padW, ceil_mode);

  input = THCTensor_(newContiguous)(state, input);
  gradOutput = THCTensor_(newContiguous)(state, gradOutput);

  int64_t nInputCols, nInputRows, nInputPlane, batchSize;
  int64_t nOutputCols, nOutputRows;
  int dimCol = 2;
  int dimRow = 1;

  if (input->dim() == 3) {
    nInputPlane = input->size(0);
    batchSize = 1;
  }
  else
  {
    dimCol = 3;
    dimRow = 2;
    nInputPlane = input->size(1);
    batchSize = input->size(0);
  }
  nInputCols = input->size(dimCol);
  nInputRows = input->size(dimRow);

  nOutputCols = pooling_output_shape<int64_t>(nInputCols, kW, padW, dW, 1, ceil_mode);
  nOutputRows = pooling_output_shape<int64_t>(nInputRows, kH, padH, dH, 1, ceil_mode);

  THCUNN_check_dim_size(state, gradOutput, input->dim(), dimRow, nOutputRows);
  THCUNN_check_dim_size(state, gradOutput, input->dim(), dimCol, nOutputCols);
  THCTensor_(resizeAs)(state, gradInput, input);

  int count = THCTensor_(nElement)(state, input);

  if(count_include_pad)
    AvePoolBackward<scalar_t, accreal, true>
      <<< GET_BLOCKS(count), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state) >>>
        (count,
        THCTensor_(data)(state, gradOutput),
        batchSize, nInputPlane, nInputRows, nInputCols, nOutputRows, nOutputCols,
        kH, kW, dH, dW, padH, padW,
        THCTensor_(data)(state, gradInput));
  else
    AvePoolBackward<scalar_t, accreal, false>
      <<< GET_BLOCKS(count), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state) >>>
        (count,
        THCTensor_(data)(state, gradOutput),
        batchSize, nInputPlane, nInputRows, nInputCols, nOutputRows, nOutputCols,
        kH, kW, dH, dW, padH, padW,
        THCTensor_(data)(state, gradInput));
  THCudaCheck(cudaGetLastError());

  // clean
  THCTensor_(free)(state, input);
  THCTensor_(free)(state, gradOutput);
}

#endif

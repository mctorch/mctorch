
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THCUNN/generic/VolumetricConvolution.cu"
#else

#include <ATen/div_rtn.h>

static inline void THNN_(VolumetricConvolution_shapeCheck)
                        (THCState *state,
                         THCTensor *input,
                         THCTensor *gradOutput,
                         THCTensor *weight,
                         THCTensor *gradWeight,
                         THCTensor *bias,
                         int dT,
                         int dW,
                         int dH,
                         int padT,
                         int padW,
                         int padH) {
  THCUNN_argCheck(state, !input->is_empty() && (input->dim() == 4 || input->dim() == 5), 2, input,
                  "non-empty 4D or 5D (batch mode) tensor expected for input, but got: %s");
  THArgCheck(!weight || THCTensor_(isContiguous)(state, weight), 4,
             "weight tensor has to be contiguous");
  THArgCheck(!bias || THCTensor_(isContiguous)(state, bias), 5,
             "bias tensor has to be contiguous");
  THArgCheck(!gradWeight || THCTensor_(isContiguous)(state, gradWeight), 5,
             "gradWeight tensor has to be contiguous");
  THArgCheck(dT > 0 && dW > 0 && dH > 0, 10,
             "stride should be greater than zero, but got dT: %d dH: %d dW: %d", dT, dH, dW);

  if (gradOutput != NULL) {
    THCUNN_argCheck(state, !gradOutput->is_empty() && (gradOutput->dim() == 4 || gradOutput->dim() == 5), 3,
                    gradOutput,
                    "non-empty 4D or 5D (batch mode) tensor expected for gradOutput, but got: %s");
  }

  if (weight != NULL) {
    THCUNN_argCheck(state, !weight->is_empty() && weight->dim() == 5, 4, weight,
                    "non-empty 5D (nOutputPlane x nInputPlane x kT x kH x kW) tensor "
                    "expected for weight, but got: %s");
  }

  if (gradWeight != NULL) {
    THCUNN_argCheck(state, !gradWeight->is_empty() && gradWeight->dim() == 5, 4, gradWeight,
                    "non-empty 5D (nOutputPlane x nInputPlane x kT x kH x kW) tensor "
                    "expected for gradWeight, but got: %s");
  }

  if (weight == NULL) {
    weight = gradWeight;
  }
  int64_t nOutputPlane = weight->size(0);
  int64_t nInputPlane  = weight->size(1);
  int64_t kT           = weight->size(2);
  int64_t kH           = weight->size(3);
  int64_t kW           = weight->size(4);

  THArgCheck(kT > 0 && kW > 0 && kH > 0, 4,
             "kernel size should be greater than zero, but got kT: %d kH: %d kW: %d", kT, kH, kW);
  int ndim = input->dim();
  int dimf = 0;
  int dimh = 1;
  int dimw = 2;
  int dimd = 3;

  if (ndim == 5)
  {
    dimf++;
    dimh++;
    dimw++;
    dimd++;
  }

  int64_t inputWidth   = input->size(dimw);
  int64_t inputHeight  = input->size(dimh);
  int64_t inputDepth   = input->size(dimd);

  int64_t exactInputDepth = inputDepth + 2*padT;
  int64_t exactInputHeight = inputHeight + 2*padH;
  int64_t exactInputWidth = inputWidth + 2*padW;

  if (exactInputDepth < kT || exactInputHeight < kH || exactInputWidth < kW) {
    THError("Calculated input size: (%d x %d x %d). "
      "Kernel size: (%d x %d x %d). Kernel size can't be greater than actual input size",
      exactInputDepth,exactInputHeight,exactInputWidth,kT,kH,kW);
  }

  int64_t outputWidth  = div_rtn<int64_t>(exactInputDepth - kH, dH) + 1;
  int64_t outputHeight = div_rtn<int64_t>(exactInputHeight - kT, dT) + 1;
  int64_t outputDepth  = div_rtn<int64_t>(exactInputWidth - kW, dW) + 1;

  if (outputWidth < 1 || outputHeight < 1 || outputDepth < 1)
  {
    THError(
      "Given input size: (%dx%dx%dx%d). Calculated output size: (%dx%dx%dx%d). Output size is too small",
      nInputPlane, inputDepth, inputHeight, inputWidth,
      nOutputPlane, outputDepth, outputHeight, outputWidth
    );
  }

  if (bias != NULL) {
    THCUNN_check_dim_size(state, bias, 1, 0, weight->size(0));
  }
  THCUNN_check_dim_size(state, input, ndim, dimf, nInputPlane);

  if (gradOutput != NULL) {
     THCUNN_check_dim_size(state, gradOutput, ndim, dimf, nOutputPlane);
     THCUNN_check_dim_size(state, gradOutput, ndim, dimh, outputHeight);
     THCUNN_check_dim_size(state, gradOutput, ndim, dimw, outputWidth);
     THCUNN_check_dim_size(state, gradOutput, ndim, dimd, outputDepth);
  }
}

void THNN_(VolumetricConvolution_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           THCTensor *weight,
           THCTensor *bias,
           THCTensor *finput,
           THCTensor *fgradInput,
           int dT, int dW, int dH,
           int padT, int padW, int padH)
{
  THCTensor *columns = finput;
  THCTensor *ones = fgradInput;
  THCUNN_assertSameGPU(state, 6, input, output, weight, bias, columns, ones);
  THNN_(VolumetricConvolution_shapeCheck)(
        state, input, NULL, weight, NULL,
        bias, dT, dW, dH, padT, padW, padH);
  input = THCTensor_(newContiguous)(state, input);

  int nOutputPlane = (int)weight->size(0);
  int nInputPlane  = (int)weight->size(1);
  int kT           = (int)weight->size(2);
  int kH           = (int)weight->size(3);
  int kW           = (int)weight->size(4);

  int batch = 1;
  if (input->dim() == 4)
  {
    // Force batch
    batch = 0;
    THCTensor_(resize5d)(state, input, 1, input->size(0), input->size(1),
                          input->size(2), input->size(3));
  }

  int64_t inputWidth   = input->size(3);
  int64_t inputHeight  = input->size(2);
  int64_t inputDepth   = input->size(4);
  int64_t outputWidth  = (inputWidth  + 2*padH - kH) / dH + 1;
  int64_t outputHeight = (inputHeight + 2*padT - kT) / dT + 1;
  int64_t outputDepth  = (inputDepth  + 2*padW - kW) / dW + 1;

  // Batch size + input planes
  int64_t batchSize = input->size(0);

  // Resize output
  THCTensor_(resize5d)(state, output, batchSize, nOutputPlane,
                        outputHeight, outputWidth, outputDepth);

  // Resize temporary columns
  THCTensor_(resize2d)(state, columns, nInputPlane*kW*kH*kT, outputDepth*outputHeight*outputWidth);

  // Define a buffer of ones, for bias accumulation
  // Note: this buffer can be shared with other modules, it only ever gets increased,
  // and always contains ones.
  if (ones->dim() != 3 || ones->size(0)*ones->size(1)*ones->size(2) < outputDepth*outputHeight*outputWidth)
  {
    // Resize plane and fill with ones...
    THCTensor_(resize3d)(state, ones, outputHeight, outputWidth, outputDepth);
    THCTensor_(fill)(state, ones, ScalarConvert<int, scalar_t>::to(1));
  }

  // Helpers
  THCTensor *input_n = THCTensor_(new)(state);
  THCTensor *output_n = THCTensor_(new)(state);

  // For each elt in batch, do:
  for (int elt = 0; elt < batchSize; elt ++)
  {
    // Matrix mulitply per output:
    THCTensor_(select)(state, input_n, input, 0, elt);
    THCTensor_(select)(state, output_n, output, 0, elt);

    // Do Bias first:
    // M,N,K are dims of matrix A and B
    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
    int64_t m_ = nOutputPlane;
    int64_t n_ = outputDepth * outputHeight * outputWidth;
    int64_t k_ = 1;

    // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
    if (bias) {
      #ifdef THC_REAL_IS_FLOAT
      THCudaBlas_Sgemm(
      #elif defined(THC_REAL_IS_HALF)
      THCudaBlas_Hgemm(
      #elif defined(THC_REAL_IS_DOUBLE)
      THCudaBlas_Dgemm(
      #endif
        state,
        't', 'n',
        n_, m_, k_,
        ScalarConvert<int, scalar_t>::to(1),
        THCTensor_(data)(state, ones), k_,
        THCTensor_(data)(state, bias), k_,
        ScalarConvert<int, scalar_t>::to(0),
        THCTensor_(data)(state, output_n), n_
      );
    } else {
      THCTensor_(zero)(state, output_n);
    }

    // Extract columns:
    im3d2col(
      THCState_getCurrentStream(state),
      THCTensor_(data)(state, input_n),
      nInputPlane, inputHeight, inputWidth, inputDepth, kT, kH, kW, padT, padH, padW, dT, dH, dW,
      THCTensor_(data)(state, columns)
    );

    // M,N,K are dims of matrix A and B
    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
    int64_t m = weight->size(0);
    int64_t n = columns->size(1);
    int64_t k = weight->size(1)*weight->size(2)*weight->size(3)*weight->size(4);

    // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
    #ifdef THC_REAL_IS_FLOAT
    THCudaBlas_Sgemm(
    #elif defined(THC_REAL_IS_HALF)
    THCudaBlas_Hgemm(
    #elif defined(THC_REAL_IS_DOUBLE)
    THCudaBlas_Dgemm(
    #endif
      state,
      'n', 'n',
      n, m, k,
      ScalarConvert<int, scalar_t>::to(1),
      THCTensor_(data)(state, columns), n,
      THCTensor_(data)(state, weight), k,
      ScalarConvert<int, scalar_t>::to(1),
      THCTensor_(data)(state, output_n), n
    );
  }

  // Free
  THCTensor_(free)(state, input_n);
  THCTensor_(free)(state, output_n);

  // Resize output
  if (batch == 0)
  {
    THCTensor_(resize4d)(state, output, nOutputPlane, outputHeight, outputWidth, outputDepth);
    THCTensor_(resize4d)(state, input, nInputPlane, inputHeight, inputWidth, inputDepth);
  }
  THCTensor_(free)(state, input);
}

void THNN_(VolumetricConvolution_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           THCTensor *weight,
           THCTensor *finput,
           int dT, int dW, int dH,
           int padT, int padW, int padH)
{

  int64_t nOutputPlane = weight->size(0);
  int64_t nInputPlane  = weight->size(1);
  int64_t kT           = weight->size(2);
  int64_t kH           = weight->size(3);
  int64_t kW           = weight->size(4);

  THCTensor *gradColumns = finput;

  THCUNN_assertSameGPU(state, 5, input, gradOutput, weight, gradColumns, gradInput);
  THNN_(VolumetricConvolution_shapeCheck)(
        state, input, gradOutput, weight, NULL,
        NULL, dT, dW, dH, padT, padW, padH);
  gradOutput = THCTensor_(newContiguous)(state, gradOutput);

  int batch = 1;
  if (input->dim() == 4)
  {
    input = THCTensor_(newContiguous)(state, input);
    // Force batch
    batch = 0;
    THCTensor_(resize5d)(state, input, 1, input->size(0), input->size(1), input->size(2), input->size(3));
    THCTensor_(resize5d)(state, gradOutput, 1, gradOutput->size(0), gradOutput->size(1), gradOutput->size(2), gradOutput->size(3));
  }

  int64_t inputWidth   = input->size(3);
  int64_t inputHeight  = input->size(2);
  int64_t inputDepth   = input->size(4);
  int64_t outputWidth  = (inputWidth  + 2*padH - kH) / dH + 1;
  int64_t outputHeight = (inputHeight + 2*padT - kT) / dT + 1;
  int64_t outputDepth  = (inputDepth  + 2*padW - kW) / dW + 1;

  // Batch size + input planes
  int64_t batchSize = input->size(0);

  // Resize output
  THCTensor_(resize5d)(state, gradInput, batchSize, nInputPlane, inputHeight, inputWidth, inputDepth);

  // Resize temporary columns
  THCTensor_(resize2d)(state, gradColumns, nInputPlane*kH*kT*kW, outputDepth*outputHeight*outputWidth);

  // Helpers
  THCTensor *gradInput_n = THCTensor_(new)(state);
  THCTensor *gradOutput_n = THCTensor_(new)(state);

  // For each elt in batch, do:
  for (int elt = 0; elt < batchSize; elt ++)
  {
    // Matrix mulitply per sample:
    THCTensor_(select)(state, gradInput_n, gradInput, 0, elt);
    THCTensor_(select)(state, gradOutput_n, gradOutput, 0, elt);

    // M,N,K are dims of matrix A and B
    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
    int64_t m = weight->size(1)*weight->size(2)*weight->size(3)*weight->size(4);
    int64_t n = gradColumns->size(1);
    int64_t k = weight->size(0);

    // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
    #ifdef THC_REAL_IS_FLOAT
    THCudaBlas_Sgemm(
    #elif defined(THC_REAL_IS_HALF)
    THCudaBlas_Hgemm(
    #elif defined(THC_REAL_IS_DOUBLE)
    THCudaBlas_Dgemm(
    #endif
      state,
      'n', 't',
      n, m, k,
      ScalarConvert<int, scalar_t>::to(1),
      THCTensor_(data)(state, gradOutput_n), n,
      THCTensor_(data)(state, weight), m,
      ScalarConvert<int, scalar_t>::to(0),
      THCTensor_(data)(state, gradColumns), n
    );

    // Unpack columns back into input:
    col2im3d<scalar_t, accreal>(
      THCState_getCurrentStream(state),
      THCTensor_(data)(state, gradColumns),
      nInputPlane, inputHeight, inputWidth, inputDepth, kT, kH, kW, padT, padH, padW, dT, dH, dW,
      THCTensor_(data)(state, gradInput_n)
    );
  }

  // Free
  THCTensor_(free)(state, gradInput_n);
  THCTensor_(free)(state, gradOutput_n);

  // Resize output
  if (batch == 0)
  {
    THCTensor_(resize4d)(state, gradOutput, nOutputPlane, outputHeight, outputWidth, outputDepth);
    THCTensor_(resize4d)(state, input, nInputPlane, inputHeight, inputWidth, inputDepth);
    THCTensor_(resize4d)(state, gradInput, nInputPlane, inputHeight, inputWidth, inputDepth);
    THCTensor_(free)(state, input);
  }
  THCTensor_(free)(state, gradOutput);

}

void THNN_(VolumetricConvolution_accGradParameters)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradWeight,
           THCTensor *gradBias,
           THCTensor *finput,
           THCTensor *fgradInput,
           int dT, int dW, int dH,
           int padT, int padW, int padH,
           accreal scale_)
{
  scalar_t scale = ScalarConvert<accreal, scalar_t>::to(scale_);
  THCTensor *columns = finput;
  THCTensor *ones = fgradInput;
  THCUNN_assertSameGPU(state, 6, input, gradOutput, gradWeight, gradBias, columns, ones);
  THNN_(VolumetricConvolution_shapeCheck)(
        state, input, gradOutput, NULL, gradWeight,
        gradBias, dT, dW, dH, padT, padW, padH);

  int nOutputPlane = (int)gradWeight->size(0);
  int nInputPlane  = (int)gradWeight->size(1);
  int kT           = (int)gradWeight->size(2);
  int kH           = (int)gradWeight->size(3);
  int kW           = (int)gradWeight->size(4);

  input = THCTensor_(newContiguous)(state, input);
  gradOutput = THCTensor_(newContiguous)(state, gradOutput);

  int batch = 1;
  if (input->dim() == 4)
  {
    // Force batch
    batch = 0;
    THCTensor_(resize5d)(state, input, 1, input->size(0), input->size(1), input->size(2), input->size(3));
    THCTensor_(resize5d)(state, gradOutput, 1, gradOutput->size(0), gradOutput->size(1), gradOutput->size(2), gradOutput->size(3));
  }

  int64_t inputWidth   = input->size(3);
  int64_t inputHeight  = input->size(2);
  int64_t inputDepth   = input->size(4);
  int64_t outputWidth  = (inputWidth  + 2*padH - kH) / dH + 1;
  int64_t outputHeight = (inputHeight + 2*padT - kT) / dT + 1;
  int64_t outputDepth  = (inputDepth  + 2*padW - kW) / dW + 1;

  // Batch size + input planes
  int64_t batchSize = input->size(0);

  // Define a buffer of ones, for bias accumulation
  if (ones->dim() != 3 || ones->size(0)*ones->size(1)*ones->size(2) < outputDepth*outputHeight*outputWidth)
  {
    // Resize plane and fill with ones...
    THCTensor_(resize3d)(state, ones, outputHeight, outputWidth, outputDepth);
    THCTensor_(fill)(state, ones, ScalarConvert<int, scalar_t>::to(1));
  }

  // Resize temporary columns
  THCTensor_(resize2d)(state, columns, nInputPlane*kH*kT*kW, outputDepth*outputHeight*outputWidth);

  // Helpers
  THCTensor *input_n = THCTensor_(new)(state);
  THCTensor *gradOutput_n = THCTensor_(new)(state);

  // For each elt in batch, do:
  for (int elt = 0; elt < batchSize; elt ++)
  {
    // Matrix mulitply per output:
    THCTensor_(select)(state, input_n, input, 0, elt);
    THCTensor_(select)(state, gradOutput_n, gradOutput, 0, elt);

    // Extract columns:
    im3d2col(
      THCState_getCurrentStream(state),
      THCTensor_(data)(state, input_n),
      nInputPlane, inputHeight, inputWidth, inputDepth, kT, kH, kW, padT, padH, padW, dT, dH, dW,
      THCTensor_(data)(state, columns)
    );

    // M,N,K are dims of matrix A and B
    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
    int64_t m = gradWeight->size(0);
    int64_t n = gradWeight->size(1)*gradWeight->size(2)*gradWeight->size(3)*gradWeight->size(4);
    int64_t k = columns->size(1);

    // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
    #ifdef THC_REAL_IS_FLOAT
    THCudaBlas_Sgemm(
    #elif defined(THC_REAL_IS_HALF)
    THCudaBlas_Hgemm(
    #elif defined(THC_REAL_IS_DOUBLE)
    THCudaBlas_Dgemm(
    #endif
      state,
      't', 'n',
      n, m, k,
      scale,
      THCTensor_(data)(state, columns), k,
      THCTensor_(data)(state, gradOutput_n), k,
      ScalarConvert<int, scalar_t>::to(1),
      THCTensor_(data)(state, gradWeight), n
    );

    // Do Bias:
    // M,N,K are dims of matrix A and B
    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
    int64_t m_ = nOutputPlane;
    int64_t k_ = outputDepth * outputHeight * outputWidth;

    // Do GEMV (note: this is a bit confusing because gemv assumes column-major matrices)
    if (gradBias) {
      #if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE)
      #ifdef THC_REAL_IS_FLOAT
      THCudaBlas_Sgemv(
      #elif defined(THC_REAL_IS_DOUBLE)
      THCudaBlas_Dgemv(
      #endif
        state,
        't',
        k_, m_,
        scale,
        THCTensor_(data)(state, gradOutput_n), k_,
        THCTensor_(data)(state, ones), 1,
        ScalarConvert<int, scalar_t>::to(1),
        THCTensor_(data)(state, gradBias), 1
      );
      #endif
      #ifdef THC_REAL_IS_HALF
      THCudaBlas_Hgemm(
        state,
        't', 'n',
        m_, 1, k_,
        scale,
        THCTensor_(data)(state, gradOutput_n), k_,
        THCTensor_(data)(state, ones), k_,
        ScalarConvert<int, scalar_t>::to(1),
        THCTensor_(data)(state, gradBias), m_
      );
      #endif
    }
  }

  // Free
  THCTensor_(free)(state, input_n);
  THCTensor_(free)(state, gradOutput_n);

  // Resize
  if (batch == 0)
  {
    THCTensor_(resize4d)(state, gradOutput, nOutputPlane, outputHeight, outputWidth, outputDepth);
    THCTensor_(resize4d)(state, input, nInputPlane, inputHeight, inputWidth, inputDepth);
  }
  THCTensor_(free)(state, input);
  THCTensor_(free)(state, gradOutput);
}

#endif

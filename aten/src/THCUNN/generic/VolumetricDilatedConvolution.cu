
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THCUNN/generic/VolumetricDilatedConvolution.cu"
#else

#include <ATen/div_rtn.h>

static inline void THNN_(VolumetricDilatedConvolution_shapeCheck)(
                         THCState *state,
                         THCTensor *input,
                         THCTensor *gradOutput,
                         THCTensor *weight,
                         THCTensor *bias,
                         int kT, int kH, int kW,
                         int dT, int dH, int dW,
                         int padT, int padH, int padW,
                         int dilationT, int dilationH, int dilationW,
                         int weight_nullable) {
  THCUNN_argCheck(state, !input->is_empty() && (input->dim() == 4 || input->dim() == 5), 2, input,
                  "non-empty 4D or 5D (batch mode) tensor expected for input, but got: %s");
  THArgCheck(kT > 0 && kW > 0 && kH > 0, 8,
             "kernel size should be greater than zero, but got kT: %d kH: %d kW: %d", kT, kH, kW);
  THArgCheck(dT > 0 && dW > 0 && dH > 0, 11,
             "stride should be greater than zero, but got dT: %d dH: %d dW: %d", dT, dH, dW);
  THArgCheck(!bias || THCTensor_(isContiguous)(state, bias), 5,
             "bias tensor has to be contiguous");
  THArgCheck(dilationT > 0 && dilationW > 0 && dilationH > 0, 15,
             "dilation should be greater than zero, but got dilationT: %d, dilationH: %d, dilationW: %d",
             dilationT, dilationH, dilationW);

   // number of input & output planes and kernel size is indirectly defined by the weight tensor
  if (weight != NULL) {
    THCUNN_argCheck(state, !weight->is_empty() && weight->dim() == 5, 4, weight,
                  "non-empty 5D (nOutputPlane x nInputPlane x kT x kH x kW) tensor "
                  "expected for weight, but got: %s");
    if (bias != NULL) {
      THCUNN_check_dim_size(state, bias, 1, 0, weight->size(0));
    }
  } else if (!weight_nullable) {
    THError("weight tensor is expected to be non-nullable");
  }

  int ndim = input->dim();
  int dimf = 0;
  int dimd = 1;
  int dimh = 2;
  int dimw = 3;

  if (ndim == 5) {
    dimf++;
    dimd++;
    dimh++;
    dimw++;
  }

  int64_t inputDepth  = input->size(dimd);
  int64_t inputHeight  = input->size(dimh);
  int64_t inputWidth   = input->size(dimw);
  int64_t outputDepth  = div_rtn<int64_t>(inputDepth  + 2*padT - (dilationT * (kT - 1) + 1), dT) + 1;
  int64_t outputHeight = div_rtn<int64_t>(inputHeight + 2*padH - (dilationH * (kH - 1) + 1), dH) + 1;
  int64_t outputWidth  = div_rtn<int64_t>(inputWidth  + 2*padW - (dilationW * (kW - 1) + 1), dW) + 1;

  if (outputDepth < 1 || outputWidth < 1 || outputHeight < 1) {
    THError("Given input size per channel: (%ld x %ld x %ld). "
      "Calculated output size per channel: (%ld x %ld x %ld). Output size is too small",
      inputDepth, inputHeight, inputWidth, outputDepth, outputHeight, outputWidth);
  }

  if (weight != NULL) {
    int64_t nInputPlane = weight->size(1);
    THCUNN_check_dim_size(state, input, ndim, dimf, nInputPlane);
  }

  if (gradOutput != NULL) {
    if (weight != NULL) {
      int64_t nOutputPlane = weight->size(0);
      THCUNN_check_dim_size(state, gradOutput, ndim, dimf, nOutputPlane);
    } else if (bias != NULL) {
      int64_t nOutputPlane = THTensor_sizeLegacyNoScalars(bias, 0);
      THCUNN_check_dim_size(state, gradOutput, ndim, dimf, nOutputPlane);
    }
    THCUNN_check_dim_size(state, gradOutput, ndim, dimd, outputDepth);
    THCUNN_check_dim_size(state, gradOutput, ndim, dimh, outputHeight);
    THCUNN_check_dim_size(state, gradOutput, ndim, dimw, outputWidth);
  }
}

void THNN_(VolumetricDilatedConvolution_updateOutput)(
           THCState *state,
           THCTensor  *input,
           THCTensor  *output,
           THCTensor  *weight,
           THCTensor  *bias,
           THCTensor  *columns,
           THCTensor  *ones,
           int kT, int kW, int kH,
           int dT, int dW, int dH,
           int padT, int padW, int padH,
           int dilationT, int dilationW, int dilationH) {

  THCUNN_assertSameGPU(state, 5, input, output, weight, columns, ones);
  if (bias) {
    THCUNN_assertSameGPU(state, 2, weight, bias);
  }
  THNN_(VolumetricDilatedConvolution_shapeCheck)(
        state, input, NULL, weight, bias,
        kT, kH, kW, dT, dH, dW, padT, padH, padW,
        dilationT, dilationH, dilationW, 0);

  // Params:
  int nInputPlane = weight->size(1);
  int nOutputPlane = weight->size(0);

  input = THCTensor_(newContiguous)(state, input);
  weight = THCTensor_(newContiguous)(state, weight);
  bias = bias ? THCTensor_(newContiguous)(state, bias) : bias;

  int is_batch = 1;
  if (input->dim() == 4) {
    // Force batch
    is_batch = 0;
    THCTensor_(resize5d)(state, input, 1, input->size(0), input->size(1), input->size(2), input->size(3));
  }

  int64_t inputDepth  = input->size(2);
  int64_t inputHeight  = input->size(3);
  int64_t inputWidth   = input->size(4);
  int64_t outputDepth  = (inputDepth  + 2*padT - (dilationT * (kT - 1) + 1)) / dT + 1;
  int64_t outputHeight = (inputHeight + 2*padH - (dilationH * (kH - 1) + 1)) / dH + 1;
  int64_t outputWidth  = (inputWidth  + 2*padW - (dilationW * (kW - 1) + 1)) / dW + 1;

  // Batch size + input planes
  int64_t batchSize = input->size(0);

  // Resize output
  THCTensor_(resize5d)(state, output, batchSize, nOutputPlane, outputDepth, outputHeight, outputWidth);

  // Resize temporary columns
  THCTensor_(resize2d)(state, columns, nInputPlane*kT*kW*kH, outputDepth*outputHeight*outputWidth);

  // Define a buffer of ones, for bias accumulation
  // Note: this buffer can be shared with other modules, it only ever gets increased,
  // and always contains ones.
  if (ones->dim() != 2 || ones->size(0)*ones->size(1)*ones->size(2) < outputDepth*outputHeight*outputWidth) {
    // Resize plane and fill with ones...
    THCTensor_(resize3d)(state, ones, outputDepth, outputHeight, outputWidth);
    THCTensor_(fill)(state, ones, ScalarConvert<int, scalar_t>::to(1));
  }

  // Helpers
  THCTensor  *input_n = THCTensor_(new)(state);
  THCTensor  *output_n = THCTensor_(new)(state);

  // For each elt in batch, do:
  for (int elt = 0; elt < batchSize; elt ++) {
    // Matrix mulitply per output:
    THCTensor_(select)(state, input_n, input, 0, elt);
    THCTensor_(select)(state, output_n, output, 0, elt);

    // Do Bias first:
    // M,N,K are dims of matrix A and B
    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
    int64_t m_ = nOutputPlane;
    int64_t n_ = outputDepth * outputHeight * outputWidth;
    int64_t k_ = 1;

    // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
    if (bias) {
      #ifdef THC_REAL_IS_FLOAT
      THCudaBlas_Sgemm(
      #elif defined(THC_REAL_IS_HALF)
      THCudaBlas_Hgemm(
      #elif defined(THC_REAL_IS_DOUBLE)
      THCudaBlas_Dgemm(
      #endif
          state,
          't', 'n',
          n_, m_, k_,
          ScalarConvert<int, scalar_t>::to(1),
          THCTensor_(data)(state, ones), k_,
          THCTensor_(data)(state, bias), k_,
          ScalarConvert<int, scalar_t>::to(0),
          THCTensor_(data)(state, output_n), n_
      );
    } else {
      THCTensor_(zero)(state, output_n);
    }

    // Extract columns:
    vol2col(
      THCState_getCurrentStream(state),
      THCTensor_(data)(state, input_n),
      nInputPlane, inputDepth, inputHeight, inputWidth,
      outputDepth, outputHeight, outputWidth,
      kT, kH, kW, padT, padH, padW, dT, dH, dW,
      dilationT, dilationH, dilationW,
      THCTensor_(data)(state, columns)
    );

    // M,N,K are dims of matrix A and B
    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
    int64_t m = nOutputPlane;
    int64_t n = columns->size(1);
    int64_t k = nInputPlane*kT*kH*kW;

    // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
    #ifdef THC_REAL_IS_FLOAT
    THCudaBlas_Sgemm(
    #elif defined(THC_REAL_IS_HALF)
    THCudaBlas_Hgemm(
    #elif defined(THC_REAL_IS_DOUBLE)
    THCudaBlas_Dgemm(
    #endif
        state,
        'n', 'n',
        n, m, k,
        ScalarConvert<int, scalar_t>::to(1),
        THCTensor_(data)(state, columns), n,
        THCTensor_(data)(state, weight), k,
        ScalarConvert<int, scalar_t>::to(1),
        THCTensor_(data)(state, output_n), n
    );
  }

  // Free
  THCTensor_(free)(state, input_n);
  THCTensor_(free)(state, output_n);

  // Resize output
  if (is_batch == 0) {
    THCTensor_(resize4d)(state, output, nOutputPlane, outputDepth, outputHeight, outputWidth);
    THCTensor_(resize4d)(state, input, nInputPlane, inputDepth, inputHeight, inputWidth);
  }

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, weight);
  if (bias) THCTensor_(free)(state, bias);
}

void THNN_(VolumetricDilatedConvolution_updateGradInput)(
           THCState *state,
           THCTensor  *input,
           THCTensor  *gradOutput,
           THCTensor  *gradInput,
           THCTensor  *weight,
           THCTensor  *gradColumns,
           int kT, int kW, int kH,
           int dT, int dW, int dH,
           int padT, int padW, int padH,
           int dilationT, int dilationW, int dilationH) {

  THCUNN_assertSameGPU(state, 5, input, gradOutput, weight,
                       gradColumns, gradInput);
  THNN_(VolumetricDilatedConvolution_shapeCheck)(
        state, input, gradOutput, weight, NULL,
        kT, kH, kW, dT, dH, dW, padT, padH, padW,
        dilationT, dilationH, dilationW, 0);

  weight = THCTensor_(newContiguous)(state, weight);

  // Params
  int nInputPlane = weight->size(1);
  int nOutputPlane = weight->size(0);

  input = THCTensor_(newContiguous)(state, input);
  gradOutput = THCTensor_(newContiguous)(state, gradOutput);
  int is_batch = 1;
  if (input->dim() == 4) {
    // Force batch
    is_batch = 0;
    THCTensor_(resize5d)(state, input, 1, input->size(0), input->size(1), input->size(2), input->size(3));
    THCTensor_(resize5d)(state, gradOutput, 1, gradOutput->size(0), gradOutput->size(1), gradOutput->size(2), gradOutput->size(3));
  }

  int64_t inputDepth  = input->size(2);
  int64_t inputWidth   = input->size(4);
  int64_t inputHeight  = input->size(3);
  int64_t outputDepth  = (inputDepth + 2*padT - (dilationT * (kT - 1) + 1)) / dT + 1;
  int64_t outputWidth  = (inputWidth + 2*padW - (dilationW * (kW - 1) + 1)) / dW + 1;
  int64_t outputHeight = (inputHeight + 2*padH - (dilationH * (kH - 1) + 1)) / dH + 1;

  // Batch size + input planes
  int64_t batchSize = input->size(0);

  // Resize output
  THCTensor_(resize5d)(state, gradInput, batchSize, nInputPlane, inputDepth, inputHeight, inputWidth);

  // Resize temporary columns
  THCTensor_(resize2d)(state, gradColumns, nInputPlane*kT*kW*kH, outputDepth*outputHeight*outputWidth);

  // Helpers
  THCTensor  *gradInput_n = THCTensor_(new)(state);
  THCTensor  *gradOutput_n = THCTensor_(new)(state);

  // For each elt in batch, do:
  for (int elt = 0; elt < batchSize; elt ++) {
    // Matrix mulitply per sample:
    THCTensor_(select)(state, gradInput_n, gradInput, 0, elt);
    THCTensor_(select)(state, gradOutput_n, gradOutput, 0, elt);

    // M,N,K are dims of matrix A and B
    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
    int64_t m = nInputPlane*kT*kW*kH;
    int64_t n = gradColumns->size(1);
    int64_t k = nOutputPlane;

    // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
    #ifdef THC_REAL_IS_FLOAT
    THCudaBlas_Sgemm(
    #elif defined(THC_REAL_IS_HALF)
    THCudaBlas_Hgemm(
    #elif defined(THC_REAL_IS_DOUBLE)
    THCudaBlas_Dgemm(
    #endif
        state,
        'n', 't',
        n, m, k,
        ScalarConvert<int, scalar_t>::to(1),
        THCTensor_(data)(state, gradOutput_n), n,
        THCTensor_(data)(state, weight), m,
        ScalarConvert<int, scalar_t>::to(0),
        THCTensor_(data)(state, gradColumns), n
    );

    // Unpack columns back into input:
    col2vol<scalar_t, accreal>(
      THCState_getCurrentStream(state),
      THCTensor_(data)(state, gradColumns),
      nInputPlane, inputDepth, inputHeight, inputWidth,
      outputDepth, outputHeight, outputWidth,
      kT, kH, kW, padT, padH, padW, dT, dH, dW,
      dilationT, dilationH, dilationW,
      THCTensor_(data)(state, gradInput_n)
    );
  }

  // Free
  THCTensor_(free)(state, gradInput_n);
  THCTensor_(free)(state, gradOutput_n);

  // Resize output
  if (is_batch == 0) {
    THCTensor_(resize4d)(state, gradOutput, nOutputPlane, outputDepth, outputHeight, outputWidth);
    THCTensor_(resize4d)(state, input, nInputPlane, inputDepth, inputHeight, inputWidth);
    THCTensor_(resize4d)(state, gradInput, nInputPlane, inputDepth, inputHeight, inputWidth);
  }

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, gradOutput);
  THCTensor_(free)(state, weight);
}

void THNN_(VolumetricDilatedConvolution_accGradParameters)(
           THCState *state,
           THCTensor  *input,
           THCTensor  *gradOutput,
           THCTensor  *gradWeight,
           THCTensor  *gradBias,
           THCTensor  *columns,
           THCTensor  *ones,
           int kT, int kW, int kH,
           int dT, int dW, int dH,
           int padT, int padW, int padH,
           int dilationT, int dilationW, int dilationH,
           accreal scale_) {

  scalar_t scale = ScalarConvert<accreal, scalar_t>::to(scale_);
  THCUNN_assertSameGPU(state, 5, input, gradOutput, gradWeight, gradBias, columns, ones);
  THNN_(VolumetricDilatedConvolution_shapeCheck)(
        state, input, gradOutput, gradWeight, gradBias,
        kT, kH, kW, dT, dH, dW, padT, padH, padW,
        dilationT, dilationH, dilationW, 1);

  // Params
  input = THCTensor_(newContiguous)(state, input);
  gradOutput = THCTensor_(newContiguous)(state, gradOutput);
  int is_batch = 1;
  if (input->dim() == 4) {
    // Force batch
    is_batch = 0;
    THCTensor_(resize5d)(state, input, 1, input->size(0), input->size(1), input->size(2), input->size(3));
    THCTensor_(resize5d)(state, gradOutput, 1, gradOutput->size(0), gradOutput->size(1), gradOutput->size(2), gradOutput->size(3));
  }

  int64_t nInputPlane = input->size(1);
  int64_t nOutputPlane = gradOutput->size(1);
  int64_t inputDepth  = input->size(2);
  int64_t inputWidth   = input->size(4);
  int64_t inputHeight  = input->size(3);
  int64_t outputDepth  = (inputDepth + 2*padT - (dilationT * (kT - 1) + 1)) / dT + 1;
  int64_t outputWidth  = (inputWidth + 2*padW - (dilationW * (kW - 1) + 1)) / dW + 1;
  int64_t outputHeight = (inputHeight + 2*padH - (dilationH * (kH - 1) + 1)) / dH + 1;

  // Batch size + input planes
  int64_t batchSize = input->size(0);

  // Define a buffer of ones, for bias accumulation
  if (ones->dim() != 3 || ones->size(0)*ones->size(1)*ones->size(2) < outputDepth*outputHeight*outputWidth) {
    // Resize plane and fill with ones...
    THCTensor_(resize3d)(state, ones, outputDepth, outputHeight, outputWidth);
    THCTensor_(fill)(state, ones, ScalarConvert<int, scalar_t>::to(1));
  }

  // Resize temporary columns
  THCTensor_(resize2d)(state, columns, nInputPlane*kT*kW*kH, outputDepth*outputHeight*outputWidth);

  // Helpers
  THCTensor  *input_n = THCTensor_(new)(state);
  THCTensor  *gradOutput_n = THCTensor_(new)(state);

  // For each elt in batch, do:
  for (int elt = 0; elt < batchSize; elt ++) {
    // Matrix mulitply per output:
    THCTensor_(select)(state, gradOutput_n, gradOutput, 0, elt);

    if (gradWeight) {
      // Matrix mulitply per output:
      THCTensor_(select)(state, input_n, input, 0, elt);

      // Extract columns:
      vol2col(
        THCState_getCurrentStream(state),
        THCTensor_(data)(state, input_n),
        nInputPlane, inputDepth, inputHeight, inputWidth,
        outputDepth, outputHeight, outputWidth,
        kT, kH, kW, padT, padH, padW, dT, dH, dW,
        dilationT, dilationH, dilationW,
        THCTensor_(data)(state, columns)
      );

      // M,N,K are dims of matrix A and B
      // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
      int64_t m = nOutputPlane;
      int64_t n = nInputPlane*kT*kW*kH;
      int64_t k = columns->size(1);

      // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
      #ifdef THC_REAL_IS_FLOAT
      THCudaBlas_Sgemm(
      #elif defined(THC_REAL_IS_HALF)
      THCudaBlas_Hgemm(
      #elif defined(THC_REAL_IS_DOUBLE)
      THCudaBlas_Dgemm(
      #endif
          state,
          't', 'n',
          n, m, k,
          scale,
          THCTensor_(data)(state, columns), k,
          THCTensor_(data)(state, gradOutput_n), k,
          ScalarConvert<int, scalar_t>::to(1),
          THCTensor_(data)(state, gradWeight), n
      );
    }

    // Do Bias:
    if (gradBias) {
      // M,N,K are dims of matrix A and B
      // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
      int64_t m_ = nOutputPlane;
      int64_t k_ = outputDepth * outputHeight * outputWidth;

      // Do GEMV (note: this is a bit confusing because gemv assumes column-major matrices)
      #if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE)
      #ifdef THC_REAL_IS_FLOAT
      THCudaBlas_Sgemv(
      #elif defined(THC_REAL_IS_DOUBLE)
      THCudaBlas_Dgemv(
      #endif
          state,
          't',
          k_, m_,
          scale,
          THCTensor_(data)(state, gradOutput_n), k_,
          THCTensor_(data)(state, ones), 1,
          ScalarConvert<int, scalar_t>::to(1),
          THCTensor_(data)(state, gradBias), 1
      );
      #endif
      #ifdef THC_REAL_IS_HALF
      THCudaBlas_Hgemm(
          state,
          't', 'n',
          m_, 1, k_,
          scale,
          THCTensor_(data)(state, gradOutput_n), k_,
          THCTensor_(data)(state, ones), k_,
          ScalarConvert<int, scalar_t>::to(1),
          THCTensor_(data)(state, gradBias), m_
      );
      #endif
    }
  }

  // Free
  THCTensor_(free)(state, input_n);
  THCTensor_(free)(state, gradOutput_n);

    // Resize output
  if (is_batch == 0) {
    THCTensor_(resize4d)(state, gradOutput, nOutputPlane, outputDepth, outputHeight, outputWidth);
    THCTensor_(resize4d)(state, input, nInputPlane, inputDepth, inputHeight, inputWidth);
  }

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, gradOutput);
}

#endif

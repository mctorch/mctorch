
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THCUNN/generic/MarginCriterion.cu"
#else

void THNN_(MarginCriterion_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *target,
           THCTensor *output,
           bool sizeAverage,
           accreal margin_)
{
  scalar_t margin = ScalarConvert<accreal, scalar_t>::to(margin_);
  THCUNN_check_nElement(state, input, target);
  THCUNN_check_dim_size(state, output, 1, 0, 1);
  THCUNN_assertSameGPU(state, 2, input, target);

  ptrdiff_t size = THCTensor_(nElement)(state, input);

  input = THCTensor_(newContiguous)(state, input);
  target = THCTensor_(newContiguous)(state, target);

  thrust::device_ptr<scalar_t> input_data(THCTensor_(data)(state, input));
  thrust::device_ptr<scalar_t> target_data(THCTensor_(data)(state, target));
  accreal sum = thrust::inner_product(input_data, input_data+size, target_data, (accreal) 0, thrust::plus<accreal>(),
      margin_functor<scalar_t, accreal>(ScalarConvert<scalar_t, accreal>::to(margin)));

  if (sizeAverage)
    sum /= size;

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, target);

  THCTensor_(set1d)(state, output, 0, ScalarConvert<accreal, scalar_t>::to(sum));
}


void THNN_(MarginCriterion_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *target,
           THCTensor *gradInput,
           bool sizeAverage,
           accreal margin_)
{
  scalar_t margin = ScalarConvert<accreal, scalar_t>::to(margin_);

  THCUNN_check_nElement(state, input, target);
  THCUNN_assertSameGPU(state, 3, input, target, gradInput);

  ptrdiff_t size = THCTensor_(nElement)(state, input);
  accreal norm = sizeAverage ? 1.f/size : 1;

  input = THCTensor_(newContiguous)(state, input);
  target = THCTensor_(newContiguous)(state, target);

  THCTensor_(resizeAs)(state, gradInput, input);

  thrust::device_ptr<scalar_t> input_data(THCTensor_(data)(state, input));
  thrust::device_ptr<scalar_t> target_data(THCTensor_(data)(state, target));
  thrust::device_ptr<scalar_t> gradInput_data(THCTensor_(data)(state, gradInput));

  thrust::transform(input_data, input_data+size, target_data, gradInput_data,
      margin_updateGradInput_functor<scalar_t, accreal>(ScalarConvert<scalar_t, accreal>::to(margin), norm));

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, target);
}

#endif

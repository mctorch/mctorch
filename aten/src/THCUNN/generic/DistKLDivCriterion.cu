
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THCUNN/generic/DistKLDivCriterion.cu"
#else

void THNN_(DistKLDivCriterion_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *target,
           THCTensor *output,
           int64_t reduction)
{
  THCUNN_check_shape(state, input, target);
  THCUNN_assertSameGPU(state, 2, input, target);

  THArgCheck(THCTensor_(nElement)(state, input) == THCTensor_(nElement)(state, target), 2,
             "input and target need to have the same number of elements");

  if (reduction == Reduction::None) {
    THCTensor_(resizeAs)(state, output, input);
    THC_pointwiseApply3<scalar_t, scalar_t, scalar_t>(state, input, target, output,
                        kl_updateOutput_no_reduce_functor<scalar_t>());
    return;
  }

  THCTensor_(resize1d)(state, output, 1);

  accreal sum;

  ptrdiff_t size = THCTensor_(nElement)(state, input);

  input = THCTensor_(newContiguous)(state, input);
  target = THCTensor_(newContiguous)(state, target);

  thrust::device_ptr<scalar_t> input_data(THCTensor_(data)(state, input));
  thrust::device_ptr<scalar_t> target_data(THCTensor_(data)(state, target));
  sum = thrust::inner_product(input_data, input_data+size, target_data, (accreal) 0, thrust::plus<accreal>(), kl_functor<scalar_t, accreal>());

  if (reduction == Reduction::Mean)
    sum /= size;

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, target);

  THCTensor_(set1d)(state, output, 0, ScalarConvert<accreal, scalar_t>::to(sum));
}

void THNN_(DistKLDivCriterion_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *target,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           int64_t reduction)
{
  THCUNN_check_shape(state, input, target);
  THCUNN_assertSameGPU(state, 4, input, target, gradInput, gradOutput);

  THArgCheck(THCTensor_(nElement)(state, input) == THCTensor_(nElement)(state, target), 2,
             "input and target need to have the same number of elements");

  THCTensor_(resizeAs)(state, gradInput, input);

  if (reduction == Reduction::None) {
    THCUNN_check_shape(state, gradOutput, input);
    THC_pointwiseApply3<scalar_t, scalar_t, scalar_t>(state, target, gradOutput, gradInput,
                        kl_updateGradInput_no_reduce_functor<scalar_t>());
    return;
  }

  THCUNN_check_dim_size(state, gradOutput, 1, 0, 1);

  ptrdiff_t size = THCTensor_(nElement)(state, input);
  scalar_t norm = (reduction == Reduction::Mean ? ScalarConvert<accreal, scalar_t>::to(accreal(1)/size) : ScalarConvert<int, scalar_t>::to(1));

  input = THCTensor_(newContiguous)(state, input);
  target = THCTensor_(newContiguous)(state, target);

  thrust::device_ptr<scalar_t> input_data(THCTensor_(data)(state, input));
  thrust::device_ptr<scalar_t> target_data(THCTensor_(data)(state, target));
  thrust::device_ptr<scalar_t> gradInput_data(THCTensor_(data)(state, gradInput));

  thrust::transform(input_data, input_data+size, target_data, gradInput_data,
                    kl_updateGradInput_functor<scalar_t>(norm, THCTensor_(get1d)(state, gradOutput, 0)));

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, target);
}

#endif

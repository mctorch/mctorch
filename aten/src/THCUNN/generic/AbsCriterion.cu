
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THCUNN/generic/AbsCriterion.cu"
#else

void THNN_(AbsCriterion_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *target,
           THCTensor *output,
           int64_t reduction)
{
  THCUNN_check_shape(state, input, target);
  THCUNN_assertSameGPU(state, 3, input, target, output);

  if (reduction == Reduction::None) {
    THCTensor_(resizeAs)(state, output, input);
    THC_pointwiseApply3<scalar_t, scalar_t, scalar_t>(state, input, target, output,
                        abs_updateOutput_no_reduce_functor<scalar_t>());
    return;
  }

  THCTensor_(resize0d)(state, output);

  ptrdiff_t size = THCTensor_(nElement)(state, input);

  input = THCTensor_(newContiguous)(state, input);
  target = THCTensor_(newContiguous)(state, target);

  thrust::device_ptr<scalar_t> input_data(THCTensor_(data)(state, input));
  thrust::device_ptr<scalar_t> target_data(THCTensor_(data)(state, target));
  accreal sum = thrust::inner_product(input_data, input_data+size, target_data, (accreal)0, thrust::plus<accreal>(), abs_functor<scalar_t, accreal>());

  if (reduction == Reduction::Mean)
    sum /= size;

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, target);

  THCTensor_(set0d)(state, output, ScalarConvert<accreal, scalar_t>::to(sum));
}

void THNN_(AbsCriterion_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *target,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           int64_t reduction)
{
  THCUNN_check_shape(state, input, target);
  THCUNN_assertSameGPU(state, 4, input, target, gradOutput, gradInput);

  THCTensor_(resizeAs)(state, gradInput, input);

  if (reduction == Reduction::None) {
    THCUNN_check_shape(state, gradOutput, input);
    THC_pointwiseApply3<scalar_t, scalar_t, scalar_t>(state, input, target, gradInput,
                        abs_updateGradInput_no_reduce_functor<scalar_t>());
    THCTensor_(cmul)(state, gradInput, gradInput, gradOutput);
    return;
  }

  THCUNN_check_dim_size(state, gradOutput, 1, 0, 1);

  ptrdiff_t size = THCTensor_(nElement)(state, input);
  scalar_t norm = ScalarConvert<double, scalar_t>::to(reduction == Reduction::Mean ? 1./size : 1.);

  input = THCTensor_(newContiguous)(state, input);
  target = THCTensor_(newContiguous)(state, target);

  thrust::device_ptr<scalar_t> input_data(THCTensor_(data)(state, input));
  thrust::device_ptr<scalar_t> target_data(THCTensor_(data)(state, target));
  thrust::device_ptr<scalar_t> gradInput_data(THCTensor_(data)(state, gradInput));

  thrust::transform(input_data, input_data+size, target_data, gradInput_data,
                    abs_updateGradInput_functor<scalar_t>(norm, THCTensor_(get0d)(state, gradOutput)));

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, target);
}

#endif

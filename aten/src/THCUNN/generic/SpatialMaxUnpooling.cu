
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THCUNN/generic/SpatialMaxUnpooling.cu"
#else

void THNN_(SpatialMaxUnpooling_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           THCIndexTensor *indices,
           int owidth, int oheight)
{
  THCUNN_assertSameGPU(state, 3, input, output, indices);
  THCUNN_argCheck(state, !input->is_empty() && (input->dim() == 3 || input->dim() == 4), 2, input,
                  "non-empty 3D or 4D (batch mode) tensor expected for input, but got: %s");
  THCUNN_check_shape_indices(state, indices, input);

  int64_t nInputCols, nInputRows, nInputPlane, batchSize;

  if (input->dim() == 3) {
    nInputCols = input->size(2);
    nInputRows = input->size(1);
    nInputPlane = input->size(0);
    batchSize = 1;
  }
  else
  {
    nInputCols = input->size(3);
    nInputRows = input->size(2);
    nInputPlane = input->size(1);
    batchSize = input->size(0);
  }

  input = THCTensor_(newContiguous)(state, input);
  indices = THCIndexTensor_(newContiguous)(state, indices);
  THCTensor_(resize4d)(state, output, batchSize, nInputPlane, oheight, owidth);
  THCTensor_(zero)(state, output);

  int count = THCTensor_(nElement)(state, input);

  MaxUnpoolForward <<< GET_BLOCKS(count), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state) >>>
      (count, THCTensor_(data)(state, input), THCIndexTensor_(data)(state, indices),
      batchSize, nInputPlane, nInputRows, nInputCols, oheight, owidth, THCTensor_(data)(state, output));
  THCudaCheck(cudaGetLastError());

  if(input->dim() == 3)
    THCTensor_(resize3d)(state, output, nInputPlane, oheight, owidth);

  THCTensor_(free)(state, input);
  THCIndexTensor_(free)(state, indices);
}

void THNN_(SpatialMaxUnpooling_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           THCIndexTensor *indices,
           int owidth, int oheight)
{
  THCUNN_assertSameGPU(state, 4, input, gradOutput, indices, gradInput);
  THCUNN_check_shape_indices(state, indices, input);

  int64_t nInputCols, nInputRows, nInputPlane, batchSize;
  int dimw = 2;
  int dimh = 1;

  if (input->dim() == 3) {
    nInputPlane = input->size(0);
    batchSize = 1;
  }
  else
  {
    ++dimw;
    ++dimh;
    nInputPlane = input->size(1);
    batchSize = input->size(0);
  }
  nInputCols = input->size(dimw);
  nInputRows = input->size(dimh);

  if(owidth!=gradOutput->size(dimw) || oheight!=gradOutput->size(dimh)){
     THError("Inconsistent gradOutput size. oheight= %d, owidth= %d, gradOutput: %dx%d",
             oheight, owidth,gradOutput->size(dimh),gradOutput->size(dimw));
  }

  input = THCTensor_(newContiguous)(state, input);
  indices = THCIndexTensor_(newContiguous)(state, indices);
  gradOutput = THCTensor_(newContiguous)(state, gradOutput);
  THCTensor_(resizeAs)(state, gradInput, input);

  int count = THCTensor_(nElement)(state, input);

  MaxUnpoolBackward <<< GET_BLOCKS(count), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state) >>>
      (count, THCTensor_(data)(state, gradOutput), THCIndexTensor_(data)(state, indices),
      batchSize, nInputPlane, nInputRows, nInputCols, oheight, owidth, THCTensor_(data)(state, gradInput));
  THCudaCheck(cudaGetLastError());

  // clean
  THCTensor_(free)(state, input);
  THCIndexTensor_(free)(state, indices);
  THCTensor_(free)(state, gradOutput);
}

#endif

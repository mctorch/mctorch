#include "hip/hip_runtime.h"
#include <THCUNN/THCUNN.h>
#include <TH/THHalf.h>
#include <THCUNN/THCHalfAutoNumerics.cuh>
#include <THC/THCApply.cuh>

template <typename T>
struct sqrtupdateOutput_functor
{
  const T bias;

  sqrtupdateOutput_functor(T bias_)
    : bias(bias_)
  {}

  __device__ void operator()(T *output, const T *input) const
  {
    *output = sqrt(*input + bias);
  }
};

template <typename T>
struct sqrtupdateGradInput_functor
{
  sqrtupdateGradInput_functor() {}

  __device__ void operator()(T *gradInput, const T *output, const T *gradOutput) const
  {
    *gradInput = (THCNumerics<T>::eq(*output,ScalarConvert<float, T>::to(0.0f))) ? ScalarConvert<float, T>::to(0.0f) : ((ScalarConvert<float, T>::to(0.5f) * *gradOutput) / *output);
  }
};

#include <THCUNN/generic/Sqrt.cu>
#include <THC/THCGenerateFloatTypes.h>

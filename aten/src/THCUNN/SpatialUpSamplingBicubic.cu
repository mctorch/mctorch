#include "hip/hip_runtime.h"
#include <THCUNN/THCUNN.h>
#include <THC/THCTensor.hpp>
#include <THCUNN/common.h>
#include <THCUNN/upsampling.h>
#include <THC/THCDeviceTensor.cuh>
#include <THC/THCDeviceTensorUtils.cuh>
#include <THC/THCDeviceUtils.cuh>
#include <TH/THHalf.h>
#include <THCUNN/THCHalfAutoNumerics.cuh>
#include <THC/THCAtomics.cuh>

template<typename Dtype, typename Acctype>
#if defined(__HIP_PLATFORM_HCC__)
__launch_bounds__(1024)
#endif
__global__ void bicubic_interp2d_kernel(
  const int num_elements,
  const Acctype height_scale,
  const Acctype width_scale,
  const bool align_corners,
  const THCDeviceTensor<Dtype, 4> in_data,
  THCDeviceTensor<Dtype, 4> out_data
) {

  int index = threadIdx.x + blockIdx.x * blockDim.x;
  const int batchsize = in_data.getSize(0);
  const int channels = in_data.getSize(1);
  const int input_height = in_data.getSize(2);
  const int input_width = in_data.getSize(3);
  const int output_height = out_data.getSize(2);
  const int output_width = out_data.getSize(3);

  if (index >= num_elements) {
    return;
  }

  // Special case: input and output are the same size, just copy
  const int output_x = index % output_width;
  const int output_y = index / output_width;
  if (input_height == output_height && input_width == output_width) {
    for (int n = 0; n < batchsize; n++){
      for (int c = 0; c < channels; c++) {
        const Dtype val = in_data[n][c][output_y][output_x];
        out_data[n][c][output_x][output_y] = val;
      }
    }
    return;
  }

  // Interpolation kernel
  Acctype real_x = area_pixel_compute_source_index(width_scale, output_x, align_corners, /*cubic=*/true);
  int in_x = floorf(real_x);
  Acctype t_x = real_x - in_x;

  Acctype real_y = area_pixel_compute_source_index(height_scale, output_y, align_corners, /*cubic=*/true);
  int in_y = floorf(real_y);
  Acctype t_y = real_y - in_y;

  for (int n = 0; n < batchsize ; n++) {
    for (int c = 0; c < channels; c++) {
      Acctype coefficients[4];

      for (int k = 0; k < 4; k++) {
        coefficients[k] = cubic_interp1d(
          upsampling_get_value_bounded<Dtype>(
            in_data, c, n, input_width, input_height, in_x - 1, in_y - 1 + k),
          upsampling_get_value_bounded<Dtype>(
            in_data, c, n, input_width, input_height, in_x + 0, in_y - 1 + k),
          upsampling_get_value_bounded<Dtype>(
            in_data, c, n, input_width, input_height, in_x + 1, in_y - 1 + k),
          upsampling_get_value_bounded<Dtype>(
            in_data, c, n, input_width, input_height, in_x + 2, in_y - 1 + k),
          t_x
        );
      }

      out_data[n][c][output_y][output_x] = ScalarConvert<Acctype, Dtype>::to(cubic_interp1d(
        coefficients[0],
        coefficients[1],
        coefficients[2],
        coefficients[3],
        t_y
      ));
    }
  }
}

// Backward (adjoint) operation 1 <- 2 (accumulates)
template <typename Dtype, typename Acctype>
#if defined(__HIP_PLATFORM_HCC__)
__launch_bounds__(1024)
#endif
__global__ void bicubic_interp2d_backward_kernel(
  const int num_elements,
  const Acctype height_scale,
  const Acctype width_scale,
  const bool align_corners,
  THCDeviceTensor<Dtype, 4> in_data,
  const THCDeviceTensor<Dtype, 4> out_data
){

  int index = threadIdx.x + blockIdx.x * blockDim.x;
  const int batchsize = in_data.getSize(0);
  const int channels = in_data.getSize(1);
  const int input_height = in_data.getSize(2);
  const int input_width = in_data.getSize(3);
  const int output_height = out_data.getSize(2);
  const int output_width = out_data.getSize(3);

  if (index >= num_elements) {
    return;
  }

  const int output_x = index % output_width;
  const int output_y = index / output_width;
  // special case: output_xust copy
  if (input_height == output_height && input_width == output_width) {
    for (int n = 0; n < batchsize ; n++){
      for (int c = 0; c < channels; ++c) {
        const Dtype val = out_data[n][c][output_y][output_x];
        in_data[n][c][output_y][output_x] += val;
      }
    }
    return;
  }

  Acctype real_x = area_pixel_compute_source_index(width_scale, output_x, align_corners, /*cubic=*/true);
  int input_x = floorf(real_x);
  Acctype t_x = real_x - input_x;

  Acctype real_y = area_pixel_compute_source_index(height_scale, output_y, align_corners, /*cubic=*/true);
  int input_y = floorf(real_y);
  Acctype t_y = real_y - input_y;

  Acctype x_coeffs[4];
  Acctype y_coeffs[4];

  get_cubic_upsampling_coefficients(x_coeffs, t_x);
  get_cubic_upsampling_coefficients(y_coeffs, t_y);

  for (int n = 0; n < batchsize ; n++){
    for (int c = 0; c < channels; ++c) {
      Dtype out_value = out_data[n][c][output_y][output_x];
      for (int i = 0; i < 4; i++) {
        for (int j = 0; j < 4; j++) {
          upsampling_increment_value_bounded<Dtype, Acctype>(
            in_data,
            c,
            n,
            input_width,
            input_height,
            input_x - 1 + j,
            input_y - 1 + i,
            out_value * y_coeffs[i] * x_coeffs[j]
          );
        }
      }
    }
  }
}


#include <THCUNN/generic/SpatialUpSamplingBicubic.cu>
#include <THC/THCGenerateFloatTypes.h>

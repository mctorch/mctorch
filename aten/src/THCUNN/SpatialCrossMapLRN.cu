#include "hip/hip_runtime.h"
#include <THCUNN/THCUNN.h>
#include <TH/THHalf.h>
#include <THCUNN/THCHalfAutoNumerics.cuh>
#include <THC/THCTensor.hpp>
#include <THC/THCStorage.hpp>
#include <THCUNN/common.h>
#include <c10/macros/Macros.h>

template <typename Dtype, typename Acctype>
__global__ void
#if __CUDA_ARCH__ >= 320 || defined __HIP_PLATFORM_HCC__
C10_LAUNCH_BOUNDS_1(CUDA_NUM_THREADS)
#endif
LRNFillScale(const int nthreads, const Dtype* const in,
    const int num, const int channels, const int height,
    const int width, const int size, const Dtype alpha_over_size,
    const Dtype k, Dtype* const scale) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local offset
    const int w = index % width;
    const int h = (index / width) % height;
    const int n = index / width / height;
    const int offset = (n * channels * height + h) * width + w;
    const int step = height * width;
    const Dtype* const in_off = in + offset;
    Dtype* const scale_off = scale + offset;
    int head = 0;
    const int pre_pad = (size - 1) / 2;
    const int post_pad = size - pre_pad - 1;
    Acctype accum_scale = Acctype(0);
    // fill the scale at [n, :, h, w]
    // accumulate values
    while (head < post_pad && head < channels) {
      accum_scale += in_off[head * step] * in_off[head * step];
      ++head;
    }
    // both add and subtract
    while (head < channels) {
      accum_scale += in_off[head * step] * in_off[head * step];
      if (head - size >= 0) {
        accum_scale -= in_off[(head - size) * step]
                       * in_off[(head - size) * step];
      }
      scale_off[(head - post_pad) * step] = ScalarConvert<Acctype, Dtype>::to(k + accum_scale * alpha_over_size);
      ++head;
    }
    // subtract only
    while (head < channels + post_pad) {
      if (head - size >= 0) {
        accum_scale -= in_off[(head - size) * step]
                       * in_off[(head - size) * step];
      }
      scale_off[(head - post_pad) * step] = ScalarConvert<Acctype, Dtype>::to(k + accum_scale * alpha_over_size);
      ++head;
    }
  }
}

template <typename Dtype>
__global__ void LRNComputeOutput(const int nthreads, const Dtype* in,
    const Dtype* scale, const Dtype negative_beta, Dtype* out) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    out[index] = in[index] * pow(scale[index], negative_beta);
  }
}

template <typename Dtype, typename Acctype>
__global__ void LRNComputeDiff(const int nthreads,
    const Dtype* const bottom_data, const Dtype* const top_data,
    const Dtype* const scale, const Dtype* const top_diff,
    const int num, const int channels, const int height,
    const int width, const int size, const Dtype negative_beta,
    const Dtype cache_ratio, Dtype* const bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local offset
    const int w = index % width;
    const int h = (index / width) % height;
    const int n = index / width / height;
    const int offset = (n * channels * height + h) * width + w;
    const int step = height * width;
    const Dtype* const bottom_off = bottom_data + offset;
    const Dtype* const top_off = top_data + offset;
    const Dtype* const scale_off = scale + offset;
    const Dtype* const top_diff_off = top_diff + offset;
    Dtype* const bottom_diff_off = bottom_diff + offset;
    int head = 0;
    const int pre_pad = size - (size + 1) / 2;
    const int post_pad = size - pre_pad - 1;
    Acctype accum_ratio = Acctype(0);
    // accumulate values
    while (head < post_pad && head < channels) {
      accum_ratio += top_diff_off[head * step] * top_off[head * step] /
          scale_off[head * step];
      ++head;
    }
    // both add and subtract
    while (head < channels) {
      accum_ratio += top_diff_off[head * step] * top_off[head * step] /
          scale_off[head * step];
      if (head - size >= 0) {
        accum_ratio -= top_diff_off[(head - size) * step] *
            top_off[(head - size) * step] / scale_off[(head - size) * step];
      }
      bottom_diff_off[(head - post_pad) * step] =
          ScalarConvert<Acctype, Dtype>::to(top_diff_off[(head - post_pad) * step]
            * pow(scale_off[(head - post_pad) * step], negative_beta)
          - cache_ratio * bottom_off[(head - post_pad) * step] * accum_ratio);
      ++head;
    }
    // subtract only
    while (head < channels + post_pad) {
      if (head - size >= 0) {
        accum_ratio -= top_diff_off[(head - size) * step] *
            top_off[(head - size) * step] / scale_off[(head - size) * step];
      }
      bottom_diff_off[(head - post_pad) * step] =
          ScalarConvert<Acctype, Dtype>::to(top_diff_off[(head - post_pad) * step]
            * pow(scale_off[(head - post_pad) * step], negative_beta)
          - cache_ratio * bottom_off[(head - post_pad) * step] * accum_ratio);
      ++head;
    }
  }
}


#include <THCUNN/generic/SpatialCrossMapLRN.cu>
#include <THC/THCGenerateFloatTypes.h>

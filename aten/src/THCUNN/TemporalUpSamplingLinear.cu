#include "hip/hip_runtime.h"
// Adapted from interp.cpp from Caffe util by Pauline Luc
// Originally developed by George Papandreou
#include <THCUNN/THCUNN.h>
#include <THC/THCTensor.hpp>
#include <THCUNN/common.h>
#include <THCUNN/upsampling.h>
#include <THC/THCDeviceTensor.cuh>
#include <THC/THCDeviceTensorUtils.cuh>
#include <THC/THCDeviceUtils.cuh>
#include <TH/THHalf.h>
#include <THCUNN/THCHalfAutoNumerics.cuh>
#include <THC/THCAtomics.cuh>

template<typename Dtype, typename Acctype>
#ifdef __HIP_PLATFORM_HCC__
C10_LAUNCH_BOUNDS_1(1024)
#endif
__global__ void caffe_gpu_interp2_kernel(const int n,
    const Acctype rwidth, const bool align_corners,
    const THCDeviceTensor<Dtype, 3> data1, THCDeviceTensor<Dtype, 3> data2) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  const int batchsize = data1.getSize(0);
  const int channels = data1.getSize(1);
  const int width1 = data1.getSize(2);
  const int width2 = data2.getSize(2);

  if (index < n) {
    const int w2 = index % width2;
    // special case: just copy
    if (width1 == width2) {
      const int w1 = w2;
      for (int n = 0; n < batchsize ; n++){
        for (int c = 0; c < channels; ++c) {
          const Dtype val = data1[n][c][w1];
          data2[n][c][w2] = val;
        }
      }
      return;
    }
    //
    const Acctype w1r = area_pixel_compute_source_index<Acctype>(rwidth, w2, align_corners, /*cubic=*/false);
    const int w1 = w1r;
    const int w1p = (w1 < width1 - 1) ? 1 : 0;
    const Acctype w1lambda = w1r - w1;
    const Acctype w0lambda = Acctype(1) - w1lambda;
    //
    for (int n = 0; n < batchsize ; n++){
        for (int c = 0; c < channels; ++c) {
        const Acctype val = w0lambda * data1[n][c][w1]
                            + w1lambda * data1[n][c][w1+w1p];
        data2[n][c][w2] = ScalarConvert<Acctype, Dtype>::to(val);
      }
    }
  }
}

// Backward (adjoint) operation 1 <- 2 (accumulates)
template <typename Dtype, typename Acctype>
#ifdef __HIP_PLATFORM_HCC__
C10_LAUNCH_BOUNDS_1(1024)
#endif
__global__ void caffe_gpu_interp2_kernel_backward(const int n,
    const Acctype rwidth, const bool align_corners,
    THCDeviceTensor<Dtype, 3> data1, const THCDeviceTensor<Dtype, 3> data2){
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  const int batchsize = data1.getSize(0);
  const int channels = data1.getSize(1);
  const int width1 = data1.getSize(2);
  const int width2 = data2.getSize(2);
  if (index < n) {
    const int w2 = index % width2;
    // special case: just copy
    if (width1 == width2) {
      const int w1 = w2;
      for (int n = 0; n < batchsize ; n++){
        for (int c = 0; c < channels; ++c) {
          const Dtype val = data2[n][c][w1];
          data1[n][c][w2] += val;
        }
      }
      return;
    }
    //
    const Acctype w1r = area_pixel_compute_source_index<Acctype>(rwidth, w2, align_corners, /*cubic=*/false);
    const int w1 = w1r;
    const int w1p = (w1 < width1 - 1) ? 1 : 0;
    const Acctype w1lambda = w1r - w1;
    const Acctype w0lambda = Acctype(1) - w1lambda;
    //
    for (int n = 0; n < batchsize ; n++){
      for (int c = 0; c < channels; ++c) {
        const Dtype d2val = data2[n][c][w2];
        atomicAdd(data1[n][c][w1].data(),
                  ScalarConvert<Acctype, Dtype>::to(w0lambda * d2val));
        atomicAdd(data1[n][c][w1+w1p].data(),
                  ScalarConvert<Acctype, Dtype>::to(w1lambda * d2val));
      }
    }
  }
}


#include <THCUNN/generic/TemporalUpSamplingLinear.cu>
#include <THC/THCGenerateFloatTypes.h>

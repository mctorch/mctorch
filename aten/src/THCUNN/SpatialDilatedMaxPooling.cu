#include "hip/hip_runtime.h"
#include <THCUNN/THCUNN.h>
#include <THC/THCTensor.hpp>
#include <TH/THHalf.h>
#include <THCUNN/THCHalfAutoNumerics.cuh>
#include <THC/THCNumerics.cuh>
#include <THCUNN/common.h>
#include <c10/macros/Macros.h>

// kernels borrowed from Caffe
template <typename Dtype, typename AccType>
__global__ void MaxPoolForward(const int nthreads, const Dtype* bottom_data,
    const int num, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const int kernel_h, const int kernel_w, const int stride_h,
    const int stride_w, const int pad_h, const int pad_w,
    const int dilation_h, const int dilation_w, Dtype* top_data,
    int64_t* top_mask) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;
    int hstart = ph * stride_h - pad_h;
    int wstart = pw * stride_w - pad_w;
    int hend = min(hstart + (kernel_h - 1) * dilation_h + 1, height);
    int wend = min(wstart + (kernel_w - 1) * dilation_w + 1, width);
    while(hstart < 0)
      hstart += dilation_h;
    while(wstart < 0)
      wstart += dilation_w;
    AccType maxval = THCNumerics<AccType>::min();
    int maxidx = -1;
    bottom_data += (n * channels + c) * height * width;
    for (int h = hstart; h < hend; h += dilation_h) {
      for (int w = wstart; w < wend; w += dilation_w) {
        Dtype val = bottom_data[h * width + w];
        if ((ScalarConvert<Dtype, AccType>::to(val) > maxval) || THCNumerics<Dtype>::isnan(val)) {
          maxidx = h * width + w;
          maxval = ScalarConvert<Dtype, AccType>::to(val);
        }
      }
    }
    top_data[index] = ScalarConvert<AccType, Dtype>::to(maxval);
    top_mask[index] = maxidx;
  }
}

const int BACKWARD_THREADS = 256;

template <typename Dtype, typename AccType>
#if defined (__HIP_PLATFORM_HCC__)
C10_LAUNCH_BOUNDS_2(BACKWARD_THREADS, 4)
#else
C10_LAUNCH_BOUNDS_2(BACKWARD_THREADS, 8)
#endif
__global__ void MaxPoolBackward(const int nthreads, const Dtype* top_diff,
    const int64_t* top_mask, const int num, const int channels,
    const int height, const int width, const int pooled_height,
    const int pooled_width, const int kernel_h, const int kernel_w,
    const int stride_h, const int stride_w, const int pad_h, const int pad_w,
    const int dilation_h, const int dilation_w,
    Dtype* bottom_diff) {
    CUDA_KERNEL_LOOP(index, height*width) {
    int h = index/width;
    int w = index - h * width;
//get some templating performance benefits without actually templating
    int phstart, phend, pwstart, pwend;
    if (stride_h == 1) {
       phstart =
        (h + pad_h < ((kernel_h - 1) * dilation_h + 1)) ? 0 : (h + pad_h - ((kernel_h - 1) * dilation_h + 1))  + 1;
       phend = min((h + pad_h)  + 1, pooled_height);
    } else if (stride_h == 2) {
       phstart =
        (h + pad_h < ((kernel_h - 1) * dilation_h + 1)) ? 0 : (h + pad_h - ((kernel_h - 1) * dilation_h + 1)) / 2  + 1;
       phend = min((h + pad_h) / 2  + 1, pooled_height);
    } else {
       phstart =
        (h + pad_h < ((kernel_h - 1) * dilation_h + 1)) ? 0 : (h + pad_h - ((kernel_h - 1) * dilation_h + 1)) / stride_h  + 1;
       phend = min((h + pad_h) / stride_h  + 1, pooled_height);
    }
    if (stride_w == 1) {
        pwstart =
        (w + pad_w < ((kernel_w - 1) * dilation_w + 1)) ? 0 : (w + pad_w - ((kernel_w - 1) * dilation_w + 1)) + 1;
        pwend = min((w + pad_w) + 1, pooled_width);
    } else if (stride_w == 2) {
        pwstart =
        (w + pad_w < ((kernel_w - 1) * dilation_w + 1)) ? 0 : (w + pad_w - ((kernel_w - 1) * dilation_w + 1)) / 2 + 1;
        pwend = min((w + pad_w) / 2 + 1, pooled_width);
    } else {
        pwstart =
        (w + pad_w < ((kernel_w - 1) * dilation_w + 1)) ? 0 : (w + pad_w - ((kernel_w - 1) * dilation_w + 1)) / stride_w + 1;
        pwend = min((w + pad_w) / stride_w + 1, pooled_width);
    }
    for (int n = blockIdx.y; n < num; n += gridDim.y)
       for (int c = blockIdx.z; c < channels; c+= gridDim.z) {

        AccType gradient = AccType(0);
        int offset = (n * channels + c) * pooled_height * pooled_width;
        top_diff += offset;
        top_mask += offset;
//get some templating performance benefits without actually templating
        if ((phstart + 1 != phend) || (pwstart + 1 != pwend)) {
        for (int ph = phstart; ph < phend; ++ph) {
          for (int pw = pwstart; pw < pwend; ++pw) {
            if (top_mask[ph * pooled_width + pw] == h * width + w) {
              gradient += ScalarConvert<Dtype, AccType>::to(top_diff[ph * pooled_width + pw]);
            }
          }
        }
        } else {
            if (top_mask[phstart * pooled_width + pwstart] == h * width + w) {
              gradient += ScalarConvert<Dtype, AccType>::to(top_diff[phstart * pooled_width + pwstart]);
            }
        }
        bottom_diff[(n*channels+c)*height*width+index] = ScalarConvert<AccType, Dtype>::to(gradient);
      }
  }
}

#include <THCUNN/generic/SpatialDilatedMaxPooling.cu>
#include <THC/THCGenerateFloatTypes.h>

#include "hip/hip_runtime.h"
#include <THCUNN/THCUNN.h>
#include <TH/THHalf.h>
#include <THCUNN/THCHalfAutoNumerics.cuh>
#include <THC/THCApply.cuh>

template <typename T>
struct softPlusupdateOutput_functor
{
  const T threshold;
  const T beta;

  softPlusupdateOutput_functor(T threshold_, T beta_)
    : threshold(threshold_)
    , beta(beta_)
  {}

  __device__ void operator()(T *output, const T *input) const {
    T betain = beta * (*input);
    *output = ((betain) > threshold) ? *input : (1/beta) * log1p(exp(betain));
  }
};

template <typename T>
struct softPlusupdateGradInput_functor
{
  const T threshold;
  const T beta;

  softPlusupdateGradInput_functor(T threshold_, T beta_)
    : threshold(threshold_)
    , beta(beta_)
  {}

  __device__ void operator()(T *gradInput, const T *output, const T *gradOutput) const
  {
    T betaout = beta * (*output);
    T exp_bo = exp(betaout);
    *gradInput = ((betaout) > threshold) ? *gradOutput : *gradOutput * (exp_bo - 1) / exp_bo;
  }
};

#include <THCUNN/generic/SoftPlus.cu>
#include <THC/THCGenerateFloatTypes.h>

#include "hip/hip_runtime.h"
#include <THCUNN/THCUNN.h>
#include <THC/THCTensor.hpp>
#include <TH/THHalf.h>
#include <THCUNN/THCHalfAutoNumerics.cuh>
#include <THC/THCAtomics.cuh>

#define START_IND(a,b,c) (int)floor((float)(a * c) / b)
#define END_IND(a,b,c) (int)ceil((float)((a + 1) * c) / b)
// #define START_IND(a,b,c) a * c / b
// #define END_IND(a,b,c)  (a + 1) * c / b + ((a + 1) * c % b > 0)?1:0


#define CUDA_MAX_THREADS 1024   // this is safe, in reality 256 is our limit

// 5d tensor B x D x T x H x W
// All kernels view batch dim B and feature dim D as collapsed.

/*
 * Description:
 *    This function adaptively average pools an input 5D tensor along dimensions
 *     2, 3 and 4.
 *
 *    gridDim.y blocks work together on a single 2D output plane specified by
 *    (blockIdx.x + offsetZ).
 */
 template <typename T>
__global__ void cunn_VolumetricAdaptiveAveragePooling_updateOutput_kernel(
                        T *input, T *output,
                        int isizeT, int isizeH, int isizeW,
                        int osizeT, int osizeH, int osizeW,
                        int64_t istrideD,
                        int64_t istrideT, int64_t istrideH, int64_t istrideW,
                        int64_t offsetZ)
{
  // iterators on output pixels
  int ot, oh, ow;

  // compute offsets based on thread/block ID
  int ostartH = blockIdx.y * blockDim.y + threadIdx.y;
  int oendH   = osizeH;
  int ostepH  = gridDim.y * blockDim.y;
  int ostartW = threadIdx.x;
  int oendW   = osizeW;
  int ostepW  = blockDim.x;

  // select output plane
  int64_t o_plane = blockIdx.x + offsetZ;
  ot = o_plane % osizeT;     // output frame/time
  int d = o_plane / osizeT;  // slice/feature

  // input frame/time ramge is fixed.
  int istartT = START_IND(ot, osizeT, isizeT);
  int iendT = END_IND(ot, osizeT, isizeT);
  int kT = iendT - istartT;

  // input offset by slice/feature and earliest relevant frame/time
  T *input_dt = input + d*istrideD + istartT*istrideT;
  // output offset by slice/feature and frame/time
  T *output_dt = output + o_plane*osizeH*osizeW;

  // For all output pixels...
  for(oh = ostartH; oh < oendH; oh += ostepH) {

    int istartH = START_IND(oh, osizeH, isizeH);
    int iendH   = END_IND(oh, osizeH, isizeH);
    int kH = iendH - istartH;

    for(ow = ostartW; ow < oendW; ow += ostepW) {

      int istartW = START_IND(ow, osizeW, isizeW);
      int iendW   = END_IND(ow, osizeW, isizeW);
      int kW = iendW - istartW;

      // Compute the average pooling from corresponding input pixels
      T *ptr_input = input_dt + istartH*istrideH + istartW*istrideW;
      T *ptr_output = output_dt + oh*osizeW + ow;
      T sum = ScalarConvert<int, T>::to(0);

      int it, ih, iw;
      for(it = 0; it < kT; ++it) {
        for(ih = 0; ih < kH; ++ih) {
          for(iw = 0; iw < kW; ++iw) {
            T val = ptr_input[ih*istrideH + iw*istrideW];
            sum += val;
          }
        }
        ptr_input += istrideT;   // next input frame
      }
      // Update output
      *ptr_output = sum / kT / kH / kW;
    }
  }
}

/*
 * Description:
 *    This function computes the gradInput from gradOutput.
 *
 *    gridDim.y blocks work together on a single 2D input plane specified by
 *    (blockIdx.x + offsetZ).
 */
 template <typename T>
__global__ void cunn_VolumetricAdaptiveAveragePooling_updateGradInput_kernel(
  T *gradInput, T *gradOutput,
  int isizeT, int isizeH, int isizeW,
  int osizeT, int osizeH, int osizeW,
  int64_t offsetZ
)
{
  // iterators on input pixels
  int it, ih, iw;

  // compute offsets based on thread/block ID
  int istartH = blockIdx.y * blockDim.y + threadIdx.y;
  int iendH   = isizeH;
  int istepH  = gridDim.y * blockDim.y;
  int istartW = threadIdx.x;
  int iendW   = isizeW;
  int istepW  = blockDim.x;

  // select input plane
  int64_t i_plane = blockIdx.x + offsetZ;
  it = i_plane % isizeT;        // output frame/time
  int d = i_plane / isizeT;     // slice/feature

  // output frame/time ramge is fixed.
  int ostartT = START_IND(it, isizeT, osizeT);
  int oendT   = END_IND(it, isizeT, osizeT);

  // gradInput offset by slice/feature and frame/time
  T *gradInput_dt = gradInput + i_plane*isizeH*isizeW;
  // gradOutput offset by slice/feature and earliest relevant frame/time
  T *gradOutput_dt = gradOutput + (d*osizeT + ostartT)*osizeH*osizeW;

  // For all input pixels...
  for(ih = istartH; ih < iendH; ih += istepH) {

    int ostartH = START_IND(ih, isizeH, osizeH);
    int oendH   = END_IND(ih, isizeH, osizeH);

    for(iw = istartW; iw < iendW; iw += istepW) {

      int ostartW = START_IND(iw, isizeW, osizeW);
      int oendW   = END_IND(iw, isizeW, osizeW);

      // Compute the gradients from corresponding output pixels
      T *ptr_gradInput = gradInput_dt + ih*isizeW + iw;
      T *ptr_gradOutput = gradOutput_dt;

      // for all relevant output pixels
      int ot, oh, ow;
      for(ot = ostartT; ot < oendT; ++ot) {
        int kT = END_IND(ot, osizeT, isizeT) - START_IND(ot, osizeT, isizeT);
        for(oh = ostartH; oh < oendH; ++oh) {
          int kH = END_IND(oh, osizeH, isizeH) - START_IND(oh, osizeH, isizeH);
          for(ow = ostartW; ow < oendW; ++ow) {
            int kW = END_IND(ow, osizeW, isizeW) - START_IND(ow, osizeW, isizeW);
            T grad_delta = ptr_gradOutput[oh*osizeW + ow] / kW / kH / kT;
            *ptr_gradInput += grad_delta;
          }
        }
        ptr_gradOutput += osizeH*osizeW;   // next output frame
      }
    }
  }
}

/*
 * Description:
 *    This function computes the gradInput from gradOutput without assuming
 *    dependencies between input pixels and output pixels.
 *
 *    gridDim.y blocks work together on a single 2D output plane specified by
 *    (blockIdx.x + offsetZ).
 *
 *    (uses atomic add)
 */
 template <typename T>
__global__ void cunn_atomic_VolumetricAdaptiveAveragePooling_updateGradInput_kernel(
  T *gradInput, T *gradOutput,
  int isizeT, int isizeH, int isizeW,
  int osizeT, int osizeH, int osizeW,
  int64_t offsetZ
)
{
  // iterators on output pixels
  int ot, oh, ow;

  // compute offsets based on thread/block ID
  int ostartH = blockIdx.y * blockDim.y + threadIdx.y;
  int oendH   = osizeH;
  int ostepH  = gridDim.y * blockDim.y;
  int ostartW = threadIdx.x;
  int oendW   = osizeW;
  int ostepW  = blockDim.x;

  // select output plane
  int64_t o_plane = blockIdx.x + offsetZ;
  ot = o_plane % osizeT;        // output frame/time
  int d = o_plane / osizeT;     // output slice/feature

  // input frame/time ramge is fixed.
  int istartT = START_IND(ot, osizeT, isizeT);
  int iendT = END_IND(ot, osizeT, isizeT);
  int kT = iendT - istartT;

  // gradInput offset by slice/feature and earliest relevant frame/time
  T *gradInput_nt = gradInput + (d*isizeT + istartT)*isizeH*isizeW;
  // gradOutput offset by slice/feature and frame/time
  T *gradOutput_nt = gradOutput + o_plane*osizeH*osizeW;

  // For all output pixels...
  for(oh = ostartH; oh < oendH; oh += ostepH) {

    int istartH = START_IND(oh, osizeH, isizeH);
    int iendH   = END_IND(oh, osizeH, isizeH);
    int kH = iendH - istartH;

    for(ow = ostartW; ow < oendW; ow += ostepW) {

      int istartW = START_IND(ow, osizeW, isizeW);
      int iendW   = END_IND(ow, osizeW, isizeW);
      int kW = iendW - istartW;

      // Compute the gradients from corresponding input pixels
      T *ptr_gradInput = gradInput_nt + istartH*isizeW + istartW;
      T *ptr_gradOutput = gradOutput_nt + oh*osizeW + ow;
      T grad_delta = *ptr_gradOutput / kT / kH / kW;

      int it, ih, iw;
      for(it = 0; it < kT; ++it) {
        for(ih = 0; ih < kH; ++ih) {
          for(iw = 0; iw < kW; ++iw) {
            atomicAdd(&(ptr_gradInput[ih*isizeW + iw]), grad_delta);
          }
        }
        ptr_gradInput += isizeH*isizeW;   // next input frame
      }
    }
  }
}

#include <THCUNN/generic/VolumetricAdaptiveAveragePooling.cu>
#include <THC/THCGenerateFloatTypes.h>

#undef CUDA_MAX_THREADS
#undef START_IND
#undef END_IND

#include <THCUNN/THCUNN.h>
#include <TH/THHalf.h>
#include <THCUNN/THCHalfAutoNumerics.cuh>
#include <THC/THCApply.cuh>

template <typename T>
struct LeakyReLUUpdateOutput
{
  const T negval_;

  LeakyReLUUpdateOutput(T negval)
    : negval_(negval)
  {}

  __device__ __forceinline__ void operator()(T *out, T *in)
  {
    T x = *in;
    *out = (x > 0) ? x : x * negval_;
  }
};

// in-place variant
template <typename T>
struct LeakyReLUUpdateOutputIP
{
  const T negval_;

  LeakyReLUUpdateOutputIP(T negval)
    : negval_(negval)
  {}

  __device__ __forceinline__ void operator()(T *x)
  {
    *x = (*x > 0) ? *x : negval_ * (*x);
  }
};

template <typename T>
struct LeakyReLUUpdateGradInput
{
  const T negval_;

  LeakyReLUUpdateGradInput(T negval)
    : negval_(negval)
  {}

  __device__ __forceinline__ void operator()(
    T* gradInput,
    T* input,
    T* gradOutput) const
  {
    *gradInput = (*input > 0) ? *gradOutput : (*gradOutput) * negval_;
  }
};

template <typename T>
struct LeakyReLUUpdateGradInputIP
{
  const T negval_;

  LeakyReLUUpdateGradInputIP(T negval)
    : negval_(negval)
  {}

  __device__ __forceinline__ void operator()(
    T* gradOutput,
    T* input) const
  {
    *gradOutput = (*input > 0) ? *gradOutput : (*gradOutput) * negval_;
  }
};

#include <THCUNN/generic/LeakyReLU.cu>
#include <THC/THCGenerateFloatTypes.h>

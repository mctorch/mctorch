#include "hip/hip_runtime.h"
#include <THCUNN/THCUNN.h>
#include <THCUNN/common.h>
#include <TH/THHalf.h>
#include <THCUNN/THCHalfAutoNumerics.cuh>
#include <THC/THCAtomics.cuh>
#include <THC/THCTensor.hpp>
#include <THC/THCStorage.hpp>

#define TEMPORAL_MAX_POOLING_THREADS 1024

template <typename Dtype>
__global__ void cunn_TemporalMaxPooling_updateOutputKernel(Dtype *input, Dtype *output, THCIndex_t *indices, int input_w, int input_n, int output_w, int kW, int dW) {
  // Block idx is the batch index, thread idx + block idx y * MAX_THREADS is the time index
  Dtype *input_data = input + blockIdx.x * input_w * input_n + (
      threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS) * input_n * dW;
  Dtype *output_data = output + blockIdx.x * output_w * input_n + (
      threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS) * input_n;
  THCIndex_t *indices_data = indices + blockIdx.x * output_w * input_n + (
      threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS) * input_n;

  int feat = 0;
  int time = 0;
  int max_time = input_n * kW;

  Dtype max_value;
  THCIndex_t max_index = 0;

  if (threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS < output_w) {
    // For all features
    for (feat = 0; feat < input_n; ++feat) {
      max_value = THCNumerics<Dtype>::min();
      // For all values in the kernel space
      for (time = 0; time < max_time; time += input_n) {
        if (max_value < input_data[time + feat]) {
          max_value = input_data[time + feat];
          max_index = time / input_n;
        }
      }
      output_data[feat] = max_value;
      indices_data[feat] = max_index;
    }
  }
}

template <typename Dtype>
__global__ void cunn_TemporalMaxPooling_updateGradInputKernel(Dtype *gradInput, Dtype *gradOutput, THCIndex_t *indices, int input_w, int input_n, int output_w, int kW, int dW) {
  // Block idx is the batch index, thread idx + block idx y * MAX_THREADS is the time index
  Dtype *gradInput_data = gradInput + blockIdx.x * input_w * input_n + (
      threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS) * input_n * dW;
  Dtype *gradOutput_data = gradOutput + blockIdx.x * output_w * input_n + (
      threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS) * input_n;
  THCIndex_t *indices_data = indices + blockIdx.x * output_w * input_n + (
      threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS) * input_n;

  int feat = 0;

  if (threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS < output_w) {
    // For all features
    for (feat = 0; feat < input_n; ++feat) {
      gradInput_data[indices_data[feat] * input_n + feat] += gradOutput_data[feat];
    }
  }
}

template <typename Dtype>
__global__ void cunn_TemporalMaxPooling_updateGradInputKernelAtomic(Dtype *gradInput, Dtype *gradOutput, THCIndex_t *indices, int input_w, int input_n, int output_w, int kW, int dW) {
  // Block idx is the batch index, thread idx + block idx y * MAX_THREADS is the time index
  Dtype *gradInput_data = gradInput + blockIdx.x * input_w * input_n + (
      threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS) * input_n * dW;
  Dtype *gradOutput_data = gradOutput + blockIdx.x * output_w * input_n + (
      threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS) * input_n;
  THCIndex_t *indices_data = indices + blockIdx.x * output_w * input_n + (
      threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS) * input_n;

  int feat = 0;

  if (threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS < output_w) {
    // For all features
    for (feat = 0; feat < input_n; ++feat) {
      atomicAdd(&gradInput_data[indices_data[feat] * input_n + feat], gradOutput_data[feat]);
    }
  }
}

#include <THCUNN/generic/TemporalMaxPooling.cu>
#include <THC/THCGenerateFloatTypes.h>

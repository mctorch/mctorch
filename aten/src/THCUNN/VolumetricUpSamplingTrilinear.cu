#include "hip/hip_runtime.h"
// Adapted from interp.cpp from Caffe util by Pauline Luc
// Originally developed by George Papandreou
#include <THCUNN/THCUNN.h>
#include <THC/THCTensor.hpp>
#include <THCUNN/common.h>
#include <THCUNN/upsampling.h>
#include <THC/THCDeviceTensor.cuh>
#include <THC/THCDeviceTensorUtils.cuh>
#include <THC/THCDeviceUtils.cuh>
#include <TH/THHalf.h>
#include <THCUNN/THCHalfAutoNumerics.cuh>
#include <THC/THCAtomics.cuh>
#include <c10/macros/Macros.h>

template<typename Dtype, typename Acctype>
C10_LAUNCH_BOUNDS_1(1024)
__global__ void caffe_gpu_interp2_kernel(const int n,
    const Acctype rdepth, const Acctype rheight, const Acctype rwidth, const bool align_corners,
    const THCDeviceTensor<Dtype, 5> data1, THCDeviceTensor<Dtype, 5> data2) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  const int batchsize = data1.getSize(0);
  const int channels = data1.getSize(1);
  const int depth1 = data1.getSize(2);
  const int height1 = data1.getSize(3);
  const int width1 = data1.getSize(4);
  const int depth2 = data2.getSize(2);
  const int height2 = data2.getSize(3);
  const int width2 = data2.getSize(4);

  if (index < n) {
    const int w2 = (index % (height2*width2)) % width2; // 0:width2-1
    const int h2 = (index % (height2*width2)) / width2; // 0:height2-1
    const int t2 = index / (height2*width2);            // 0:depth2-1
    // special case: just copy
    if (depth1 == depth2 && height1 == height2 && width1 == width2) {
      const int t1 = t2;
      const int h1 = h2;
      const int w1 = w2;
      for (int n = 0; n < batchsize ; n++){
        for (int c = 0; c < channels; ++c) {
          const Dtype val = data1[n][c][t1][h1][w1];
          data2[n][c][t2][h2][w2] = val;
        }
      }
      return;
    }
    //
    const Acctype t1r = area_pixel_compute_source_index<Acctype>(rdepth, t2, align_corners, /*cubic=*/false);
    const int t1 = t1r;
    const int t1p = (t1 < depth1 - 1) ? 1 : 0;
    const Acctype t1lambda = t1r - t1;
    const Acctype t0lambda = Acctype(1) - t1lambda;
    //
    const Acctype h1r = area_pixel_compute_source_index<Acctype>(rheight, h2, align_corners, /*cubic=*/false);
    const int h1 = h1r;
    const int h1p = (h1 < height1 - 1) ? 1 : 0;
    const Acctype h1lambda = h1r - h1;
    const Acctype h0lambda = Acctype(1) - h1lambda;
    //
    const Acctype w1r = area_pixel_compute_source_index<Acctype>(rwidth, w2, align_corners, /*cubic=*/false);
    const int w1 = w1r;
    const int w1p = (w1 < width1 - 1) ? 1 : 0;
    const Acctype w1lambda = w1r - w1;
    const Acctype w0lambda = Acctype(1) - w1lambda;
    //
    for (int n = 0; n < batchsize ; n++){
        for (int c = 0; c < channels; ++c) {
        const Acctype val = t0lambda * (h0lambda * (w0lambda * data1[n][c][t1][h1][w1]
                                                  + w1lambda * data1[n][c][t1][h1][w1+w1p])
                                      + h1lambda * (w0lambda * data1[n][c][t1][h1+h1p][w1]
                                                  + w1lambda * data1[n][c][t1][h1+h1p][w1+w1p]))
                          + t1lambda * (h0lambda * (w0lambda * data1[n][c][t1+t1p][h1][w1]
                                                  + w1lambda * data1[n][c][t1+t1p][h1][w1+w1p])
                                      + h1lambda * (w0lambda * data1[n][c][t1+t1p][h1+h1p][w1]
                                                  + w1lambda * data1[n][c][t1+t1p][h1+h1p][w1+w1p]));
        data2[n][c][t2][h2][w2] = ScalarConvert<Acctype, Dtype>::to(val);
      }
    }
  }
}

// Backward (adjoint) operation 1 <- 2 (accumulates)
template <typename Dtype, typename Acctype>
C10_LAUNCH_BOUNDS_1(1024)
__global__ void caffe_gpu_interp2_kernel_backward(const int n,
    const Acctype rdepth, const Acctype rheight, const Acctype rwidth, const bool align_corners,
    THCDeviceTensor<Dtype, 5> data1, const THCDeviceTensor<Dtype, 5> data2){
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  const int batchsize = data1.getSize(0);
  const int channels = data1.getSize(1);
  const int depth1 = data1.getSize(2);
  const int height1 = data1.getSize(3);
  const int width1 = data1.getSize(4);
  const int depth2 = data2.getSize(2);
  const int height2 = data2.getSize(3);
  const int width2 = data2.getSize(4);
  if (index < n) {
    const int w2 = (index % (height2*width2)) % width2; // 0:width2-1
    const int h2 = (index % (height2*width2)) / width2; // 0:height2-1
    const int t2 = index / (height2*width2);            // 0:depth2-1
    // special case: just copy
    if (depth1 == depth2 && height1 == height2 && width1 == width2) {
      const int t1 = t2;
      const int h1 = h2;
      const int w1 = w2;
      for (int n = 0; n < batchsize ; n++){
        for (int c = 0; c < channels; ++c) {
          const Dtype val = data2[n][c][t1][h1][w1];
          data1[n][c][t2][h2][w2] += val;
        }
      }
      return;
    }
    //
    const Acctype t1r = area_pixel_compute_source_index<Acctype>(rdepth, t2, align_corners, /*cubic=*/false);
    const int t1 = t1r;
    const int t1p = (t1 < depth1 - 1) ? 1 : 0;
    const Acctype t1lambda = t1r - t1;
    const Acctype t0lambda = Acctype(1) - t1lambda;
    //
    const Acctype h1r = area_pixel_compute_source_index<Acctype>(rheight, h2, align_corners, /*cubic=*/false);
    const int h1 = h1r;
    const int h1p = (h1 < height1 - 1) ? 1 : 0;
    const Acctype h1lambda = h1r - h1;
    const Acctype h0lambda = Acctype(1) - h1lambda;
    //
    const Acctype w1r = area_pixel_compute_source_index<Acctype>(rwidth, w2, align_corners, /*cubic=*/false);
    const int w1 = w1r;
    const int w1p = (w1 < width1 - 1) ? 1 : 0;
    const Acctype w1lambda = w1r - w1;
    const Acctype w0lambda = Acctype(1) - w1lambda;
    //
    for (int n = 0; n < batchsize ; n++){
      for (int c = 0; c < channels; ++c) {
        const Dtype d2val = data2[n][c][t2][h2][w2];
        atomicAdd(data1[n][c][t1][h1][w1].data(),
                  ScalarConvert<Acctype, Dtype>::to(t0lambda * h0lambda * w0lambda * d2val));
        atomicAdd(data1[n][c][t1][h1][w1+w1p].data(),
                  ScalarConvert<Acctype, Dtype>::to(t0lambda * h0lambda * w1lambda * d2val));
        atomicAdd(data1[n][c][t1][h1+h1p][w1].data(),
                  ScalarConvert<Acctype, Dtype>::to(t0lambda * h1lambda * w0lambda * d2val));
        atomicAdd(data1[n][c][t1][h1+h1p][w1+w1p].data(),
                  ScalarConvert<Acctype, Dtype>::to(t0lambda * h1lambda * w1lambda * d2val));
        atomicAdd(data1[n][c][t1+t1p][h1][w1].data(),
                  ScalarConvert<Acctype, Dtype>::to(t1lambda * h0lambda * w0lambda * d2val));
        atomicAdd(data1[n][c][t1+t1p][h1][w1+w1p].data(),
                  ScalarConvert<Acctype, Dtype>::to(t1lambda * h0lambda * w1lambda * d2val));
        atomicAdd(data1[n][c][t1+t1p][h1+h1p][w1].data(),
                  ScalarConvert<Acctype, Dtype>::to(t1lambda * h1lambda * w0lambda * d2val));
        atomicAdd(data1[n][c][t1+t1p][h1+h1p][w1+w1p].data(),
                  ScalarConvert<Acctype, Dtype>::to(t1lambda * h1lambda * w1lambda * d2val));
      }
    }
  }
  /////////////////////////////////////////////////////////
}


#include <THCUNN/generic/VolumetricUpSamplingTrilinear.cu>
#include <THC/THCGenerateFloatTypes.h>

#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/detail/TensorInfo.cuh>
#include <c10/macros/Macros.h>
#include <hiprand/hiprand_kernel.h>

#include <THC/THCGeneral.h>
#include <THC/THCTensorRandom.h>
#include <THC/THCGenerator.hpp>


THCGenerator* THCRandom_getGenerator(THCState* state);

namespace at{
namespace native{

namespace {

// philox generates 128 bits of randomness at a time. Kernel uses this explicitly by putting suitably transformed result into float4
// for all members of float4 to be consumed UNROLL has to be 4. Don't change!
const int UNROLL = 4;

std::pair<uint64_t, uint64_t> next_philox_seed(at::Generator* gen, uint64_t increment) {
  auto gen_ = THCRandom_getGenerator(at::globalContext().getTHCState());
  uint64_t offset = gen_->state.philox_seed_offset.fetch_add(increment);
  return std::make_pair(gen_->state.initial_seed, offset);
}


template <
          typename scalar_t,
          typename accscalar_t,
          typename IndexType,
          int ADims>
#if __CUDA_ARCH__ >= 350
C10_LAUNCH_BOUNDS_2(256, 8)
#elif defined (__HIP_PLATFORM_HCC__)
C10_LAUNCH_BOUNDS_2(256, 4)
#endif
__global__ void
fused_dropout_kernel(cuda::detail::TensorInfo<scalar_t, IndexType> a,
                      cuda::detail::TensorInfo<scalar_t, IndexType> b,
                      cuda::detail::TensorInfo<uint8_t, IndexType> c,
                      IndexType totalElements, accscalar_t p, std::pair<uint64_t, uint64_t> seeds
                      ) {

  accscalar_t pinv = accscalar_t(1)/p;
  IndexType idx = blockIdx.x * blockDim.x + threadIdx.x;
  hiprandStatePhilox4_32_10_t state;
    hiprand_init(
        seeds.first,
        idx,
        seeds.second,
        &state);
  IndexType rounded_size = ((totalElements - 1)/(blockDim.x * gridDim.x * UNROLL)+1) *
        blockDim.x * gridDim.x * UNROLL;
  for (IndexType linearIndex = idx;
       linearIndex < rounded_size;
       linearIndex += gridDim.x * blockDim.x*UNROLL) {
//hiprand_uniform_double was pure evil anyway, not doing what it promises, and there's nothing for halfs, so generate float for everything
       float4 rand = hiprand_uniform4(&state);
       scalar_t src[UNROLL];
       rand.x = rand.x < p;
       rand.y = rand.y < p;
       rand.z = rand.z < p;
       rand.w = rand.w < p;
       for (int ii = 0; ii < UNROLL; ii++) {
           IndexType li = linearIndex + blockDim.x * gridDim.x * ii;
           if (li < totalElements) {
    // Convert `linearIndex` into an offset of `a`
               const IndexType aOffset =
                   cuda::detail::IndexToOffset<scalar_t, IndexType, ADims>::get(li, a);
               src[ii] = a.data[aOffset];
           }
       }
       for (int ii = 0; ii < UNROLL; ii++) {
           IndexType li = linearIndex + blockDim.x * gridDim.x * ii;
           if (li < totalElements) {
    // Convert `linearIndex` into an offset of `b`
               const IndexType bOffset =
                   cuda::detail::IndexToOffset<scalar_t, IndexType, 1>::get(li, b);
               b.data[bOffset] = src[ii]*(&rand.x)[ii]*pinv;
               c.data[bOffset] = (uint8_t)(&rand.x)[ii];
           }
       }
       __syncthreads();
  }
}

template<typename scalar_t, typename accscalar_t>
void masked_scale_kernel(at::Tensor& ret, const at::Tensor src, const at::Tensor mask, accscalar_t scale){
   at::cuda::CUDA_tensor_apply3<scalar_t, scalar_t, uint8_t>(ret, src, mask, [scale]__device__(scalar_t& ret_val, const scalar_t& src_val, const uint8_t mask_val){
       ret_val = (float)mask_val * src_val * scale;
  });
}
} //anonymous namespace

std::tuple<Tensor,Tensor>
fused_dropout_cuda(const Tensor& self, double p, Generator * gen){
  Tensor ret = at::empty_like(self);
  Tensor mask = at::empty(self.sizes(), self.options().dtype(kByte));
  const int64_t nelem = self.numel();
  const int64_t block_size = 256;
  unsigned int blocks_per_sm = at::cuda::getCurrentDeviceProperties()->maxThreadsPerMultiProcessor/block_size;
  dim3 dim_block(block_size);
  dim3 grid((nelem + block_size -1)/block_size);
  grid.x = std::min((unsigned int)at::cuda::getCurrentDeviceProperties()->multiProcessorCount * blocks_per_sm, grid.x);
//number of times random will be generated per thread, to offset philox counter in thc random state
  int64_t counter_offset = ((nelem - 1)/(block_size*grid.x*UNROLL)+1)*UNROLL;
  if (cuda::detail::canUse32BitIndexMath(self)){
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(self.scalar_type(), "fused_dropout", [&] {
      using accscalar_t = acc_type<scalar_t, true>;
      accscalar_t pa = (accscalar_t)(p);
      auto self_info = cuda::detail::getTensorInfo<scalar_t, unsigned int>(self);
      auto ret_info = cuda::detail::getTensorInfo<scalar_t, unsigned int>(ret);
      auto mask_info = cuda::detail::getTensorInfo<uint8_t, unsigned int>(mask);
      self_info.collapseDims();
      ret_info.collapseDims();
      mask_info.collapseDims(); //ret and mask are collapsed to 1d contiguous tensor
      switch (self_info.dims) {
        case 1:
            fused_dropout_kernel<scalar_t, accscalar_t, unsigned int, 1><<<grid, dim_block, 0, at::cuda::getCurrentCUDAStream()>>>(self_info, ret_info, mask_info, nelem, pa, next_philox_seed(gen,counter_offset));
            break;
        default:
            fused_dropout_kernel<scalar_t, accscalar_t, unsigned int, -1><<<grid, dim_block, 0, at::cuda::getCurrentCUDAStream()>>>(self_info, ret_info, mask_info, nelem, pa, next_philox_seed(gen,counter_offset));
      }
   });
  } else {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(self.scalar_type(), "fused_dropout", [&] {
      using accscalar_t = acc_type<scalar_t, true>;
      accscalar_t pa = (accscalar_t)(p);
      auto self_info = cuda::detail::getTensorInfo<scalar_t, uint64_t>(self);
      auto ret_info = cuda::detail::getTensorInfo<scalar_t, uint64_t>(ret);
      auto mask_info = cuda::detail::getTensorInfo<uint8_t, uint64_t>(mask);
      self_info.collapseDims();
      ret_info.collapseDims();
      mask_info.collapseDims(); //ret and mask are collapsed to 1d contiguous tensor
      switch (self_info.dims) {
        case 1:
            fused_dropout_kernel<scalar_t, accscalar_t, uint64_t, 1><<<grid, dim_block, 0, at::cuda::getCurrentCUDAStream()>>>(self_info, ret_info, mask_info, nelem, pa, next_philox_seed(gen,counter_offset));
            break;
        default:
            fused_dropout_kernel<scalar_t, accscalar_t, uint64_t, -1><<<grid, dim_block, 0, at::cuda::getCurrentCUDAStream()>>>(self_info, ret_info, mask_info, nelem, pa, next_philox_seed(gen,counter_offset));
      }
   });
  }
  THCudaCheck(hipGetLastError());
  return std::tuple<Tensor,Tensor>(ret, mask);
}

Tensor masked_scale_cuda(const Tensor& self, const Tensor& mask, double scale){
   Tensor ret = at::empty_like(self);
   AT_CHECK(mask.scalar_type() == at::ScalarType::Byte, "mask should be torch.uint8 dtype");
   AT_DISPATCH_FLOATING_TYPES_AND_HALF(ret.scalar_type(), "masked_scale", [&] {
      using accscalar_t = acc_type<scalar_t, true>;
      accscalar_t pa = (accscalar_t)(scale);
    masked_scale_kernel<scalar_t>(ret, self, mask, pa);
  });
  return ret;
}

}
}

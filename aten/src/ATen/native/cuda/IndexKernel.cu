#include <ATen/native/Indexing.h>

#include <ATen/ATen.h>
#include <ATen/Dispatch.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/cuda/Array.h>

namespace at { namespace native {

template <int N>
static OffsetCalculator<N> index_make_offset_calculator(const TensorIterator& iter) {
  AT_ASSERT(N <= iter.ntensors());
  std::array<const int64_t*, N> strides;
  for (int i = 0; i < N; i++) {
    strides[i] = iter.strides(i).data();
  }
  return OffsetCalculator<N>(iter.ndim(), iter.shape().data(), strides.data());
}

template <typename func_t>
void gpu_index_kernel(TensorIterator& iter, IntArrayRef index_size, IntArrayRef index_stride, const func_t& f) {
  int num_indices = index_size.size();
  AT_ASSERT(num_indices == index_stride.size());
  AT_ASSERT(num_indices == iter.ntensors() - 2);

  if (iter.numel() == 0) {
    return;
  }

  auto sizes = cuda::Array<int64_t, 25>(0);
  auto strides = cuda::Array<int64_t, 25>(0);
  auto index_ptrs = cuda::Array<char*, 25>(nullptr);
  for (int i = 0; i < num_indices; i++) {
    sizes[i] = index_size[i];
    strides[i] = index_stride[i];
    index_ptrs[i] = (char*)iter.data_ptr(i + 2);
  }

  char* out_ptr = (char*)iter.data_ptr(0);
  char* in_ptr = (char*)iter.data_ptr(1);

  auto offset_calc = index_make_offset_calculator<3>(iter);
  launch_kernel<launch_size_nd, launch_bound2>(iter.numel(), [=]__device__(int idx) {
    auto offsets = offset_calc.get(idx);
    char* out_data = out_ptr + offsets[0];
    char* in_data = in_ptr + offsets[1];

    int64_t offset = 0;
    #pragma unroll
    for (int i = 0; i < num_indices; i++) {
      int64_t index = *(int64_t*)(index_ptrs[i] + offsets[2]);
      assert(index >= -sizes[i] && index < sizes[i] && "index out of bounds");
      if (index < 0) {
        index += sizes[i];
      }
      offset += index * strides[i];
    }

    f(out_data, in_data, offset);
  });
}

// The kernels are templated on an opaque, self-aligned type of the correct
// size to avoid redundant kernels for different types of the same size.
template <int N> struct alignas(N) OpaqueType { char data[N]; };


template <typename scalar_t>
void index_kernel_impl(TensorIterator& iter, IntArrayRef index_size, IntArrayRef index_stride) {
  gpu_index_kernel(iter, index_size, index_stride, []C10_DEVICE(char* out_data, char* in_data, int64_t offset) {
    *(scalar_t*)out_data = *(scalar_t*)(in_data + offset);
  });
}

template <typename scalar_t>
void index_put_kernel_impl(TensorIterator& iter, IntArrayRef index_size, IntArrayRef index_stride) {
  gpu_index_kernel(iter, index_size, index_stride, []C10_DEVICE(char* out_data, char* in_data, int64_t offset) {
    *(scalar_t*)(out_data + offset) = *(scalar_t*)in_data;
  });
}

static void index_kernel(TensorIterator& iter, IntArrayRef index_size, IntArrayRef index_stride) {
  AT_DISPATCH_ALL_TYPES_AND2(at::ScalarType::Half, at::ScalarType::Bool, iter.dtype(), "index_cuda", [&] {
    using dtype = OpaqueType<sizeof(scalar_t)>;
    index_kernel_impl<dtype>(iter, index_size, index_stride);
  });
}


static void index_put_kernel(TensorIterator& iter, IntArrayRef index_size, IntArrayRef index_stride, bool accumulate) {
  AT_ASSERTM(!accumulate, "index_put does not support accumulate=true");
  AT_DISPATCH_ALL_TYPES_AND2(at::ScalarType::Half, at::ScalarType::Bool, iter.dtype(), "index_put", [&] {
    using dtype = OpaqueType<sizeof(scalar_t)>;
    index_put_kernel_impl<dtype>(iter, index_size, index_stride);
  });
}

REGISTER_DISPATCH(index_stub, &index_kernel);
REGISTER_DISPATCH(index_put_stub, &index_put_kernel);

}} // namespace at::native

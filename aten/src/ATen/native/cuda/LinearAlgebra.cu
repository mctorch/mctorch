#include <ATen/ATen.h>

namespace at { namespace native {

Tensor baddbmm_cuda(const Tensor& self, const Tensor& batch1, const Tensor& batch2, Scalar beta, Scalar alpha) {
  return _th_baddbmm(self, batch1, batch2, beta, alpha);
}

Tensor& baddbmm_out_cuda(Tensor &result, const Tensor& self, const Tensor& batch1, const Tensor& batch2, Scalar beta, Scalar alpha) {
  return _th_baddbmm_out(result, self, batch1, batch2, beta, alpha);
}

Tensor& baddbmm__cuda(Tensor& self, const Tensor& batch1, const Tensor& batch2, Scalar beta, Scalar alpha) {
  return _th_baddbmm_out(self, self, batch1, batch2, beta, alpha);
}

Tensor bmm_cuda(const Tensor& self, const Tensor& mat2) {
  return _th_bmm(self, mat2);
}

Tensor& bmm_out_cuda(Tensor &result, const Tensor& batch1, const Tensor& batch2) {
  return _th_bmm_out(result, batch1, batch2);
}

} }

#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/NativeFunctions.h>
#include <ATen/Dispatch.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/native/Activation.h>
#include <ATen/native/cuda/Loops.cuh>


namespace at { namespace native {

// -----------------------------------
// prelu forward
// -----------------------------------
template <typename scalar_t>
void prelu_cuda_kernel_share_weights(
  const Tensor& input,
  Tensor& result,
  const scalar_t* weight_data) {

  at::cuda::CUDA_tensor_apply2<scalar_t, scalar_t>(
    input,
    result,
    [=] __device__ (
      const scalar_t& input_val,
      scalar_t& result_val) {
        result_val = (input_val > 0) ? input_val : *weight_data * input_val;
  });
}

template <typename scalar_t>
__global__ void prelu_cuda_kernel_multi_weights(
  scalar_t* result_data,
  const scalar_t* input_data,
  const scalar_t* weight_data,
  int64_t input_stride0,
  int64_t input_stride1,
  int64_t input_numel) {

  int64_t linearId = blockIdx.x * blockDim.x + threadIdx.x;
  if (linearId >= input_numel) return;

  // multiply values at each channel with weight[channel_index]
  int64_t channel = (linearId % input_stride0) / input_stride1;
  scalar_t input_data_val = input_data[linearId];
  result_data[linearId] = (input_data_val > 0) ? input_data_val : weight_data[channel] * input_data_val;
}

Tensor prelu_cuda(const Tensor& self, const Tensor& weight_) {
  AT_CHECK(self.is_cuda());
  AT_CHECK(weight_.is_cuda());

  auto input = self.contiguous();
  auto weight = weight_.contiguous();

  AT_CHECK(input.is_contiguous());
  AT_CHECK(weight.is_contiguous());

  int64_t weight_num = weight.numel();
  Tensor result = at::empty_like(input);
  auto strides = input.strides();

  // case1: shared weight for all channels
  if (weight_num == 1) {
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "prelu_cuda", [&] {
      prelu_cuda_kernel_share_weights<scalar_t>(
        input,
        result,
        weight.data<scalar_t>());
    });
  }
  else { // case2: multiple weights, one for each channel
    int64_t input_ndim = input.dim();
    AT_CHECK(input_ndim > 0, "Not allow zero-dim input tensor.");

    int64_t channel_size = 1; // channel_size default to 1
    int64_t input_stride0 = 1, input_stride1 = 1;

    if (input_ndim > 1) {
      channel_size = input.size(1); // channel is the 2nd dim of input
      input_stride0 = strides[0];
      input_stride1 = strides[1];
    }
    AT_CHECK(channel_size == weight_num,
      "Mismatch of parameter numbers and input channel size. Found parameter numbers = ", weight_num,
      " and channel size = ", channel_size, ".");

    // config to run cuda kernel
    int64_t input_numel = input.numel();
    const dim3 block = dim3(std::min(static_cast<int64_t>(cuda::getApplyBlock().x), input_numel));
    dim3 grid;
    int curDevice = -1;
    hipGetDevice(&curDevice);
    hipStream_t stream = at::cuda::getCurrentCUDAStream(curDevice);
    AT_CHECK(cuda::getApplyGrid(input_numel, grid, curDevice), "prelu: input too large or too many dimensions");

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "prelu_cuda", [&] {
      prelu_cuda_kernel_multi_weights<scalar_t>
      <<<grid, block, 0, stream>>>(
        result.data<scalar_t>(),
        input.data<scalar_t>(),
        weight.data<scalar_t>(),
        input_stride0,
        input_stride1,
        input_numel);
    });
  }
  return result;
}

// -----------------------------------
// prelu backward
// -----------------------------------
template <typename scalar_t>
void prelu_cuda_backward_kernel_share_weights(
  const Tensor& input,
  const Tensor& grad_out,
  Tensor& input_grad,
  Tensor& weight_grad_collector,
  const scalar_t* weight_data) {

  at::cuda::CUDA_tensor_apply4<scalar_t, scalar_t, scalar_t, scalar_t>(
    input,
    grad_out,
    input_grad,
    weight_grad_collector,
    [=] __device__ (
      const scalar_t& input_val,
      const scalar_t& grad_out_val,
      scalar_t& input_grad_val,
      scalar_t& weight_grad_collector_val) {
        input_grad_val = (input_val > 0) ? grad_out_val : *weight_data * grad_out_val;
        weight_grad_collector_val = (input_val > 0) ? scalar_t(0) : input_val * grad_out_val;
  });
}

template <typename scalar_t>
__global__ void prelu_cuda_backward_kernel_multi_weights(
  const scalar_t* input_data,
  const scalar_t* weight_data,
  const scalar_t* grad_out_data,
  scalar_t* input_grad_data,
  scalar_t* weight_grad_collector,
  int64_t input_stride0,
  int64_t input_stride1,
  int64_t input_numel) {

  int64_t linearId = blockIdx.x * blockDim.x + threadIdx.x;
  if (linearId >= input_numel) return;
  int64_t channel = (linearId % input_stride0) / input_stride1;
  scalar_t input_data_val = input_data[linearId];
  scalar_t grad_out_data_val = grad_out_data[linearId];
  input_grad_data[linearId] = (input_data_val > 0) ? grad_out_data_val : weight_data[channel] * grad_out_data_val;
  weight_grad_collector[linearId] = (input_data_val > 0) ? scalar_t(0) : input_data_val * grad_out_data_val;
}

std::tuple<Tensor, Tensor> prelu_backward_cuda(const Tensor& grad_out_, const Tensor& self, const Tensor& weight_) {
  AT_CHECK(grad_out_.is_cuda());
  AT_CHECK(self.is_cuda());
  AT_CHECK(weight_.is_cuda());

  auto input = self.contiguous();
  auto grad_out = grad_out_.contiguous();
  auto weight = weight_.contiguous();

  AT_CHECK(input.is_contiguous());
  AT_CHECK(weight.is_contiguous());
  AT_CHECK(grad_out.is_contiguous());

  int64_t weight_num = weight.numel();
  auto strides = input.strides();
  auto dims = input.dim();
  Tensor input_grad = at::empty_like(input);
  Tensor weight_grad = at::empty_like(weight);
  Tensor weight_grad_collector = at::empty_like(input);
  // case1: shared parameter for all channels
  if (weight_num == 1) {
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "prelu_backward_cuda", [&] {
      prelu_cuda_backward_kernel_share_weights<scalar_t>(
        input,
        grad_out,
        input_grad,
        weight_grad_collector,
        weight.data<scalar_t>());
    });
    weight_grad.fill_(weight_grad_collector.sum());
  }
  else { // case2: multiple parameters, one for each channel
    int64_t input_ndim = input.dim();
    AT_CHECK(input_ndim > 0, "Not allow zero-dim input tensor.");

    int64_t channel_size = 1; // channel_size default to 1
    int64_t input_stride0 = 1, input_stride1 = 1;

    if (input_ndim > 1) {
      channel_size = input.size(1); // channel is the 2nd dim of input
      input_stride0 = strides[0];
      input_stride1 = strides[1];
    }
    AT_CHECK(channel_size == weight_num,
      "Mismatch of parameter numbers and input channel size. Found parameter numbers = ", weight_num,
      " and channel size = ", channel_size, ".");

    // config to run cuda kernel
    int64_t input_numel = input.numel();
    const dim3 block = dim3(std::min(static_cast<int64_t>(cuda::getApplyBlock().x), input_numel));
    dim3 grid;
    int curDevice = -1;
    hipGetDevice(&curDevice);
    hipStream_t stream = at::cuda::getCurrentCUDAStream(curDevice);
    AT_CHECK(cuda::getApplyGrid(input_numel, grid, curDevice), "prelu_backward_cuda: input too large or too many dimensions");

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "prelu_backward_cuda", [&] {
      prelu_cuda_backward_kernel_multi_weights<scalar_t>
      <<<grid, block, 0, stream>>>(
        input.data<scalar_t>(),
        weight.data<scalar_t>(),
        grad_out.data<scalar_t>(),
        input_grad.data<scalar_t>(),
        weight_grad_collector.data<scalar_t>(),
        input_stride0,
        input_stride1,
        input_numel);
    });
    // update weight_grad
    std::vector<int64_t> reduce_dims;
    reduce_dims.push_back(0);
    if (dims > 2) {
      for(int64_t i = 2; i < dims; i++) reduce_dims.push_back(i);
    }
    weight_grad = weight_grad_collector.sum(reduce_dims);
  }
  return std::tuple<Tensor, Tensor>{input_grad, weight_grad};
}

// -----------------------------------
// hardshrink
// -----------------------------------
template <typename scalar_t>
void hardshrink_cuda_kernel(const Tensor& self, Tensor& out_tensor, scalar_t lambd) {
  at::cuda::CUDA_tensor_apply2<scalar_t, scalar_t>(
    self,
    out_tensor,
    [=] __device__ (
      scalar_t& self_val,
      scalar_t& out_tensor_val) {
        out_tensor_val = (self_val >= -lambd && self_val <= lambd) ? scalar_t(0) : self_val;
  });
}

template <typename scalar_t>
void hardshrink_backward_cuda_kernel(const Tensor& self, Tensor& out_tensor, scalar_t lambd, const Tensor& grad) {
  at::cuda::CUDA_tensor_apply3<scalar_t, scalar_t, scalar_t>(
    self,
    grad,
    out_tensor,
    [=] __device__ (
      scalar_t& self_val,
      scalar_t& grad_val,
      scalar_t& out_tensor_val) {
        out_tensor_val = (self_val >= -lambd && self_val <= lambd) ? scalar_t(0) : grad_val;
  });
}

Tensor hardshrink_cuda(const Tensor & self, Scalar lambd) {
  auto out_tensor = at::empty_like(self);
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(self.scalar_type(), "hardshrink_cuda", [&] {
    hardshrink_cuda_kernel<scalar_t>(self, out_tensor, lambd.to<scalar_t>());
  });
  return out_tensor;
}

Tensor hardshrink_backward_cuda(const Tensor & grad, const Tensor & self, Scalar lambd) {
  auto out_tensor = at::empty_like(grad);
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(self.scalar_type(), "hardshrink_backward_cuda", [&] {
    hardshrink_backward_cuda_kernel<scalar_t>(self, out_tensor, lambd.to<scalar_t>(), grad);
  });
  return out_tensor;
}

template <typename scalar_t>
void threshold_kernel_impl(TensorIterator& iter, scalar_t threshold, scalar_t value) {
  gpu_binary_kernel(iter, [=]GPU_LAMBDA(scalar_t x, scalar_t other) -> scalar_t {
    return x <= threshold ? value : other;
  });
}

static void threshold_kernel(TensorIterator& iter, Scalar threshold, Scalar value) {
  AT_DISPATCH_ALL_TYPES_AND(at::ScalarType::Half, iter.dtype(), "threshold_cuda", [&] {
    threshold_kernel_impl<scalar_t>(iter, threshold.to<scalar_t>(), value.to<scalar_t>());
  });
}

REGISTER_DISPATCH(threshold_stub, &threshold_kernel);

}}  // namespace at::native

#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/detail/KernelUtils.h>
#include <ATen/NativeFunctions.h>
#include <ATen/TensorUtils.h>
#include <ATen/Utils.h>
#include <c10/util/Exception.h>

#include <algorithm>
#include <cfloat>
#include <cmath>

namespace at {
namespace native {

using namespace at::cuda::detail;

namespace {

template <typename scalar_t, typename accscalar_t>
__device__ inline int get_interval(accscalar_t sample,
  int index, int inputSize, int outputSize, int poolSize) {
  accscalar_t alpha = static_cast<accscalar_t>(inputSize - poolSize) /
    static_cast<accscalar_t>(outputSize - 1);
  if (index == outputSize - 1) {
    return inputSize - poolSize;
  } else {
    return static_cast<int>((index + sample) * alpha) -
      static_cast<int>(sample * alpha);
  }
}

template <typename scalar_t>
__global__ void fractional_max_pool2d_out_cuda_frame(
  PackedTensorAccessor<scalar_t, 4> output,
  PackedTensorAccessor<int64_t, 4> indices,
  PackedTensorAccessor<scalar_t, 4> input,
  PackedTensorAccessor<scalar_t, 3> samples,
  int poolSizeH, int poolSizeW) {

  using accscalar_t = at::acc_type<scalar_t, /*is_cuda=*/true>;

  int ourOutputPoint = threadIdx.x + blockIdx.x * blockDim.x;
  int plane = blockIdx.y;
  int batch = blockIdx.z;

  // Each thread generates a specific output point
  if (ourOutputPoint < output.size(2) * output.size(3)) {
    int outputW = ourOutputPoint % output.size(3);
    int outputH = ourOutputPoint / output.size(3);

    int poolW = get_interval<scalar_t, accscalar_t>(
      static_cast<accscalar_t>(samples[batch][plane][0]),
        outputW, input.size(3), output.size(3), poolSizeW);
    int poolH = get_interval<scalar_t, accscalar_t>(
      static_cast<accscalar_t>(samples[batch][plane][1]),
        outputH, input.size(2), output.size(2), poolSizeH);

    scalar_t maxVal = at::numeric_limits<scalar_t>::lowest();
    int maxIndex = -1;

    for (int h = poolH; h < poolH + poolSizeH; ++h) {
      if (poolSizeW < 2 || poolSizeW > 7) {
        for (int w = poolW; w < poolW + poolSizeW; ++w) {
          scalar_t val = input[batch][plane][h][w];
          // for consistency with THNN, favor the first max
          if (val > maxVal) {
            maxIndex = h * input.size(3) + w;
            maxVal = val;
          }
        }
      } else {
        for (int i = 0; i < poolSizeW; ++i) {
          int w = i + poolW;
          scalar_t val = input[batch][plane][h][w];
          // for consistency with THNN, favor the first max
          if (val > maxVal) {
            maxIndex = h * input.size(3) + w;
            maxVal = val;
          }
        }
      }
    }

    assert(maxVal != at::numeric_limits<scalar_t>::lowest());
    assert(maxIndex != -1);

    indices[batch][plane][outputH][outputW] = maxIndex;
    output[batch][plane][outputH][outputW] = maxVal;
  }
}

template <typename scalar_t>
__global__ void fractional_max_pool2d_backward_out_cuda_frame(
  PackedTensorAccessor<scalar_t, 4> gradInput,
  PackedTensorAccessor<scalar_t, 4> gradOutput,
  PackedTensorAccessor<int64_t, 4> indices) {
  // Output (h, w) point that this thread is responsible for
  int ourOutputPoint = threadIdx.x + blockIdx.x * blockDim.x;
  int plane = blockIdx.y;
  int batch = blockIdx.z;

  // Each thread generates a specific output point
  if (ourOutputPoint < gradOutput.size(2) *
    gradOutput.size(3)) {
    int outputW = ourOutputPoint % gradOutput.size(3);
    int outputH = ourOutputPoint / gradOutput.size(3);

    int index = indices[batch][plane][outputH][outputW];
    assert(index >= 0);
    int inputW = index % gradInput.size(3);
    int inputH = index / gradInput.size(3);
    assert(inputH < gradInput.size(2));

    atomicAdd(
      &gradInput[batch][plane][inputH][inputW],
      gradOutput[batch][plane][outputH][outputW]
    );
  }
}

void fractional_max_pool2d_out_cuda_template(
  Tensor & output,
  Tensor& indices,
  const Tensor& input,
  IntArrayRef pool_size,
  IntArrayRef output_size,
  const Tensor& randomSamples) {
  int planeDim = 0;
  int dimh = 1;
  int dimw = 2;
  int numBatch = 1;

  int ndims = input.ndimension();
  AT_CHECK(input.numel() > 0,
    "fractional_max_pool2d(): expected input to have non-empty ",
    "spatial dimensions.");

  AT_CHECK((ndims == 3 || ndims == 4),
     "non-empty 3D or 4D (batch mode) tensor expected for input");

  if (ndims == 4) {
    numBatch = input.size(0);
    planeDim++;
    dimh++;
    dimw++;
  }

  /* sizes */
  int numPlanes = input.size(planeDim);
  int inputH = input.size(dimh);
  int inputW = input.size(dimw);

  int outputH = output_size[0];
  int outputW = output_size[1];
  int poolSizeH = pool_size[0];
  int poolSizeW = pool_size[1];

  AT_CHECK(outputH + poolSizeH - 1 <= inputH,
             "fractional_max_pool2d(): pool_size height ", poolSizeH,
             " too large relative to input height ", inputH);
  AT_CHECK(outputW + poolSizeW - 1 <= inputW,
           "pool_size width ", poolSizeW,
           " too large relative to input width ", inputW);

  if (ndims == 3) {
    /* resize output */
    output.resize_({numPlanes, outputH, outputW});
    /* indices will contain the locations for each output point */
    indices.resize_({numPlanes, outputH, outputW});
  } else {
    output.resize_({numBatch, numPlanes, outputH, outputW});
    indices.resize_({numBatch, numPlanes, outputH, outputW});
  }

  auto output_ = output;
  auto input_ = input;
  auto indices_ = indices;

  if(ndims == 3) {
    output_ = output_.reshape({1, numPlanes, outputH, outputW});
    indices_ = indices_.reshape({1, numPlanes, outputH, outputW});
    input_ = input_.reshape({1, input.size(0), input.size(1), input.size(2)});
  }

  // block is limited to 4 warps
  // grid handles overflow per each plane
  int outputPlaneSize = output_.size(2) *
    output_.size(3);
  dim3 grid((outputPlaneSize + 127) / 128, // ceil(outputPlaneSize / 128)
            input_.size(1),
            input_.size(0));
  dim3 block(outputPlaneSize > 128 ? 128 : outputPlaneSize);

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(),
    "fractional_max_pool2d_out_cuda_frame",
    [&] {
      auto devInput = input_.packed_accessor<scalar_t, 4>();
      auto devOutput = output_.packed_accessor<scalar_t, 4>();
      auto devIndices = indices_.packed_accessor<int64_t, 4>();
      auto devSamples = randomSamples.packed_accessor<scalar_t, 3>();
      fractional_max_pool2d_out_cuda_frame<scalar_t>
        <<<grid, block, 0, at::cuda::getCurrentCUDAStream()>>>(
          devOutput, devIndices, devInput, devSamples,
          poolSizeH, poolSizeW);
       }
     );
  AT_CHECK(hipGetLastError() == hipSuccess,
     "fractional_max_pool2d_out_cuda_frame failed with error code ",
     hipGetLastError());
}

void fractional_max_pool2d_backward_out_cuda_template(
  Tensor& gradInput,
  const Tensor& gradOutput,
  const Tensor& input,
  IntArrayRef pool_size /* unused */,
  IntArrayRef output_size,
  const Tensor& indices)
{
  int dimh = 1;
  int dimw = 2;

  int ndims = input.ndimension();
  if (ndims == 4) {
    dimh++;
    dimw++;
  }

  /* sizes */
  int inputH = input.size(dimh);
  int inputW = input.size(dimw);

  int outputH = output_size[0];
  int outputW = output_size[1];

  AT_CHECK(outputH == gradOutput.size(dimh),
           "fractional_max_pool2d(): gradOutput height unexpected");
  AT_CHECK(outputW == gradOutput.size(dimw),
           "fractional_max_pool2d(): gradOutput width unexpected");

  /* resize */
  gradInput.resize_as_(input);
  gradInput.zero_();

  auto gradInput_ = gradInput;
  auto gradOutput_ = gradOutput;
  auto indices_ = indices;

  if(ndims == 3) {
    gradInput_ = gradInput_.reshape({1, input.size(0), inputH, inputW});
    gradOutput_ = gradOutput_.reshape({1, gradOutput.size(0), outputH, outputW});
    indices_ = indices_.reshape({1, indices_.size(0), outputH, outputW});
  }

  /* backprop */
  // block is limited to 4 warps
  // grid handles overflow per each plane
  int outputPlaneSize = gradOutput_.size(2) *
    gradOutput_.size(3);
  dim3 grid((outputPlaneSize + 127) / 128, // ceil(outputPlaneSize / 128)
            gradInput_.size(1),
            gradInput_.size(0));
  dim3 block(outputPlaneSize > 128 ? 128 : outputPlaneSize);

  auto devIndices = indices.packed_accessor<int64_t, 4>();
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(gradOutput.scalar_type(),
    "fractional_max_pool2d_backward_out_cuda_frame",
    [&] {
      auto devGradInput = gradInput_.packed_accessor<scalar_t, 4>();
      auto devGradOutput = gradOutput_.packed_accessor<scalar_t, 4>();
      fractional_max_pool2d_backward_out_cuda_frame<scalar_t>
        <<<grid, block, 0, at::cuda::getCurrentCUDAStream()>>>(
        devGradInput, devGradOutput, devIndices);
      }
    );
  AT_CHECK(hipGetLastError() == hipSuccess,
    "fractional_max_pool2d_backward_out_cuda_frame failed with error code ",
    hipGetLastError());
}

}// namespace

std::tuple<Tensor&, Tensor&> fractional_max_pool2d_out_cuda(
  at::Tensor& output,
  at::Tensor& indices,
  const at::Tensor& input,
  IntArrayRef pool_size,
  IntArrayRef output_size,
  const at::Tensor& randomSamples)
{
  fractional_max_pool2d_out_cuda_template(
    output,
    indices,
    input,
    pool_size,
    output_size,
    randomSamples);
  return std::tuple<Tensor&, Tensor&>(output, indices);
}

std::tuple<Tensor, Tensor> fractional_max_pool2d_cuda(
  const at::Tensor& input,
  IntArrayRef pool_size,
  IntArrayRef output_size,
  const at::Tensor& randomSamples)
{
  Tensor output = at::empty({0}, input.options());
  Tensor indices = at::empty({0}, input.options().dtype(kLong));
  fractional_max_pool2d_out_cuda_template(
    output,
    indices,
    input,
    pool_size,
    output_size,
    randomSamples);
  return std::tuple<Tensor, Tensor>(output, indices);
}

Tensor& fractional_max_pool2d_backward_out_cuda(
  at::Tensor& gradInput,
  const at::Tensor& gradOutput_,
  const at::Tensor& input,
  IntArrayRef pool_size,
  IntArrayRef output_size,
  const at::Tensor& indices)
{
  fractional_max_pool2d_backward_out_cuda_template(
    gradInput,
    gradOutput_,
    input,
    pool_size,
    output_size,
    indices);
  return gradInput;
}

Tensor fractional_max_pool2d_backward_cuda(
  const at::Tensor& gradOutput_,
  const at::Tensor& input,
  IntArrayRef pool_size,
  IntArrayRef output_size,
  const at::Tensor& indices)
{
  Tensor gradInput = at::empty({0}, input.options());
  fractional_max_pool2d_backward_out_cuda_template(
    gradInput,
    gradOutput_,
    input,
    pool_size,
    output_size,
    indices);
  return gradInput;
}

}// at::native
}// at

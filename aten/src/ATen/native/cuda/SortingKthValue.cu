#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/native/SortingUtils.h>
#include <assert.h>
#include <c10/macros/Macros.h>
#include <stdlib.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/cuda/detail/TensorInfo.cuh>
#include <THC/THCDeviceUtils.cuh> // only for THCRoundUp?
#include <THC/THCNumerics.cuh>
#include <THC/THCScanUtils.cuh>
#include <THC/THCTensorMathReduce.cuh> // AddOp

#include <thrust/device_ptr.h>
#include <thrust/sort.h>

#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/extrema.h>
#include <thrust/inner_product.h>
#include <thrust/sequence.h>
#include <THC/THCThrustAllocator.cuh>
#include <ATen/native/cuda/SortingCommon.cuh>
#include <ATen/native/cuda/SortingRadixSelect.cuh>

namespace at {
namespace native {

namespace {

template <typename scalar_t, typename index_t, int Dim>
__global__ void gatherKthValue(
    cuda::detail::TensorInfo<scalar_t, index_t> input,
    index_t inputSliceSize,
    index_t k,

    index_t numInputSlices,
    index_t inputWithinSliceStride,

    cuda::detail::TensorInfo<scalar_t, index_t> kthValue,
    cuda::detail::TensorInfo<int64_t, index_t> indices) {
  // Indices are limited to integer fp precision, so counts can fit in
  // int32, regardless of index_t
  __shared__ int smem[WARP_SIZE]; // one per each warp, up to warp limit

  index_t slice = getLinearBlockId<index_t>();
  if (slice >= numInputSlices) {
    return;
  }

  // Find the start offset for our slice
  index_t sliceStartIndex =
      cuda::detail::IndexToOffset<scalar_t, index_t, Dim>::get(slice, input);
  index_t kthValueSliceStartIndex =
      cuda::detail::IndexToOffset<scalar_t, index_t, Dim>::get(slice, kthValue);
  index_t indicesSliceStartIndex =
      cuda::detail::IndexToOffset<int64_t, index_t, Dim>::get(slice, indices);

  scalar_t* inputSliceStart = &input.data[sliceStartIndex];
  scalar_t* kthValueSliceStart = &kthValue.data[kthValueSliceStartIndex];
  int64_t* indicesSliceStart = &indices.data[indicesSliceStartIndex];

  // Find the k-th highest element in our input
  scalar_t kValue = static_cast<scalar_t>(0);
  radixSelect<
      scalar_t,
      typename TopKTypeConfig<scalar_t>::RadixType,
      index_t,
      false>(
      inputSliceStart,
      k,
      inputSliceSize,
      inputWithinSliceStride,
      smem,
      &kValue);

  // Find the index of the k-th highest element
  index_t kValueIndex = 0;
  bool foundKValue = false;

  for (index_t i = threadIdx.x; i < inputSliceSize; i += blockDim.x) {
    bool inRange = (i < inputSliceSize);
    scalar_t v = inRange ? doLdg(&inputSliceStart[i * inputWithinSliceStride])
                         : static_cast<scalar_t>(0);
    bool isKValue = inRange && THCNumerics<scalar_t>::eq_with_nan(v, kValue);

    if (isKValue) {
      kValueIndex = i;
      foundKValue = true;
      break;
    }
  }

  if (foundKValue) {
    kthValueSliceStart[0] = kValue;
    indicesSliceStart[0] = kValueIndex;
  }
}

struct KthValueLauncher {
  int64_t k;

  KthValueLauncher(int64_t k) : k(k) {}

  template <typename scalar_t, typename index_t, int all_dims>
  inline void launch(
      cuda::detail::TensorInfo<scalar_t, index_t> values_info,
      int collapse_values_dim,
      cuda::detail::TensorInfo<int64_t, index_t> indices_info,
      int collapse_indices_dim,
      cuda::detail::TensorInfo<scalar_t, index_t> self_info,
      int collapse_self_dim,
      int64_t num_slices,
      int64_t slice_size) {
    dim3 grid;
    if (!getGridFromTiles(num_slices, grid)) {
      AT_ERROR("slices are too many");
    }

    dim3 block(
        std::min(THCRoundUp(slice_size, (int64_t)WARP_SIZE), (int64_t)1024));
    auto stream = at::cuda::getCurrentCUDAStream();
    gatherKthValue<scalar_t, index_t, all_dims><<<grid, block, 0, stream>>>(
        self_info,
        slice_size,
        k,
        num_slices,
        /* The actual dimension that the k-selection is running in */
        /* may have changed from collapseDims() */
        self_info.strides[collapse_self_dim],
        values_info,
        indices_info);
  }
};

template <typename scalar_t>
void kthvalue_cuda_template(
    Tensor& values,
    Tensor& indices,
    const Tensor& self,
    int64_t k,
    int64_t dim_,
    bool keepdim) {
  int64_t dim = maybe_wrap_dim(dim_, self.dim());
  int64_t slicesize = self.size(dim);
  // FIXME: This seems bogus, I only do this because it was the old behaviour.
  //        The reductions are fine, as long as the axis being reduced along
  //        isn't of 0 elements (and the output has elements).
  AT_CHECK(
      self.numel() > 0,
      "cannot perform reduction function kthvalue",
      " on tensor with no elements because the operation does not have an identity");
  AT_CHECK(k >= 1 && k <= slicesize, "selected number k out of range");

  _reduction_with_indices_allocate_or_resize_output(
      values, indices, self, dim, keepdim);
  if (self.dim() == 0 && self.numel() == 1) {
    values.copy_(self);
    indices.zero_();
    return;
  }

  AT_CHECK(
      self.dim() <= MAX_TENSORINFO_DIMS,
      "cannot operate on more than ",
      MAX_TENSORINFO_DIMS,
      " dimensions");

  // Based on required index size, run the algorithm with the
  // appropriate index type
  if (cuda::detail::canUse32BitIndexMath(self) &&
      cuda::detail::canUse32BitIndexMath(values) &&
      cuda::detail::canUse32BitIndexMath(indices)) {
    run_launcher<scalar_t, uint32_t>(
        values, indices, self, dim, KthValueLauncher(k));
  } else {
    run_launcher<scalar_t, uint64_t>(
        values, indices, self, dim, KthValueLauncher(k));
  }

  if (!keepdim) {
    values.squeeze_(dim);
    indices.squeeze_(dim);
  }

  AT_CUDA_CHECK(hipGetLastError());
}

// this does not reduce to median with dim beause we don't want to copy twice
template <typename scalar_t>
Tensor median_cuda_template(const Tensor& self) {
  AT_CHECK(self.numel() > 0, "median cannot be called with empty tensor");
  if (self.dim() == 0 && self.numel() == 1) {
    return self.clone();
  }
  auto self_copy = self.clone().view(-1);
  auto values = at::empty({1}, self.options());
  auto indices = at::empty({1}, self.options().dtype(kLong));
  AT_CHECK(
      self.dim() <= MAX_TENSORINFO_DIMS,
      "cannot operate on more than ",
      MAX_TENSORINFO_DIMS,
      " dimensions");

  // Based on required index size, run the algorithm with the
  // appropriate index type
  if (cuda::detail::canUse32BitIndexMath(self) &&
      cuda::detail::canUse32BitIndexMath(values) &&
      cuda::detail::canUse32BitIndexMath(indices)) {
    run_launcher<scalar_t, uint32_t>(
        values,
        indices,
        self_copy,
        0,
        KthValueLauncher((self_copy.size(0) + 1) / 2)); // KthValue is 1-based
  } else {
    run_launcher<scalar_t, uint64_t>(
        values,
        indices,
        self_copy,
        0,
        KthValueLauncher((self_copy.size(0) + 1) / 2)); // KthValue is 1-based
  }
  return values.view({});
}

} // namespace

std::tuple<Tensor&, Tensor&> kthvalue_out_cuda(
    Tensor& values,
    Tensor& indices,
    const Tensor& self,
    int64_t k,
    int64_t dim,
    bool keepdim) {
  AT_DISPATCH_ALL_TYPES_AND(at::ScalarType::Half, self.scalar_type(), "kthvalue_cuda", [&] {
    kthvalue_cuda_template<scalar_t>(values, indices, self, k, dim, keepdim);
  });
  return std::forward_as_tuple(values, indices);
}

Tensor median_cuda(const Tensor& self) {
  return AT_DISPATCH_ALL_TYPES_AND(at::ScalarType::Half, self.scalar_type(), "median", [&] {
    return median_cuda_template<scalar_t>(self);
  });
}

} // namespace native
} // namespace at

#include "hip/hip_runtime.h"
#include <ATen/Dispatch.h>
#include <ATen/NativeFunctions.h>
#include <ATen/AccumulateType.h>
#include <ATen/cuda/Exceptions.h>
#include <cmath>
#include <limits>

#include <thrust/device_ptr.h>
#include <thrust/sequence.h>

namespace at {
namespace native {

template<typename T, typename accT = T>
struct LinspaceOp {
  __host__ __device__ LinspaceOp(accT start, accT step):
    start_(start), step_(step) { }
  __device__ __forceinline__ T operator()(ptrdiff_t index) {
    accT increment = step_ * static_cast<accT>(index);
    accT value = start_ + increment;
    return static_cast<T>(value);
  }

  const accT start_, step_;
};

template<typename T, typename accT = T>
struct LogspaceOp {
  __host__ __device__ LogspaceOp(accT start, accT step, accT base):
    start_(start), step_(step), base_(base) { }
  __device__ __forceinline__ T operator()(ptrdiff_t index) {
    accT increment = step_ * static_cast<accT>(index);
    accT value = std::pow(base_, start_ + increment);
    return static_cast<T>(value);
  }

  const accT start_, step_, base_;
};

Tensor& linspace_cuda_out(Tensor& result, Scalar start, Scalar end, int64_t steps) {
  AT_CHECK(steps >= 0, "number of steps must be non-negative");

  if (result.numel() != steps) {
    result.resize_({steps});
  }
  Tensor r = result.is_contiguous() ? result : result.contiguous();

  if (steps == 0) {
    // skip
  } else if (steps == 1) {
    r.fill_(start);
  } else {
    AT_DISPATCH_FLOATING_TYPES(r.scalar_type(), "linspace_cuda", [&]() {
      scalar_t scalar_start = start.to<scalar_t>();
      scalar_t scalar_end = end.to<scalar_t>();
      scalar_t step = (scalar_end - scalar_start) / static_cast<scalar_t>(steps - 1);
      LinspaceOp<scalar_t> linspace_method(scalar_start, step);
      thrust::device_ptr<scalar_t> data_(r.data<scalar_t>());
      thrust::tabulate(data_, data_ + steps, linspace_method);
    });
  }

  if (!result.is_contiguous()) {
    result.copy_(r);
  }
  AT_CUDA_CHECK(hipGetLastError());
  return result;
}

Tensor& logspace_cuda_out(Tensor& result, Scalar start, Scalar end, int64_t steps, double base) {
  AT_CHECK(steps >= 0, "number of steps must be non-negative");

  if (result.numel() != steps) {
    result.resize_({steps});
  }
  Tensor r = result.is_contiguous() ? result : result.contiguous();

  if (steps == 0) {
    // skip
  } else if (steps == 1) {
    r.fill_(std::pow(base, start.to<double>()));
  } else {
    AT_DISPATCH_FLOATING_TYPES(r.scalar_type(), "logspace_cuda", [&]() {
      scalar_t scalar_base = static_cast<scalar_t>(base);
      scalar_t scalar_start = start.to<scalar_t>();
      scalar_t scalar_end = end.to<scalar_t>();
      scalar_t step = (scalar_end - scalar_start) / static_cast<scalar_t>(steps - 1);
      LogspaceOp<scalar_t> logspace_method(scalar_start, step, scalar_base);
      thrust::device_ptr<scalar_t> data_(r.data<scalar_t>());
      thrust::tabulate(data_, data_ + steps, logspace_method);
    });
  }

  if (!result.is_contiguous()) {
    result.copy_(r);
  }
  AT_CUDA_CHECK(hipGetLastError());
  return result;
}

Tensor& range_cuda_out(Tensor& result, Scalar start, Scalar end, Scalar step) {
  AT_DISPATCH_ALL_TYPES_AND(at::ScalarType::Half, result.scalar_type(), "range_cuda", [&]() {
    using accscalar_t = at::acc_type<scalar_t, true>;
    auto xstart = start.to<accscalar_t>();
    auto xend = end.to<accscalar_t>();
    auto xstep = step.to<accscalar_t>();

    AT_CHECK(xstep > 0 || xstep < 0, "step must be nonzero");
    AT_CHECK(std::isfinite(static_cast<double>(xstart)) &&
             std::isfinite(static_cast<double>(xend)),
             "unsupported range: ", xstart, " -> ", xend);
    AT_CHECK(((xstep > 0) && (xend >= xstart)) || ((xstep < 0) && (xend <= xstart)),
             "upper bound and larger bound inconsistent with step sign");
    int64_t size = static_cast<int64_t>(((xend - xstart) / xstep) + 1);
    if (result.numel() != size) {
      result.resize_({size});
    }
    Tensor r = result.is_contiguous() ? result : result.contiguous();
    LinspaceOp<scalar_t, accscalar_t> linspace_method(xstart, xstep);
    thrust::device_ptr<scalar_t> data_ptr(r.data<scalar_t>());
    thrust::tabulate(data_ptr, data_ptr + size, linspace_method);

    if (!result.is_contiguous()) {
      result.copy_(r);
    }
  });

  AT_CUDA_CHECK(hipGetLastError());
  return result;
}

Tensor& arange_cuda_out(Tensor& result, Scalar start, Scalar end, Scalar step) {
  AT_DISPATCH_ALL_TYPES_AND(at::ScalarType::Half, result.scalar_type(), "arange_cuda", [&]() {
    using accscalar_t = at::acc_type<scalar_t, true>;
    auto xstart = start.to<accscalar_t>();
    auto xend = end.to<accscalar_t>();
    auto xstep = step.to<accscalar_t>();

    // we use double precision for (start - end) / step
    // to compute size_d for consistency across devices.
    // The problem with using accscalar_t is that accscalar_t might be float32 on gpu for a float32 scalar_t,
    // but double on cpu for the same,
    // and the effective output size starts differing on CPU vs GPU because of precision issues, which
    // we dont want.
    // the corner-case we do want to take into account is int64_t, which has higher precision than double
    double size_d;
    if (std::is_same<scalar_t, int64_t>::value) {
      size_d = std::ceil(static_cast<double>(end.to<accscalar_t>() - start.to<accscalar_t>())
                         / step.to<accscalar_t>());
    } else {
      size_d = std::ceil(static_cast<double>(end.to<double>() - start.to<double>())
                         / step.to<double>());
    }

    AT_CHECK(xstep > 0 || xstep < 0, "step must be nonzero");
    AT_CHECK(std::isfinite(static_cast<double>(xstart)) &&
             std::isfinite(static_cast<double>(xend)),
             "unsupported range: ", xstart, " -> ", xend);
    AT_CHECK(((xstep > 0) && (xend >= xstart)) || ((xstep < 0) && (xend <= xstart)),
             "upper bound and larger bound inconsistent with step sign");

    AT_CHECK(size_d >= 0 && size_d <= static_cast<double>(std::numeric_limits<int64_t>::max()),
             "invalid size, possible overflow?");
    int64_t size = static_cast<int64_t>(size_d);

    if (result.numel() != size) {
      result.resize_({size});
    }
    Tensor r = result.is_contiguous() ? result : result.contiguous();
    LinspaceOp<scalar_t, accscalar_t> linspace_method(xstart, xstep);
    thrust::device_ptr<scalar_t> data_ptr(r.data<scalar_t>());
    thrust::tabulate(data_ptr, data_ptr + size, linspace_method);

    if (!result.is_contiguous()) {
      result.copy_(r);
    }
  });

  AT_CUDA_CHECK(hipGetLastError());
  return result;
}

}} // namespace at::native

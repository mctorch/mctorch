#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/Context.h>
#include <ATen/Dispatch.h>
#include <ATen/NativeFunctions.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAEvent.h>
#include <c10/cuda/CUDAStream.h>
#include <ATen/native/Copy.h>

namespace {

using namespace at;
using namespace at::cuda;

// Copy operator for the pointwise apply kernel
template <typename dst_T, typename src_T>
struct CopyOp {
  static void apply(Tensor& dst, const Tensor& src) {
    CUDA_tensor_apply2<dst_T, src_T>(
        dst, src, [] __device__(dst_T & dst_val, const src_T& src_val) {
#if __CUDA_ARCH__ >= 350
          dst_val = static_cast<dst_T>(
              static_cast<native::inter_copy_type_t<dst_T>>(__ldg(&src_val)));
#else
          dst_val = static_cast<dst_T>(static_cast<native::inter_copy_type_t<dst_T>>(src_val));
#endif
      });
  }
};

template<typename dst_T>
struct CopyOp<dst_T, bool> {
  static void apply(Tensor& dst, const Tensor& src) {
    CUDA_tensor_apply2<dst_T, bool>(
      dst, src, [] __device__(dst_T & dst_val, const bool& src_val) {
        dst_val = static_cast<dst_T>(static_cast<native::inter_copy_type_t<dst_T>>(src_val));
      });
  }
};

// device-to-device copy, does type conversion
template <typename dst_T, typename src_T>
void copy_device_to_device(Tensor& dst, const Tensor& src) {
  auto numel = dst.numel();
  if (dst.is_same(src) || numel == 0) {
    return;
  }

  // We can memcpy the memory if:
  // -both tensors are contiguous; or,
  // -there is only one element to copy; or,
  // -FIXME: if both tensors have matching size and stride arrays, and no
  // holes within (in other words, there is some permutation that can be applied
  // to the size/strides such that the resulting tensor is
  // contiguous).
  // -AND: both tensors have the same type.
  bool same_type = std::is_same<dst_T, src_T>::value;
  bool memcpy_eligible =
      ((src.is_contiguous() && dst.is_contiguous()) || (numel == 1)) &&
      same_type;

  Device src_device = src.device();
  Device dst_device = dst.device();

  CUDAGuard device_guard(src_device);

  // Try to enable p2p access. This also handles the case src_device ==
  // dst_device.
  bool p2pEnabled = THCState_getPeerToPeerAccess(
      globalContext().getTHCState(), src_device.index(), dst_device.index());

  // We always perform the copy on the source device, using the current stream
  // on the source device, and we fully synchronize on both src and dst's
  // current streams for completion of the copy. We have to explicitly do this
  // for non-contig copies. This mimics the behavior of cross-device
  // hipMemcpyAsync on the default stream.
  CUDAStream copy_stream = getCurrentCUDAStream(src_device.index());
  if (src_device != dst_device) {
    // This is a cross-device copy on the src current stream and dst current
    // stream. We perform a two-way barrier between both devices' streams
    // before the copy. This ensures that any write-after-write and
    // write-after-read dependencies on the destination side are handled, so
    // that no one is operating on the dst memory when we perform the copy.
    // src waits on dst barrier (src already waits on src)
    CUDAEvent dst_ready;
    device_guard.set_device(dst_device);
    dst_ready.record(getCurrentCUDAStream(dst_device.index()));

    device_guard.set_device(src_device);
    dst_ready.block(copy_stream);
  }

  if (memcpy_eligible) {
    // Perform the copy
    AT_CUDA_CHECK(hipMemcpyAsync(
        dst.data<dst_T>(),
        src.data<src_T>(),
        numel * sizeof(dst_T),
        hipMemcpyDeviceToDevice,
        copy_stream));
  } else {
    // Non-contiguous copy or a type-conversion copy

    // We avoid creating temporary memory copies if possible.
    // If both src and dst are on the same device, or if they are on
    // different devices and p2p access is enabled, perform the copy
    // by a pointwise copy kernel.
    // Otherwise, we'll have to make contiguous (which will in fact
    // invoke copy() again), and then perform the copy.
    // FIXME: might want to consider only running the pointwise kernel
    // if both src and dst innermost dimensions are contiguous. If
    // they are not, then taking the hit of the memory allocation/free
    // might be worth it to avoid non-coalesced reads or writes.
    if (p2pEnabled) {
      CopyOp<dst_T, src_T>::apply(dst, src);
    } else {
      // GPUs can't access each other directly, but the tensors
      // involved are non-contiguous and/or are different types.

      // Make sure the src is contiguous and in the same type as dst
      Tensor src_contig;
      if (same_type) {
        src_contig = src.contiguous();
      } else {
        // Types are different
        // Copy into the new format, contiguous, on the source device
        src_contig = at::empty_like(dst, src.options().dtype(dst.dtype()));

        CopyOp<dst_T, src_T>::apply(src_contig, src);
      }

      // Make sure the dst is contiguous
      device_guard.set_device(dst_device);
      Tensor dst_contig = dst.contiguous();

      // Now, we are ready for a cross-device memcpy of contiguous
      // data, of the same layout and type
      device_guard.set_device(src_device);

      AT_CUDA_CHECK(hipMemcpyAsync(
          dst_contig.data<dst_T>(),
          src_contig.data<dst_T>(),
          numel * sizeof(dst_T),
          hipMemcpyDeviceToDevice,
          copy_stream));

      if (!dst.is_contiguous()) {
        copy_device_to_device<dst_T, dst_T>(dst, dst_contig);
      }
    }
  }

  if (src_device != dst_device) {
    // dst waits on src barrier (dst already waits on dst). We cannot
    // operate on dst's copy until the copy is complete.

    // Still on src_device, record stream event
    CUDAEvent src_ready;
    src_ready.record(copy_stream);

    device_guard.set_device(dst_device);
    src_ready.block(getCurrentCUDAStream(dst_device.index()));
  }

  AT_CUDA_CHECK(hipGetLastError());
}

void copy_from_cpu(Tensor& dst, const Tensor& src) {
  Tensor dst_contig = dst.contiguous();
  Tensor src_contig = src.contiguous();

  CUDAStream stream = getCurrentCUDAStream();

  AT_CUDA_CHECK(hipMemcpyAsync(
      dst_contig.data_ptr(),
      src_contig.data_ptr(),
      src.numel() * src.element_size(),
      hipMemcpyHostToDevice,
      stream));
  AT_CUDA_CHECK(hipStreamSynchronize(stream));
  AT_DISPATCH_ALL_TYPES_AND2(at::ScalarType::Half, at::ScalarType::Bool, src.scalar_type(), "copy_from_cpu", [&]() {
    copy_device_to_device<scalar_t, scalar_t>(dst, dst_contig);
  });
}

void copy_to_cpu(Tensor& dst, const Tensor& src) {
  Tensor dst_contig = dst.contiguous();
  Tensor src_contig = src.contiguous();

  CUDAGuard device_guard(src.device());
  CUDAStream stream = getCurrentCUDAStream();

  AT_CUDA_CHECK(hipMemcpyAsync(
      dst_contig.data_ptr(),
      src_contig.data_ptr(),
      src.numel() * src.element_size(),
      hipMemcpyDeviceToHost,
      stream));
  AT_CUDA_CHECK(hipStreamSynchronize(stream));
  _copy_same_type_(dst, dst_contig);
}

void copy_from_cpu_async_(Tensor& dst, const Tensor& src) {
  AT_CHECK(dst.is_contiguous(), "Target tensor must be contiguous.");
  AT_CHECK(src.is_contiguous(), "Source tensor must be contiguous.");

  if (dst.numel() == 0) {
    return;
  }

  CUDAGuard device_guard(dst.device());
  CUDAStream stream = getCurrentCUDAStream();

  AT_DISPATCH_ALL_TYPES_AND2(at::ScalarType::Half, at::ScalarType::Bool, src.scalar_type(), "copy_from_cpu_async", [&]() {
    AT_CUDA_CHECK(hipMemcpyAsync(
        dst.data<scalar_t>(),
        src.data<scalar_t>(),
        src.numel() * sizeof(scalar_t),
        hipMemcpyHostToDevice,
        stream));
    AT_CUDA_CHECK(THCCachingHostAllocator_recordEvent(
        src.storage().data<scalar_t>(), stream));
  });
}

void copy_to_cpu_async_(Tensor& dst, const Tensor& src) {
  AT_CHECK(dst.is_contiguous(), "Target tensor must be contiguous.");
  AT_CHECK(src.is_contiguous(), "Source tensor must be contiguous.");

  if (dst.numel() == 0) {
    return;
  }

  CUDAGuard device_guard(src.device());
  CUDAStream stream = getCurrentCUDAStream();

  AT_DISPATCH_ALL_TYPES_AND2(at::ScalarType::Half, at::ScalarType::Bool, src.scalar_type(), "copy_to_cpu_async", [&]() {
    AT_CUDA_CHECK(hipMemcpyAsync(
        dst.data<scalar_t>(),
        src.data<scalar_t>(),
        src.numel() * sizeof(scalar_t),
        hipMemcpyDeviceToHost,
        stream));
    AT_CUDA_CHECK(THCCachingHostAllocator_recordEvent(
        src.storage().data<scalar_t>(), stream));
  });
}

template <typename dst_T>
void _copy__cuda(Tensor& dst, const Tensor& src, bool non_blocking) {
  AT_CHECK(dst.numel() == src.numel(), "sizes do not match");
  AT_DISPATCH_ALL_TYPES_AND2(at::ScalarType::Half, at::ScalarType::Bool, src.scalar_type(), "_copy__cuda", [&]() {
    if (dst.is_cuda() && src.is_cuda()) {
      copy_device_to_device<dst_T, scalar_t>(dst, src);
    } else if (dst.is_cuda()) {
      if (std::is_same<dst_T, scalar_t>::value) {
        if (non_blocking) {
          copy_from_cpu_async_(dst, src);
        } else {
          copy_from_cpu(dst, src);
        }
      } else {
        // Do a dtype converting copy on the CPU, then copy to device
        Tensor srcf = at::empty_like(src, src.options().dtype(dst.dtype()));
        s_copy_(srcf, src);
        copy_from_cpu(dst, srcf);
      }
    } else {
      if (std::is_same<dst_T, scalar_t>::value) {
        if (non_blocking) {
          copy_to_cpu_async_(dst, src);
        } else {
          copy_to_cpu(dst, src);
        }
      } else {
        // Copy to CPU as the same dtype, then do a dtype converting copy
        Tensor srcf = at::empty_like(src, dst.options().dtype(src.dtype()));
        copy_to_cpu(srcf, src);
        s_copy_(dst, srcf);
      }
    }
  });
}

} // namespace

namespace at {
namespace native {

Tensor& _s_copy__cuda(Tensor& self, const Tensor& src, bool non_blocking) {
  AT_DISPATCH_ALL_TYPES_AND2(at::ScalarType::Half, at::ScalarType::Bool, self.scalar_type(), "_copy__cuda", [&]() {
    ::_copy__cuda<scalar_t>(self, src, non_blocking);
  });
  return self;
}

Tensor _s_copy_from_cuda(
    const Tensor& self,
    const Tensor& dst,
    bool non_blocking) {
  Tensor dst_ = dst;
  _s_copy__cuda(dst_, self);
  return dst;
}

} // namespace native
} // namespace at

#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/TensorUtils.h>
#include <ATen/NativeFunctions.h>
#include <ATen/WrapDimUtils.h>
#include <THC/THCTensorMathReduce.cuh>
#include <THC/THCTensorSort.cuh>
#include <THC/THCThrustAllocator.cuh>

#include <ATen/AccumulateType.h>
#include <ATen/cuda/NumericLimits.cuh>
#include <type_traits>

namespace at {
namespace native {

namespace {

template<typename T, typename AccumT, typename OutT>
struct LogSoftMaxForwardEpilogue {
  __device__ __forceinline__ LogSoftMaxForwardEpilogue(AccumT max_input, AccumT sum)
    : logsum(max_input + std::log(sum)) {}

  __device__ __forceinline__ OutT operator()(T input) const {
    return static_cast<OutT>(input - logsum);
}

  const AccumT logsum;
};

template<typename T, typename AccumT, typename OutT>
struct LogSoftMaxBackwardEpilogue {
  __device__ __forceinline__ LogSoftMaxBackwardEpilogue(AccumT sum)
    : sum(sum) {}

  __device__ __forceinline__ T operator()(OutT gradOutput, OutT output) const {
    return static_cast<T>(gradOutput - std::exp(static_cast<AccumT>(output)) * sum);
  }

  const AccumT sum;
};

template<typename T, typename AccumT, typename OutT>
struct SoftMaxForwardEpilogue {
  __device__ __forceinline__ SoftMaxForwardEpilogue(AccumT max_input, AccumT sum)
    : max_input(max_input)
    , sum(sum) {}

  __device__ __forceinline__ OutT operator()(T input) const {
    return static_cast<OutT>(std::exp(input - max_input) / sum);
  }

  const AccumT max_input;
  const AccumT sum;
};

template<typename T, typename AccumT, typename OutT>
struct SoftMaxBackwardEpilogue {
  __device__ __forceinline__ SoftMaxBackwardEpilogue(AccumT sum)
    : sum(sum) {}

  // XXX: gradOutput that we get here is really gradOutput * output
  // Look for cmul in SoftMax_updateGradInput
  __device__ __forceinline__ T operator()(OutT gradOutput, OutT output) const {
    return static_cast<T>(gradOutput - output * sum);
  }

  const AccumT sum;
};




////////////////////////////////////////////////////////////////////////////////
// Spatial kernel (fast with large inner_size and small dim_size)
////////////////////////////////////////////////////////////////////////////////
// Let's assume that our input has been flattened to have only three dimension:
//     outer x dim x inner
// The spatial algorithm tries to paralellize along all of them.
// Within a 2d block threadIdx.y paralellizes over dim slices, and threads that
// share it will speed up reductions over dim (along axis x).
// The 2d grid is used to paralellize inner dimension over y axis and outer over x.
inline dim3 SpatialSoftMax_getGridSize(
    dim3 block, uint32_t max_active_blocks,
    uint64_t outer_size, uint64_t dim_size, uint64_t inner_size) {
  // First, tile as many blocks as we can over the y axis
  uint32_t inner_blocks = (inner_size + block.y - 1) / block.y;
  if (inner_blocks > max_active_blocks)
    inner_blocks = max_active_blocks;
  // Fill the x axis with as many blocks as we can fit (a little more is ok too)
  uint32_t outer_blocks = (max_active_blocks + inner_blocks - 1) / inner_blocks;
  if (outer_blocks > outer_size)
    outer_blocks = outer_size;
  return dim3(outer_blocks, inner_blocks);
}

const int max_threads = 1024;

inline dim3 SpatialSoftMax_getBlockSize(
  uint64_t outer_size, uint64_t dim_size, uint64_t inner_size) {
  uint32_t inner_threads = inner_size;
  inner_threads = std::min(inner_threads, static_cast<uint32_t>(max_threads));
  uint32_t dim_threads = 1;
  if (inner_threads <= 64 && dim_size >= 64) {
    while (inner_threads * dim_threads <= max_threads && dim_threads <= dim_size)
      dim_threads *= 2;
    dim_threads /= 2;
  }
  return dim3(dim_threads, inner_threads);
}


template<typename accscalar_t, typename Kernel>
void SpatialSoftMax_getLaunchSizes(
    Kernel k,
    uint64_t outer_size, uint64_t dim_size, uint64_t inner_size,
    dim3& grid, dim3& block, uint32_t& smem_size) {
  block = SpatialSoftMax_getBlockSize(outer_size, dim_size, inner_size);
  uint32_t block_threads = block.x * block.y;
  smem_size = block.x == 1 ? 0 : block_threads * sizeof(accscalar_t);
  int max_active_blocks;
#ifdef __HIP_PLATFORM_HCC__
  max_active_blocks = 16;
#else
  hipOccupancyMaxActiveBlocksPerMultiprocessor(&max_active_blocks,
                                                k, block_threads, smem_size);
#endif
  max_active_blocks *= at::cuda::getCurrentDeviceProperties()->multiProcessorCount;
  grid = SpatialSoftMax_getGridSize(block, max_active_blocks, outer_size, dim_size, inner_size);
}

inline dim3 SoftMax_getBlockSize(int ILP, uint64_t dim_size) {
  uint64_t block_size = 1;
  uint64_t max_block_size = std::min(dim_size / ILP, static_cast<uint64_t>(max_threads));
  while (block_size < max_block_size) block_size *= 2;
  // Launch at least a single warp - the kernel assumes that.
  block_size = std::max(block_size, static_cast<uint64_t>(32));
  return dim3(block_size);
}

template<typename T>
struct Add {
  __device__ __forceinline__ T operator()(T a, T b) const {
    return a + b;
  }
};

template<typename T>
struct Max {
  __device__ __forceinline__ T operator()(T a, T b) const {
    return a < b ? b : a;
  }
};

// Note that it's not a complete block-wide reduction.
// Only threads that share threadIdx.y reduce values.
template<typename T, template<typename> class ReduceOp>
__forceinline__ __device__
T spatialBlockReduceX(T *shared, T val) {
  ReduceOp<T> r;
  shared += threadIdx.y * blockDim.x;

  __syncthreads();

  shared[threadIdx.x] = val;

  // NOTE: loop starts with __syncthreads()
  int offset = blockDim.x / 2;
  while (offset > 0) {
    __syncthreads();
    if (threadIdx.x < offset)
      shared[threadIdx.x] = r(shared[threadIdx.x], shared[threadIdx.x + offset]);
    offset /= 2;
  }

  __syncthreads();

  return shared[0];
}

template <typename scalar_t, typename accscalar_t, typename outscalar_t, template<typename, typename, typename> class Epilogue>
__global__ void cunn_SpatialSoftMaxForward(
    outscalar_t *output, scalar_t *input,
    uint32_t outer_size, uint32_t dim_size, uint32_t inner_size)
{
  extern __shared__ unsigned char smem[];
  auto sdata = reinterpret_cast<accscalar_t*>(smem);
  const uint32_t outer_stride = inner_size * dim_size;
  const uint32_t dim_stride = inner_size;

  for (uint32_t outer_index = blockIdx.x; outer_index < outer_size; outer_index += gridDim.x) {
    const uint32_t outer_offset = outer_index * outer_stride;
    for (uint32_t inner_index = blockIdx.y * blockDim.y + threadIdx.y; inner_index < inner_size; inner_index += blockDim.y * gridDim.y) {
      const uint32_t data_offset = outer_offset + inner_index;
      ////////////////////////////////////////////////////////////
      // These two blocks are really eqivalent, but specializing on
      // blockDim.x == 1 makes the kernel faster when it's unused.
      // I didn't want to thread an extra template parameter, and nvcc
      // seems to be smart enough to hoist the if outside of the loops.
      ////////////////////////////////////////////////////////////

      if (blockDim.x > 1) {
        accscalar_t max_input = at::numeric_limits<accscalar_t>::lowest();
        for (uint32_t d = threadIdx.x; d < dim_size; d += blockDim.x) {
          const accscalar_t value = static_cast<accscalar_t>(input[data_offset + d * dim_stride]);
          max_input = Max<accscalar_t>()(max_input, value);
        }
        max_input = spatialBlockReduceX<accscalar_t, Max>(sdata,max_input);

        accscalar_t sum = 0;
        for (uint32_t d = threadIdx.x; d < dim_size; d += blockDim.x)
          sum += std::exp(static_cast<accscalar_t>(input[data_offset + d * dim_stride])
                 - max_input);
        sum = spatialBlockReduceX<accscalar_t, Add>(sdata, sum);

        Epilogue<scalar_t, accscalar_t, outscalar_t> epilogue(max_input, sum);
        for (uint32_t d = threadIdx.x; d < dim_size; d += blockDim.x)
          output[data_offset + d * dim_stride] = epilogue(input[data_offset + d * dim_stride]);
      } else {
        accscalar_t max_input = at::numeric_limits<accscalar_t>::lowest();
        for (uint32_t d = threadIdx.x; d < dim_size; d += blockDim.x) {
          const accscalar_t value = static_cast<accscalar_t>(input[data_offset + d * dim_stride]);
          max_input = Max<accscalar_t>()(max_input, value);
        }
        accscalar_t sum = 0;
        for (uint32_t d = threadIdx.x; d < dim_size; d += blockDim.x)
          sum += std::exp(static_cast<accscalar_t>(input[data_offset + d * dim_stride])
                 - max_input);
        Epilogue<scalar_t, accscalar_t, outscalar_t> epilogue(max_input, sum);
        for (uint32_t d = threadIdx.x; d < dim_size; d += blockDim.x)
          output[data_offset + d * dim_stride] = epilogue(input[data_offset + d * dim_stride]);
      }
    }
  }
}



template <typename scalar_t, typename accscalar_t, typename outscalar_t, template<typename, typename, typename> class Epilogue>
__global__ void cunn_SpatialSoftMaxBackward(
    scalar_t *gradInput, outscalar_t *output, outscalar_t *gradOutput,
    uint32_t outer_size, uint32_t dim_size, uint32_t inner_size)
{
  extern __shared__ unsigned char smem[];
  auto sdata = reinterpret_cast<accscalar_t*>(smem);
  const uint32_t outer_stride = inner_size * dim_size;
  const uint32_t dim_stride = inner_size;

  for (uint32_t outer_index = blockIdx.x; outer_index < outer_size; outer_index += gridDim.x) {
    const uint32_t outer_offset = outer_index * outer_stride;
    for (uint32_t inner_index = blockIdx.y * blockDim.y + threadIdx.y; inner_index < inner_size; inner_index += blockDim.y * gridDim.y) {
      const uint32_t data_offset = outer_offset + inner_index;
      // See the comment in forward kernel
      if (blockDim.x > 1) {
        accscalar_t sum = 0;
        for (uint32_t d = threadIdx.x; d < dim_size; d += blockDim.x)
          sum += gradOutput[data_offset + d * dim_stride];
        sum = spatialBlockReduceX<accscalar_t, Add>(sdata, sum);

        Epilogue<scalar_t, accscalar_t, outscalar_t> epilogue(sum);
        for (uint32_t d = threadIdx.x; d < dim_size; d += blockDim.x) {
          gradInput[data_offset + d * dim_stride] =
            epilogue(gradOutput[data_offset + d * dim_stride],
                    output[data_offset + d * dim_stride]);
        }
      } else {
        accscalar_t sum = 0;
        for (uint32_t d = 0; d < dim_size; d++)
          sum += gradOutput[data_offset + d * dim_stride];

        Epilogue<scalar_t, accscalar_t, outscalar_t> epilogue(sum);
        for (uint32_t d = 0; d < dim_size; d++) {
          gradInput[data_offset + d * dim_stride] =
            epilogue(gradOutput[data_offset + d * dim_stride],
                    output[data_offset + d * dim_stride]);
        }
      }
    }
  }
}


////////////////////////////////////////////////////////////////////////////////
// Regular kernel (fast when dim_size is large; requires inner_size == 1)
////////////////////////////////////////////////////////////////////////////////


template <typename T, typename AccumT>
struct MaxFloat
{
  __device__ __forceinline__ AccumT operator()(AccumT max, T v) const {
    return ::max(max, (AccumT)v);
  }
};

template<typename T, typename AccumT>
struct AddFloat
{
  __device__ __forceinline__ AccumT operator()(AccumT sum, T v) const {
    return sum + v;
  }
};

template<typename T, typename AccumT>
struct SumExpFloat
{
  __device__ __forceinline__ SumExpFloat(AccumT v)
    : max_k(v) {}

  __device__ __forceinline__ AccumT operator()(AccumT sum, T v) const {
    return sum + std::exp(v - max_k);
  }

  const AccumT max_k;
};

template <template<typename> class Reduction, typename AccumT>
__device__ __forceinline__ AccumT
blockReduce(AccumT* smem, AccumT val,
            const Reduction<AccumT>& r,
            AccumT defaultVal)
{
  // To avoid RaW races from chaining blockReduce calls together, we need a sync here
  __syncthreads();

  smem[threadIdx.x] = val;

  __syncthreads();

  AccumT warpVal = defaultVal;

  // First warp will perform per-warp reductions for the remaining warps
  if (threadIdx.x < 32) {
    int lane = threadIdx.x % 32;
    if (lane < blockDim.x / 32) {
#pragma unroll
      for (int i = 0; i < 32; ++i) {
        warpVal = r(warpVal, smem[lane * 32 + i]);
      }
      smem[lane] = warpVal;
    }
  }

  __syncthreads();

  // First thread will perform a reduction of the above per-warp reductions
  AccumT blockVal = defaultVal;

  if (threadIdx.x == 0) {
    for (int i = 0; i < blockDim.x / 32; ++i) {
      blockVal = r(blockVal, smem[i]);
    }
    smem[0] = blockVal;
  }

  // Sync and broadcast
  __syncthreads();
  return smem[0];
}

template <template<typename, typename> class Reduction, int ILP, typename T, typename AccumT>
__device__ __forceinline__ AccumT
ilpReduce(T* data,
          int size,
          const Reduction<T, AccumT>& r,
          AccumT defaultVal)
{
  AccumT threadVal = defaultVal;
  int offset = threadIdx.x;

  int last = size % (ILP * blockDim.x);

  // Body (unroll by ILP times)
  for (; offset < size - last; offset += blockDim.x * ILP) {
    T tmp[ILP];

#pragma unroll
    for (int j = 0; j < ILP; ++j)
      tmp[j] = data[offset + j * blockDim.x];

#pragma unroll
    for (int j = 0; j < ILP; ++j)
      threadVal = r(threadVal, tmp[j]);
  }

  // Epilogue
  for (; offset < size; offset += blockDim.x)
    threadVal = r(threadVal, data[offset]);

  return threadVal;
}

template <int ILP, typename scalar_t, typename accscalar_t, typename outscalar_t, template <typename, typename, typename> class Epilogue>
__global__ void
cunn_SoftMaxForward(outscalar_t *output, scalar_t *input, int classes)
{
  extern __shared__ unsigned char smem[];
  auto sdata = reinterpret_cast<accscalar_t*>(smem);
  // forward pointers to batch[blockIdx.x]
  // each block handles a sample in the mini-batch
  input += blockIdx.x * classes;
  output += blockIdx.x * classes;

  // find the max
  accscalar_t threadMax = ilpReduce<MaxFloat, ILP, scalar_t, accscalar_t>(
      input, classes, MaxFloat<scalar_t, accscalar_t>(), -at::numeric_limits<accscalar_t>::max());
  accscalar_t max_k = blockReduce<Max, accscalar_t>(
      sdata, threadMax, Max<accscalar_t>(), -at::numeric_limits<accscalar_t>::max());

  // reduce all values
  accscalar_t threadExp = ilpReduce<SumExpFloat, ILP, scalar_t, accscalar_t>(
      input, classes, SumExpFloat<scalar_t, accscalar_t>(max_k), static_cast<accscalar_t>(0));
  accscalar_t sumAll = blockReduce<Add, accscalar_t>(
      sdata, threadExp, Add<accscalar_t>(), static_cast<accscalar_t>(0));

  Epilogue<scalar_t, accscalar_t, outscalar_t> epilogue(max_k, sumAll);
  int offset = threadIdx.x;
  int last = classes % (ILP * blockDim.x);
  for (; offset < classes - last; offset += blockDim.x * ILP) {
    scalar_t tmp[ILP];

#pragma unroll
    for (int j = 0; j < ILP; ++j)
      tmp[j] = input[offset + j * blockDim.x];

#pragma unroll
    for (int j = 0; j < ILP; ++j)
      output[offset + j * blockDim.x] = epilogue(tmp[j]);
  }

  for (; offset < classes; offset += blockDim.x)
    output[offset] = epilogue(input[offset]);
}

template <int ILP, typename scalar_t, typename accscalar_t, typename outscalar_t, template<typename, typename, typename> class Epilogue>
__global__ void
cunn_SoftMaxBackward(scalar_t *gradInput, outscalar_t *output, outscalar_t *gradOutput, int classes)
{
  extern __shared__ unsigned char smem[];
  auto sdata = reinterpret_cast<accscalar_t*>(smem);
  gradInput += blockIdx.x * classes;
  output += blockIdx.x * classes;
  gradOutput += blockIdx.x * classes;

  accscalar_t threadSum = ilpReduce<AddFloat, 4, outscalar_t, accscalar_t>(
      gradOutput, classes, AddFloat<outscalar_t, accscalar_t>(), accscalar_t(0));
  accscalar_t sum_k = blockReduce<Add, accscalar_t>(
        sdata, threadSum, Add<accscalar_t>(), accscalar_t(0));

  Epilogue<scalar_t, accscalar_t, outscalar_t> epilogue(sum_k);
  int offset = threadIdx.x;
  int last = classes % (ILP * blockDim.x);
  for (; offset < classes - last; offset += blockDim.x * ILP) {
    outscalar_t tmpGradOutput[ILP];
    outscalar_t tmpOutput[ILP];

#pragma unroll
    for (int j = 0; j < ILP; ++j) {
      tmpGradOutput[j] = gradOutput[offset + j * blockDim.x];
      tmpOutput[j] = output[offset + j * blockDim.x];
    }

#pragma unroll
    for (int j = 0; j < ILP; ++j)
      gradInput[offset + j * blockDim.x] = epilogue(tmpGradOutput[j], tmpOutput[j]);
  }

  for (; offset < classes; offset += blockDim.x)
    gradInput[offset] = epilogue(gradOutput[offset], output[offset]);
}






template<template<typename, typename, typename> class Epilogue>
Tensor host_softmax(const Tensor & input_, const int64_t dim_, const bool half_to_float){
  if (half_to_float) AT_ASSERTM(input_.scalar_type() == ScalarType::Half,"conversion is supported for Half type only");
  auto input = input_.contiguous();
  Tensor output = half_to_float ? at::empty_like(input, input.options().dtype(ScalarType::Float)) : at::empty_like(input);
  static_assert(std::is_same<acc_type<at::Half, true>, float>::value, "accscalar_t for half should be float");
  if (input.dim() == 0) input = input.view(1);
  int64_t dim = maybe_wrap_dim(dim_, input.dim());
  AT_CHECK(dim >=0 && dim < input.dim(), "dim must be non-negative and less than input dimensions");
  int64_t outer_size = 1;
  int64_t dim_size = input.size(dim);

  if (input.numel() > 0) {
    int64_t inner_size = 1;
    hipStream_t stream = at::cuda::getCurrentCUDAStream();
    for (int64_t i = 0; i < dim; ++i)
      outer_size *= input.size(i);
    for (int64_t i = dim + 1; i < input.dim(); ++i)
      inner_size *= input.size(i);
    // This kernel spawns a block per each element in the batch.
    // XXX: it assumes that inner_size == 1
    if (inner_size == 1) {
      const int ILP = 2;
      dim3 grid(outer_size);
      dim3 block = SoftMax_getBlockSize(ILP, dim_size);
      AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "host_softmax", [&] {
      using accscalar_t = acc_type<scalar_t, true>;
      if (!half_to_float) {
          cunn_SoftMaxForward<ILP, scalar_t, accscalar_t, scalar_t, Epilogue>
            <<<grid, block, block.x * sizeof(accscalar_t), stream>>>(
              output.data<scalar_t>(), input.data<scalar_t>(), dim_size
          );
      } else {
          cunn_SoftMaxForward<ILP, scalar_t, accscalar_t, accscalar_t, Epilogue>
            <<<grid, block, block.x * sizeof(accscalar_t), stream>>>(
              output.data<accscalar_t>(), input.data<scalar_t>(), dim_size
          );
      }
      });
    // This kernel runs in a 2D grid, where each application along y dimension has a fixed
    // outer_size, and runs in parallel over inner_size. Dimension x is parallel over outer_size.
    // Reductions over dim are done in a single-threaded manner.
    } else {
      uint32_t smem_size;
      dim3 grid, block;
      AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "host_softmax", [&] {
      using accscalar_t = acc_type<scalar_t, true>;
      if (!half_to_float) {
          SpatialSoftMax_getLaunchSizes<accscalar_t>(
              &cunn_SpatialSoftMaxForward<scalar_t, accscalar_t, scalar_t, Epilogue>,
              outer_size, dim_size, inner_size,
              grid, block, smem_size);
          cunn_SpatialSoftMaxForward<scalar_t, accscalar_t, scalar_t, Epilogue>
            <<<grid, block, smem_size, stream>>>(
             output.data<scalar_t>(), input.data<scalar_t>(), outer_size, dim_size, inner_size
      );
      } else {
          SpatialSoftMax_getLaunchSizes<accscalar_t>(
              &cunn_SpatialSoftMaxForward<scalar_t, accscalar_t, accscalar_t, Epilogue>,
              outer_size, dim_size, inner_size,
              grid, block, smem_size);
          cunn_SpatialSoftMaxForward<scalar_t, accscalar_t, accscalar_t, Epilogue>
            <<<grid, block, smem_size, stream>>>(
             output.data<accscalar_t>(), input.data<scalar_t>(), outer_size, dim_size, inner_size
      );
      }
      });
    }
    THCudaCheck(hipGetLastError());
  }
  return output;
}

template<template<typename, typename, typename> class Epilogue>
Tensor host_softmax_backward(const Tensor &grad_, const Tensor &output_, int64_t dim_, bool half_to_float){
  int64_t dim = maybe_wrap_dim(dim_, grad_.dim());
  Tensor gI = half_to_float ? at::empty_like(grad_, grad_.options().dtype(ScalarType::Half)) : at::empty_like(grad_);
  if (grad_.numel() == 0) {
    return gI;
  }
  auto grad = grad_.contiguous();
  static_assert(std::is_same<acc_type<at::Half, true>, float>::value, "accscalar_t for half should be float");
  if (grad.dim() == 0) grad = grad.view(1);
  AT_CHECK(dim >=0 && dim < grad.dim(), "dim must be non-negative and less than input dimensions");
  auto output = output_.contiguous();
  if (output.dim() == 0) output = output.view(1);
  int64_t outer_size = 1;
  int64_t dim_size = output.size(dim);
  int64_t inner_size = 1;
  for (int64_t i = 0; i < dim; ++i)
    outer_size *= output.size(i);
  for (int64_t i = dim + 1; i < output.dim(); ++i)
    inner_size *= output.size(i);
// See descriptions of kernels above.
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  if (inner_size == 1) {
    const int ILP = 2;
    dim3 grid(outer_size);
    dim3 block = SoftMax_getBlockSize(ILP, dim_size);
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(gI.scalar_type(), "host_softmax_backward", [&] {
    using accscalar_t = acc_type<scalar_t, true>;
    if (!half_to_float) {
        cunn_SoftMaxBackward<ILP, scalar_t, accscalar_t, scalar_t, Epilogue>
         <<<grid, block, block.x * sizeof(accscalar_t), stream>>>(
            gI.data<scalar_t>(), output.data<scalar_t>(), grad.data<scalar_t>(), dim_size
    );
    } else {
        cunn_SoftMaxBackward<ILP, scalar_t, accscalar_t, accscalar_t, Epilogue>
         <<<grid, block, block.x * sizeof(accscalar_t), stream>>>(
            gI.data<scalar_t>(), output.data<accscalar_t>(), grad.data<accscalar_t>(), dim_size
    );
    }
    });
  } else {
    uint32_t smem_size;
    dim3 grid, block;
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(grad.scalar_type(), "host_softmax_backward", [&] {
    using accscalar_t = acc_type<scalar_t, true>;
    if (!half_to_float) {
        SpatialSoftMax_getLaunchSizes<accscalar_t>(
            &cunn_SpatialSoftMaxBackward<scalar_t, accscalar_t, scalar_t, Epilogue>,
            outer_size, dim_size, inner_size,
            grid, block, smem_size);

        cunn_SpatialSoftMaxBackward<scalar_t, accscalar_t, scalar_t, Epilogue>
          <<<grid, block, smem_size, stream>>>(
            gI.data<scalar_t>(), output.data<scalar_t>(), grad.data<scalar_t>(),
            outer_size, dim_size, inner_size
        );
    } else {
        SpatialSoftMax_getLaunchSizes<accscalar_t>(
            &cunn_SpatialSoftMaxBackward<scalar_t, accscalar_t, accscalar_t, Epilogue>,
            outer_size, dim_size, inner_size,
            grid, block, smem_size);

        cunn_SpatialSoftMaxBackward<scalar_t, accscalar_t, accscalar_t, Epilogue>
          <<<grid, block, smem_size, stream>>>(
            gI.data<scalar_t>(), output.data<accscalar_t>(), grad.data<accscalar_t>(),
            outer_size, dim_size, inner_size
        );
    }
    });
  }
  THCudaCheck(hipGetLastError());
  return gI;
}
}

Tensor log_softmax_cuda(const Tensor &input, const int64_t dim, const bool half_to_float){
  return host_softmax<LogSoftMaxForwardEpilogue>(input, dim, half_to_float);
}

Tensor log_softmax_backward_cuda(const Tensor &grad, const Tensor &output, int64_t dim, const Tensor &input){
  bool half_to_float = grad.scalar_type() != input.scalar_type();
  if (half_to_float) {
     AT_ASSERTM((grad.scalar_type() == ScalarType::Float && input.scalar_type() == ScalarType::Half), "expected input and grad types to match, or input to be at::Half and grad to be at::Float");
  }
  return host_softmax_backward<LogSoftMaxBackwardEpilogue>(grad, output, dim, half_to_float);
}

Tensor softmax_cuda(const Tensor &input, const int64_t dim, const bool half_to_float){
  return host_softmax<SoftMaxForwardEpilogue>(input, dim, half_to_float);
}

Tensor softmax_backward_cuda(const Tensor &grad, const Tensor &output, int64_t dim, const Tensor &input){
  bool half_to_float = grad.scalar_type() != input.scalar_type();
  if (half_to_float) {
     AT_ASSERTM((grad.scalar_type() == ScalarType::Float && input.scalar_type() == ScalarType::Half), "expected input and grad types to match, or input to be at::Half and grad to be at::Float");
  }
  Tensor tmp = grad * output;
  return host_softmax_backward<SoftMaxBackwardEpilogue>(tmp, output, dim, half_to_float);
}

}
}

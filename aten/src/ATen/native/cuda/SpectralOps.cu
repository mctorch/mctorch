#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/Config.h>
#include <ATen/Dispatch.h>
#include <ATen/Utils.h>
#include <ATen/NativeFunctions.h>
#include <ATen/detail/CUDAHooksInterface.h>
#include <ATen/native/SpectralOpsUtils.h>
#include <ATen/native/cuda/CuFFTUtils.h>
#include <ATen/native/cuda/CuFFTPlanCache.h>
#include <THC/THCTensorSort.cuh>
#include <THC/THCThrustAllocator.cuh>

#include <thrust/execution_policy.h>
#include <thrust/unique.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <vector>
#include <cmath>

namespace at { namespace native {

using namespace at::native::detail;

// In real-to-complex transform, cuFFT only fills half of the values due to
// conjugate symmetry. See native/SpectralUtils.h for more details.
// The following structs are used to fill in the other half with symmetry in
// case of real-to-complex transform with onesided=False flag.
// See NOTE [ Fourier Transform Conjugate Symmetry ] in native/SpectralOpsUtils.h.

// counting_iterator => index to fill
struct cnt_to_dst_idx_functor : public thrust::unary_function<int64_t, int64_t>
{
  int64_t last_dim_size;
  int64_t last_dim_start_slice;
  int64_t last_dim_to_fill_size;

  cnt_to_dst_idx_functor(int64_t last_dim_size, int64_t last_dim_start_slice) :
    last_dim_size(last_dim_size), last_dim_start_slice(last_dim_start_slice),
    last_dim_to_fill_size(last_dim_size - last_dim_start_slice) {}

  // HIP wants __host__ __device__ tag, CUDA does not
#ifdef __HIP_PLATFORM_HCC__
  __host__ __device__
#endif
  cnt_to_dst_idx_functor & operator=(const cnt_to_dst_idx_functor&) = default;

  __host__ __device__ __forceinline__
  int64_t operator()(const int64_t& i) const
  {
    int64_t imag = i % 2;
    int64_t idx = i / 2;
    int64_t num_dim = idx / last_dim_to_fill_size;
    int64_t slice_idx = idx % last_dim_to_fill_size;
    return (num_dim * last_dim_size + last_dim_start_slice + slice_idx) * 2 + imag;
  }
};

// index to fill => index to read from
template <typename scalar_t>
struct dst_idx_to_src_functor : public thrust::unary_function<int64_t, scalar_t>
{
  // output can have at most dim 5 (batch + 3 signal dim + real/imag)
  int64_t sizes[max_rank + 2], strides[max_rank + 2];
  const int64_t signal_ndim;
  scalar_t *data;  // device ptr

  dst_idx_to_src_functor(const Tensor& batched_complex_signal)
    : signal_ndim(batched_complex_signal.dim() - 1),
      data(batched_complex_signal.data<scalar_t>()) {
    for (int64_t i = 0; i < signal_ndim; i++) {
      sizes[i] = batched_complex_signal.size(i);
      strides[i] = batched_complex_signal.stride(i);
    }
  }

  __device__ __forceinline__
  scalar_t operator()(const int64_t& write_idx_with_imag) const
  {
    int64_t imag = write_idx_with_imag % 2;
    // all but first (batch) and last (real/imag) dims need to be reflected
    int64_t read_idx = 0;
    int64_t remainder = write_idx_with_imag - imag;
    int64_t dim_idx, dim_stride;
    for (int64_t i = 0; i < signal_ndim; i++) {
      dim_stride = strides[i];
      dim_idx = remainder / dim_stride;
      if (i == 0) {
        read_idx += dim_idx * dim_stride;
      } else if (dim_idx != 0) {
        read_idx += (sizes[i] - dim_idx) * dim_stride;
      }
      remainder = remainder % dim_stride;
    }
    if (imag) {
      return -data[read_idx + 1];
    } else {
      return data[read_idx];
    }
  }
};

// input should be a contiguous batched tensor of same size as full (twosided)
// signals, but only contains half (onesided) of the values.
// This function modifies inplace.
__forceinline__
static void _fft_fill_with_conjugate_symmetry_(Tensor& input,
                      int64_t size_last_dim, int64_t last_dim_start_slice) {
  if (last_dim_start_slice >= size_last_dim) {
    return;
  }

  // copy
  int64_t n = input.numel() / size_last_dim * (size_last_dim - last_dim_start_slice);

  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  auto allocator = THCThrustAllocator(globalContext().lazyInitCUDA());
  auto policy = thrust::cuda::par(allocator).on(stream);
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "_fft_fill_with_conjugate_symmetry_", [&] {
    typedef thrust::device_ptr<scalar_t> device_ptr;
    typedef thrust::counting_iterator<int64_t> counter;
    typedef thrust::transform_iterator<cnt_to_dst_idx_functor, counter> dst_idx_iterator;
    typedef thrust::permutation_iterator<device_ptr, dst_idx_iterator> dst_iterator;
    typedef thrust::transform_iterator<dst_idx_to_src_functor<scalar_t>, dst_idx_iterator> src_iterator;

    dst_idx_iterator dst_idxs(counter(0), cnt_to_dst_idx_functor(size_last_dim, last_dim_start_slice));

    auto data = device_ptr(input.data<scalar_t>());
    dst_iterator dsts(data, dst_idxs);
    src_iterator srcs(dst_idxs, dst_idx_to_src_functor<scalar_t>(input));
    thrust::copy_n(policy, srcs, n, dsts);
  });
}

// NOTE [ cuFFT Embedded Strides ]
//
// cuFFT supports a subset of arbitrary strides via their "advanced data layout"
// option (http://docs.nvidia.com/cuda/cufft/index.html#advanced-data-layout).
// Specifically, these are tensors that can be viewed as subtensors resulted
// from slicing a larger contiguous tensors. For such input tensors, let the
// sizes of the enclosing tensor be `inembed`, and we can have in 3d case:
//
//     input[x, y, z] = input[((x * inembed[1] + y) * inembed[2] + z)]
//
// Above is the simplified formula ignoring the batch dimension. In fact, the
// last dimension of the enclosing tensor doesn't have to be contiguous, i.e.,
// it can be greater than 1. Then one can set the base stride for the enclosing
// tensor with `istride`. Then we have
//
//     input[x, y, z] = input[((x * inembed[1] + y) * inembed[2] + z) * istride]
//
// For example, consider
//
//     enclosing = torch.zeros(6, 8, 10)  # contiguous
//     input = enclosing[:4, 2:6, 6:]
//     input.size()                       # [ 4,  4,  4]
//     input.stride()                     # [80, 10,  1]
//     # inembed = [6, 8, 10]
//     input[2, 1, 3] = input[((2 * 8) + 1) * 10 + 3]   # using above formula
//                    = input[173]
//                    = input[2 * 80 + 1 * 10 + 1 * 3]  # using strides directly
//
// Generally, the embedded strides can be computed as
//
//     embed[i] = stride[i - 1] / stride[i].
//
// Note that the value of embed[0] isn't used to compute indices and doesn't
// matter.
//
// Contrary to advanced data layout, simple layout means that *embeds have
// unit-strides. In particular, unit-stride refers to that the input and output
// tensors being contiguous, and that the strides at the innermost signal
// dimension being unit (1) w.r.t. the corresponding data type.

static inline Tensor _run_cufft(
    const CuFFTConfig &config, Tensor& input, int64_t signal_ndim,
    bool complex_input, bool complex_output, bool inverse,
    IntArrayRef checked_signal_sizes, bool normalized, bool onesided,
    IntArrayRef output_sizes, bool input_was_cloned
) {
  if (config.should_clone_input() && !input_was_cloned) {
    input = input.clone();
  }

  auto& plan = config.plan();
  auto& ctx = at::globalContext();

  // set output
  auto output = at::empty(output_sizes, input.options());

  // set to current stream
  CUFFT_CHECK(hipfftSetStream(plan, at::cuda::getCurrentCUDAStream()));

  auto ws = at::empty({ config.workspace_size() }, at::device(at::kCUDA).dtype(at::kByte));
  CUFFT_CHECK(hipfftSetWorkArea(plan, ws.data_ptr()));

  // run
#ifdef __HIP_PLATFORM_HCC__
  if (input.scalar_type() == ScalarType::Float) {
      if (complex_input && complex_output) {
        CUFFT_CHECK(hipfftExecC2C(plan, static_cast<hipfftComplex*>(input.data_ptr()),
          static_cast<hipfftComplex*>(output.data_ptr()),
          inverse ? HIPFFT_BACKWARD : HIPFFT_FORWARD));
      } else if (complex_input && !complex_output) {
        CUFFT_CHECK(hipfftExecC2R(plan, static_cast<hipfftComplex*>(input.data_ptr()),
          static_cast<hipfftReal*>(output.data_ptr())));
      } else if (!complex_input && complex_output) {
        CUFFT_CHECK(hipfftExecR2C(plan, static_cast<hipfftReal*>(input.data_ptr()),
          static_cast<hipfftComplex*>(output.data_ptr())));
      } else {
        AT_ERROR("hipFFT doesn't support r2r (float)");
      }
    } else if (input.scalar_type() == ScalarType::Double) {
      if (complex_input && complex_output) {
        CUFFT_CHECK(hipfftExecZ2Z(plan, static_cast<hipfftDoubleComplex*>(input.data_ptr()),
          static_cast<hipfftDoubleComplex*>(output.data_ptr()),
          inverse ? HIPFFT_BACKWARD : HIPFFT_FORWARD));
      } else if (complex_input && !complex_output) {
        CUFFT_CHECK(hipfftExecZ2D(plan, static_cast<hipfftDoubleComplex*>(input.data_ptr()),
          static_cast<hipfftDoubleReal*>(output.data_ptr())));
      } else if (!complex_input && complex_output) {
        CUFFT_CHECK(hipfftExecD2Z(plan, static_cast<hipfftDoubleReal*>(input.data_ptr()),
          static_cast<hipfftDoubleComplex*>(output.data_ptr())));
      } else {
        AT_ERROR("hipFFT doesn't support r2r (double)");
      }
    } else {
      std::ostringstream ss;
      ss << "hipFFT doesn't support tensor of type: "
         << toString(input.scalar_type());
      AT_ERROR(ss.str());
    }
#else
  CUFFT_CHECK(hipfftXtExec(plan, input.data_ptr(), output.data_ptr(),
    inverse ? HIPFFT_BACKWARD : HIPFFT_FORWARD));
#endif

  // rescale if needed by normalized flag or inverse transform
  auto size_last_signal_dim = checked_signal_sizes[signal_ndim - 1];
  if (normalized || inverse) {
    auto signal_numel = at::prod_intlist(checked_signal_sizes);
    double scale_denom;
    if (normalized) {
      scale_denom = std::sqrt(static_cast<double>(signal_numel));
    } else {
      scale_denom = static_cast<double>(signal_numel);
    }
    if (!complex_input && complex_output && !onesided) {
      auto end_data_slice = infer_ft_real_to_complex_onesided_size(size_last_signal_dim);
      output.narrow(signal_ndim, 0, end_data_slice).div_(scale_denom);
    } else {
      output.div_(scale_denom);
    }
  }

  // if needed, fill out the other half using conjugate symmetry
  if (!complex_input && complex_output && !onesided) {
    auto start_slice = infer_ft_real_to_complex_onesided_size(size_last_signal_dim);
    _fft_fill_with_conjugate_symmetry_(output, size_last_signal_dim, start_slice);
  }
  return output;
}

// The cuFFT plan cache, defined in CuFFTUtils.h
std::vector<optional<CuFFTParamsLRUCache>> plan_caches;
std::mutex plan_caches_mutex;

static inline
CuFFTParamsLRUCache &cufft_get_plan_cache(int64_t device_index) {
  std::lock_guard<std::mutex> guard(plan_caches_mutex);

  AT_ASSERT(device_index >= 0);

  if (device_index >= plan_caches.size()) {
    plan_caches.resize(device_index + 1);
  }

  if (!plan_caches[device_index]) {
    plan_caches[device_index].emplace();
  }

  return *plan_caches[device_index];
}


namespace detail {

int64_t cufft_get_plan_cache_max_size_impl(int64_t device_index) {
  AT_CHECK(0 <= device_index && device_index < at::detail::getCUDAHooks().getNumGPUs(),
    "cufft_get_plan_cache_max_size: expected 0 <= device_index < ",
    at::detail::getCUDAHooks().getNumGPUs(), "], but got device_index=",
    device_index);
  return cufft_get_plan_cache(device_index).max_size();
}

void cufft_set_plan_cache_max_size_impl(int64_t device_index, int64_t max_size) {
  AT_CHECK(0 <= device_index && device_index < at::detail::getCUDAHooks().getNumGPUs(),
    "cufft_set_plan_cache_max_size: expected 0 <= device_index < ",
    at::detail::getCUDAHooks().getNumGPUs(), "], but got device_index=",
    device_index);
  return cufft_get_plan_cache(device_index).resize(max_size);
}

int64_t cufft_get_plan_cache_size_impl(int64_t device_index) {
  AT_CHECK(0 <= device_index && device_index < at::detail::getCUDAHooks().getNumGPUs(),
    "cufft_get_plan_cache_size: expected 0 <= device_index < ",
    at::detail::getCUDAHooks().getNumGPUs(), "], but got device_index=",
    device_index);
  return cufft_get_plan_cache(device_index).size();
}

void cufft_clear_plan_cache_impl(int64_t device_index) {
  AT_CHECK(0 <= device_index && device_index < at::detail::getCUDAHooks().getNumGPUs(),
    "cufft_clear_plan_cache: expected 0 <= device_index < ",
    at::detail::getCUDAHooks().getNumGPUs(), "], but got device_index=",
    device_index);
  return cufft_get_plan_cache(device_index).clear();
}

} // namespace at::native::detail

// cuFFT
// Currently not utilizing multi GPUs so this can be potentially sped up.
Tensor _fft_cufft(const Tensor& self, int64_t signal_ndim,
                  bool complex_input, bool complex_output, bool inverse,
                  IntArrayRef checked_signal_sizes, bool normalized, bool onesided,
                  IntArrayRef output_sizes) {

  CuFFTParamsLRUCache& plan_cache = cufft_get_plan_cache(self.device().index());

  Tensor input = self;
  bool input_was_cloned = false;

  // Slice when twosided complex-to-real. This is not always needed because we
  // calculate the inembed. But it will benefit us in certain cases where we
  // clone the input tensor.
  //
  // See NOTE [ cuFFT Embedded Strides ].
  // See NOTE [ Fourier Transform Conjugate Symmetry ] in native/SpectralOpsUtils.h.
  if (complex_input && !complex_output && !onesided) {
    auto onesided_size = infer_ft_real_to_complex_onesided_size(checked_signal_sizes[signal_ndim - 1]);
    input = input.narrow(signal_ndim, 0, onesided_size);
  }

  // cuFFT requires input and output data pointers to complex type aligned.
  // Our newly allocated output tensor is always 512 bytes aligned so it is fine
  // (see kRoundSmall and kRoundLarge in THCCachingAllocator.cpp), but we do
  // need to check input tensor to make sure that it is not unaligned, e.g.,
  // from a slicing.
  auto complex_size_bytes = 2 * input.element_size();
  if (reinterpret_cast<std::uintptr_t>(input.data_ptr()) % complex_size_bytes != 0) {
    input = input.clone();
    input_was_cloned = true;
  }

  // Now that we have done error check and data_ptr checks, we delegate all
  // futher cuFFT parameter computation and plan creation to the helper class
  // CuFFTConfig in CuFFTUtils.h.

  // If plan caching is enabled, we check the cache. Note that this accesses
  // plan_cache.max_size() and thus makes this function less functional.
  // However, integrating additional arguments into the "public" level c++ APIs,
  // e.g., irfft, is difficult as we have a long call sequence looking like
  //   irfft --> _fft --> _fft_with_size --dispatching-to-> _fft_cufft

  // This read is not locked for perf reason. Shouldn't matter too much because
  // we check again after acquiring the lock.
  if (plan_cache.max_size() > 0) {
    CuFFTParams params;
    setCuFFTParams(&params, input, signal_ndim, complex_input,
      complex_output, checked_signal_sizes, onesided);
    std::lock_guard<std::mutex> guard(plan_cache.mutex);
    if (plan_cache.max_size() > 0) {  // check again after acquiring the lock
      const CuFFTConfig &config = plan_cache.try_emplace_value(std::move(params),
                                             input, signal_ndim, complex_input,
                                             complex_output, checked_signal_sizes,
                                             onesided, output_sizes);
      return _run_cufft(config, input, signal_ndim, complex_input,
                        complex_output, inverse, checked_signal_sizes, normalized,
                        onesided, output_sizes, input_was_cloned);
    }
  }
  CuFFTConfig config(input, signal_ndim, complex_input, complex_output,
                     checked_signal_sizes, onesided, output_sizes);
  return _run_cufft(config, input, signal_ndim, complex_input,
                    complex_output, inverse, checked_signal_sizes, normalized,
                    onesided, output_sizes, input_was_cloned);
}

}} // at::native

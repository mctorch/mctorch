#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2016-present, Facebook, Inc.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cfloat>

#include "caffe2/core/context_gpu.h"
#include "modules/detectron/group_spatial_softmax_op.h"

namespace caffe2 {

namespace {

__global__ void GroupSpatialSoftmaxKernel(const int num, const int A, const int W,
    const int H, const float* Xdata, float* Pdata, const int num_classes) {
  // Loop throuh labels (N x A x H x W)
  CUDA_1D_KERNEL_LOOP(index, num * A * H * W) {
    int D = num_classes * A;
    int x = index % W;
    int y = (index / W) % H;
    int a = (index / (W * H)) % A;
    int i = index / W / H / A;

    // Subtract max on each cell for numerical reasons
    float max_val = -FLT_MAX;
    for(int c = a * num_classes; c < (a + 1) * num_classes; ++c) {
      int idx = i * (H * W * D) +  c * (H * W) + y * W + x;
      max_val = max(max_val, Xdata[idx]);
    }
    // Exponentiate
    float expsum = 0.0f;
    for(int c = a * num_classes; c < (a + 1) * num_classes; ++c) {
      int idx = i * (H * W * D) + c * (H * W) + y * W + x;
      float expx = exp(Xdata[idx] - max_val);
      Pdata[idx] = expx;
      expsum += expx;
    }

    // Normalize
    for(int c = a * num_classes; c < (a + 1) * num_classes; ++c) {
      int idx = i * (H * W * D) + c * (H * W) + y * W + x;
      Pdata[idx] /= expsum;
    }

  }
}

__global__ void SumProbsKernel(const int N, const int A, const int W,
    const int H, const float* Ydata, const float* dYdata,
    float* sum_probs_data, const int num_classes) {
  CUDA_1D_KERNEL_LOOP(i, N * A * W * H) {
    int D = num_classes * A;
    int x = i % W;
    int y = (i / W) % H;
    int a = (i / (W * H)) % A;
    int n = i / (W * H * A);

    sum_probs_data[i] = 0.0;
    for(int c = a * num_classes; c < (a + 1) * num_classes; ++c) {
      int idx = n * (H * W * D) + c * (H * W) + y * W + x;
      sum_probs_data[i] += (Ydata[idx] * dYdata[idx]);
    }
  }
}

__global__ void SubSumKernel(
    const int N, const int A, const int W, const int H,
    const float* sum_probs_data, float* dXdata, const int num_classes) {
  CUDA_1D_KERNEL_LOOP(i, N * (A * num_classes) * W * H) {
    int D = num_classes * A;
    int x = i % W;
    int y = (i / W) % H;
    int a = ((i / (W * H)) % D) / num_classes;
    int n = i / W / H / D;
    int idx = n * (H * W * A) + a * (H * W) + y * W + x;
    dXdata[i] = (dXdata[i] - sum_probs_data[idx]);
  }
}

} // namespace


template <>
bool GroupSpatialSoftmaxOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0);  // Logits

  int N = X.dim32(0);
  int D = X.dim32(1);
  int H = X.dim32(2);
  int W = X.dim32(3);
  int A = D / num_classes_;

  auto* P = Output(0, X.sizes(), at::dtype<float>()); // Probabilities from softmax
  DCHECK_EQ(X.ndim(), 4);

  const float* Xdata = X.data<float>();
  float* Pdata = P->mutable_data<float>();

  // Softmax for each x,y location
  GroupSpatialSoftmaxKernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS,
                         0, context_.hip_stream()>>>(
      N, A, W, H, Xdata, Pdata, num_classes_);
  return true;
}


template<>
bool GroupSpatialSoftmaxGradientOp<float, HIPContext>::RunOnDevice() {
  auto& Y = Input(0);  // Probabilities from softmax
  auto& dY = Input(1);


  DCHECK_EQ(Y.ndim(), 4);

  int N = Y.dim32(0);
  int D = Y.dim32(1);
  int H = Y.dim32(2);
  int W = Y.dim32(3);
  int A = D / num_classes_;

  auto* dX = Output(0, Y.sizes(), at::dtype<float>());

  if (sum_probs_.size() != N * A * H * W) {
    ReinitializeTensor(&sum_probs_, {N * A * H * W}, at::dtype<float>().device(CUDA));
  }

  const float* Ydata = Y.data<float>();
  const float* dYdata = dY.data<float>();
  float* dXdata = dX->mutable_data<float>();

  float* sum_probs_data = sum_probs_.mutable_data<float>();
  math::Set<float, HIPContext>(
      sum_probs_.size(), 0.0f, sum_probs_data, &context_);

  // Complete math:
  // J_ij = h_i (delta_ij - h_j)
  // d x_i = sum_j d h_ij = sum_j J_ij * dy_j
  //       = sum_j h_i (delta_ij - h_j) * dy_j
  //       = h_i dy_i - (sum_j h_i h_j dy_j)
  //       = h_i dy_i - h_i sum_j h_j dy_j

  // Step 0: dx = dy
  context_.Copy<float, HIPContext, HIPContext>(Y.size(), dYdata, dXdata);

  // Step 1: s = Sum(dY[j] * Y[j])
  SumProbsKernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS, 0,
                   context_.hip_stream()>>>(
    N, A, W, H, Ydata, dYdata, sum_probs_data, num_classes_);

  // Step 2: dX[i] = dX[i] - s
  SubSumKernel<<<CAFFE_GET_BLOCKS(Y.size()), CAFFE_CUDA_NUM_THREADS, 0,
                  context_.hip_stream()>>>(
    N, A, W, H, sum_probs_.data<float>(), dXdata, num_classes_);

  // Step 3: dX[i] = Y[i] * dX[i]
  math::Mul<float, HIPContext>(Y.size(), dXdata, Ydata, dXdata, &context_);

  return true;
}


REGISTER_HIP_OPERATOR(GroupSpatialSoftmax,
                       GroupSpatialSoftmaxOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(GroupSpatialSoftmaxGradient,
                       GroupSpatialSoftmaxGradientOp<float, HIPContext>);
} // namespace caffe2

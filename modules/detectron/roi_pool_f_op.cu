#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2016-present, Facebook, Inc.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cfloat>

#include "caffe2/core/context_gpu.h"
#include "modules/detectron/roi_pool_f_op.h"

namespace caffe2 {

namespace {

template <typename T>
inline __device__ T gpu_atomic_add(const T val, T* address);

template <>
inline __device__
float gpu_atomic_add(const float val, float* address) {
  return atomicAdd(address, val);
}

template <typename T>
__global__ void RoIPoolFForward(const int nthreads, const T* bottom_data,
    const T spatial_scale, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const T* bottom_rois, T* top_data, int* argmax_data) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    const T* offset_bottom_rois = bottom_rois + n * 5;
    int roi_batch_ind = offset_bottom_rois[0];
    int roi_start_w = roundf(offset_bottom_rois[1] * spatial_scale);
    int roi_start_h = roundf(offset_bottom_rois[2] * spatial_scale);
    int roi_end_w = roundf(offset_bottom_rois[3] * spatial_scale);
    int roi_end_h = roundf(offset_bottom_rois[4] * spatial_scale);

    // Force malformed ROIs to be 1x1
    int roi_width = max(roi_end_w - roi_start_w + 1, 1);
    int roi_height = max(roi_end_h - roi_start_h + 1, 1);
    T bin_size_h = static_cast<T>(roi_height)
                       / static_cast<T>(pooled_height);
    T bin_size_w = static_cast<T>(roi_width)
                       / static_cast<T>(pooled_width);

    int hstart = static_cast<int>(floor(static_cast<T>(ph)
                                        * bin_size_h));
    int wstart = static_cast<int>(floor(static_cast<T>(pw)
                                        * bin_size_w));
    int hend = static_cast<int>(ceil(static_cast<T>(ph + 1)
                                     * bin_size_h));
    int wend = static_cast<int>(ceil(static_cast<T>(pw + 1)
                                     * bin_size_w));

    // Add roi offsets and clip to input boundaries
    hstart = min(max(hstart + roi_start_h, 0), height);
    hend = min(max(hend + roi_start_h, 0), height);
    wstart = min(max(wstart + roi_start_w, 0), width);
    wend = min(max(wend + roi_start_w, 0), width);
    bool is_empty = (hend <= hstart) || (wend <= wstart);

    // Define an empty pooling region to be zero
    T maxval = is_empty ? 0 : -FLT_MAX;
    // If nothing is pooled, argmax = -1 causes nothing to be backprop'd
    int maxidx = -1;
    const T* offset_bottom_data =
        bottom_data + (roi_batch_ind * channels + c) * height * width;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        int bottom_index = h * width + w;
        if (offset_bottom_data[bottom_index] > maxval) {
          maxval = offset_bottom_data[bottom_index];
          maxidx = bottom_index;
        }
      }
    }
    top_data[index] = maxval;
    argmax_data[index] = maxidx;
  }
}

template <typename T>
__global__ void RoIPoolFBackward(const int nthreads, const T* top_diff,
    const int* argmax_data, const int num_rois, const T spatial_scale,
    const int channels, const int height, const int width,
    const int pooled_height, const int pooled_width, T* bottom_diff,
    const T* bottom_rois) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    const T* offset_bottom_rois = bottom_rois + n * 5;
    int roi_batch_ind = offset_bottom_rois[0];
    int bottom_offset = (roi_batch_ind * channels + c) * height * width;
    int top_offset    = (n * channels + c) * pooled_height * pooled_width;
    const T* offset_top_diff = top_diff + top_offset;
    T* offset_bottom_diff = bottom_diff + bottom_offset;
    const int* offset_argmax_data = argmax_data + top_offset;

    int argmax = offset_argmax_data[ph * pooled_width + pw];
    if (argmax != -1) {
      gpu_atomic_add(
          static_cast<T>(offset_top_diff[ph * pooled_width + pw]),
          offset_bottom_diff + argmax);
    }
  }
}

} // namespace

template<>
bool RoIPoolFOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0);  // Input data to pool
  auto& R = Input(1);  // RoIs

  if (R.size() == 0) {
    // Handle empty rois
    std::vector<int64_t> sizes = {0, X.dim32(1), pooled_height_, pooled_width_};
    /* auto* Y = */ Output(0, sizes, at::dtype<float>());
    /* auto* A = */ Output(1, sizes, at::dtype<int>());
    return true;
  }

  auto* Y = Output(0, {R.dim32(0), X.dim32(1), pooled_height_, pooled_width_}, at::dtype<float>()); // RoI pooled data
  auto* A = Output(1, Y->sizes(), at::dtype<int>()); // argmaxes
  int output_size = Y->size();
  RoIPoolFForward<float><<<CAFFE_GET_BLOCKS(output_size),
                          CAFFE_CUDA_NUM_THREADS,
                          0, context_.hip_stream()>>>(
      output_size, X.data<float>(), spatial_scale_, X.dim32(1), X.dim32(2),
      X.dim32(3), pooled_height_, pooled_width_, R.data<float>(),
      Y->mutable_data<float>(), A->mutable_data<int>());
  return true;
}


template<>
bool RoIPoolFGradientOp<float, HIPContext>::RunOnDevice() {
  auto& X  = Input(0);  // Input data to pool
  auto& R  = Input(1);  // RoIs
  auto& A  = Input(2);  // argmaxes
  auto& dY = Input(3);  // Gradient of net w.r.t. output of "forward" op
                        // (aka "gradOutput")

  auto* dX = Output(0, X.sizes(), at::dtype<float>());    // Gradient of net w.r.t. input to "forward" op
                        // (aka "gradInput")
  // Must zero-out dX before accumulating gradients
  math::Set<float, HIPContext>(
      dX->size(), 0.f, dX->mutable_data<float>(), &context_);
  if (dY.size() > 0) {  // Handle possibly empty gradient if there were no rois
    RoIPoolFBackward<float><<<CAFFE_GET_BLOCKS(dY.size()),
                             CAFFE_CUDA_NUM_THREADS,
                             0, context_.hip_stream()>>>(
        dY.size(), dY.data<float>(), A.data<int>(), R.dim32(0), spatial_scale_,
        X.dim32(1), X.dim32(2), X.dim32(3), pooled_height_, pooled_width_,
        dX->mutable_data<float>(), R.data<float>());
  }
  return true;
}


REGISTER_HIP_OPERATOR(RoIPoolF,
                       RoIPoolFOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(RoIPoolFGradient,
                       RoIPoolFGradientOp<float, HIPContext>);
} // namespace caffe2

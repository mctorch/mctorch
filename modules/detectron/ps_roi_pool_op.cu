#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2016-present, Facebook, Inc.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

// Based on https://github.com/daijifeng001/caffe-rfcn/blob/r-fcn/src/caffe/layers/psroi_pooling_layer.cu
//
// ------------------------------------------------------------------
// R-FCN
// Copyright (c) 2016 Microsoft
// Licensed under The MIT License [see r-fcn/LICENSE for details]
// Written by Yi Li
// ------------------------------------------------------------------
//
// COPYRIGHT
//
// All contributions by the University of California:
// Copyright (c) 2014, 2015, The Regents of the University of California
// (Regents)
// All rights reserved.
//
// All other contributions:
// Copyright (c) 2014, 2015, the respective contributors
// All rights reserved.
//
// Caffe uses a shared copyright model: each contributor holds copyright over
// their contributions to Caffe. The project versioning records all such
// contribution and copyright details. If a contributor wants to further mark
// their specific copyright on a particular contribution, they should indicate
// their copyright solely in the commit message of the change when it is
// committed.
//
// LICENSE
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice,
//    this list of conditions and the following disclaimer.
// 2. Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
// ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE
// LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
// CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
// SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
// INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
// CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
// ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
// POSSIBILITY OF SUCH DAMAGE.
//
// CONTRIBUTION AGREEMENT
//
// By contributing to the BVLC/caffe repository through pull-request, comment,
// or otherwise, the contributor releases their content to the
// license and copyright terms herein.

#include <cfloat>

#include "caffe2/core/context_gpu.h"
#include "modules/detectron/ps_roi_pool_op.h"

namespace caffe2 {

namespace {

template <typename T>
inline __device__ T gpu_atomic_add(const T val, T* address);

template <>
inline __device__
float gpu_atomic_add(const float val, float* address) {
  return atomicAdd(address, val);
}

template <typename T>
__global__ void PSRoIPoolForward(
    const int nthreads,
    const T* bottom_data,
    const T spatial_scale,
    const int channels,
    const int height,
    const int width,
    const int pooled_height,
    const int pooled_width,
    const T* bottom_rois,
    const int output_dim,
    const int group_size,
    T* top_data,
    int* mapping_channel) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // The output is in order (n, ctop, ph, pw)
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int ctop = (index / pooled_width / pooled_height) % output_dim;
    int n = index / pooled_width / pooled_height / output_dim;

    // [start, end) interval for spatial sampling
    const T* offset_bottom_rois = bottom_rois + n * 5;
    int roi_batch_ind = offset_bottom_rois[0];
    T roi_start_w = static_cast<T>(
      roundf(offset_bottom_rois[1])) * spatial_scale;
    T roi_start_h = static_cast<T>(
      roundf(offset_bottom_rois[2])) * spatial_scale;
    T roi_end_w = static_cast<T>(
      roundf(offset_bottom_rois[3]) + 1.) * spatial_scale;
    T roi_end_h = static_cast<T>(
      roundf(offset_bottom_rois[4]) + 1.) * spatial_scale;

    // Force too small ROIs to be 1x1
    T roi_width = c10::cuda::compat::max(roi_end_w - roi_start_w, static_cast<T>(0.1));  // avoid 0
    T roi_height = c10::cuda::compat::max(roi_end_h - roi_start_h, static_cast<T>(0.1));

    // Compute w and h at bottom
    T bin_size_h = roi_height / static_cast<T>(pooled_height);
    T bin_size_w = roi_width / static_cast<T>(pooled_width);

    // Add roi offsets and clip to input boundaries
    int hstart = floor(
      static_cast<T>(ph) * bin_size_h + roi_start_h);
    int wstart = floor(
      static_cast<T>(pw)* bin_size_w + roi_start_w);
    int hend = ceil(
      static_cast<T>(ph + 1) * bin_size_h + roi_start_h);
    int wend = ceil(
      static_cast<T>(pw + 1) * bin_size_w + roi_start_w);

    hstart = min(max(hstart, 0), height);
    hend = min(max(hend, 0), height);
    wstart = min(max(wstart, 0),width);
    wend = min(max(wend, 0), width);
    bool is_empty = (hend <= hstart) || (wend <= wstart);

    int gw = pw;
    int gh = ph;
    int c = (ctop * group_size + gh) * group_size + gw;

    const T* offset_bottom_data =
      bottom_data + (roi_batch_ind * channels + c) * height * width;
    T out_sum = 0;
    for (int h = hstart; h < hend; ++h){
     for (int w = wstart; w < wend; ++w){
       int bottom_index = h*width + w;
       out_sum += offset_bottom_data[bottom_index];
     }
    }

    T bin_area = (hend - hstart) * (wend - wstart);
    top_data[index] = is_empty ? 0. : out_sum / bin_area;
    mapping_channel[index] = c;
  }
}

template <typename T>
__global__ void PSRoIPoolBackward(
    const int nthreads,
    const T* top_diff,
    const int* mapping_channel,
    const int num_rois,
    const T spatial_scale,
    const int channels,
    const int height,
    const int width,
    const int pooled_height,
    const int pooled_width,
    const int output_dim,
    T* bottom_diff,
    const T* bottom_rois) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // The output is in order (n, ctop, ph, pw)
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int n = index / pooled_width / pooled_height / output_dim;

    // [start, end) interval for spatial sampling
    const T* offset_bottom_rois = bottom_rois + n * 5;
    int roi_batch_ind = offset_bottom_rois[0];
    T roi_start_w = static_cast<T>(
      roundf(offset_bottom_rois[1])) * spatial_scale;
    T roi_start_h = static_cast<T>(
      roundf(offset_bottom_rois[2])) * spatial_scale;
    T roi_end_w = static_cast<T>(
      roundf(offset_bottom_rois[3]) + 1.) * spatial_scale;
    T roi_end_h = static_cast<T>(
      roundf(offset_bottom_rois[4]) + 1.) * spatial_scale;

    // Force too small ROIs to be 1x1
    T roi_width = c10::cuda::compat::max(roi_end_w - roi_start_w, static_cast<T>(0.1)); //avoid 0
    T roi_height = c10::cuda::compat::max(roi_end_h - roi_start_h, static_cast<T>(0.1));

    // Compute w and h at bottom
    T bin_size_h = roi_height / static_cast<T>(pooled_height);
    T bin_size_w = roi_width / static_cast<T>(pooled_width);

    int hstart = floor(
      static_cast<T>(ph)* bin_size_h + roi_start_h);
    int wstart = floor(
      static_cast<T>(pw)* bin_size_w + roi_start_w);
    int hend = ceil(
      static_cast<T>(ph + 1) * bin_size_h + roi_start_h);
    int wend = ceil(
      static_cast<T>(pw + 1) * bin_size_w + roi_start_w);
    // Add roi offsets and clip to input boundaries
    hstart = min(max(hstart, 0), height);
    hend = min(max(hend, 0), height);
    wstart = min(max(wstart, 0), width);
    wend = min(max(wend, 0), width);
    bool is_empty = (hend <= hstart) || (wend <= wstart);

    // Compute c at bottom
    int c = mapping_channel[index];
    T* offset_bottom_diff =
      bottom_diff + (roi_batch_ind * channels + c) * height * width;
    T bin_area = (hend - hstart) * (wend - wstart);
    T diff_val = is_empty ? 0. : top_diff[index] / bin_area;
    for (int h = hstart; h < hend; ++h){
      for (int w = wstart; w < wend; ++w){
        int bottom_index = h * width + w;
        gpu_atomic_add(diff_val, offset_bottom_diff + bottom_index);
      }
    }
  }
}

} // namespace

template<>
bool PSRoIPoolOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0);  // Input data to pool
  auto& R = Input(1);  // RoIs

  auto* Y = Output(0, {R.dim32(0), output_dim_, pooled_height_, pooled_width_}, at::dtype<float>()); // PSRoI pooled data
  auto* A = Output(1, Y->sizes(), at::dtype<int>()); // mapping_channel
  int output_size = Y->numel();
  PSRoIPoolForward<float><<<CAFFE_GET_BLOCKS(output_size),
                            CAFFE_CUDA_NUM_THREADS,
                            0, context_.hip_stream()>>>(
      output_size, X.data<float>(), spatial_scale_, X.dim32(1), X.dim32(2),
      X.dim32(3), pooled_height_, pooled_width_, R.data<float>(), output_dim_,
      group_size_, Y->mutable_data<float>(), A->mutable_data<int>());
  return true;
}


template<>
bool PSRoIPoolGradientOp<float, HIPContext>::RunOnDevice() {
  auto& X  = Input(0);  // Input data to pool
  auto& R  = Input(1);  // RoIs
  auto& A  = Input(2);  // mapping channels
  auto& dY = Input(3);  // Gradient of net w.r.t. output of "forward" op
                        // (aka "gradOutput")

  auto* dX = Output(0, X.sizes(), at::dtype<float>()); // Gradient of net w.r.t. input to "forward" op
                                                       // (aka "gradInput")
  // Must zero-out dX before accumulating gradients
  math::Set<float, HIPContext>(
      dX->size(), 0.f, dX->mutable_data<float>(), &context_);
  PSRoIPoolBackward<float><<<CAFFE_GET_BLOCKS(dY.size()),
                             CAFFE_CUDA_NUM_THREADS,
                             0, context_.hip_stream()>>>(
      dY.size(), dY.data<float>(), A.data<int>(), R.dim32(0), spatial_scale_,
      X.dim32(1), X.dim32(2), X.dim32(3), pooled_height_, pooled_width_,
      output_dim_, dX->mutable_data<float>(), R.data<float>());
  return true;
}


REGISTER_HIP_OPERATOR(PSRoIPool,
                       PSRoIPoolOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(PSRoIPoolGradient,
                       PSRoIPoolGradientOp<float, HIPContext>);
} // namespace caffe2

/**
 * Copyright (c) 2016-present, Facebook, Inc.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/* SampleAs by Kaiming He for Mask R-CNN
X.dim32(0) = L.dim32(0)
Y's output samples are the samples of X for which L > 0.
*/
#include <cfloat>

#include "caffe2/core/context_gpu.h"
#include "modules/detectron/sample_as_op.h"

#include <stdio.h>

namespace caffe2 {

template <>
bool SampleAsOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0); // Input data to be sliced
  auto& L = Input(1); // Target data that provide the identity

  CAFFE_ENFORCE(
      X.dim32(0) == L.dim32(0),
      "X.dim32(0) must be equal to L.dim32(0)",
      "(",
      X.dim32(0),
      " vs. ",
      L.dim32(0),
      ")");

  // copy L to CPU:
  std::vector<int> labels(L.dim32(0));
  context_.CopyBytes<HIPContext, CPUContext>(
      L.dim32(0) * sizeof(int), L.data<int>(), &labels[0]);
  // Make sure that the copy is finished
  context_.FinishDeviceComputation();

  int count = 0;
  for (int i = 0; i < L.dim32(0); i++) {
    if (labels[i] > 0) {
      count++;
    }
  }
  assert(count > 0);

  // resize Y
  vector<int64_t> out_shape(X.sizes().vec());
  out_shape[0] = count;
  auto* Y = Output(0, out_shape, at::dtype<float>()); // Sliced data (Y.dim32(0) = num of (L > 0))

  const int len = X.size() / X.dim32(0);

  float* output = Y->mutable_data<float>();
  for (int i = 0; i < L.dim32(0); i++) {
    if (labels[i] > 0) {
      context_.CopyBytes<HIPContext, HIPContext>(
          len * sizeof(float), X.data<float>() + i * len, output);
      output += len;
    } // if
  } // i

  return true;
}

template <>
bool SampleAsGradientOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0);
  auto& L = Input(1);
  auto& dY = Input(2);


  auto* dX = Output(0, X.sizes(), at::dtype<float>());

  // copy L to CPU:
  std::vector<int> labels(L.dim32(0));
  context_.CopyBytes<HIPContext, CPUContext>(
      L.dim32(0) * sizeof(int), L.data<int>(), &labels[0]);
  // Make sure that the copy is finished
  context_.FinishDeviceComputation();

  // zero-out dX
  math::Set<float, HIPContext>(
      dX->size(), 0.f, dX->mutable_data<float>(), &context_);

  const int len = X.size() / X.dim32(0);

  const float* input = dY.data<float>();
  for (int i = 0; i < L.dim32(0); i++) {
    if (labels[i] > 0) {
      context_.CopyBytes<HIPContext, HIPContext>(
          len * sizeof(float), input, dX->mutable_data<float>() + i * len);
      input += len;
    } // if
  } // i

  return true;
}

REGISTER_HIP_OPERATOR(SampleAs, SampleAsOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(
    SampleAsGradient,
    SampleAsGradientOp<float, HIPContext>);
} // namespace caffe2

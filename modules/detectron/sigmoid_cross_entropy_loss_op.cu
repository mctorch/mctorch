#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2016-present, Facebook, Inc.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "caffe2/core/context_gpu.h"
#include "modules/detectron/sigmoid_cross_entropy_loss_op.h"

namespace caffe2 {

namespace {
__global__ void ElementwiseMaxKernel(const int n, float* data, const float a) {
  CUDA_1D_KERNEL_LOOP(index, n) {
    data[index] = (data[index] > a) ? data[index] : a;
  }
}

__global__ void SigmoidCrossEntropyLossKernel(
    const int n,
    const float* logits,
    const int* targets,
    float* losses,
    float* counts) {
  CUDA_1D_KERNEL_LOOP(index, n) {
    if (targets[index] == -1) {
      losses[index] = 0.;
      counts[index] = 0.;
    } else {
      losses[index] =
          -1. * logits[index] * (targets[index] - (logits[index] >= 0)) +
          logf(
              1 +
              expf(logits[index] - 2 * logits[index] * (logits[index] >= 0)));
      counts[index] = 1.;
    }
  }
}

__global__ void SigmoidCrossEntropyLossGradientKernel(
    const int n,
    const float* logits,
    const int* targets,
    float* d_logits,
    float* counts) {
  CUDA_1D_KERNEL_LOOP(index, n) {
    if (targets[index] == -1) {
      d_logits[index] = 0.;
      counts[index] = 0.;
    } else {
      d_logits[index] = 1. / (1. + expf(-logits[index])) - targets[index];
      counts[index] = 1.;
    }
  }
}
} // namespace

template <>
bool SigmoidCrossEntropyLossOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0);
  auto& T = Input(1);


  CAFFE_ENFORCE(
      X.size() == T.size(),
      "Logit and target must have the same size",
      "(",
      X.size(),
      " vs. ",
      T.size(),
      ")");
  auto* avg_loss = Output(0, vector<int64_t>(), at::dtype<float>());
  counts_.ResizeLike(X);
  losses_.ResizeLike(X);
  ReinitializeTensor(&normalizer_, vector<int64_t>(), at::dtype<float>().device(CUDA));
  SigmoidCrossEntropyLossKernel<<<
      CAFFE_GET_BLOCKS(X.size()),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(
      X.size(),
      X.data<float>(),
      T.data<int>(),
      losses_.mutable_data<float>(),
      counts_.mutable_data<float>());
  float* avg_loss_data = avg_loss->mutable_data<float>();
  math::Sum<float, HIPContext>(
      losses_.size(), losses_.data<float>(), avg_loss_data, &context_);
  if (normalize_) {
    float* normalizer_data = normalizer_.mutable_data<float>();
    math::Sum<float, HIPContext>(
        counts_.size(), counts_.data<float>(), normalizer_data, &context_);
    // Prevent division by zero is all counts are zero
    ElementwiseMaxKernel<<<
        CAFFE_GET_BLOCKS(normalizer_.size()),
        CAFFE_CUDA_NUM_THREADS,
        0,
        context_.hip_stream()>>>(normalizer_.size(), normalizer_data, 1e-5);
    math::Div<float, HIPContext>(
        1, avg_loss_data, normalizer_data, avg_loss_data, &context_);
  }
  math::Scale<float, float, HIPContext>(
      1, scale_, avg_loss_data, avg_loss_data, &context_);

  return true;
}

template <>
bool SigmoidCrossEntropyLossGradientOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0);
  auto& T = Input(1);
  auto& d_avg_loss = Input(2);


  auto* dX = Output(0, X.sizes(), at::dtype<float>());
  counts_.ResizeLike(X);
  ReinitializeTensor(&normalizer_, vector<int64_t>(), at::dtype<float>().device(CUDA));
  SigmoidCrossEntropyLossGradientKernel<<<
      CAFFE_GET_BLOCKS(X.size()),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(
      X.size(),
      X.data<float>(),
      T.data<int>(),
      dX->mutable_data<float>(),
      counts_.mutable_data<float>());
  if (normalize_) {
    float* normalizer_data = normalizer_.mutable_data<float>();
    math::Sum<float, HIPContext>(
        counts_.size(), counts_.data<float>(), normalizer_data, &context_);
    // Prevent division by zero is all counts are zero
    ElementwiseMaxKernel<<<
        CAFFE_GET_BLOCKS(normalizer_.size()),
        CAFFE_CUDA_NUM_THREADS,
        0,
        context_.hip_stream()>>>(normalizer_.size(), normalizer_data, 1e-5);
    math::Div<float, HIPContext>(
        1,
        d_avg_loss.data<float>(),
        normalizer_data,
        normalizer_data,
        &context_);
    math::Scale<float, float, HIPContext>(
        1, scale_, normalizer_data, normalizer_data, &context_);
    math::Scale<float, float, HIPContext>(
        dX->size(),
        normalizer_data,
        dX->data<float>(),
        dX->mutable_data<float>(),
        &context_);
  } else {
    math::Scale<float, float, HIPContext>(
        dX->size(),
        scale_,
        dX->data<float>(),
        dX->mutable_data<float>(),
        &context_);
    math::Scale<float, float, HIPContext>(
        dX->size(),
        d_avg_loss.data<float>(),
        dX->data<float>(),
        dX->mutable_data<float>(),
        &context_);
  }
  return true;
}

REGISTER_HIP_OPERATOR(
    SigmoidCrossEntropyLoss,
    SigmoidCrossEntropyLossOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(
    SigmoidCrossEntropyLossGradient,
    SigmoidCrossEntropyLossGradientOp<float, HIPContext>);
} // namespace caffe2

#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2016-present, Facebook, Inc.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cfloat>

#include "caffe2/core/context_gpu.h"
#include "modules/detectron/softmax_focal_loss_op.h"

namespace caffe2 {

namespace {

__global__ void SpatialSoftmaxKernel(const int N, const int A,
    const int H, const int W, const float* Xdata, float* Pdata,
    const int num_classes) {
  CUDA_1D_KERNEL_LOOP(index, N * A * H * W) {
    int D = num_classes * A;
    int x = index % W;
    int y = (index / W) % H;
    int a = (index / (W * H)) % A;
    int i = index / W / H / A;

    // Subtract max on each cell for numerical reasons
    float max_val = -FLT_MAX;
    for(int c = a * num_classes; c < (a + 1) * num_classes; ++c) {
      int idx = i * (H * W * D) +  c * (H * W) + y * W + x;
      max_val = max(max_val, Xdata[idx]);
    }
    // Exponentiate
    float expsum = 0.0f;
    for(int c = a * num_classes; c < (a + 1) * num_classes; ++c) {
      int idx = i * (H * W * D) + c * (H * W) + y * W + x;
      float expx = exp(Xdata[idx] - max_val);
      Pdata[idx] = expx;
      expsum += expx;
    }
    // Normalize
    for(int c = a * num_classes; c < (a + 1) * num_classes; ++c) {
      int idx = i * (H * W * D) + c * (H * W) + y * W + x;
      Pdata[idx] /= expsum;
    }
  }
}


__global__ void SoftmaxFocalLossKernel(
    const int N, const int A, const int H, const int W,
    const float* Pdata, const int* targets, float* losses,
    const float* weight_pos, const float gamma, const float alpha,
    const int num_classes) {
  CUDA_1D_KERNEL_LOOP(i, N * A * H * W) {
    int D = A * num_classes;
    int x = i % W;
    int y = (i / W) % H;
    int a = (i / (W * H)) % A;
    int n = i / (W * H * A);
    const int label = static_cast<int>(targets[i]);

    float Np = c10::cuda::compat::max(weight_pos[0], static_cast<float>(1.0));
    float z = (label == 0) * (1 - alpha) / Np +
              (label >= 1) * alpha / Np;

    losses[i] = 0.0;
    if (label >= 0) {
      int offset = a * num_classes;
      int idx = n * (H * W * D) + (offset + label) * (H * W) + y * W + x;
      losses[i] =
          -(pow(1.0f - Pdata[idx], gamma) *
          log(c10::cuda::compat::max(Pdata[idx], FLT_MIN))) * z;
    }
  }
}


__global__ void SoftmaxFocalLossGradientWeightKernel(
    const int N, const int A, const int H, const int W,
    const float* Pdata, const int* targets, float* buff,
    const float* weight_pos, const float gamma, const float alpha,
    const int num_classes) {
  CUDA_1D_KERNEL_LOOP(i, N * A * H * W) {
    int D = A * num_classes;
    int x = i % W;
    int y = (i / W) % H;
    int a = (i / (W * H)) % A;
    int n = i / (W * H * A);
    const int label = static_cast<int>(targets[i]);
    float Np = c10::cuda::compat::max(weight_pos[0], static_cast<float>(1.0));
    float z =  (label == 0) * (1 - alpha) / Np +
               (label >= 1) * alpha / Np;

    buff[i] = 0.0;
    if (label >= 0) {
      int offset = a * num_classes;
      int idx = n * (H * W * D) + (offset + label) * (H * W) + y * W + x;
      float onemp = 1. - Pdata[idx];
      float p = Pdata[idx];
      buff[i] =
          (-pow(onemp, gamma) +
          gamma * pow(onemp, gamma - 1) * p * log(c10::cuda::compat::max(p, FLT_MIN))) * z;
    }
  }
}


__global__ void SoftmaxFocalLossGradientKernel(
    const int N, const int D, const int H, const int W,
    const float* Pdata, const int* targets, const float* buff,
    const float* d_loss_data, float* dX, const int num_classes) {
  CUDA_1D_KERNEL_LOOP(i, N * D * H * W) {
    int A = D / num_classes;
    int x = i % W;
    int y = (i / W) % H;
    int d = (i / (W * H)) % D;
    int a = d / num_classes;
    int c = d % num_classes;
    int n = i / (W * H * D);
    float d_loss = *d_loss_data;

    int ind = n * (H * W * A) + a * (H * W) + y * W + x;
    const int label = static_cast<int>(targets[ind]);

    float c1 = (label >= 0) * 1.0;
    float c2 = (label == c) * 1.0;
    dX[i] = 0.0;
    dX[i] = c1 * d_loss * buff[ind] * (c2 - Pdata[i]);
  }
}

} // namespace


template <>
bool SoftmaxFocalLossOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0);         // Logits
  auto& T = Input(1);         // Labels
  auto& wp = Input(2);        // num of foregound
   // average loss as output
          // softmax probability, going to be re-used in gradient

  int N = X.dim32(0);
  int D = X.dim32(1);
  int H = X.dim32(2);
  int W = X.dim32(3);
  int A = D / num_classes_;

  ReinitializeTensor(&losses_, {N * A * H * W}, at::dtype<float>().device(CUDA));
  auto* P = Output(1, {N * D * H * W}, at::dtype<float>());
  auto* avg_loss = Output(0, vector<int64_t>(), at::dtype<float>());
  math::Set<float, HIPContext>(
      avg_loss->size(), 0.f, avg_loss->mutable_data<float>(), &context_);
  math::Set<float, HIPContext>(
      P->size(), 0.f, P->mutable_data<float>(), &context_);
  math::Set<float, HIPContext>(
      losses_.size(), 0.f, losses_.mutable_data<float>(), &context_);
  DCHECK_EQ(X.ndim(), 4);

  const float* Xdata = X.data<float>();
  const float* Wdata = wp.data<float>();


  // Spatial Softmax Kernel
  SpatialSoftmaxKernel
      <<<CAFFE_GET_BLOCKS(N * A * H * W), CAFFE_CUDA_NUM_THREADS,
         0, context_.hip_stream()>>>(
    N, A, H, W, Xdata, P->mutable_data<float>(), num_classes_);

  // Compute loss for each x,y location
  const int* Tdata = T.data<int>();
  SoftmaxFocalLossKernel
  <<<CAFFE_GET_BLOCKS(N * A * H * W), CAFFE_CUDA_NUM_THREADS,
      0, context_.hip_stream()>>>(
    N, A, H, W, P->data<float>(), Tdata, losses_.mutable_data<float>(),
    Wdata, gamma_, alpha_, num_classes_);

  // sum the losses
  float* avg_loss_data = avg_loss->mutable_data<float>();
  math::Sum<float, HIPContext>(
      losses_.size(), losses_.data<float>(), avg_loss_data, &context_);
  math::Scale<float, float, HIPContext>(
      1, scale_, avg_loss_data, avg_loss_data, &context_);

  return true;
}


template<>
bool SoftmaxFocalLossGradientOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0);    // Logits
  auto& T = Input(1);    // Label
  auto& wp = Input(2);   // num of foreground example
  auto& P = Input(3);    // Softmax Probability
  auto& d_avg_loss = Input(4);


  int N = X.dim32(0);
  int D = X.dim32(1);
  int H = X.dim32(2);
  int W = X.dim32(3);
  int A = D / num_classes_;

  ReinitializeTensor(&buff_, {N * A * H * W}, at::dtype<float>().device(CUDA));

  auto* dX = Output(0, X.sizes(), at::dtype<float>()); // gradient wrt logits

  const float* Xdata = X.data<float>();
  const int* Tdata = T.data<int>();
  const float* Pdata = P.data<float>();
  const float* Wdata = wp.data<float>();


  // Compute the weight for gradients
  SoftmaxFocalLossGradientWeightKernel
      <<<CAFFE_GET_BLOCKS(N * A * H * W), CAFFE_CUDA_NUM_THREADS,
         0, context_.hip_stream()>>>(
    N, A, H, W, Pdata, Tdata, buff_.mutable_data<float>(),
    Wdata, gamma_, alpha_, num_classes_);
  // Compute the gradient with the weights
  const float* Bdata = buff_.data<float>();
  SoftmaxFocalLossGradientKernel
      <<<CAFFE_GET_BLOCKS(N * D * H * W), CAFFE_CUDA_NUM_THREADS,
         0, context_.hip_stream()>>>(
    N, D, H, W, Pdata, Tdata, Bdata, d_avg_loss.data<float>(),
    dX->mutable_data<float>(), num_classes_);
  math::Scale<float, float, HIPContext>(
      dX->size(),
      scale_,
      dX->data<float>(),
      dX->mutable_data<float>(),
      &context_);
  return true;
}


REGISTER_HIP_OPERATOR(SoftmaxFocalLoss,
                       SoftmaxFocalLossOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(SoftmaxFocalLossGradient,
                       SoftmaxFocalLossGradientOp<float, HIPContext>);
} // namespace caffe2

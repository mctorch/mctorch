#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2016-present, Facebook, Inc.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cfloat>

#include "caffe2/core/context_gpu.h"
#include "modules/detectron/sigmoid_focal_loss_op.h"

namespace caffe2 {

namespace {

__global__ void SigmoidFocalLossKernel(
    const int N, const int D, const int H, const int W, const float* logits,
    const int* targets, const float* weight_pos,
    const float gamma, const float alpha,
    const int num_classes, float* losses) {
  CUDA_1D_KERNEL_LOOP(i, N * D * H * W) {
    int x = i % W;
    int y = (i / W) % H;
    int c = (i / (W * H)) % D;  // channel, here D is channel dim in input NxDxHxW
    int n = i / (W * H * D);    // n in NxDxHxW

    int A = D / num_classes;   // num_anchors = A
    int a = c / num_classes;   // current anchor out of A anchors in D = A * num_cls
    int d = c % num_classes;   // current class
    int t = targets[n * (H * W * A) + a * (H * W) + y * W + x];   // target

    // check whether the class is true class or not.
    // The target classes are in range 1 - 81 and the d is in range 0-80
    // because we predict A*80 dim, so for comparison purpose, compare t and (d+1)
    float c1 = (t == (d + 1));
    float c2 = (t != -1 & t != (d + 1));

    float Np = c10::cuda::compat::max(weight_pos[0], static_cast<float>(1.0));
    float zn = (1.0 - alpha) / Np;
    float zp = alpha / Np;

    // p = 1. / 1. + expf(-x)
    float p = 1. / (1. + expf(-logits[i]));

    // (1 - p)**gamma * log(p) where
    float term1 = powf((1. - p), gamma) * logf(c10::cuda::compat::max(p, FLT_MIN));
    // p**gamma * log(1 - p)
    float term2 =
        powf(p, gamma) *
        (-1. * logits[i] * (logits[i] >= 0) -
         logf(1. + expf(logits[i] - 2. * logits[i] * (logits[i] >= 0))));

    losses[i] = 0.0;
    losses[i] += -c1 * term1 * zp;
    losses[i] += -c2 * term2 * zn;
  }
}

__global__ void SigmoidFocalLossGradientKernel(
    const int N, const int D, const int H, const int W, const float* logits,
    const int* targets, float* dX_data, const float* weight_pos,
    const float gamma, const float alpha, const int num_classes,
    const float* avg_loss) {
  CUDA_1D_KERNEL_LOOP(i, N * D * H * W) {
      float a_loss = avg_loss[0];
      int x = i % W;
      int y = (i / W) % H;
      int c = (i / (W * H)) % D;
      int n = i / (W * H * D);

      int A = D / num_classes;   // num_anchors
      int a = c / num_classes;   // current anchor
      int d = c % num_classes;   // current class

      float Np = c10::cuda::compat::max(weight_pos[0], static_cast<float>(1.0));
      float zn = (1.0 - alpha) / Np;
      float zp = alpha / Np;
      int t = targets[n * (H * W * A) + a * (H * W) + y * W + x];

      float c1 = (t == (d + 1));
      float c2 = (t != -1 & t != (d + 1));
      float p = 1. / (1. + expf(-logits[i]));

      // (1-p)**g * (1 - p - g*p*log(p))
      float term1 =
          powf((1. - p), gamma) *
          (1. - p - (p * gamma * logf(c10::cuda::compat::max(p, FLT_MIN))));
      // (p**g) * (g*(1-p)*log(1-p) - p)
      float term2 =
          powf(p, gamma) *
          ((-1. * logits[i] * (logits[i] >= 0) -
           logf(1. + expf(logits[i] - 2. * logits[i] * (logits[i] >= 0)))) *
           (1. - p) * gamma - p);
      dX_data[i] = 0.0;
      dX_data[i] += -c1 * zp * term1;
      dX_data[i] += -c2 * zn * term2;
      dX_data[i] = dX_data[i] * a_loss;
  }
}
} // namespace

template<>
bool SigmoidFocalLossOp<float, HIPContext>::RunOnDevice() {
  // Input logits, for example: N x (A * 80) x H x W in cls-agnostic
  auto& X = Input(0);
  // Target, for example: N x A x H x W
  auto& T = Input(1);
  // Number of positive examples: scalar
  auto& wp = Input(2);
  // output avg Sigmoid focal loss as mentioned in RetinaNet paper


  int N = X.dim32(0);
  int D = X.dim32(1);
  int H = X.dim32(2);
  int W = X.dim32(3);

  auto* avg_loss = Output(0, vector<int64_t>(), at::dtype<float>());
  losses_.ResizeLike(X);
  float* avg_loss_data = avg_loss->mutable_data<float>();

  SigmoidFocalLossKernel<<<CAFFE_GET_BLOCKS(X.size()),
          CAFFE_CUDA_NUM_THREADS, 0, context_.hip_stream()>>>(
      N, D, H, W, X.data<float>(), T.data<int>(),
      wp.data<float>(), gamma_, alpha_, num_classes_,
      losses_.mutable_data<float>());

  math::Sum<float, HIPContext>(
      losses_.size(), losses_.data<float>(), avg_loss_data, &context_);
  math::Scale<float, float, HIPContext>(
      1, scale_, avg_loss_data, avg_loss_data, &context_);

  return true;
}


template<>
bool SigmoidFocalLossGradientOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0);
  auto& T = Input(1);
  auto& wp = Input(2);
  auto& d_avg_loss = Input(InputSize() - 1);


  // get input shape
  int N = X.dim32(0);
  int D = X.dim32(1);
  int H = X.dim32(2);
  int W = X.dim32(3);

  auto* dX = Output(0, X.sizes(), at::dtype<float>());

  SigmoidFocalLossGradientKernel<<<CAFFE_GET_BLOCKS(X.size()),
          CAFFE_CUDA_NUM_THREADS, 0, context_.hip_stream()>>>(
      N, D, H, W, X.data<float>(), T.data<int>(), dX->mutable_data<float>(),
      wp.data<float>(), gamma_, alpha_, num_classes_,
      d_avg_loss.data<float>());
  math::Scale<float, float, HIPContext>(
      dX->size(),
      scale_,
      dX->data<float>(),
      dX->mutable_data<float>(),
      &context_);

  return true;
}


REGISTER_HIP_OPERATOR(SigmoidFocalLoss,
                       SigmoidFocalLossOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(SigmoidFocalLossGradient,
                       SigmoidFocalLossGradientOp<float, HIPContext>);
} // namespace caffe2

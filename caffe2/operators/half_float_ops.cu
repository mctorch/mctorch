#include "hip/hip_runtime.h"
#include "caffe2/operators/half_float_ops.h"

#include "caffe2/core/context_gpu.h"

#ifdef CAFFE_HAS_CUDA_FP16

namespace caffe2 {
namespace {
__global__ void FloatToHalfKernel(const int N, const float* X, half* Y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    Y[i] = __float2half(X[i]);
  }
}

__global__ void HalfToFloatKernel(const int N, const half* X, float* Y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    Y[i] = __half2float(X[i]);
  }
}
}

template <>
bool FloatToHalfOp<HIPContext>::RunOnDevice() {
  auto& X = Input(0);

  auto* Y = Output(0, X.sizes(), at::dtype<at::Half>());
  FloatToHalfKernel<<<
      CAFFE_GET_BLOCKS(X.numel()),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(
      X.numel(),
      X.data<float>(),
      reinterpret_cast<half*>(Y->template mutable_data<at::Half>()));
  return true;
}

template <>
bool HalfToFloatOp<HIPContext>::RunOnDevice() {
  auto& X = Input(0);

  auto* Y = Output(0, X.sizes(), at::dtype<float>());
  HalfToFloatKernel<<<
      CAFFE_GET_BLOCKS(X.numel()),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(
      X.numel(),
      reinterpret_cast<const half*>(X.data<at::Half>()),
      Y->template mutable_data<float>());
  return true;
}

REGISTER_HIP_OPERATOR(FloatToHalf, FloatToHalfOp<HIPContext>);
REGISTER_HIP_OPERATOR(HalfToFloat, HalfToFloatOp<HIPContext>);
} // namespace caffe2

#endif // CAFFE_HAS_CUDA_FP16

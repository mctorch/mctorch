#include "hip/hip_runtime.h"
#include "caffe2/core/common_gpu.h"
#include "caffe2/core/context_gpu.h"
#include "caffe2/sgd/adam_op.h"

namespace caffe2 {

__global__ void AdamUpdate(
    int N,
    const float* g,
    const float* m,
    const float* v,
    float* ng,
    float* nm,
    float* nv,
    float beta1,
    float beta2,
    float eps_hat,
    float correction,
    const float* lr) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    float gi = g[i];
    float mi = nm[i] = m[i] * beta1 + gi * (1 - beta1);
    float vi = nv[i] = v[i] * beta2 + gi * gi * (1 - beta2);
    ng[i] = lr[0] * correction * mi / (sqrtf(vi) + eps_hat);
  }
}

template <>
void adam_update<HIPContext>(
    int N,
    const float* g,
    const float* m,
    const float* v,
    float* ng,
    float* nm,
    float* nv,
    float beta1,
    float beta2,
    float eps_hat,
    float correction,
    const float* lr,
    HIPContext* context) {
  AdamUpdate<<<
      CAFFE_GET_BLOCKS(N),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context->hip_stream()>>>(
      N, g, m, v, ng, nm, nv, beta1, beta2, eps_hat, correction, lr);
}

__global__ void AdamCompute(
    int N,
    const float* w,
    const float* g,
    const float* m,
    const float* v,
    float* nw,
    float* nm,
    float* nv,
    float beta1,
    float beta2,
    float eps_hat,
    float correction,
    const float* lr) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    float gi = g[i];
    float mi = nm[i] = m[i] * beta1 + gi * (1 - beta1);
    float vi = nv[i] = v[i] * beta2 + gi * gi * (1 - beta2);
    float ng = lr[0] * correction * mi / (sqrtf(vi) + eps_hat);
    nw[i] = w[i] + ng;
  }
}

template <>
void adam_compute<HIPContext>(
    int N,
    const float* w,
    const float* g,
    const float* m,
    const float* v,
    float* nw,
    float* nm,
    float* nv,
    float beta1,
    float beta2,
    float eps_hat,
    float correction,
    const float* lr,
    HIPContext* context) {
  AdamCompute<<<
      CAFFE_GET_BLOCKS(N),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context->hip_stream()>>>(
      N, w, g, m, v, nw, nm, nv, beta1, beta2, eps_hat, correction, lr);
}

__global__ void AdamComputeOutputGrad(
    int N,
    const float* w,
    const float* g,
    const float* m,
    const float* v,
    float* nw,
    float* nm,
    float* nv,
    float* ng,
    float beta1,
    float beta2,
    float eps_hat,
    float correction,
    const float* lr) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    float gi = g[i];
    float mi = nm[i] = m[i] * beta1 + gi * (1 - beta1);
    float vi = nv[i] = v[i] * beta2 + gi * gi * (1 - beta2);
    float ngi = ng[i] = correction * mi / (sqrtf(vi) + eps_hat);
    nw[i] = w[i] + lr[0] * ngi;
  }
}

template <>
void adam_compute_output_grad<HIPContext>(
    int N,
    const float* w,
    const float* g,
    const float* m,
    const float* v,
    float* nw,
    float* nm,
    float* nv,
    float* ng,
    float beta1,
    float beta2,
    float eps_hat,
    float correction,
    const float* lr,
    HIPContext* context) {
  AdamComputeOutputGrad<<<
      CAFFE_GET_BLOCKS(N),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context->hip_stream()>>>(
      N, w, g, m, v, nw, nm, nv, ng, beta1, beta2, eps_hat, correction, lr);
}

template <typename SIndex>
__global__ void SparseAdamKernel(
    const size_t N,
    const size_t grad_slice_sz,
    const float beta1,
    const float beta2,
    const float epsilon,
    float* param,
    float* mom1,
    float* mom2,
    const SIndex* indices,
    const float* grad,
    const float correction,
    const float* lr,
    const float iter) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    const size_t gradIdx = i;
    const SIndex index = indices[i / grad_slice_sz];
    const size_t paramIdx = index * grad_slice_sz + (i % grad_slice_sz);

    float m1n = mom1[paramIdx] =
        mom1[paramIdx] * beta1 + grad[gradIdx] * (1.0f - beta1);
    float m2n = mom2[paramIdx] =
        mom2[paramIdx] * beta2 + grad[gradIdx] * grad[gradIdx] * (1.0f - beta2);
    param[paramIdx] += lr[0] * correction * m1n / (sqrt(m2n) + epsilon);
  }
}

template <typename SIndex>
__global__ void SparseAdamOutputGradKernel(
    const size_t N,
    const size_t grad_slice_sz,
    const float beta1,
    const float beta2,
    const float epsilon,
    float* param,
    float* mom1,
    float* mom2,
    float* output_grad,
    const SIndex* indices,
    const float* grad,
    const float correction,
    const float* lr,
    const float iter) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    const size_t gradIdx = i;
    const SIndex index = indices[i / grad_slice_sz];
    const size_t paramIdx = index * grad_slice_sz + (i % grad_slice_sz);

    float m1n = mom1[paramIdx] =
        mom1[paramIdx] * beta1 + grad[gradIdx] * (1.0f - beta1);
    float m2n = mom2[paramIdx] =
        mom2[paramIdx] * beta2 + grad[gradIdx] * grad[gradIdx] * (1.0f - beta2);
    float gradOut = output_grad[gradIdx] =
        correction * m1n / (sqrt(m2n) + epsilon);
    param[paramIdx] += lr[0] * gradOut;
  }
}

template <>
template <typename SIndex>
bool SparseAdamOp<float, HIPContext>::DoRunWithType() {
  Output(OUTPUT_PARAM)->ResizeLike(Input(PARAM));
  Output(OUTPUT_MOMENT_1)->ResizeLike(Input(MOMENT_1));
  Output(OUTPUT_MOMENT_2)->ResizeLike(Input(MOMENT_2));

  auto N = Input(GRAD).size();
  auto grad_slice_sz = Input(GRAD).size_from_dim(Input(INDICES).ndim());
  const auto iter =
      OperatorBase::Input<Tensor>(ITER, CPU).template data<int64_t>()[0];
  const float correction = sqrtf(1.0f - std::pow(beta2_, iter + 1)) /
      (1.0f - std::pow(beta1_, iter + 1));

  if (OutputSize() == 3) {
    SparseAdamKernel<SIndex>
        <<<CAFFE_GET_BLOCKS(N),
           CAFFE_CUDA_NUM_THREADS,
           0,
           context_.hip_stream()>>>(
            N,
            grad_slice_sz,
            beta1_,
            beta2_,
            epsilon_,
            Output(OUTPUT_PARAM)->template mutable_data<float>(),
            Output(OUTPUT_MOMENT_1)->template mutable_data<float>(),
            Output(OUTPUT_MOMENT_2)->template mutable_data<float>(),
            Input(INDICES).template data<SIndex>(),
            Input(GRAD).template data<float>(),
            correction,
            Input(LR).template data<float>(),
            iter);
  } else {
    Output(OUTPUT_GRAD)->ResizeLike(Input(GRAD));
    SparseAdamOutputGradKernel<SIndex>
        <<<CAFFE_GET_BLOCKS(N),
           CAFFE_CUDA_NUM_THREADS,
           0,
           context_.hip_stream()>>>(
            N,
            grad_slice_sz,
            beta1_,
            beta2_,
            epsilon_,
            Output(OUTPUT_PARAM)->template mutable_data<float>(),
            Output(OUTPUT_MOMENT_1)->template mutable_data<float>(),
            Output(OUTPUT_MOMENT_2)->template mutable_data<float>(),
            Output(OUTPUT_GRAD)->template mutable_data<float>(),
            Input(INDICES).template data<SIndex>(),
            Input(GRAD).template data<float>(),
            correction,
            Input(LR).template data<float>(),
            iter);
  }

  return true;
}

REGISTER_HIP_OPERATOR(Adam, AdamOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(SparseAdam, SparseAdamOp<float, HIPContext>);

} // namespace caffe2

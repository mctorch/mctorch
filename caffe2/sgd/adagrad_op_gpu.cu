#include "hip/hip_runtime.h"
#include <cub/block/block_reduce.cuh>
#include "caffe2/sgd/adagrad_op.h"
#include "caffe2/core/common_gpu.h"
#include "caffe2/core/context_gpu.h"

namespace caffe2 {

__global__ void AdagradUpdate(
    int N,
    const float* w,
    const float* g,
    const float* h,
    float* nw,
    float* nh,
    float epsilon,
    float decay,
    const float* lr) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    float gi = g[i];
    float hi = nh[i] = decay * h[i] + gi * gi;
    nw[i] = w[i] + lr[0] * gi / (sqrtf(hi) + epsilon);
  }
}

template <>
void adagrad_update<HIPContext>(
    int N,
    const float* w,
    const float* g,
    const float* h,
    float* nw,
    float* nh,
    float epsilon,
    float decay,
    const float* lr,
    HIPContext* context) {
  AdagradUpdate<<<
      CAFFE_GET_BLOCKS(N),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context->hip_stream()>>>(N, w, g, h, nw, nh, epsilon, decay, lr);
}

template <typename SIndex, typename THalf>
__global__ void SparseAdagradKernel(
    const size_t N,
    const size_t grad_slice_sz,
    const float epsilon,
    THalf* param,
    THalf* param_mom,
    const SIndex* indices,
    const float* grad,
    const float* lr) {
  const float LR = lr[0];
  CUDA_1D_KERNEL_LOOP(i, N)
  {
    const size_t gradIdx = i;
    const SIndex index = indices[i / grad_slice_sz];
    const size_t paramIdx = index * grad_slice_sz + (i % grad_slice_sz);

    float mom_new = grad[gradIdx] * grad[gradIdx] + param_mom[paramIdx];
    param_mom[paramIdx] = mom_new;
    float param_new =
        LR * grad[gradIdx] / (sqrtf(mom_new) + epsilon) + param[paramIdx];
    param[paramIdx] = param_new;
  }
}

/**
 * Calculate RowwiseSparseAdagrad
 * M: gradients.dims[0]
 * N: gradients.size_from_dim(1)
 * grad: pointer to the gradients
 * param: pointer to weights
 * param_mom: pointer to the momentum
 * indices: keys
 */
template <typename SIndex>
__global__ void RowWiseSparseAdagradKernel(
    const int M,
    const int N,
    const float epsilon,
    float* param,
    float* param_mom,
    const SIndex* indices,
    const float* grad,
    const float* lr) {
  typedef hipcub::BlockReduce<float, CAFFE_CUDA_NUM_THREADS> BlockReduce;
  __shared__ BlockReduce::TempStorage temp_storage;
  // in case gridDim is smaller than M
  for (int i = blockIdx.x; i < M; i += gridDim.x) {
    const SIndex index = indices[i];
    float sum_squares = 0.0;
    __shared__ float row_sum_squares_avg;

    // in case N is bigger than block size which is 512 by default
    for (int j = threadIdx.x; j < N; j += blockDim.x) {
      const float x_ij = grad[i * N + j];
      sum_squares += x_ij * x_ij;
    }
    float reduce_result = BlockReduce(temp_storage).Sum(sum_squares);
    if (threadIdx.x == 0) {
      row_sum_squares_avg = reduce_result / (float)N;
      param_mom[index] += row_sum_squares_avg;
    }
    __syncthreads();
    // update param
    float step = lr[0] / (sqrtf(param_mom[index]) + epsilon);
    for (int j = threadIdx.x; j < N; j += blockDim.x) {
      param[index * N + j] = param[index * N + j] + grad[i * N + j] * step;
    }
  }
}

template <typename T, class Context>
class CUDASparseAdagradOp final : public Operator<Context> {
 public:
  USE_OPERATOR_CONTEXT_FUNCTIONS;
  CUDASparseAdagradOp(const OperatorDef& operator_def, Workspace* ws)
      : Operator<Context>(operator_def, ws),
        epsilon_(this->template GetSingleArgument<float>("epsilon", 1e-5f)) {
    const T decay = this->template GetSingleArgument<T>("decay", 1.0f);
    CAFFE_ENFORCE_EQ(decay, 1.0, "Decay is not supported for SparseAdagradOp");
  }

  bool RunOnDevice() override {
    // Enforce shapes
    CAFFE_ENFORCE_EQ(Input(PARAM).size(), Input(MOMENT_1).size());
    CAFFE_ENFORCE_EQ(Input(LR).size(), 1);
    CAFFE_ENFORCE_EQ(
        Input(PARAM).size_from_dim(1),
        Input(GRAD).size_from_dim(Input(INDICES).ndim()));

    return DispatchHelper<TensorTypes<int32_t, int64_t>>::call(
        this, Input(INDICES));
  }

  template <typename IndexType>
  bool DoRunWithType() {
    auto n = Input(INDICES).size();
    if (n == 0) {
      return true;
    }
    return DispatchHelper<TensorTypes2<float, at::Half>, IndexType>::call(
        this, Input(PARAM));
  }

  template <typename IndexType, typename THalf>
  bool DoRunWithType2() {
    const auto* lr = Input(LR).template data<T>();
    const auto* indices = Input(INDICES).template data<IndexType>();
    const auto* gradIn = Input(GRAD).template data<T>();
    const auto* paramIn = Input(PARAM).template data<THalf>();
    const auto* momentIn = Input(MOMENT_1).template data<THalf>();
    auto* paramOut = Output(OUTPUT_PARAM)->template mutable_data<THalf>();
    auto* momentOut = Output(OUTPUT_MOMENT_1)->template mutable_data<THalf>();

    auto N = Input(GRAD).size();
    auto grad_slice_sz = Input(GRAD).size_from_dim(Input(INDICES).ndim());
    if (N == 0) {
      // empty grad, nothing to do here, not even launching the kernel
      return true;
    }
    SparseAdagradKernel<IndexType, THalf>
        <<<CAFFE_GET_BLOCKS(N),
           CAFFE_CUDA_NUM_THREADS,
           0,
           context_.hip_stream()>>>(
            N,
            grad_slice_sz,
            epsilon_,
            Output(OUTPUT_PARAM)->template mutable_data<THalf>(),
            Output(OUTPUT_MOMENT_1)->template mutable_data<THalf>(),
            Input(INDICES).template data<IndexType>(),
            Input(GRAD).template data<float>(),
            Input(LR).template data<float>());
    return true;
  }

 protected:
  T epsilon_;
  INPUT_TAGS(PARAM, MOMENT_1, INDICES, GRAD, LR);
  OUTPUT_TAGS(OUTPUT_PARAM, OUTPUT_MOMENT_1);
};

template <>
template <typename SIndex>
bool RowWiseSparseAdagradOp<float, HIPContext>::DoRunWithType() {
  auto N = Input(GRAD).size();
  if (N == 0) {
    // empty grad, nothing to do here, not even launching the kernel
    return true;
  }
  // size of the 1st dimension of the input gradient
  auto GRAD_M = Input(GRAD).dim32(0);
  auto GRAD_N = N / GRAD_M;

  // each thread block will handle multiple rows of the input and output
  RowWiseSparseAdagradKernel<<<
      min(GRAD_M, CAFFE_MAXIMUM_NUM_BLOCKS),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(
      GRAD_M,
      GRAD_N,
      epsilon_,
      Output(OUTPUT_PARAM)->template mutable_data<float>(),
      Output(OUTPUT_MOMENT_1)->template mutable_data<float>(),
      Input(INDICES).template data<SIndex>(),
      Input(GRAD).template data<float>(),
      Input(LR).template data<float>());
  return true;
}

REGISTER_HIP_OPERATOR(Adagrad, AdagradOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(SparseAdagrad, CUDASparseAdagradOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(
    RowWiseSparseAdagrad,
    RowWiseSparseAdagradOp<float, HIPContext>);
}
